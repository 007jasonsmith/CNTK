#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "MatrixQuantizerGPU.h"
#include "MatrixQuantizer_kernel.cu"
#include "GPUMatrix.h"

#pragma comment (lib, "cudart.lib")     // instruct linker to reference these libs
#pragma comment (lib, "cublas.lib")
#pragma comment (lib, "cusparse.lib")
#pragma comment (lib, "hiprand.lib")

#pragma warning (disable: 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning (disable: 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this
#pragma warning (disable: 4702) // unreachable code; triggered for unknown reasons


namespace Microsoft { namespace MSR { namespace CNTK {

    // CUDA failed
    // Since the outer code sometimes does not recover properly, as an option we log and die right away.
    // This is needed for our GCD farm which has intermittent CUDA errors that sometimes cause the DBN tool, when running with MPI, to hang instead of terminating.
    void cudafail(const char * msg)
    {
        // TODO: get from an env variable
        bool dieoncudafailure = true;       
        if (!dieoncudafailure)
        {
            throw std::runtime_error(msg);
        }
        fprintf(stderr, "%s\n", msg);
        fprintf(stderr, "cudafail: terminating\n"), fflush(stderr);
#ifdef WIN32
        TerminateProcess(GetCurrentProcess(), EXIT_FAILURE);   // fail the hard way to ensure it won't hang elsewhere
#else
        exit(1);
#endif
    }

    // allows to write cudaFunction() || "error"   (CUDA runtime)
    static 
#ifdef WIN32
    __declspec(noinline)
#endif
    void operator|| (hipError_t rc, const char * msg)
    {
        if (rc != hipSuccess)
        {
            char buf[1000];
            sprintf_s(buf, "%s: %s (cuda error %d)", msg, hipGetErrorString(rc), rc);
            cudafail(buf);
        }
    }

    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::Sync()
    {
        hipDeviceSynchronize() || "hipDeviceSynchronize failed";
    }

    // wait until stream has completed all scheduled operations
    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::SyncStream(hipStream_t stream)
    {
        hipStreamSynchronize(stream) || "hipStreamSynchronize failed";
    }

    // same but for event
    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::SyncEvent(hipEvent_t ev)
    {
        auto rc = hipEventQuery(ev);
        if (rc != hipErrorNotReady)
        {
            // if Event is ready then no need to wait
            rc || "hipEventQuery failed";
            return;
        }
        // we must wait
        hipEventSynchronize(ev) || "hipEventSynchronize failed";
    }


    //lazy initialization 
    template<class ElemType>
    int MatrixQuantizerGPU<ElemType>::numDevices = -1;
    
    template<class ElemType>
    size_t MatrixQuantizerGPU<ElemType>::GetNumDevice()
    {
        if (numDevices < 0)
        {
            hipGetDeviceCount(&numDevices) || "hipGetDeviceCount failed";
            fprintf(stderr, "MatrixQuantizerGPU::GetNumDevice: %d physical CUDA devices detected\n", numDevices);
        }
        return numDevices;
    }

    //streams
    template<class ElemType>
    std::vector<hipStream_t> MatrixQuantizerGPU<ElemType>::m_fetchStreams;
    
    template<class ElemType>
    std::vector<hipStream_t> MatrixQuantizerGPU<ElemType>::m_assignStreams;
    
    template<class ElemType>
    hipStream_t MatrixQuantizerGPU<ElemType>::GetComputeStream() const
    {
        return NULL;
    }
    
    template<class ElemType>
    hipStream_t MatrixQuantizerGPU<ElemType>::GetFetchStream()  const
    {
        return  m_fetchStreams[this->GetDeviceId()]; 
    }
    
    template<class ElemType>
    hipStream_t MatrixQuantizerGPU<ElemType>::GetAssignStream() const
    {
        return  m_assignStreams[this->GetDeviceId()];
    }

    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // computestream: the stream the caller issued the quant op on
    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::FlagQuantizeCompleteEvent(hipStream_t computestream) const
    {
        // schedule to flag the quantize-complete event (on main stream)
        hipEventRecord(m_quantizeCompleteEvent, computestream) || "hipEventRecord failed";
        
        // when running synchronously (for time measurements), then we (CPU) wait right here
        if (m_forceSync)
        {
            SyncStream(computestream);
        }
    }

    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::SyncQuantizeCompleEventAndFetchAndFlagFetchCompleteEvent(char *cpuBuffer, char*gpuBuffer, size_t size) const
    {
        // schedule fetch stream to wait until the last quantize op is complete, i.e. the data in the buffer is now valid
        // wait until commencement
        hipStreamWaitEvent(GetFetchStream(), m_quantizeCompleteEvent, 0/*flags 'must be 0'*/) || "hipStreamWaitEvent failed";    
        
        // schedule to fetch that quantized data into CPU buffer (on a separate transfer stream)
        hipMemcpyAsync(cpuBuffer, gpuBuffer, size, hipMemcpyDeviceToHost, GetFetchStream()) || "hipMemcpyAsync failed";
        
        hipEventRecord(m_fetchCompleteEvent, GetFetchStream()) || "hipEventRecord failed";  // for next GPU operation
        
        // when running synchronously (for time measurements), then we (CPU) wait right here
        if (m_forceSync)
        {
            SyncStream(GetFetchStream());
        }
    }

    // schedule main stream to wait until fetch is complete, i.e. buffer is free again to be written to by GPU code
    // computestream: the stream the caller issued the quant op on
    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::SyncFetchCompleteEvent(hipStream_t computestream) const
    {
        hipStreamWaitEvent(computestream, m_fetchCompleteEvent, 0/*flags 'must be 0'*/) || "hipStreamWaitEvent failed";    // wait until commencement
    }

    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::SyncAssignCompleteEvent(hipStream_t computestream) const
    {
            // schedule to wait for the assign-complete event (on main/compute stream)     --CPU buffer free once main stream does anything after this
            hipStreamWaitEvent(computestream, m_assignCompleteEvent, 0/*flags 'must be 0'*/) || "hipStreamWaitEvent failed";
            
            // Note that the NVidia doc says somewhat confusingly:
            //  * If \p stream is NULL, any future work submitted in any stream will wait for
            //  * \p event to complete before beginning execution. This effectively creates a
            //  * barrier for all future work submitted to the device on this thread.
            // -> it says that this may bring the whole machinery to stall. Or does hipStreamWaitEvent() honor hipStreamNonBlocking?
            // According to NVidia (Jiri Kraus), this works as expected.
    }

    template<class ElemType>
    QuantizedMatrix<ElemType>& MatrixQuantizerGPU<ElemType>::GetTempGPUQuantizedMatrix(size_t nBits)
    {
        // Check if the existing one is good for our needs
        if ((m_tempGPUQuantizedMatrix != nullptr) && (m_tempGPUQuantizedMatrix->GetNumBits() == nBits))
        {
            return *m_tempGPUQuantizedMatrix;
        }
        
        if (m_tempGPUQuantizedMatrix != nullptr)
        {
            delete m_tempGPUQuantizedMatrix;
            m_tempGPUQuantizedMatrix = nullptr;
        }
        
        m_tempGPUQuantizedMatrix = new QuantizedMatrix<ElemType>(this->m_inMatrix.GetNumRows(), this->m_inMatrix.GetNumCols(), nBits, this->GetDeviceId());
        
        return *m_tempGPUQuantizedMatrix;
    }    
    
    /////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    ///cpubuffer should be page-locked memory allocated, otherwise CUDA will not be efficient (hence we don't use STL)
    template<class ElemType>
    MatrixQuantizerGPU<ElemType>::MatrixQuantizerGPU(const Matrix<ElemType>& inMatrix, bool forceSync /*= false*/) 
    : MatrixQuantizer<ElemType>(inMatrix), m_quantizeCompleteEvent(NULL), m_fetchCompleteEvent(NULL),
    m_assignCompleteEvent(NULL), m_forceSync(forceSync), m_tempGPUQuantizedMatrix(nullptr), m_quantizeOpIncludedFetch(false)
    {
        // events
        // Note: Do NOT use hipEventBlockingSync (which supposedly yields the process)--it will totally break hipEventSynchronize(), causing it to take 50 or 100 ms randomly.
        hipEventCreateWithFlags(&m_quantizeCompleteEvent, hipEventDisableTiming) || "hipEventCreateWithFlags failed";
        hipEventCreateWithFlags(&m_fetchCompleteEvent, hipEventDisableTiming) || "hipEventCreateWithFlags failed";
        hipEventCreateWithFlags(&m_assignCompleteEvent, hipEventDisableTiming) || "hipEventCreateWithFlags failed";

        // lazily create the shared transfer streams
        // Using one stream for now for each purpose, shared per device (we can only do one transfer at a time with one stream). For model parallelism, they need to be device-specific.
        if (m_fetchStreams.empty())
        {
            m_fetchStreams.resize(GetNumDevice(), NULL);
            m_assignStreams.resize(GetNumDevice(), NULL);
        }
        
#pragma warning (disable: 4127)
        if (!m_fetchStreams[this->GetDeviceId()])
        {
            hipStreamCreateWithFlags(&m_fetchStreams[this->GetDeviceId()], hipStreamNonBlocking) || "hipStreamCreateWithFlags failed";
            hipStreamCreateWithFlags(&m_assignStreams[this->GetDeviceId()], hipStreamNonBlocking) || "hipStreamCreateWithFlags failed";
        }
    }

    template<class ElemType>
    MatrixQuantizerGPU<ElemType>::~MatrixQuantizerGPU()
    {
        if (nullptr != m_tempGPUQuantizedMatrix)
        {
            delete m_tempGPUQuantizedMatrix;
            m_tempGPUQuantizedMatrix = nullptr;
        }
        
        try
        {
            // BUGBUG: we don't destroy our streams (they are static variables); we need a static destructor, I am too lazy now
            hipEventDestroy(m_assignCompleteEvent);
            hipEventDestroy(m_fetchCompleteEvent);
            hipEventDestroy(m_quantizeCompleteEvent);
            Sync();
        }
        catch (const std::exception &)
        {
            fflush(stderr);        // needed?
            throw;
        }
    }

    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::QuantizeAsync(QuantizedMatrix<ElemType>& outQMatrix)
    {
        // Verify various input matrix parameter's dimensions
        assert((this->m_inMatrix.GetNumRows() == outQMatrix.GetNumRows()) && (this->m_inMatrix.GetNumCols() == outQMatrix.GetNumCols()));
        
        size_t nBits = outQMatrix.GetNumBits();

        PrepareDevice(this->GetDeviceId());
        if (m_forceSync) 
        {
            Sync();             
        }
        
        QuantizedMatrix<ElemType>& outQMatrixGPU = (outQMatrix.GetDeviceId() == CPUDEVICE) ? GetTempGPUQuantizedMatrix(nBits) : outQMatrix;

        // Do the quantization on compute sstream and insert event into stream
        _QuantizeMatrix<ElemType>(this->m_inMatrix.BufferPointer(), this->m_residual->BufferPointer(),
                                  this->m_inMatrix.GetNumRows(), this->m_inMatrix.GetNumCols(),
                                  outQMatrixGPU.GetArray(), nBits, GetComputeStream(),
                                  this->m_residual->BufferPointer());
        
        FlagQuantizeCompleteEvent(GetComputeStream());            

        // copy from gpu to cpu if needed
        m_quantizeOpIncludedFetch = false;
        if (outQMatrix.GetDeviceId() == CPUDEVICE)
        {
            SyncQuantizeCompleEventAndFetchAndFlagFetchCompleteEvent(outQMatrix.GetArray(), outQMatrixGPU.GetArray(), outQMatrixGPU.GetSize());
            m_quantizeOpIncludedFetch = true;
        }
    }

    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::WaitQuantizeAsyncDone()
    {
        PrepareDevice(this->GetDeviceId());
        
        if (m_quantizeOpIncludedFetch)
        {
            SyncEvent(m_fetchCompleteEvent);
        }
        else
        {
            SyncEvent(m_quantizeCompleteEvent);
        }
    }

    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::UnquantizeAsync(QuantizedMatrix<ElemType>& inQMatrix, Matrix<ElemType>& outMatrix, bool add /*= false*/)
    {
        // The outMatrix should be on the same GPU as m_inMatrix
        assert(outMatrix.GetDeviceId() == this->GetDeviceId());
        
        PrepareDevice(this->GetDeviceId());
        
        size_t nBits = inQMatrix.GetNumBits();
        
        // Verify  input matrix parameter's dimensions
        assert((inQMatrix.GetNumRows() == outMatrix.GetNumRows()) && (inQMatrix.GetNumCols() == outMatrix.GetNumCols()));                
        
        QuantizedMatrix<ElemType>& inQMatrixGPU = (inQMatrix.GetDeviceId() == CPUDEVICE) ? GetTempGPUQuantizedMatrix(nBits) : inQMatrix;
        
        if (inQMatrix.GetDeviceId() == CPUDEVICE)
        {
            // schedule assign to GPU (on transfer stream)
            hipMemcpyAsync(inQMatrixGPU.GetArray(), inQMatrix.GetArray(), inQMatrix.GetSize(), hipMemcpyHostToDevice, GetAssignStream()) || "hipMemcpyAsync failed";
            
            // schedule to flag the assign-complete event
            hipEventRecord(m_assignCompleteEvent, GetAssignStream()) || "hipEventRecord failed";    // for subsequent GPU operation to consume this buffer
            
            if (m_forceSync)
            {
                SyncStream(GetAssignStream());
            }
            
            // let the computing stream wait for the assign complete
            SyncAssignCompleteEvent(GetComputeStream());            
        }            
        
        //do the actually unquantization 
        _UnquantizeMatrix(inQMatrixGPU.GetArray(), inQMatrixGPU.GetSize(),
            outMatrix.BufferPointer(), outMatrix.GetNumRows(), outMatrix.GetNumCols(),
            nBits, add, GetComputeStream());

        //flag the event of quantization
        FlagQuantizeCompleteEvent(GetComputeStream());
    }

    template<class ElemType>
    void MatrixQuantizerGPU<ElemType>::WaitUnquantizeAsyncDone()
    {
        PrepareDevice(this->GetDeviceId());
        SyncEvent(m_quantizeCompleteEvent);
    }

    //explicit 
    template class MatrixQuantizerGPU<float>;
    template class MatrixQuantizerGPU<double>;

}}}
