#include "hip/hip_runtime.h"
//
// <copyright file="GPUSparseMatrix.cu" company="Microsoft">
//     Copyright (c) Microsoft Corporation.  All rights reserved.
// </copyright>
//

#include "BestGpu.h"

#ifndef CPUONLY

#include "GPUSparseMatrix.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"
#include "GPUMatrixCUDAKernels.cu"
#include <functional>
#include "CommonMatrix.h"
#include <iostream> // for cout
#include <assert.h>

#pragma warning (disable: 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning (disable: 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this

#ifdef    _WIN32
// thread local storage to access the current stream, initalize to default stream
extern __declspec (thread)
#endif
hipStream_t t_stream;

void CUDACALL(hipError_t x) 
{
    if(x!=hipSuccess) 
    { 
        const char* errmsg = hipGetErrorString(x);
        std::cerr<< "!!!!!!!!CUDA EXCEPTION: " << errmsg << std::endl;

        throw std::runtime_error(errmsg);
    }    
}

void CUSPARSECALL(hipsparseStatus_t x) 
{
    if(x!= HIPSPARSE_STATUS_SUCCESS) 
    {         
        std::cerr << "!!!!!!!!CUSPARSE EXCEPTION: " << std::endl;
        throw std::runtime_error("CUSPARSE EXCEPTION");
    }    
}

namespace Microsoft { namespace MSR { namespace CNTK {
    void PrepareDevice(DEVICEID_TYPE deviceId);

#pragma region Constructors and Destructor

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ZeroInit(const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/, 
        const DEVICEID_TYPE computeDevice /*= AUTOPLACEMATRIX*/)
    {
        if (matrixFormat != MatrixFormat::matrixFormatSparseCSC && matrixFormat != MatrixFormat::matrixFormatSparseCSR &&
            matrixFormat != MatrixFormat::matrixFormatSparseBlockCol && matrixFormat != MatrixFormat::matrixFormatSparseBlockRow)
        {
            throw std::logic_error("GPUSparseMatrix:  unsupported sparse matrix format");
        }

        m_computeDevice = computeDevice; //current GPU device Id
        m_numRows=0;  
        m_numCols=0;
        m_elemSizeAllocated = m_nz = 0; //Number of non-zero elements
        m_totalBufferSizeAllocated = 0;
        m_format = matrixFormat;
        m_externalBuffer = false;
        m_pArray=nullptr; 
        m_matrixName=nullptr;

        //MajorIndexLocation() = nullptr;
        //SecondaryIndexLocation() = nullptr;
        m_rowIdx = nullptr;

        m_blockSize = 0;
        m_blockVal = nullptr;
        m_blockIds = nullptr;

        m_expandedSize = 0;
        m_block2Id = nullptr;
        m_block2UniqId = nullptr;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const MatrixFormat matrixFormat /*= MatrixFormat::matrixFormatSparseCSR*/,
        const DEVICEID_TYPE computeDevice /*= AUTOPLACEMATRIX*/)
    {
        ZeroInit(matrixFormat, computeDevice)
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUMatrix<ElemType>& deepCopy, const MatrixFormat matrixFormat = MatrixFormat::matrixFormatSparseCSR)
    {
        ZeroInit(matrixFormat, deepCopy.GetComputeDeviceId());
        if (!deepCopy.IsEmpty()) 
            SetValue(deepCopy);
    }


    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        DeepCopy(deepCopy);
    }

    // PrepareDevice - Setup the correct cuda context for an operation
    // deviceId - the device on which the operation will take place
    //            defaults to -1, which means use matrices current device
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::PrepareDevice(DEVICEID_TYPE deviceId /*=-1*/) const
    {
        // if default value use current compute device
        if (deviceId == -1)
            deviceId = (DEVICEID_TYPE)m_computeDevice;
        Microsoft::MSR::CNTK::PrepareDevice(deviceId);
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::DeepCopy(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        m_computeDevice=deepCopy.m_computeDevice;
        deepCopy.PrepareDevice();

        Resize(deepCopy.m_numRows, deepCopy.m_numCols, deepCopy.m_nz, deepCopy.m_format);
        CUDACALL(hipMemcpy(NzValues(), deepCopy.NzValues(), NzSize(), hipMemcpyDeviceToDevice));
        CUDACALL(hipMemcpy(MajorIndexLocation(), deepCopy.MajorIndexLocation(), MajorIndexSize(), hipMemcpyDeviceToDevice));
        CUDACALL(hipMemcpy(SecondaryrIndexLocation(), deepCopy.SecondaryrIndexLocation(), SecondaryIndexSize(), hipMemcpyDeviceToDevice));

        m_externalBuffer = deepCopy.m_externalBuffer;
        SetMatrixName(deepCopy.m_matrixName);

        //TODO: to copy other varibles used only for class based LM
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetValue(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        DeepCopy(deepCopy);
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::CopyToDenseMatrix()
    {
        GPUMatrix<ElemType> res(GetComputeDeviceId());
        if (IsEmpty())
            return res;

        PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descr = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO);

        res.Resize(m_numRows, m_numCols);

        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));
        CUSPARSECALL(hipsparseSetStream(cusparseHandle, t_stream));
        if (m_format == MatrixType::matrixFormatSparseCSR)
        {
            if (sizeof(ElemType) == sizeof(float))
            {
                CUSPARSECALL(hipsparseScsr2dense(cusparseHandle, int(m_numRows), int(m_numCols), descr, (float*)NzValues(), RowLocation(), ColLocation(), (float*)res.BufferPointer(), int(m_numRows)));
            }
            else
            {
                CUSPARSECALL(hipsparseDcsr2dense(cusparseHandle, int(m_numRows), int(m_numCols), descr, (double*)NzValues(), RowLocation(), ColLocation(), (double*)res.BufferPointer(), int(m_numRows)));
            }
        }
        else if (m_format == MatrixType::matrixFormatSparseCSC)
        {
            if (sizeof(ElemType) == sizeof(float))
            {
                CUSPARSECALL(hipsparseScsc2dense(cusparseHandle, int(m_numRows), int(m_numCols), descr, (float*)NzValues(), RowLocation(), ColLocation(), (float*)res.BufferPointer(), int(m_numRows)));
            }
            else
            {
                CUSPARSECALL(hipsparseDcsc2dense(cusparseHandle, int(m_numRows), int(m_numCols), descr, (double*)NzValues(), RowLocation(), ColLocation(), (double*)res.BufferPointer(), int(m_numRows)));
            }
        }
        else
        {
            NOT_IMPLEMENTED;
        }

        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));

        res.SetMatrixName(m_matrixName);        
        return res;            
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& denseMatrix, const MatrixFormat matrixFormat/* = matrixFormatSparseCSR*/)
    {
        if (matrixFormat != matrixFormatSparseCSR && matrixFormat != matrixFormatSparseCSC)
        {
            NOT_IMPLEMENTED;
        }

        PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descr = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

        int numRows = (int)denseMatrix.GetNumRows(); //m
        int numCols = (int)denseMatrix.GetNumCols(); //n

        int *nnzPerRowOrCol = nullptr;
        CUDACALL(hipMalloc((void**)&nnzPerRowOrCol, sizeof(int)*((matrixFormat&matrixFormatRowMajor) ? numRows : numCols)));

        int nnzTotalDevHostPtr = -1;

        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));

        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseSnnz(cusparseHandle, (matrixFormat&matrixFormatRowMajor) ? HIPSPARSE_DIRECTION_ROW : HIPSPARSE_DIRECTION_COLUMN, (int)numRows, (int)numCols, descr,
                reinterpret_cast<float*>(denseMatrix.BufferPointer()), (int)numRows, nnzPerRowOrCol, &nnzTotalDevHostPtr));
        }
        else
        {
            CUSPARSECALL(hipsparseDnnz(cusparseHandle, (matrixFormat&matrixFormatRowMajor) ? HIPSPARSE_DIRECTION_ROW : HIPSPARSE_DIRECTION_COLUMN, (int)numRows, (int)numCols, descr,
                reinterpret_cast<double*>(denseMatrix.BufferPointer()), (int)numRows, nnzPerRowOrCol, &nnzTotalDevHostPtr));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));

        Resize(numRows, numCols, nnzTotalDevHostPtr, matrixFormat);

        CUDACALL(hipEventCreate(&done));
        if (m_format == MatrixType::matrixFormatSparseCSR)
        {
            if (sizeof(ElemType) == sizeof(float))
            {
                CUSPARSECALL(hipsparseSdense2csr(cusparseHandle, (int)m_numRows, (int)m_numCols, descr, reinterpret_cast<float*>(denseMatrix.BufferPointer()),
                    (int)m_numRows, nnzPerRowOrCol, reinterpret_cast<float*>(NzValues()), RowLocation(), ColLocation()));
            }
            else
            {
                CUSPARSECALL(hipsparseDdense2csr(cusparseHandle, (int)m_numRows, (int)m_numCols, descr, reinterpret_cast<double*>(denseMatrix.BufferPointer()),
                    (int)m_numRows, nnzPerRowOrCol, reinterpret_cast<double*>(NzValues()), RowLocation(), ColLocation()));
            }
        }
        else if (m_format == MatrixType::matrixFormatSparseCSC)
        {
            if (sizeof(ElemType) == sizeof(float))
            {
                CUSPARSECALL(hipsparseSdense2csr(cusparseHandle, (int)m_numRows, (int)m_numCols, descr, reinterpret_cast<float*>(denseMatrix.BufferPointer()),
                    (int)m_numRows, nnzPerRowOrCol, reinterpret_cast<float*>(NzValues()), RowLocation(), ColLocation()));
            }
            else
            {
                CUSPARSECALL(hipsparseDdense2csr(cusparseHandle, (int)m_numRows, (int)m_numCols, descr, reinterpret_cast<double*>(denseMatrix.BufferPointer()),
                    (int)m_numRows, nnzPerRowOrCol, reinterpret_cast<double*>(NzValues()), RowLocation(), ColLocation()));
            }
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        SetMatrixName(denseMatrix.GetMatrixName());
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        if (this != &deepCopy)
        {
            SetValue(deepCopy);
        }
        return *this;       
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(GPUSparseMatrix<ElemType>&& moveFrom)
    {
        m_computeDevice=moveFrom.m_computeDevice;
        m_numRows=moveFrom.m_numRows;  
        m_numCols=moveFrom.m_numCols;
        m_nz=moveFrom.m_nz; 
        m_elemSizeAllocated = moveFrom.m_elemSizeAllocated;
        m_totalBufferSizeAllocated = moveFrom.m_totalBufferSizeAllocated;
        m_pArray = moveFrom.m_pArray;
        m_format = moveFrom.m_format;
        m_externalBuffer = moveFrom.m_externalBuffer;
        m_matrixName=moveFrom.m_matrixName;

        m_rowIdx = moveFrom.m_rowIdx;

        m_blockSize = moveFrom.m_blockSize;
        m_blockVal = moveFrom.m_blockVal;
        m_blockIds = moveFrom.m_blockIds;

        m_expandedSize = moveFrom.m_expandedSize;
        m_block2Id = moveFrom.m_block2Id;
        m_block2UniqId = moveFrom.m_block2UniqId;

        moveFrom.ZeroInit(moveFrom.m_format, moveFrom.m_computeDevice);
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(GPUSparseMatrix<ElemType>&& moveFrom)
    {
        Clear();
        m_computeDevice=moveFrom.m_computeDevice;
        m_numRows=moveFrom.m_numRows;
        m_numCols=moveFrom.m_numCols;
        m_nz=moveFrom.m_nz;
        m_elemSizeAllocated = moveFrom.m_elemSizeAllocated;
        m_totalBufferSizeAllocated = moveFrom.m_totalBufferSizeAllocated;
        m_pArray = moveFrom.m_pArray;
        m_format = moveFrom.m_format;
        m_externalBuffer = moveFrom.m_externalBuffer;

        m_matrixName=moveFrom.m_matrixName;

        m_rowIdx = moveFrom.m_rowIdx;

        m_blockSize = moveFrom.m_blockSize;
        m_blockVal = moveFrom.m_blockVal;
        m_blockIds = moveFrom.m_blockIds;

        m_expandedSize = moveFrom.m_expandedSize;
        m_block2Id = moveFrom.m_block2Id;
        m_block2UniqId = moveFrom.m_block2UniqId;

        moveFrom.ZeroInit(moveFrom.m_format, moveFrom.m_computeDevice);

        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::~GPUSparseMatrix()
    {
        Clear();
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Clear()
    {
        if (m_matrixName!=nullptr) 
        {
            delete[] m_matrixName;
            m_matrixName = NULL;
        }

        if(m_pArray != nullptr) 
            CUDACALL(hipFree(m_pArray));

        if(m_blockVal != nullptr) 
            CUDACALL(hipFree(m_blockVal));
        if(m_blockIds != nullptr) 
            CUDACALL(hipFree(m_blockIds));

        ZeroInit(m_format, m_computeDevice);
    }

    //ResizeAsAndCopyIndexFrom - Resize this sparse matrix to have the same element structure as the passed matrix
    // a - sparse matrix whose structure we want to clone
    // remark: this was done for element wise operations where the structure will be identical after an operation
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ResizeAsAndCopyIndexFrom(const GPUSparseMatrix<ElemType>& a, const bool growOnly /*= true*/)
    {
        Resize(a.m_numRows, a.m_numCols, a.m_nz, a.m_format, growOnly);

        // copy over the non-zero locations from the source matrix
        CUDACALL(hipMemcpy(ColLocation(),a.ColLocation(),ColSize(),hipMemcpyDeviceToDevice));
        CUDACALL(hipMemcpy(RowLocation(),a.RowLocation(),RowSize(),hipMemcpyDeviceToDevice));

        CUDACALL(hipMemcpy(MajorIndexLocation(), a.MajorIndexLocation(), MajorIndexSize(), hipMemcpyDeviceToDevice));
        CUDACALL(hipMemcpy(SecondaryrIndexLocation(), a.SecondaryrIndexLocation(), SecondaryIndexSize(), hipMemcpyDeviceToDevice));
    }

    //-------------------------------------------------------------------------
    // Start of new GPU Sparse Matrix code 
    //-------------------------------------------------------------------------

    template<class ElemType>
    ElemType* GPUSparseMatrix<ElemType>::BufferPointer() const
    {
        if(m_format == matrixFormatSparseCSC || m_format == matrixFormatSparseCSR) 
        {
            return m_pArray;
        }  
        else
        {
            return m_blockVal;
        }
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, const size_t numNZ, const MatrixFormat matrixFormat, const bool growOnly /*= true*/)
    {               
        m_numRows = numRows;
        m_numCols = numCols; 
        m_nz = numNZ;

        if(m_format == MatrixFormat::matrixFormatSparseCSC || m_format == MatrixFormat::matrixFormatSparseCSR) 
        {
            bool reallocate = (m_totalBufferSizeAllocated < BufferSize() || (!growOnly && m_totalBufferSizeAllocated > BufferSize()));

            m_totalBufferSizeAllocated = BufferSize();
            m_elemSizeAllocated = ElemCountFromBufferSize();

            if (reallocate)
            {
                if (m_pArray != nullptr)
                    CUDACALL(hipFree(m_pArray));
                if (m_rowIdx != nullptr)
                    CUDACALL(hipFree(m_rowIdx));
                if (m_block2Id != nullptr)
                    CUDACALL(hipFree(m_block2Id));
                if (m_block2UniqId != nullptr)
                    CUDACALL(hipFree(m_block2UniqId));

                PrepareDevice();

                CUDACALL(hipMalloc((void **)&m_pArray, m_totalBufferSizeAllocated));
                CUDACALL(hipMalloc((void **)&m_rowIdx, sizeof(size_t)*m_elemSizeAllocated));
                CUDACALL(hipMalloc((void **)&m_block2Id, sizeof(size_t)*(numCols * 2)));
                CUDACALL(hipMalloc((void **)&m_block2UniqId, sizeof(size_t)*(numCols * 2)));
            }
        } 
        else if (m_format == MatrixFormat::matrixFormatSparseBlockCol || m_format == MatrixFormat::matrixFormatSparseBlockRow)
        {
            if (m_blockVal != nullptr)
                CUDACALL(hipFree(m_blockVal));
            if (m_blockIds != nullptr)
                CUDACALL(hipFree(m_blockIds));
            PrepareDevice();
            CUDACALL(hipMalloc((void **)&m_blockVal, sizeof(ElemType)*numNZ));
            int max = numCols > numRows ? numCols : numRows;
            CUDACALL(hipMalloc((void **)&m_blockIds, sizeof(size_t)*max));

        }
        else
            NOT_IMPLEMENTED;
    }

    //Reset matrix so it can be reused
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Reset()
    {                
        m_nz = 0;
        m_blockSize = 0;
    }

#pragma endregion Constructors and Destructor

#pragma region Static BLAS Functions
    
    // copy features to GPU matrix 
     template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetMatrixFromCSCFormat(size_t *h_row, size_t *h_rowIdx, size_t size, size_t blockSize)
    {
        if(m_format != matrixFormatSparseCSC) 
        {
            throw std::logic_error("CPUSparseMatrix: unsupported SetValue() call.");
        }

        if(m_elemSizeAllocated < size) 
        {
            throw std::logic_error("CPUSparseMatrix:  allocated size is too small.");
        }

        Reset();
        m_nz = size;
        m_blockSize = blockSize;
        PrepareDevice();
        CUDACALL(hipMemcpy(MajorIndexLocation(), h_row, sizeof(size_t)*size,hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(m_rowIdx, h_rowIdx, sizeof(size_t)*size,hipMemcpyHostToDevice));   
    }
       
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetMatrixFromLabelAndClass(size_t *h_row, size_t *h_block2Id, size_t *h_block2UniqId, size_t labelSize, size_t expandedSize, size_t blockSize)
    {
        if(m_format != matrixFormatSparseCSC) 
        {
            throw std::logic_error("CPUSparseMatrix: unsupported SetValue() call.");
        }

        if(m_elemSizeAllocated < labelSize) 
        {
            throw std::logic_error("CPUSparseMatrix:  allocated size is too small.");
        }
        
        Reset();
        m_nz = labelSize;
        m_expandedSize = expandedSize;
        m_blockSize = blockSize;
        PrepareDevice();
        
        CUDACALL(hipMemcpy(MajorIndexLocation(), h_row, sizeof(size_t)*labelSize,hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(m_block2Id, h_block2Id, sizeof(size_t)*labelSize,hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(m_block2UniqId, h_block2UniqId, sizeof(size_t)*labelSize,hipMemcpyHostToDevice));   
    }

    // forward pass from feature to hidden layer
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA, 
        const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, ElemType beta, GPUMatrix<ElemType>& c)

    {
        if (lhs.GetComputeDeviceId()!=rhs.GetComputeDeviceId()||(lhs.GetComputeDeviceId()!=c.GetComputeDeviceId()))
            throw std::runtime_error("MultiplyAndWeightedAddStD: All matrices must be on the same GPU");

        if (lhs.IsEmpty() || rhs.IsEmpty())
            throw std::logic_error("LeftMultiplyAndAdd:  one of the input matrix is empty.");

        int m = transposeA? (int)lhs.GetNumCols(): (int)lhs.GetNumRows();
        int k = transposeA? (int)lhs.GetNumRows(): (int)lhs.GetNumCols();
        int l = transposeB? (int)rhs.GetNumCols(): (int)rhs.GetNumRows();
        int n = transposeB? (int)rhs.GetNumRows(): (int)rhs.GetNumCols();

        assert (m>0 && k>0 && l>0 && n>0);  //converting from size_t to int may cause overflow
        assert (k == l);
        if (k != l) 
        {
            throw std::invalid_argument("CPUSparseMatrix::MultiplyAndAdd: The inner dimensions of a and b must match.");
        }

        if (c.GetNumRows() != m || c.GetNumCols() != n) 
        {
            c.Resize(m,n);
        }         

        if (beta == 0)
        {
            c.SetValue(0.0);
        }
        else 
        {
            c *= beta;
        }

        size_t blocksPerGrid = rhs.m_nz;
        size_t p = (threadsPerBlock < lhs.GetNumRows())? threadsPerBlock : lhs.GetNumRows();
        
        if (!transposeA && !transposeB)
        {
            hipEvent_t done = nullptr; 
            CUDACALL(hipEventCreate(&done));
            _denseMulSparseToDense<ElemType><<<blocksPerGrid, p>>>(
                alpha,
                reinterpret_cast<ElemType*>(lhs.BufferPointer()),
                m,
                k,
                rhs.MajorIndexLocation(),
                reinterpret_cast<ElemType*>(c.BufferPointer()));
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));
            CUDACALL(hipEventDestroy(done));
        }
        else if (!transposeA && transposeB)
        {           
            NOT_IMPLEMENTED;
        }
        else if (transposeA && !transposeB)
        {
            NOT_IMPLEMENTED;
        }
        else 
        {
            NOT_IMPLEMENTED;
        }
    }

    // backward pass from hidden layer to feature weight
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::MultiplyAndAdd(ElemType /*alpha*/, const GPUMatrix<ElemType>& lhs, const bool transposeA, 
        const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, GPUSparseMatrix<ElemType>& c)
    {
        if (lhs.GetComputeDeviceId()!=rhs.GetComputeDeviceId())
            throw std::runtime_error("GPUSparseMatrix::MultiplyAndAdd: All matrices must be on the same GPU");
        
        int m = transposeA? (int)lhs.GetNumCols(): (int)lhs.GetNumRows();
        int k = transposeA? (int)lhs.GetNumRows(): (int)lhs.GetNumCols();
        int l = transposeB? (int)rhs.GetNumCols(): (int)rhs.GetNumRows();
        int n = transposeB? (int)rhs.GetNumRows(): (int)rhs.GetNumCols();

        assert(m>0 && k>0 && l>0 && n>0); (void)m; (void)n;  //converting from size_t to int may cause overflow
        assert (k == l);
        if (k != l) 
        {
            throw std::invalid_argument("GPUSparseMatrix::MultiplyAndAdd: The inner dimensions of a and b must match.");
        }

        c.SetFormat(matrixFormatSparseBlockCol);  
        size_t nz = rhs.m_blockSize * c.GetNumRows();        
        //allocate enough memory
        if(c.m_elemSizeAllocated < nz) 
        {
            c.Resize(c.GetNumRows(), c.GetNumCols(), nz);
        }
        c.m_blockSize = rhs.m_blockSize;      
        c.m_nz = nz;
        CUDACALL(hipMemset(c.m_blockVal,0,sizeof(ElemType)*(c.m_nz)));
        CUDACALL(hipMemset(c.m_blockIds,0,sizeof(size_t)*(c.m_blockSize)));
                
        if (!transposeA && !transposeB)
        {
            NOT_IMPLEMENTED;
        }
        else if (!transposeA && transposeB)
        {   
            hipEvent_t done = nullptr;
            CUDACALL(hipEventCreate(&done));
            size_t blocksPerGrid =rhs.GetNZElements();  
            _denseMulSparseToSparse<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
                lhs.BufferPointer(),
                lhs.GetNumRows(),
                rhs.MajorIndexLocation(),
                rhs.m_rowIdx,          
                c.m_blockVal, 
                c.m_blockIds);
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));
            CUDACALL(hipEventDestroy(done));
        }
        else if (transposeA && !transposeB)
        {
            NOT_IMPLEMENTED;
        }
        else 
        {
            NOT_IMPLEMENTED;
        }
    }

    // used for gradients udpate
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(const ElemType alpha, const GPUSparseMatrix<ElemType>& lhs, GPUMatrix<ElemType>& rhs)
    {
        if (lhs.GetComputeDeviceId()!=rhs.GetComputeDeviceId())
            throw std::runtime_error("GPUSparseMatrix::ScaleAndAdd: All matrices must be on the same GPU");

        if (lhs.m_format == matrixFormatSparseBlockCol || lhs.m_format == matrixFormatSparseBlockRow) 
        {
            size_t len = (lhs.m_format == matrixFormatSparseBlockCol) ? lhs.GetNumRows(): lhs.GetNumCols();
            bool blockCol = (lhs.m_format == matrixFormatSparseBlockCol);

            hipEvent_t done = nullptr;
            CUDACALL(hipEventCreate(&done));
            size_t blocksPerGrid = lhs.m_blockSize;
            _scaleAndAdd<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
                alpha,
                blockCol,
                lhs.m_blockVal,
                lhs.m_blockIds,
                len,
                rhs.BufferPointer(),
                rhs.GetNumRows());
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));
            CUDACALL(hipEventDestroy(done));
        } 
        else 
        {
            throw std::runtime_error("GPUSparseMatrix:: ScaleAndAdd() Not implemented");
        }
    }

    // a: H x No: H is hidden layer size and No is mini-batch size
    // weight: V x H, V is vocab size
    // label: V x No
    // cls: 2 x Nc, Nc is number of classes, each col is start and end word ids of a class
    // idx2cls: V x 1, mapping from word to class id
    // etp: V x No, stores predicted values
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropy(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& weight,
        const GPUSparseMatrix<ElemType> & label, const GPUMatrix<ElemType>& cls, 
        const GPUMatrix<ElemType>& idx2cls, GPUSparseMatrix<ElemType>& etp, GPUMatrix<ElemType>& entropyScore)
    {
        int deviceId = a.GetComputeDeviceId();
        if (weight.GetComputeDeviceId()!=deviceId || label.GetComputeDeviceId()!=deviceId || cls.GetComputeDeviceId()!=deviceId 
            || idx2cls.GetComputeDeviceId()!=deviceId || etp.GetComputeDeviceId()!=deviceId )
            throw std::runtime_error("GPUSparseMatrix:: ClassEntropy() All matrices must be on the same GPU");  

        size_t nC = cls.GetNumCols();
        size_t nV = label.GetNumRows() - nC;

        if (nV != idx2cls.GetNumRows() || idx2cls.GetNumCols() != 1 || cls.GetNumCols() + idx2cls.GetNumRows() != label.GetNumRows())
            throw std::logic_error("ClassEntropy: check matrix dimension");        
        
        //allocate enough memory
        if(etp.m_elemSizeAllocated < label.m_expandedSize) 
        {
            etp.Resize(etp.GetNumRows(), etp.GetNumCols(), label.m_expandedSize);
        }
        etp.m_nz = label.m_expandedSize;
        CUDACALL(hipMemset(etp.m_pArray,0,sizeof(ElemType)*(etp.m_nz)));
        entropyScore.SetValue((ElemType)0);     

        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));
        size_t blocksPerGrid = label.m_expandedSize;

        //_computePrediction<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
        _computePrediction<ElemType><<<blocksPerGrid, 20>>>(
            idx2cls.GetNumRows(),
            a.BufferPointer(),
            a.GetNumRows(),
            weight.BufferPointer(),
            weight.GetNumRows(),
            label.m_nz,
            label.MajorIndexLocation(),
            label.m_block2Id,
            cls.BufferPointer(),
            idx2cls.BufferPointer(),            
            etp.m_pArray,
            etp.MajorIndexLocation(),
            etp.SecondaryIndexLocation());

        blocksPerGrid = label.m_nz;
        _normalizePrediction<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
            label.m_nz,
            label.m_expandedSize,
            label.MajorIndexLocation(),
            label.m_block2Id, 
            etp.MajorIndexLocation(),
            etp.m_pArray,
            entropyScore.BufferPointer());

        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
   }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropyError(GPUSparseMatrix<ElemType>& a)
    {
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));

        int N = a.m_nz;
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock); 

        _computePredictionError<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
            a.m_pArray,
            N);

        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropyGradientOfInput(const GPUSparseMatrix<ElemType>& error, const GPUMatrix<ElemType>& weight,  GPUMatrix<ElemType>& grd)
    {
        int deviceId = error.GetComputeDeviceId();
        if (weight.GetComputeDeviceId()!=deviceId || grd.GetComputeDeviceId()!=deviceId )
            throw std::runtime_error("GPUSparseMatrix::ClassEntropyGradientOfInput() All matrices must be on the same GPU");

        grd.SetValue((ElemType)0); 
        hipEvent_t done = nullptr; 
        CUDACALL(hipEventCreate(&done));

        size_t blocksPerGrid = grd.GetNumElements();
        //_computeGradientOfInput<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
        _computeGradientOfInput<ElemType><<<blocksPerGrid, 20>>>(
            error.m_pArray,
            error.MajorIndexLocation(),
            error.SecondaryIndexLocation(),
            weight.BufferPointer(),
            weight.GetNumRows(),
            grd.BufferPointer(), 
            grd.GetNumRows());
        CUDACALL(hipEventRecord(done));  
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }
    
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropyGradientOfWeight(const GPUSparseMatrix<ElemType>& error,  const GPUMatrix<ElemType>& input, const GPUSparseMatrix<ElemType> & label, const GPUMatrix<ElemType>& cls, 
        const GPUMatrix<ElemType>& idx2cls, GPUSparseMatrix<ElemType>& grd)
    {
        int deviceId = error.GetComputeDeviceId();
        if (input.GetComputeDeviceId()!=deviceId || label.GetComputeDeviceId()!=deviceId || cls.GetComputeDeviceId()!=deviceId  || idx2cls.GetComputeDeviceId()!=deviceId || grd.GetComputeDeviceId()!=deviceId )
            throw std::runtime_error("GPUSparseMatrix::ClassEntropyGradientOfWeight() All matrices must be on the same GPU");

        grd.SetFormat(matrixFormatSparseBlockRow);  
        size_t nz = label.m_blockSize * grd.GetNumCols();        
        //allocate enough memory
        if(grd.m_elemSizeAllocated < nz) 
        {
            grd.Resize(grd.GetNumRows(), grd.GetNumCols(), nz);
        }
        grd.m_blockSize = label.m_blockSize;      
        grd.m_nz = nz;
        CUDACALL(hipMemset(grd.m_blockVal,0,sizeof(ElemType)*(grd.m_nz)));
        CUDACALL(hipMemset(grd.m_blockIds,0,sizeof(size_t)*(grd.m_blockSize)));

        hipEvent_t done = nullptr;  
        CUDACALL(hipEventCreate(&done));

        size_t blocksPerGrid = error.m_nz;
        _computeGradientOfWeight<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
            error.m_pArray,
            error.MajorIndexLocation(),
            error.SecondaryIndexLocation(),
            input.GetNumCols(),
            idx2cls.GetNumRows(),
            label.MajorIndexLocation(),
            label.m_block2UniqId,
            cls.BufferPointer(),
            idx2cls.BufferPointer(),              
            input.BufferPointer(),
            input.GetNumRows(),
            grd.m_blockVal, 
            grd.m_blockIds);
        CUDACALL(hipEventRecord(done)); 
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncate (const ElemType threshold)
    {
        if(m_format == matrixFormatSparseBlockCol || m_format == matrixFormatSparseBlockRow) 
        {
            long N=(long)GetNZElements();
            int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
            hipEvent_t done = nullptr;
            CUDACALL(hipEventCreate(&done));        
            _inplaceTruncate<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_blockVal,threshold,N);
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));   
            CUDACALL(hipEventDestroy(done));
        } 
        else 
        {
            throw std::runtime_error("GPUSparseMatrix:: InplaceTruncate() only support block based sparse matrix");
        }
        return *this;
    } 

    // normal update for smoothed gradients c and current gradients (this)
    template<class ElemType> 
    void GPUSparseMatrix<ElemType>::NormalGrad(GPUMatrix<ElemType>& c, const ElemType momentum)
    {
        if (c.IsEmpty())
        {
            c.Resize(GetNumRows(), GetNumCols());
            c.SetValue(0.0);
        }

        if(m_format == matrixFormatSparseBlockCol || m_format == matrixFormatSparseBlockRow) 
        {
            size_t blocksPerGrid = m_blockSize;
            bool isBlockCol = (m_format == MatrixFormat::matrixFormatSparseBlockCol);
            size_t len = isBlockCol ? GetNumRows(): GetNumCols();
            hipEvent_t done = nullptr;
            CUDACALL(hipEventCreate(&done));        
            _normalGrad<ElemType><<<blocksPerGrid,threadsPerBlock>>>(
                isBlockCol,
                len,
                momentum,
                m_blockIds,
                m_blockVal,
                c.BufferPointer(),
                c.GetNumRows());                        
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));    
            CUDACALL(hipEventDestroy(done));
        } 
        else 
        {
            throw std::runtime_error("GPUSparseMatrix:: NormalGrad() only support block sparse format");
        }
    }

    //-------------------------------------------------------------------------
    // End of new GPU Sparse Matrix code 
    //-------------------------------------------------------------------------

    template<class ElemType>
    void  GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, const bool transposeA, 
        const GPUMatrix<ElemType>& b, ElemType beta, GPUMatrix<ElemType>& c)
    {
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId()||(b.GetComputeDeviceId()!=a.GetComputeDeviceId()))
            throw std::runtime_error("MultiplyAndWeightedAddStD: All matrices must be on the same GPU");
        a.PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descr = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseOperation_t oper = transposeA ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

        int m = (int)a.GetNumRows();
        int n = (int)b.GetNumCols();
        assert(n==(int)c.GetNumCols());
        int k = (int)a.GetNumCols();

        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsrmm(cusparseHandle,oper,m,n,k,(int)a.GetNZElements(),reinterpret_cast <float*>(&alpha),descr,reinterpret_cast <const float*>(a.NzValues()),
                a.RowLocation(), a.ColLocation(), reinterpret_cast <float*>(b.BufferPointer()),
                (int)b.GetNumRows(),reinterpret_cast <float*>(&beta),reinterpret_cast <float*>(c.BufferPointer()),(int)c.GetNumRows()));
        }
        else 
        {
            CUSPARSECALL(hipsparseDcsrmm(cusparseHandle,oper,m,n,k,(int)a.GetNZElements(),reinterpret_cast <double*>(&alpha),descr,reinterpret_cast <const double*>(a.NzValues()),
                a.RowLocation(), a.ColLocation(), reinterpret_cast <double*>(b.BufferPointer()),
                (int)b.GetNumRows(),reinterpret_cast <double*>(&beta),reinterpret_cast <double*>(c.BufferPointer()),(int)c.GetNumRows()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));        
    }
       

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S, const GPUMatrix<ElemType>& D, GPUMatrix<ElemType>& C)
    {
        if (C.GetNumRows()!=S.GetNumRows() || C.GetNumCols()!=D.GetNumRows())
        {
            GPUMatrix<ElemType> tmp(S.GetNumRows(),D.GetNumCols(),S.GetComputeDeviceId());
            C=tmp;
        }
        MultiplyAndWeightedAdd(1,S,false,D,0,C);
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Multiply(const GPUMatrix<ElemType>& D, const GPUSparseMatrix<ElemType>& S, GPUMatrix<ElemType>& C)
    {   
        GPUMatrix<ElemType> Res(S.GetNumCols(),D.GetNumRows());
        MultiplyAndWeightedAdd(1,S,true,D.Transpose(),0,Res);
        C.AssignTransposeOf(Res);       
    }

    // ElemCountFromBufferSize - Return the elemCountAllocated for a particular buffersize
    // totalBufferSize - total buffer we have to use
    // return: size of allocated elements/index slots available
    template<class ElemType>
    size_t GPUSparseMatrix<ElemType>::ElemCountFromBufferSize(const size_t totalBufferSize)
    {
        size_t elemSizeAllocated;
        if (m_format & matrixFormatCompressed)
        {
            elemSizeAllocated = (totalBufferSize-SecondaryIndexSize())/(sizeof(int)+sizeof(ElemType));
        }
        else // uncompressed COO format
        {
            elemSizeAllocated = totalBufferSize/(2*sizeof(int)+sizeof(ElemType));
        }
        return elemSizeAllocated;
    }

    // PrepareBuffer - Get the dimensions start buffer, computes the starting row/column of each value
    // m - rows in the source
    // n - cols in the source
    // canReuseBuffer - target matrix can be reused for temporary space
    // func - function to call to count elements in the result (returns count, and fills csrRowPtr array)
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::PrepareBuffer(size_t m, size_t n, bool canReuseBuffer, std::function<size_t (int* csrRowPtrC)> func)
    {
        int* csrRowPtrC=nullptr;
        GPUSparseMatrix<ElemType>& c = *this;
        size_t cSize = c.BufferSize();
        size_t rowBufferRequired = (m + 1)*sizeof(int);
        // determine the size of the buffer and align the final location of the row index buffer
        size_t nzBufSize = cSize-rowBufferRequired;
        nzBufSize -= nzBufSize%(sizeof(int)+sizeof(ElemType));
        bool allocatedBuffer = false;

        // do we have enough memory to store just the row buffer?
        if (cSize >= rowBufferRequired && c.NzValues() != nullptr && canReuseBuffer)
        {
            // determine the final location if we reuse the buffer
            csrRowPtrC = (int*)((byte*)c.NzValues() + nzBufSize);
        }
        else
        {
            CUDACALL(hipMalloc((void **)&csrRowPtrC,(m+1)*sizeof(int)));
            allocatedBuffer = true;
        }

        // get the non-zero count from the function (and 
        size_t nnzC = func(csrRowPtrC);

        // now we know the number of Non-zeros in the result set, set the output size
        c.m_elemSizeAllocated = c.m_nz = nnzC;
        c.m_numRows = m;
        c.m_numCols = n;
        size_t requiredSize = c.BufferSize();
        // see if the buffer we already have is big enough
        if (cSize >= requiredSize)
        {
            // compute the allocated size, to take up any additional space in the memory block 
            c.m_elemSizeAllocated = c.ElemCountFromBufferSize(cSize);
            // copy the rowPtr array to the proper location
            CUDACALL(hipMemcpy(c.SecondaryIndexLocation(),csrRowPtrC,c.SecondaryIndexSize(),hipMemcpyDeviceToDevice));
        }
        else
        {
            void* oldBuffer = c.m_pArray;
            // allocate required array space
            CUDACALL(hipMalloc((void **)&c.m_pArray,requiredSize));      
            // copy over 
            CUDACALL(hipMemcpy(c.SecondaryIndexLocation(),csrRowPtrC,c.SecondaryIndexSize(),hipMemcpyDeviceToDevice));
            // release the previous buffer since we just reallocated it
            if (oldBuffer != nullptr)
                CUDACALL(hipFree(oldBuffer));
        }
        // if we allocated the buffer, free it here
        if (allocatedBuffer)
            CUDACALL(hipFree(csrRowPtrC));
    }

    // Multiply - multiply one spares matrix by another sparse matrix
    // S1 - first sparse matrix
    // transposeS1 - transpose first matrix?
    // S2 - second sparse matrix
    // transposeS2 - tanspose second matrix?
    // c - result matrix
    // NOTE: if c has enough space allocated, it will be reused, otherwise it will be freed and a new memory block used
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S1, bool transposeS1, const GPUSparseMatrix<ElemType>& S2, bool transposeS2, GPUSparseMatrix<ElemType> &c)
    {
        if (S1.GetComputeDeviceId()!=S2.GetComputeDeviceId())
            throw std::runtime_error("Sparse matrix multiply: both matrices must be on the same device");

        S1.PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descrA)); CUSPARSECALL(hipsparseCreateMatDescr(&descrB)); CUSPARSECALL(hipsparseCreateMatDescr(&descrC));        
        hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrC,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrB,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseOperation_t operA = transposeS1 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t operB = transposeS2 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

        int m = int(transposeS1 ? S1.GetNumCols() : S1.GetNumRows());
        int n = int(transposeS2 ? S2.GetNumRows() : S2.GetNumCols());
        int k = int(transposeS1 ? S1.GetNumRows() : S1.GetNumCols());
        int l = int(transposeS2 ? S2.GetNumCols() : S2.GetNumRows());
        if (k!=l)
            throw std::runtime_error("Sparse matrix multiply: dimensionality mismatch");

        int nnzA = (int)S1.GetNZElements();
        int nnzB = (int)S2.GetNZElements();

        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));
        //Step 1 
        c.PrepareBuffer(m, n, true, // true means we can reuse the "c" buffer if it exists for temporaries
            [&](int* csrRowPtrC) -> size_t
        {
            int nnzTotal = -1; 
            CUSPARSECALL(hipsparseXcsrgemmNnz(cusparseHandle,operA,operB,m,n,k,descrA,nnzA,S1.RowLocation(),S1.ColLocation(),descrB,nnzB,
                S2.RowLocation(),S2.ColLocation(),descrC,csrRowPtrC,&nnzTotal));
            return nnzTotal;
        });


        //Step 2
        if (sizeof(float)==sizeof(ElemType))
        {
            CUSPARSECALL(hipsparseScsrgemm(cusparseHandle,operA,operB,m,n,k,descrA,nnzA,(const float*)S1.NzValues(),S1.RowLocation(),S1.ColLocation(),
                descrB,nnzB,(const float*)S2.NzValues(),S2.RowLocation(),S2.ColLocation(),
                descrC,(float*)c.NzValues(),c.RowLocation(),c.ColLocation()));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsrgemm(cusparseHandle,operA,operB,m,n,k,descrA,nnzA,(const double*)S1.NzValues(),S1.RowLocation(),S1.ColLocation(),
                descrB,nnzB,(const double*)S2.NzValues(),S2.RowLocation(),S2.ColLocation(),
                descrC,(double*)c.NzValues(),c.RowLocation(),c.ColLocation()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        hipsparseDestroy(cusparseHandle);   
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignProductOf(const GPUSparseMatrix<ElemType>& a, const bool transposeA, const GPUSparseMatrix<ElemType>& b, const bool transposeB)
    {
        Multiply(a,transposeA,b,transposeB,*this);
        return *this;
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha,const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUSparseMatrix<ElemType>& c)
    {
        if (a.GetNumCols()!=b.GetNumCols() || a.GetNumRows()!=b.GetNumRows())
            throw std::runtime_error("Dimensions mismatch in ScaleAndAdd");
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId())
            throw std::runtime_error("ScaleAndAdd: matrices must be on the same device");

        int m = (int)a.GetNumRows();
        int n = (int)a.GetNumCols();
        int nnzA = (int)a.GetNZElements();
        int nnzB = (int)b.GetNZElements();

        a.PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descrA)); CUSPARSECALL(hipsparseCreateMatDescr(&descrB)); CUSPARSECALL(hipsparseCreateMatDescr(&descrC));
        hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrB,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ZERO);

        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));
        //Step 1 
        bool inOutParameter = (&b == &c);
        c.PrepareBuffer(m, n, !inOutParameter, [&] (int* csrRowPtrC) -> size_t
        {
            int nnzTotal = -1;
            CUSPARSECALL(hipsparseXcsrgeamNnz(cusparseHandle,m,n,descrA,nnzA,a.RowLocation(),a.ColLocation(),descrB,nnzB,b.RowLocation(),b.ColLocation(),descrC,csrRowPtrC,&nnzTotal));
            return nnzTotal;
        });

        //Step 2
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsrgeam(cusparseHandle,m,n,reinterpret_cast <const float*>(&alpha),descrA,nnzA,reinterpret_cast <const float*>(a.NzValues()),a.RowLocation(),a.ColLocation(),
                reinterpret_cast <const float*>(&beta),descrB,nnzB,reinterpret_cast <const float*>(b.NzValues()),b.RowLocation(),b.ColLocation(),descrC,reinterpret_cast <float*>(c.NzValues()),c.RowLocation(),c.ColLocation()));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsrgeam(cusparseHandle,m,n,reinterpret_cast <const double*>(&alpha),descrA,nnzA,reinterpret_cast <const double*>(a.NzValues()),a.RowLocation(),a.ColLocation(),
                reinterpret_cast <const double*>(&beta),descrB,nnzB,reinterpret_cast <const double*>(b.NzValues()),b.RowLocation(),b.ColLocation(),descrC,reinterpret_cast <double*>(c.NzValues()),c.RowLocation(),c.ColLocation()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        hipsparseDestroy(cusparseHandle);   
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha,const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        if (a.GetNumRows()!=b.GetNumRows()||a.GetNumRows()!=c.GetNumRows()||a.GetNumCols()!=b.GetNumCols()||a.GetNumCols()!=c.GetNumCols())
            throw std::logic_error("ScaleAndAdd: dimension mismatch");
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId()||a.GetComputeDeviceId()!=c.GetComputeDeviceId())
            throw std::runtime_error("ScaleAndAdd: matrices must be on the same device");
        b.PrepareDevice();
        //copy b to c
        CUDACALL(hipMemcpy(c.BufferPointer(),b.BufferPointer(),sizeof(ElemType)*b.GetNumElements(),hipMemcpyDeviceToDevice));
        if (beta!=1)
        {
            c*=beta;
        }
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));
        long M=(long)a.GetNumRows();
        int blocksPerGrid =(int)ceil(1.0*M/threadsPerBlock);        
        _sparsePlusDense<ElemType><<<blocksPerGrid,threadsPerBlock>>>(alpha,a.NzValues(),a.RowLocation(),a.ColLocation(),c.BufferPointer(),M);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha,const GPUMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        ScaleAndAdd(beta,b,alpha,a,c);
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Scale(ElemType alpha, GPUSparseMatrix<ElemType>& a)
    {
        if (a.IsEmpty())
            return;

        long N=(long)a.GetNZElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));        
        _scaleArray<ElemType><<<blocksPerGrid,threadsPerBlock>>>(alpha,a.NzValues(),N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));        
        CUDACALL(hipEventDestroy(done));        
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ElementWisePower (ElemType alpha, const GPUSparseMatrix<ElemType>& a, GPUSparseMatrix<ElemType>& c)
    {
        if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }
        else 
        {
            if (a.IsEmpty())
                throw std::logic_error("ElementWisePower:  The input matrix a is empty.");
            if (a.GetNumRows()!=c.GetNumRows() || a.GetNumCols()!=c.GetNumCols() || a.GetNZElements()!=c.GetNZElements())
                c.ResizeAsAndCopyIndexFrom(a);

            hipEvent_t done = nullptr;
            CUDACALL(hipEventCreate(&done));
            a.PrepareDevice();
            long N=(long)a.GetNZElements();
            int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
            _elementWisePowerOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(alpha,a.NzValues(),c.NzValues(),N);             
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));   
        }
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId())
            throw std::runtime_error("a and b must be on the same device");

        //This implementation requires additional memory
        //need to put a in ColumnMajor format
        int m = (int)a.GetNumRows();
        int n = (int)a.GetNumCols();
        int nnz = (int)a.GetNZElements();
        hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
        ElemType* cscValA = nullptr;
        int* cscRowIndA = nullptr;
        int* cscColPtrA = nullptr;
        a.PrepareDevice();
        CUDACALL(hipMalloc((void **)&cscValA,nnz*sizeof(ElemType)));
        CUDACALL(hipMalloc((void **)&cscRowIndA,nnz*sizeof(int)));        
        CUDACALL(hipMalloc((void **)&cscColPtrA,(n+1)*sizeof(int)));
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const float*>(a.NzValues()),a.RowLocation(),a.ColLocation(),reinterpret_cast<float*>(cscValA),cscRowIndA,cscColPtrA,cpVals,idxBase));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const double*>(a.NzValues()),a.RowLocation(),a.ColLocation(),reinterpret_cast<double*>(cscValA),cscRowIndA,cscColPtrA,cpVals,idxBase));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        CUDACALL(hipEventDestroy(done));

        //Given sparse matrix in column major format, calculate indices for corresponding sparse vector
        int* vectArray=nullptr;
        CUDACALL(hipMalloc((void**)&vectArray,sizeof(int)*a.m_nz));
        long M=n;
        long N=m;
        //int* h_vectArray= new int[a.m_nz];
        int blocksPerGrid =(int)ceil(1.0*M/threadsPerBlock);   
        CUDACALL(hipEventCreate(&done));
        _getSparseVectorRepresntationForMatrix<ElemType><<<blocksPerGrid,threadsPerBlock>>>(cscColPtrA,cscRowIndA,vectArray,M,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        CUDACALL(hipFree(cscRowIndA));
        CUDACALL(hipFree(cscColPtrA));
        //CUDACALL(hipMemcpy(h_vectArray,vectArray,sizeof(int)*a.m_nz,hipMemcpyDeviceToHost));    

        //Actual dot product
        ElemType res=0;
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseSdoti(cusparseHandle,(int)a.m_nz,reinterpret_cast<float*>(cscValA),vectArray,
                reinterpret_cast<float*>(b.BufferPointer()),
                reinterpret_cast<float*>(&res),idxBase));
        }
        else
        {
            CUSPARSECALL(hipsparseDdoti(cusparseHandle,(int)a.m_nz,reinterpret_cast<double*>(cscValA),vectArray,
                reinterpret_cast<double*>(b.BufferPointer()),
                reinterpret_cast<double*>(&res),idxBase));
        }       
        CUDACALL(hipFree(vectArray));
        CUDACALL(hipFree(cscValA));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));   
        return res;        
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
    {
        return GPUSparseMatrix<ElemType>::InnerProductOfMatrices(b,a);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b, 
        const ElemType threshold)
    {
        if (a.GetNZElements()!=b.GetNZElements() || a.GetNumRows()  != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            return false;

        a.PrepareDevice();
        long *res = new long[3];
        res[0]=1;
        res[1]=1;
        res[2]=1;
        long *d_res = nullptr;
        CUDACALL(hipMalloc((void**)&d_res,sizeof(long)*3)); 
        CUDACALL(hipMemcpy(d_res,res,sizeof(long)*3,hipMemcpyHostToDevice));

        int blocksPerGrid =(int)ceil(1.0*a.GetNZElements()/threadsPerBlock); 
        _areEqual<ElemType><<<blocksPerGrid,threadsPerBlock>>>(a.NzValues(),b.NzValues(),(long)a.GetNZElements(),threshold,d_res);        
        _areEqual<int><<<blocksPerGrid,threadsPerBlock>>>(a.ColLocation(),b.ColLocation(),(long)a.GetNZElements(),(int)threshold,d_res+1);
        blocksPerGrid =(int)ceil((1.0*a.GetNumRows()+1.0)/threadsPerBlock); 
        _areEqual<int><<<blocksPerGrid,threadsPerBlock>>>(a.RowLocation(),b.RowLocation(),(long)a.GetNumRows()+1,(int)threshold,d_res+2);

        CUDACALL(hipMemcpy(res,d_res,sizeof(long)*3,hipMemcpyDeviceToHost));        
        if (res[0]*res[1]*res[2]==1)
            return true;
        else
            return false;
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::AreEqual(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b, 
        const ElemType threshold)
    {
        if (a.GetNumElements()!=b.GetNZElements() || a.GetNumRows()  != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            return false;
        GPUSparseMatrix<ElemType> c;
        c.SetValue(a);
        return AreEqual(c,b,threshold);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, 
        const ElemType threshold)
    {
        if (a.GetNZElements()!=b.GetNumElements() || a.GetNumRows()  != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            return false;
        GPUSparseMatrix<ElemType> c;
        c.SetValue(b);
        return AreEqual(a,c,threshold);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUSparseMatrix<ElemType>& a, const ElemType threshold) const
    {
        return AreEqual(*this,a,threshold);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUMatrix<ElemType>& a, const ElemType threshold) const
    {
        return AreEqual(*this,a,threshold);
    }
#pragma endregion Static BLAS Functions

#pragma region Member BLAS Functions

    template<class ElemType>
    int GPUSparseMatrix<ElemType>::GetComputeDeviceId() const 
    {
        // for externally managed memory the CUDA context will have the current device
        if (m_computeDevice == MANAGEDEXTERN)
        {
            int devId;
            assert(m_externalBuffer);
            CUDACALL(hipGetDevice(&devId));
            return devId;
        }
        return m_computeDevice;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf (const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.GetNumRows()!=b.GetNumRows()||a.GetNumCols()!=b.GetNumCols())
            throw std::logic_error("ElementProductOf: matrix dimensions mismatch");

        b.PrepareDevice();        
        GPUMatrix<ElemType> c(b.GetNumRows(),b.GetNumCols(),b.GetComputeDeviceId());

        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));
        long M=(long)a.GetNumRows();
        int blocksPerGrid =(int)ceil(1.0*M/threadsPerBlock);        
        _sparseMulDense<ElemType><<<blocksPerGrid,threadsPerBlock>>>(a.NzValues(),a.RowLocation(),a.ColLocation(),b.BufferPointer(),c.BufferPointer(),M);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf (const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
    {
        return GPUSparseMatrix<ElemType>::ElementProductOf(b,a);        
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator+ (const GPUSparseMatrix<ElemType>& a) const
    {
        GPUSparseMatrix<ElemType> res;
        GPUSparseMatrix<ElemType>::ScaleAndAdd(1,*this,1,a,res);
        return res;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator- (const GPUSparseMatrix<ElemType>& a) const
    {
        GPUSparseMatrix<ElemType> res;
        GPUSparseMatrix<ElemType>::ScaleAndAdd(1,*this,-1,a,res);
        return res;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator^=(ElemType alpha)
    {
        GPUSparseMatrix<ElemType>& us = *this;
        ElementWisePower(alpha, us, us);
        return us;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator^ (ElemType alpha) const
    {
        GPUSparseMatrix<ElemType> c;
        c.ResizeAsAndCopyIndexFrom(*this);
        ElementWisePower(alpha, *this, c);
        return c;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator*=(ElemType alpha)
    {
        GPUSparseMatrix<ElemType>& us = *this;
        if (alpha!=1)            
            Scale(alpha,us);
        return us;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator* (ElemType alpha) const
    {
        GPUSparseMatrix<ElemType> c(*this);
        if (alpha!=1)
            Scale(alpha, c);
        return c;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementPowerOf(const GPUSparseMatrix<ElemType>& a, const ElemType power)
    {
        ElementWisePower(power, a, *this);
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::Transpose() const
    {
        int m = (int)GetNumRows();
        int n = (int)GetNumCols();
        int nnz = (int)GetNZElements();
        hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;

        assert(GetFormat()&matrixFormatCompressed); // for now this only supports compressed formats
        PrepareDevice();
        GPUSparseMatrix c(n, m, nnz, nullptr, GetFormat(), GetComputeDeviceId(), m_elemSizeAllocated);
        CUDACALL(hipMalloc((void **)&c.m_pArray,c.BufferSize()));

        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));

        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const float*>(this->NzValues()),this->SecondaryIndexLocation(),this->MajorIndexLocation(),
                reinterpret_cast<float*>(c.NzValues()),c.MajorIndexLocation(),c.SecondaryIndexLocation(),cpVals,idxBase));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const double*>(this->NzValues()),this->SecondaryIndexLocation(),this->MajorIndexLocation(),
                reinterpret_cast<double*>(c.NzValues()),c.MajorIndexLocation(),c.SecondaryIndexLocation(),cpVals,idxBase));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        CUDACALL(hipEventDestroy(done));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));        
        return c;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTransposeOf(const GPUSparseMatrix<ElemType>& a)
    {
        if (this == &a)
            throw std::logic_error("AssignTransposeOf: a is the same as [this]. Does not support inplace transpose.");

        if (a.IsEmpty())
            throw std::logic_error("AssignTransposeOf: Matrix a is empty.");

        *this = a.Transpose();
        return *this;
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::InplaceTranspose()
    {
        if (IsEmpty())
            return;
        // transfer converted block over to this pointer
        *this = std::move(Transpose());
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::SumOfAbsElements() const
    {
        if (IsEmpty())
            throw std::logic_error("SumOfAbsElements: Matrix is empty");

        hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(GetComputeDeviceId());
        if (sizeof(ElemType)==sizeof(float))
        {
            float res=0;
            hipblasSasum(cuHandle,(int)GetNZElements(),reinterpret_cast<float*>(m_pArray),1,&res);
            return res;
        }
        else
        {
            double res=0;
            hipblasDasum(cuHandle,(int)GetNZElements(),reinterpret_cast<double*>(m_pArray),1,&res);
            return ElemType(res);
        }         
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::SumOfElements() const
    {
        if (IsEmpty())
            throw std::logic_error("SumOfElements: Matrix is empty");

        PrepareDevice();
        ElemType* d_sum = nullptr;
        ElemType h_sum;
        CUDACALL(hipMalloc((void**)&d_sum,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSum<ElemType><<<1,1024>>>(m_pArray,d_sum,(LONG64)GetNZElements());
        CUDACALL(hipMemcpy(&h_sum,d_sum,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDACALL(hipFree(d_sum));               
        return h_sum;        
    }


    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::FrobeniusNorm() const 
    {
        if (IsEmpty())
            throw std::logic_error("FrobeniusNorm: Matrix is empty.");

        ElemType* d_sum = nullptr;
        ElemType h_sum=0;
        CUDACALL(hipMalloc((void**)&d_sum,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSum2<ElemType><<<1,1024>>>(m_pArray,d_sum,(int)GetNZElements());
        CUDACALL(hipMemcpy(&h_sum,d_sum,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDACALL(hipFree(d_sum));               
        if (sizeof(ElemType)==sizeof(float))
            return (ElemType)sqrtf((float)h_sum);
        else
            return (ElemType)sqrt((double)h_sum);
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::MatrixNormInf() const
    {
        if (IsEmpty())
            throw std::logic_error("MatrixNorm1: Matrix is empty.");

        ElemType* d_maxAbs = nullptr;
        ElemType h_maxAbs=0;
        CUDACALL(hipMalloc((void**)&d_maxAbs,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionMatrixNormInf<ElemType><<<1,1024>>>(m_pArray,d_maxAbs,(int)GetNZElements());
        CUDACALL(hipMemcpy(&h_maxAbs,d_maxAbs,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDACALL(hipFree(d_maxAbs));               
        if (sizeof(ElemType)==sizeof(float))
            return h_maxAbs;
        else
            return h_maxAbs; 
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::MatrixNorm1() const
    {
        if (IsEmpty())
            throw std::logic_error("MatrixNorm1: Matrix is empty.");
        return SumOfAbsElements();              
    }

#pragma endregion Member BLAS Functions

#pragma region Other Functions

    template<class ElemType>    
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::ElementInverse ()
    {
        if (IsEmpty())
            throw std::logic_error("ElementInverse: Matrix is empty.");

        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));        
        _elemInverse<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementInverseOf (const GPUSparseMatrix<ElemType>& a)
    {
        SetValue(a);
        return ElementInverse();
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSigmoid()
    {
        performInplaceFunction(0);                    
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSigmoidOf (const GPUSparseMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceSigmoid();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLinearRectifierDerivative()
    {
        performInplaceFunction(6);                    
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLinearRectifierDerivativeOf (const GPUSparseMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceLinearRectifierDerivative();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTanh()
    {
        performInplaceFunction(1);
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTanhOf (const GPUSparseMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceTanh();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSqrt()
    {
        performInplaceFunction(2);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSqrtOf (const GPUSparseMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceSqrt();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceExp()
    {
        performInplaceFunction(3);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignExpOf (const GPUSparseMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceExp();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLog()
    {
        performInplaceFunction(4);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLogOf (const GPUSparseMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceLog();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceAbs()
    {
        performInplaceFunction(5);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignAbsOf (const GPUSparseMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceAbs();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateBottom (const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("InplaceTruncateBottom: Matrix is empty.");
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));        
        _inplaceTruncateBottom<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,threshold,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateBottomOf (const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignTruncateBottomOf: Matrix a is empty.");

        if (this!=&a)
        {
            //Resize(a.GetNumRows(), a.GetNumCols());           
            ResizeAsAndCopyIndexFrom(a);  
        }
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));        
        _assignTruncateBottom<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,a.NzValues(),threshold,N);                        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        return *this;
    }   

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateTop (const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("InplaceTruncateTop: Matrix is empty.");
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));        
        _inplaceTruncateTop<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,threshold,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        return *this;        
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateTopOf (const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignTruncateTopOf: Matrix a is empty.");

        if (this!=&a)
        {
            ResizeAsAndCopyIndexFrom(a);
        }

        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));        
        _assignTruncateTop<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,a.NzValues(),threshold,N);                        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        return *this;        
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::SetToZeroIfAbsLessThan (const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("SetToZeroIfAbsLessThan: Matrix is empty.");
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));        
        _setToZeroIfAbsLessThan<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,threshold,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        return *this;  
    }
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Unrolling (//GPUSparseMatrix<ElemType>& debugMatrix, 
        GPUMatrix<ElemType>& /*UnrolledMatrix*/, const GPUMatrix<ElemType>& /*InMatrix*/, GPUSparseMatrix<ElemType>& /*UnrollMapping*/, 
        const int /*inputWidth*/, const int /*inputHeight*/, const int /*inputChannelNum*/,
        const int /*FltWidth*/,const int /*FltHeight*/, const int /*FltChannel*/,
        const int /*FltStepW*/,  const int /*FltStepH*/)
    {
        ////if ((UnrolledMatrix.m_computeDevice!=InMatrix.m_computeDevice) ||(InMatrix.m_computeDevice!=UnrollMapping.m_computeDevice)) //different GPUs
        ////{
        ////    throw std::invalid_argument("All matrices must be on the same GPU");
        ////}
        ////else
        ////{ 
        //    //m_computeDevice = deviceId;

        //    const int inPatchSize = inputWidth * inputHeight;// * inputChannelNum;
        //    const int inRowHeight = InMatrix.GetNumRows();//m_inSampleNum;
        //    const int inColWidth = InMatrix.GetNumCols();
        //    const int inChannelNum = inputChannelNum;//column as sample VS column as channel//inColWidth;
        //    const int inSampleNum = inColWidth;// //inRowHeight / inPatchSize ;
        //    const int filterPatchSize = FltWidth * FltHeight;
        //    const int outWidth = inputWidth + 2 * (FltWidth - 1); // - FltWidth + 1; // Filter Width Step = 1; with padding
        //    const int outHeight = inputHeight + 2 * (FltHeight -1);//inputHeight - FltHeight + 1; 
        //    const int outWidthFltNum = ceil( double(outWidth - FltWidth + 1) / FltStepW);
        //    const int outHeightFltNum = ceil( double(outHeight - FltHeight + 1) /FltStepH);
        //    //const int convNum = outWidth * outHeight;
        //    //auto& UnrolledMatrix=*this;

        //    const int unrolledRowNum = outHeightFltNum * outWidthFltNum * inChannelNum;//Number of Filters Per Sample//outHeightFltNum * outWidthFltNum;
        //    const int unrolledColNum = filterPatchSize * inSampleNum;//filterPatchSize * inChannelNum;
        //    if (UnrolledMatrix.IsEmpty())
        //        UnrolledMatrix = GPUMatrix<ElemType>::Zeros(unrolledRowNum, unrolledColNum);//UnrolledMatrix.ZeroInit();
        //    //UnrollMapping.SetValue(-1);
        //    long N = inRowHeight * inColWidth; //total number of threads
        //    int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        //    //CUDA_CALL(hipSetDevice(InMatrix.m_computeDevice));
        //    ElemType* d_unrolledMatrix;
        //    ElemType* d_unrollMapping;
        //    const int outArraySize = unrolledRowNum * unrolledColNum;
        //    UnrollMapping.ZeroInit();

        //    //GPUSparseMatrix<ElemType>UnrollMapping;// = ZeroInit();//GPUSparseMatrix(InMatrix.GetNumElements(), UnrolledMatrix.GetNumElements());

        //    //const int _debugSize = unrolledRowNum * unrolledColNum;

        //    //int* d_debugArray; 
        //    //CUDA_CALL(hipMalloc((void**)&d_debugArray, _debugSize * sizeof(int)));
        //    //CUDA_CALL(hipMemcpy(d_debugArray, debugMatrix, _debugSize *sizeof(int),hipMemcpyHostToDevice)); 



        //    if (FltStepW == 1 && FltStepH == 1)
        //        _unrollElem_noStride<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
        //        UnrolledMatrix.BufferPointer(), InMatrix.BufferPointer(), UnrollMapping.m_pArray,
        //        inRowHeight, inColWidth, 
        //        inputWidth, inputHeight, inputChannelNum,
        //        FltWidth,FltHeight, FltChannel,
        //        inPatchSize, outWidthFltNum,outHeightFltNum,
        //        unrolledRowNum, unrolledColNum);
        //    else
        //    {
        //        _unrollElem_Stride<ElemType><<<blocksPerGrid, threadsPerBlock>>> (
        //            UnrolledMatrix.BufferPointer(), InMatrix.BufferPointer(), UnrollMapping.m_pArray,
        //            inRowHeight, inColWidth, 
        //            inputWidth, inputHeight, inputChannelNum,
        //            FltWidth,FltHeight, FltChannel,
        //            outWidthFltNum, outHeightFltNum,
        //            FltStepW,  FltStepH,
        //            unrolledRowNum, unrolledColNum);
        //    }           
        //    //CUDA_CALL(hipMemcpy(debugMatrix, d_debugArray, _debugSize *sizeof(int),hipMemcpyDeviceToHost)); 

        ////}
    }

#pragma endregion

#pragma region Helper Functions

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::performInplaceFunction(int kind)
    {        
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done = nullptr;
        CUDACALL(hipEventCreate(&done));        
        switch (kind)
        {
        case 0:
            _inplaceSigmoidOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);
            break;
        case 1:
            _inplaceTanhOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);
            break;
        case 2:
            _inplaceSqrtOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);
            break;
        case 3:
            _inplaceExpOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);
            break;
        case 4:
            _inplaceLogOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);
            break;
        case 5:
            _inplaceAbsOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);
            break;
        case 6:
            _inplaceLinRectDerivative<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);
        } 
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));        
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetMatrixFromCSRFormat(const GPUSPARSE_INDEX_TYPE *h_CSRRow, const GPUSPARSE_INDEX_TYPE *h_Col, const ElemType *h_Val,
        const size_t nz, const size_t numRows, const size_t numCols, const bool IsOnDevice /*= false*/, const DEVICEID_TYPE devId /*= 0*/)
    {
        m_computeDevice = devId;
        m_elemSizeAllocated = m_nz = nz;
        m_numCols=numCols;
        m_numRows=numRows;  
        m_format=matrixFormatSparseCSR;
        m_externalBuffer = false;

        if (OwnBuffer() && m_pArray != NULL)
        {
            CUDACALL(hipFree(m_pArray));            
        }

        PrepareDevice();
        CUDACALL(hipMalloc((void **)&m_pArray,BufferSize()));

        hipMemcpyKind kind = IsOnDevice?hipMemcpyDeviceToDevice:hipMemcpyHostToDevice;
        CUDACALL(hipMemcpy(RowLocation(),h_CSRRow,RowSize(),kind));
        CUDACALL(hipMemcpy(ColLocation(),h_Col,ColSize(),kind));
        CUDACALL(hipMemcpy(NzValues(),h_Val,NzSize(),kind));
    }

    // NOTE: we should change this to just use a single buffer, and return pointers into it
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::GetMatrixFromCSRFormat(int*& h_CSRRow, int*& h_Col, ElemType*& h_Val, size_t &nz, size_t &numRows, size_t &numCols) const
    {
        if (h_CSRRow!=nullptr || h_Col!=nullptr || h_Val!=nullptr)
            throw std::exception("Passed pointers must be nullptr");
        nz = GetNZElements();
        numRows = GetNumRows();
        numCols = GetNumCols();

        if (IsEmpty())
            return;
        else
        {
            PrepareDevice();
            h_Val = new ElemType[nz];
            h_CSRRow = new int[m_numRows + 1];
            h_Col = new int[nz];

            CUDACALL(hipMemcpy(h_CSRRow,RowLocation(),RowSize(),hipMemcpyDeviceToHost));
            CUDACALL(hipMemcpy(h_Col,   ColLocation(),ColSize(),hipMemcpyDeviceToHost));
            CUDACALL(hipMemcpy(h_Val,   NzValues(), NzSize(), hipMemcpyDeviceToHost));
        }
    }

#pragma endregion Helper Functions

    template class GPUSparseMatrix<float>; 
    template class GPUSparseMatrix<double>;    

    template <class ElemType>
    MATH_API File& operator>>(File& stream, GPUSparseMatrix<ElemType>& us)
    {
        stream.GetMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
        size_t elsize;
        stream>>elsize;
        if (sizeof(ElemType)!=elsize)
            throw std::runtime_error("Template argument size doesn't match those in file");
        std::wstring matrixName;

        // save off the buffer size being passed in
        ElemType* deviceBuffer = us.m_pArray;
        size_t deviceBufferSize = us.BufferSize();

        // now prepare this header to receive the data being read
        // Once CPUSpareMatrix uses same format, should use that class
        size_t nz, colnum, rownum;
        int format;

        // read in the header information
        stream>>matrixName>>format>>nz>>colnum>>rownum;
        us.m_format = (MatrixFormat)format;
        us.m_numCols = colnum;
        us.m_numRows = rownum;
        us.m_elemSizeAllocated = us.m_nz = nz;
        us.m_externalBuffer = false;

        // temporarily allocate a CPU side array here (could use CPUSparseMatrix when has same format)
        ElemType* hostBuffer = new ElemType[us.BufferSize()];
        us.m_pArray = hostBuffer;
        ElemType *dVal=us.NzValues();
        int* idx=us.MajorIndexLocation();
        int* cidx=us.SecondaryIndexLocation();
        size_t ncidx = us.SecondaryIndexCount();

        // read in the sparse matrix info
        for (int i=0;i<nz;++i)
        {
            stream>>dVal[i];
        }
        for (int i=0;i<nz;++i)
        {
            stream>>idx[i];
        }
        for (int i=0;i<ncidx;++i)
        {
            stream>>cidx[i];
        }  

        // decide if we have enough room in the current buffer
        if (deviceBufferSize >= us.BufferSize())
        {
            us.m_elemSizeAllocated = us.ElemCountFromBufferSize(deviceBufferSize);
        }
        else
        {
            us.PrepareDevice();
            if (deviceBufferSize > 0)
                CUDACALL(hipFree((void **)&deviceBuffer));
            CUDACALL(hipMalloc((void **)&us.m_pArray, us.BufferSize()));
        }

        // copy over the different sections data
        CUDACALL(hipMemcpy(us.NzValues(),dVal,us.NzSize(),hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(us.MajorIndexLocation(),idx,us.MajorIndexSize(),hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(us.SecondaryIndexLocation(),cidx,us.SecondaryIndexSize(),hipMemcpyHostToDevice));

        // copy over the name if necessary
        if (us.m_matrixName != nullptr)
            delete us.m_matrixName;
        us.m_matrixName = new wchar_t[matrixName.length()+1];
        wmemcpy(us.m_matrixName,matrixName.c_str(),matrixName.length()+1);

        return stream;
    }

    template MATH_API File& operator>>(File& stream, GPUSparseMatrix<float>& us);
    template MATH_API File& operator>>(File& stream, GPUSparseMatrix<double>& us);

    template <class ElemType>
    MATH_API File& operator<<(File& stream, const GPUSparseMatrix<ElemType>& us)
    {
        stream.PutMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
        stream<<sizeof(ElemType);
        if (us.GetMatrixName()==nullptr)
        {
            std::wstring s(L"nnmatrix");
            stream<<s;
        }
        else
        {
            stream<<us.GetMatrixName();
        }

        // What we would like to do here, is transfer to CPUSparse and save, do that when the format is the same
        char* hostBuffer = new char[us.BufferSize()];       // TODO: use std::shared_ptr
        GPUSparseMatrix<ElemType> hostSide(us.GetNumRows(), us.GetNumCols(), us.NzCount(), (ElemType*)hostBuffer, us.GetFormat());
        CUDACALL(hipMemcpy(hostBuffer, us.NzValues(),us.BufferSize(),hipMemcpyDeviceToHost));

        // now setup all the stuff pointing to the CPU side info
        const ElemType *dVal=hostSide.NzValues();
        int* idx=hostSide.MajorIndexLocation();
        int* cidx=hostSide.SecondaryIndexLocation();
        size_t nz=us.NzCount();
        size_t ncidx=us.SecondaryIndexCount();
        int format = us.GetFormat();
        stream<<format<<nz<<us.GetNumCols()<<us.GetNumRows();
        for (int i=0;i<nz;++i)
        {
            stream<<dVal[i];
        }
        for (int i=0;i<nz;++i)
        {
            stream<<idx[i];
        }
        for (int i=0;i<ncidx;++i)
        {
            stream<<cidx[i];
        }
        stream.PutMarker(fileMarkerEndSection, std::wstring(L"EMAT"));

        // now release the hostSide buffer
        delete hostBuffer;
        hostSide.m_pArray = nullptr;

        return stream;
    }
    template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<float>& us);
    template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<double>& us);

}}}

#endif  // CPUONLY
