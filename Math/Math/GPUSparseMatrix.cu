#include "hip/hip_runtime.h"
//
// <copyright file="GPUSparseMatrix.cu" company="Microsoft">
//     Copyright (c) Microsoft Corporation.  All rights reserved.
// </copyright>
//
#pragma once
#include "GPUSparseMatrix.cuh"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"
#include "GPUMatrixCUDAKernels.cu"
#include <functional>
#include "CommonMatrix.h"
#include <iostream> // for cout
#include <assert.h>

// thread local storage to access the current stream, initalize to default stream
extern __declspec( thread ) hipStream_t t_stream;

void CUDACALL(hipError_t x) 
{
    if(x!=hipSuccess) 
    { 
        const char* errmsg = hipGetErrorString(x);
        std::cout<<"!!!!!!!!CUDA EXCEPTION: "<<errmsg<<std::endl;

        throw std::exception(errmsg);
    }    
}

void CUSPARSECALL(hipsparseStatus_t x) 
{
    if(x!= HIPSPARSE_STATUS_SUCCESS) 
    {         
        std::cout<<"!!!!!!!!CUSPARSE EXCEPTION: "<<std::endl;
        throw std::exception("CUSPARSE EXCEPTION");
    }    
}

namespace Microsoft { namespace MSR { namespace CNTK {
    void PrepareDevice(short deviceId);

#pragma region Constructors and Destructor

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix()
    {
        ZeroInit();
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ZeroInit()
    {
        m_legacy = true;
        m_computeDevice=0; //current GPU device Id
        m_numRows=0;  
        m_numCols=0;
        m_elemSizeAllocated = m_nz = 0; //Number of non-zero elements
        m_format = matrixFormatSparseCSR;
        m_externalBuffer = false;
        m_pArray=NULL; 
        m_matrixName=NULL;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUMatrix<ElemType>& deepCopy)
    {
        ZeroInit();
        if (!deepCopy.IsEmpty()) 
            SetValue(deepCopy);
    }


    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        m_legacy = true;
        DeepCopy(deepCopy);
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const size_t numRows, const size_t numCols, const size_t nz, ElemType* pArray, 
        const size_t matrixFlags /*=matrixFormatSparseCSR*/, int deviceId /*=MANAGEDEXTERN*/, const size_t elemSizeAllocated /*=0*/)
    {
        m_legacy = true;
        m_computeDevice=deviceId;
        m_numRows=numRows;  
        m_numCols=numCols;
        m_nz=nz; 
        m_elemSizeAllocated=elemSizeAllocated?elemSizeAllocated:nz; 
        m_pArray = pArray;
        m_format = (MatrixFormat)(matrixFormatMask&matrixFlags);
        m_externalBuffer = true;
    }

    // legacy code
    /*template<class ElemType>
    void GPUSparseMatrix<ElemType>::Resize(const size_t nR, const size_t nC)
    {
        if (!this->IsEmpty())
        {
            Clear();
        }
        m_numRows=nR;  
        m_numCols=nC;
        m_nz=0; 
        m_elemSizeAllocated=m_nz; 
        m_pArray = nullptr;
    }*/

    // PrepareDevice - Setup the correct cuda context for an operation
    // deviceId - the device on which the operation will take place
    //            defaults to -1, which means use matrices current device
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::PrepareDevice(short deviceId /*=-1*/) const
    {
        // if default value use current compute device
        if (deviceId == -1)
            deviceId = m_computeDevice;
        Microsoft::MSR::CNTK::PrepareDevice(deviceId);
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::DeepCopy(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        m_computeDevice=deepCopy.m_computeDevice;
        m_numRows=deepCopy.m_numRows;  
        m_numCols=deepCopy.m_numCols;
        m_nz=deepCopy.m_nz; 
        m_elemSizeAllocated=deepCopy.m_elemSizeAllocated; 
        m_format = deepCopy.m_format;

        deepCopy.PrepareDevice();

        // about to overwrite this buffer, so free it if we own it
        if (OwnBuffer() && m_pArray!=NULL)
        {
            CUDACALL(hipFree(m_pArray));
        }
        else if (!deepCopy.OwnBuffer())
        {
            // just copy over the pointer, this assumses duplicate non-owned buffers are valid
            m_pArray = deepCopy.m_pArray;
        }
        else if (deepCopy.m_pArray!=NULL)
        {
            CUDACALL(hipMalloc((void **)&m_pArray,BufferSize()));
            CUDACALL(hipMemcpy(m_pArray,deepCopy.m_pArray,BufferSize(),hipMemcpyDeviceToDevice));
        }
        else
            m_pArray = NULL;
        m_externalBuffer = deepCopy.m_externalBuffer;

        if (deepCopy.m_matrixName!=NULL)
        {
            m_matrixName = new wchar_t[wcslen(deepCopy.m_matrixName)+1];
            wmemcpy(m_matrixName,deepCopy.m_matrixName,wcslen(deepCopy.m_matrixName)+1);
        }
        else
            m_matrixName=NULL;
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetValue(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        if (!this->IsEmpty())
        {
            Clear();
        }

        DeepCopy(deepCopy);
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::CopyToDenseMatrix()
    {
        GPUMatrix<ElemType> res;
        if (this->IsEmpty())
            return res;

        PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descr = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

        ElemType* pArrayDev = NULL;
        CUDACALL(hipMalloc((void**)&pArrayDev,sizeof(ElemType)*m_numCols*m_numRows));
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        CUSPARSECALL(hipsparseSetStream(cusparseHandle, t_stream));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsr2dense(cusparseHandle,int(m_numRows),int(m_numCols),descr,(float*)NzLocation(),RowLocation(),ColLocation(),(float*)pArrayDev,int(m_numRows)));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsr2dense(cusparseHandle,int(m_numRows),int(m_numCols),descr,(double*)NzLocation(),RowLocation(),ColLocation(),(double*)pArrayDev,int(m_numRows)));
        }        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));
        res.SetValue(m_numRows,m_numCols,pArrayDev,(matrixFlagNormal|matrixFlagSetValueOnDevice));
        if (pArrayDev!=NULL)
            CUDACALL(hipFree(pArrayDev));
        res.SetMatrixName(m_matrixName);        
        return res;            
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& denseMatrix)
    {
        if (!this->IsEmpty())
        {
            Clear();
        }

        PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descr = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

        m_numRows = denseMatrix.GetNumRows(); //m
        m_numCols = denseMatrix.GetNumCols(); //n
        m_format = matrixFormatSparseCSR;

        int *nnzPerRow = NULL;
        CUDACALL(hipMalloc((void**)&nnzPerRow,sizeof(int)*m_numCols));            

        int nnzTotalDevHostPtr = -1;

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseSnnz(cusparseHandle,(m_format&matrixFormatRowMajor)?HIPSPARSE_DIRECTION_ROW:HIPSPARSE_DIRECTION_COLUMN,(int)m_numRows,(int)m_numCols,descr,
                reinterpret_cast<float*>(denseMatrix.BufferPointer()), (int)m_numRows,nnzPerRow,&nnzTotalDevHostPtr));
        }
        else
        {
            CUSPARSECALL(hipsparseDnnz(cusparseHandle,(m_format&matrixFormatRowMajor)?HIPSPARSE_DIRECTION_ROW:HIPSPARSE_DIRECTION_COLUMN,(int)m_numRows,(int)m_numCols,descr,
                reinterpret_cast<double*>(denseMatrix.BufferPointer()), (int)m_numRows,nnzPerRow,&nnzTotalDevHostPtr));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));

        // about to overwrite this buffer, so free it if we own it
        if (OwnBuffer() && m_pArray!=NULL)
        {
            CUDACALL(hipFree(m_pArray));
        }

        //allocate memory for sparse matrix
        m_elemSizeAllocated = m_nz = nnzTotalDevHostPtr;
        CUDACALL(hipMalloc((void**)&m_pArray,BufferSize()));
        m_externalBuffer = false;

        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseSdense2csr(cusparseHandle,(int)m_numRows,(int)m_numCols,descr,reinterpret_cast<float*>(denseMatrix.BufferPointer()),
                (int)m_numRows,nnzPerRow,reinterpret_cast<float*>(NzLocation()),RowLocation(),ColLocation()));
        }
        else
        {
            CUSPARSECALL(hipsparseDdense2csr(cusparseHandle,(int)m_numRows,(int)m_numCols,descr,reinterpret_cast<double*>(denseMatrix.BufferPointer()),
                (int)m_numRows,nnzPerRow,reinterpret_cast<double*>(NzLocation()),RowLocation(),ColLocation()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        this->SetMatrixName(denseMatrix.GetMatrixName());
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        if (this != &deepCopy)
        {
            SetValue(deepCopy);
        }
        return *this;       
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(GPUSparseMatrix<ElemType>&& moveFrom)
    {
        m_computeDevice=moveFrom.m_computeDevice;
        m_numRows=moveFrom.m_numRows;  
        m_numCols=moveFrom.m_numCols;
        m_nz=moveFrom.m_nz; 
        m_elemSizeAllocated = moveFrom.m_elemSizeAllocated;
        m_pArray = moveFrom.m_pArray;
        m_format = moveFrom.m_format;
        m_externalBuffer = moveFrom.m_externalBuffer;
        m_matrixName=moveFrom.m_matrixName;

        moveFrom.ZeroInit();
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(GPUSparseMatrix<ElemType>&& moveFrom)
    {
        Clear();
        m_computeDevice=moveFrom.m_computeDevice;
        m_numRows=moveFrom.m_numRows;
        m_numCols=moveFrom.m_numCols;
        m_nz=moveFrom.m_nz;
        m_elemSizeAllocated = moveFrom.m_elemSizeAllocated;
        m_pArray = moveFrom.m_pArray;
        m_format = moveFrom.m_format;
        m_externalBuffer = moveFrom.m_externalBuffer;

        m_matrixName=moveFrom.m_matrixName;

        moveFrom.m_pArray = NULL;
        moveFrom.m_matrixName=NULL;
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::~GPUSparseMatrix()
    {
        if(m_legacy) 
        {
            Clear();
        }
        else 
        {
            ClearNew();
        }
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClearNew()
    {
        if (m_matrixName!=NULL) 
        {
            delete[] m_matrixName;
            m_matrixName = nullptr;
        }
        if(m_format == MatrixFormat::matrixFormatSparseCSC || m_format == MatrixFormat::matrixFormatSparseCSR) 
        {
            if(m_val != NULL) 
                CUDACALL(hipFree(m_val));
            if(m_row != NULL) 
                CUDACALL(hipFree(m_row));
            if(m_pb != NULL)
                CUDACALL(hipFree(m_pb));
        }  
        else if (m_format == MatrixFormat::matrixFormatSparseBlockCol || m_format == MatrixFormat::matrixFormatSparseBlockRow) 
        {
            if(m_blockVal != NULL) 
                CUDACALL(hipFree(m_blockVal));
            if(m_blockIds != NULL) 
                CUDACALL(hipFree(m_blockIds));
        }
    }


    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Clear()
    {
        if (m_pArray!=NULL)
            CUDACALL(hipFree(m_pArray));
        if (m_matrixName!=NULL)
            delete[] m_matrixName;
        ZeroInit();
    }

    //ResizeAs - Resize this sparse matrix to have the same element structure as the passed matrix
    // a - sparse matrix whose structure we want to clone
    // remark: this was done for element wise operations where the structure will be identical after an operation
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ResizeAs(const GPUSparseMatrix<ElemType>& a)
    {
        bool reallocate = (BufferSize() != a.BufferSize());

        m_numRows=a.m_numRows;
        m_numCols=a.m_numCols;
        m_nz=a.m_nz; 
        m_elemSizeAllocated = a.m_elemSizeAllocated;
        m_format = a.m_format;

        if (reallocate)
        {
            if (!OwnBuffer())
                throw runtime_error("cannot reallocate a buffer not owned by the matrix");
            if (m_pArray!=NULL)
                CUDACALL(hipFree(m_pArray));
            CUDACALL(hipMalloc((void **)&m_pArray,BufferSize()));                  
        }

        // copy over the non-zero locations from the source matrix
        CUDACALL(hipMemcpy(ColLocation(),a.ColLocation(),ColSize(),hipMemcpyDeviceToDevice));
        CUDACALL(hipMemcpy(RowLocation(),a.RowLocation(),RowSize(),hipMemcpyDeviceToDevice));
    }

    //-------------------------------------------------------------------------
    // Start of new GPU Sparse Matrix code 
    //-------------------------------------------------------------------------

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Init()
    {
        m_legacy = false;
        m_numRows = 0;
        m_numCols = 0;
        m_elemSizeAllocated = 0;
        m_externalBuffer = false;
        m_pArray = NULL;        
        PrepareDevice();
        m_nz = 0;
        m_matrixName = NULL;   

        if(m_format == MatrixFormat::matrixFormatSparseCSC || m_format == MatrixFormat::matrixFormatSparseCSR) 
        {
            m_colIdx = -1;
            m_val = NULL;
            m_row = NULL;
            m_pb = NULL;
            m_rowIdx = NULL;
            m_col = NULL;

            m_block2Id = NULL;
            m_block2UniqId = NULL;
        } 
        else if (m_format == MatrixFormat::matrixFormatSparseBlockCol || m_format == MatrixFormat::matrixFormatSparseBlockRow) 
        {
            m_blockSize = 0;      
            m_blockVal = NULL;
            m_blockIds = NULL;
        }
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const MatrixFormat format, const int deviceId)
    {
        if(format != MatrixFormat::matrixFormatSparseCSC && format != MatrixFormat::matrixFormatSparseCSR && format != MatrixFormat::matrixFormatSparseBlockCol && format != MatrixFormat::matrixFormatSparseBlockRow) 
        {
            throw std::logic_error("GPUSparseMatrix:  unsupported sparse matrix format");
        }
        m_format = format;
        m_computeDevice = deviceId;
        Init();
    }

    template<class ElemType>
    ElemType* GPUSparseMatrix<ElemType>::BufferPointer() const
    {
        if(m_format == MatrixFormat::matrixFormatSparseCSC || m_format == MatrixFormat::matrixFormatSparseCSR) 
        {
            return m_val;
        }  
        else
        {
            return m_blockVal;
        }
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, size_t size)
    {               
        m_nz = 0; 
        m_colIdx = -1;
        m_numRows = numRows;
        m_numCols = numCols; 
        if(m_elemSizeAllocated < size) 
        {    
            m_elemSizeAllocated = size;
            if(m_format == MatrixFormat::matrixFormatSparseCSC || m_format == MatrixFormat::matrixFormatSparseCSR) 
            {
                if(m_val != NULL) 
                    CUDACALL(hipFree(m_val));
                if(m_row != NULL) 
                    CUDACALL(hipFree(m_row));
                if(m_pb != NULL) 
                    CUDACALL(hipFree(m_pb));                
                if(m_rowIdx != NULL) 
                    CUDACALL(hipFree(m_rowIdx));
                if(m_col != NULL) 
                    CUDACALL(hipFree(m_col));
                if(m_block2Id != NULL) 
                    CUDACALL(hipFree(m_block2Id));
                if(m_block2UniqId != NULL) 
                    CUDACALL(hipFree(m_block2UniqId));

                PrepareDevice();
                CUDACALL(hipMalloc((void **)&m_val,sizeof(ElemType)*size));
                CUDACALL(hipMalloc((void **)&m_row,sizeof(size_t)*size));
                int len = m_format == MatrixFormat::matrixFormatSparseCSC ? numCols : numRows;
                CUDACALL(hipMalloc((void **)&m_pb,sizeof(size_t)*(len+1)));
                CUDACALL(hipMalloc((void **)&m_rowIdx,sizeof(size_t)*size));
                CUDACALL(hipMalloc((void **)&m_col,sizeof(size_t)*size));                
                CUDACALL(hipMalloc((void **)&m_block2Id,sizeof(size_t)*(numCols*2)));
                CUDACALL(hipMalloc((void **)&m_block2UniqId,sizeof(size_t)*(numCols*2)));
            } 
            else if(m_format == MatrixFormat::matrixFormatSparseBlockCol || m_format == MatrixFormat::matrixFormatSparseBlockRow) 
            {
                if(m_blockVal != NULL) 
                    CUDACALL(hipFree(m_blockVal));
                if(m_blockIds != NULL) 
                    CUDACALL(hipFree(m_blockIds));
                PrepareDevice();
                CUDACALL(hipMalloc((void **)&m_blockVal,sizeof(ElemType)*size));
                int max = numCols > numRows ? numCols : numRows;
                CUDACALL(hipMalloc((void **)&m_blockIds,sizeof(size_t)*max));
            }
        }
    }

    //Reset matrix so it can be reused
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Reset()
    {                
        m_nz = 0;
        m_colIdx = -1;
        m_blockSize = 0;
    }

#pragma endregion Constructors and Destructor

#pragma region Static BLAS Functions
    
    // copy features to GPU matrix 
     template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetMatrixFromCSCFormat(size_t *h_row, size_t *h_rowIdx, size_t size, size_t blockSize)
    {
        if(m_format != MatrixFormat::matrixFormatSparseCSC) 
        {
            throw std::logic_error("CPUSparseMatrix: unsupported SetValue() call.");
        }

        if(m_elemSizeAllocated < size) 
        {
            throw std::logic_error("CPUSparseMatrix:  allocated size is too small.");
        }

        Reset();
        m_nz = size;
        m_blockSize = blockSize;
        PrepareDevice();
        CUDACALL(hipMemcpy(m_row, h_row, sizeof(size_t)*size,hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(m_rowIdx, h_rowIdx, sizeof(size_t)*size,hipMemcpyHostToDevice));   
    }
       
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetMatrixFromLabelAndClass(size_t *h_row, size_t *h_block2Id, size_t *h_block2UniqId, size_t labelSize, size_t expandedSize, size_t blockSize)
    {
        if(m_format != MatrixFormat::matrixFormatSparseCSC) 
        {
            throw std::logic_error("CPUSparseMatrix: unsupported SetValue() call.");
        }

        if(m_elemSizeAllocated < labelSize) 
        {
            throw std::logic_error("CPUSparseMatrix:  allocated size is too small.");
        }
        
        Reset();
        m_nz = labelSize;
        m_expandedSize = expandedSize;
        m_blockSize = blockSize;
        PrepareDevice();
        
        CUDACALL(hipMemcpy(m_row, h_row, sizeof(size_t)*labelSize,hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(m_block2Id, h_block2Id, sizeof(size_t)*labelSize,hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(m_block2UniqId, h_block2UniqId, sizeof(size_t)*labelSize,hipMemcpyHostToDevice));   
    }

    // forward pass from feature to hidden layer
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA, 
        const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, ElemType beta, GPUMatrix<ElemType>& c)

    {
        if (lhs.GetComputeDeviceId()!=rhs.GetComputeDeviceId()||(lhs.GetComputeDeviceId()!=c.GetComputeDeviceId()))
            throw std::exception("MultiplyAndWeightedAddStD: All matrices must be on the same GPU");

        if (lhs.IsEmpty() || rhs.IsEmpty())
            throw std::logic_error("LeftMultiplyAndAdd:  one of the input matrix is empty.");

        int m = transposeA? (int)lhs.GetNumCols(): (int)lhs.GetNumRows();
        int k = transposeA? (int)lhs.GetNumRows(): (int)lhs.GetNumCols();
        int l = transposeB? (int)rhs.GetNumCols(): (int)rhs.GetNumRows();
        int n = transposeB? (int)rhs.GetNumRows(): (int)rhs.GetNumCols();

        assert (m>0 && k>0 && l>0 && n>0);  //converting from size_t to int may cause overflow
        assert (k == l);
        if (k != l) 
        {
            throw std::invalid_argument("CPUSparseMatrix::MultiplyAndAdd: The inner dimensions of a and b must match.");
        }

        if (c.GetNumRows() != m || c.GetNumCols() != n) 
        {
            c.Resize(m,n);
        }         

        if (beta == 0)
        {
            c.SetValue(0.0);
        }
        else 
        {
            c *= beta;
        }

        int blocksPerGrid = rhs.m_nz;
        int p = (threadsPerBlock < lhs.GetNumRows())? threadsPerBlock : lhs.GetNumRows();
        
        if (!transposeA && !transposeB)
        {
            hipEvent_t done; 
            CUDACALL(hipEventCreate(&done));
            _denseMulSparseToDense<ElemType><<<blocksPerGrid, p>>>(
                alpha,
                reinterpret_cast<ElemType*>(lhs.BufferPointer()),
                m,
                k,
                rhs.m_row,
                reinterpret_cast<ElemType*>(c.BufferPointer()));
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));
            CUDACALL(hipEventDestroy(done));
        }
        else if (!transposeA && transposeB)
        {           
            NOT_IMPLEMENTED;
        }
        else if (transposeA && !transposeB)
        {
            NOT_IMPLEMENTED;
        }
        else 
        {
            NOT_IMPLEMENTED;
        }
    }

    // backward pass from hidden layer to feature weight
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::MultiplyAndAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA, 
        const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, GPUSparseMatrix<ElemType>& c)
    {
        if (lhs.GetComputeDeviceId()!=rhs.GetComputeDeviceId())
            throw std::exception("GPUSparseMatrix::MultiplyAndAdd: All matrices must be on the same GPU");
        
        int m = transposeA? (int)lhs.GetNumCols(): (int)lhs.GetNumRows();
        int k = transposeA? (int)lhs.GetNumRows(): (int)lhs.GetNumCols();
        int l = transposeB? (int)rhs.GetNumCols(): (int)rhs.GetNumRows();
        int n = transposeB? (int)rhs.GetNumRows(): (int)rhs.GetNumCols();

        assert (m>0 && k>0 && l>0 && n>0);  //converting from size_t to int may cause overflow
        assert (k == l);
        if (k != l) 
        {
            throw std::invalid_argument("GPUSparseMatrix::MultiplyAndAdd: The inner dimensions of a and b must match.");
        }

        c.SetFormat(matrixFormatSparseBlockCol);  
        size_t nz = rhs.m_blockSize * c.GetNumRows();        
        //allocate enough memory
        if(c.m_elemSizeAllocated < nz) 
        {
            c.Resize(c.GetNumRows(), c.GetNumCols(), nz);
        }
        c.m_blockSize = rhs.m_blockSize;      
        c.m_nz = nz;
        CUDACALL(hipMemset(c.m_blockVal,0,sizeof(ElemType)*(c.m_nz)));
        CUDACALL(hipMemset(c.m_blockIds,0,sizeof(size_t)*(c.m_blockSize)));
                
        if (!transposeA && !transposeB)
        {
            NOT_IMPLEMENTED;
        }
        else if (!transposeA && transposeB)
        {   
            hipEvent_t done;       
            CUDACALL(hipEventCreate(&done));
            int blocksPerGrid =rhs.GetNZElements();  
            _denseMulSparseToSparse<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
                lhs.BufferPointer(),
                lhs.GetNumRows(),
                rhs.m_row,
                rhs.m_rowIdx,          
                c.m_blockVal, 
                c.m_blockIds);
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));
            CUDACALL(hipEventDestroy(done));
        }
        else if (transposeA && !transposeB)
        {
            NOT_IMPLEMENTED;
        }
        else 
        {
            NOT_IMPLEMENTED;
        }
    }

    // used for gradients udpate
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(const ElemType alpha, const GPUSparseMatrix<ElemType>& lhs, GPUMatrix<ElemType>& rhs)
    {
        if (lhs.GetComputeDeviceId()!=rhs.GetComputeDeviceId())
            throw std::exception("GPUSparseMatrix::ScaleAndAdd: All matrices must be on the same GPU");

        if (lhs.m_format == MatrixFormat::matrixFormatSparseBlockCol || lhs.m_format == MatrixFormat::matrixFormatSparseBlockRow) 
        {
            size_t len = (lhs.m_format == MatrixFormat::matrixFormatSparseBlockCol) ? lhs.GetNumRows(): lhs.GetNumCols();
            bool blockCol = (lhs.m_format == MatrixFormat::matrixFormatSparseBlockCol);

            hipEvent_t done;       
            CUDACALL(hipEventCreate(&done));
            int blocksPerGrid =lhs.m_blockSize;  
            _scaleAndAdd<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
                alpha,
                blockCol,
                lhs.m_blockVal,
                lhs.m_blockIds,
                len,
                rhs.BufferPointer(),
                rhs.GetNumRows());
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));
            CUDACALL(hipEventDestroy(done));
        } 
        else 
        {
            throw std::exception("GPUSparseMatrix:: ScaleAndAdd() Not implemented");
        }
    }

    // a: H x No: H is hidden layer size and No is mini-batch size
    // weight: V x H, V is vocab size
    // label: V x No
    // cls: 2 x Nc, Nc is number of classes, each col is start and end word ids of a class
    // idx2cls: V x 1, mapping from word to class id
    // etp: V x No, stores predicted values
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropy(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& weight,
        const GPUSparseMatrix<ElemType> & label, const GPUMatrix<ElemType>& cls, 
        const GPUMatrix<ElemType>& idx2cls, GPUSparseMatrix<ElemType>& etp, GPUMatrix<ElemType>& entropyScore)
    {
        int deviceId = a.GetComputeDeviceId();
        if (weight.GetComputeDeviceId()!=deviceId || label.GetComputeDeviceId()!=deviceId || cls.GetComputeDeviceId()!=deviceId 
            || idx2cls.GetComputeDeviceId()!=deviceId || etp.GetComputeDeviceId()!=deviceId )
            throw std::exception("GPUSparseMatrix:: ClassEntropy() All matrices must be on the same GPU");  

        size_t nC = cls.GetNumCols();
        size_t nV = label.GetNumRows() - nC;

        if (nV != idx2cls.GetNumRows() || idx2cls.GetNumCols() != 1 || cls.GetNumCols() + idx2cls.GetNumRows() != label.GetNumRows())
            throw std::logic_error("ClassEntropy: check matrix dimension");        
        
        //allocate enough memory
        if(etp.m_elemSizeAllocated < label.m_expandedSize) 
        {
            etp.Resize(etp.GetNumRows(), etp.GetNumCols(), label.m_expandedSize);
        }
        etp.m_nz = label.m_expandedSize;
        CUDACALL(hipMemset(etp.m_val,0,sizeof(ElemType)*(etp.m_nz)));
        entropyScore.SetValue((ElemType)0);     

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        int blocksPerGrid = label.m_expandedSize;

        //_computePrediction<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
        _computePrediction<ElemType><<<blocksPerGrid, 20>>>(
            idx2cls.GetNumRows(),
            a.BufferPointer(),
            a.GetNumRows(),
            weight.BufferPointer(),
            weight.GetNumRows(),
            label.m_nz,
            label.m_row,
            label.m_block2Id,
            cls.BufferPointer(),
            idx2cls.BufferPointer(),            
            etp.m_val,
            etp.m_row,
            etp.m_pb);

        blocksPerGrid = label.m_nz;
        _normalizePrediction<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
            label.m_nz,
            label.m_expandedSize,
            label.m_row,
            label.m_block2Id, 
            etp.m_row,
            etp.m_val,
            entropyScore.BufferPointer());

        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
   }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropyError(GPUSparseMatrix<ElemType>& a)
    {
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));

        int N = a.m_nz;
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock); 

        _computePredictionError<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
            a.m_val,
            N);

        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropyGradientOfInput(const GPUSparseMatrix<ElemType>& error, const GPUMatrix<ElemType>& weight,  GPUMatrix<ElemType>& grd)
    {
        int deviceId = error.GetComputeDeviceId();
        if (weight.GetComputeDeviceId()!=deviceId || grd.GetComputeDeviceId()!=deviceId )
            throw std::exception("GPUSparseMatrix::ClassEntropyGradientOfInput() All matrices must be on the same GPU");

        grd.SetValue((ElemType)0); 
        hipEvent_t done; 
        CUDACALL(hipEventCreate(&done));

        int blocksPerGrid =grd.GetNumElements();
        //_computeGradientOfInput<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
        _computeGradientOfInput<ElemType><<<blocksPerGrid, 20>>>(
            error.m_val,
            error.m_row,
            error.m_pb,
            weight.BufferPointer(),
            weight.GetNumRows(),
            grd.BufferPointer(), 
            grd.GetNumRows());
        CUDACALL(hipEventRecord(done));  
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }
    
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropyGradientOfWeight(const GPUSparseMatrix<ElemType>& error,  const GPUMatrix<ElemType>& input, const GPUSparseMatrix<ElemType> & label, const GPUMatrix<ElemType>& cls, 
        const GPUMatrix<ElemType>& idx2cls, GPUSparseMatrix<ElemType>& grd)
    {
        int deviceId = error.GetComputeDeviceId();
        if (input.GetComputeDeviceId()!=deviceId || label.GetComputeDeviceId()!=deviceId || cls.GetComputeDeviceId()!=deviceId  || idx2cls.GetComputeDeviceId()!=deviceId || grd.GetComputeDeviceId()!=deviceId )
            throw std::exception("GPUSparseMatrix::ClassEntropyGradientOfWeight() All matrices must be on the same GPU");

        grd.SetFormat(matrixFormatSparseBlockRow);  
        size_t nz = label.m_blockSize * grd.GetNumCols();        
        //allocate enough memory
        if(grd.m_elemSizeAllocated < nz) 
        {
            grd.Resize(grd.GetNumRows(), grd.GetNumCols(), nz);
        }
        grd.m_blockSize = label.m_blockSize;      
        grd.m_nz = nz;
        CUDACALL(hipMemset(grd.m_blockVal,0,sizeof(ElemType)*(grd.m_nz)));
        CUDACALL(hipMemset(grd.m_blockIds,0,sizeof(size_t)*(grd.m_blockSize)));

        hipEvent_t done;  
        CUDACALL(hipEventCreate(&done));

        int blocksPerGrid =error.m_nz; 
        _computeGradientOfWeight<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
            error.m_val,
            error.m_row,
            error.m_pb,
            input.GetNumCols(),
            idx2cls.GetNumRows(),
            label.m_row,
            label.m_block2UniqId,
            cls.BufferPointer(),
            idx2cls.BufferPointer(),              
            input.BufferPointer(),
            input.GetNumRows(),
            grd.m_blockVal, 
            grd.m_blockIds);
        CUDACALL(hipEventRecord(done)); 
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncate (const ElemType threshold)
    {
        if(m_format == MatrixFormat::matrixFormatSparseBlockCol || m_format == MatrixFormat::matrixFormatSparseBlockRow) 
        {
            long N=(long)GetNZElements();
            int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
            hipEvent_t done;       
            CUDACALL(hipEventCreate(&done));        
            _inplaceTruncate<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_blockVal,threshold,N);                        
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));   
            CUDACALL(hipEventDestroy(done));
        } 
        else 
        {
            throw std::exception("GPUSparseMatrix:: InplaceTruncate() only support block based sparse matrix");
        }
        return *this;
    } 

    // normal update for smoothed gradients c and current gradients (this)
    template<class ElemType> 
    void GPUSparseMatrix<ElemType>::NormalGrad(GPUMatrix<ElemType>& c, const ElemType momentum)
    {
        if (c.IsEmpty())
        {
            c.Resize(this->GetNumRows(), this->GetNumCols());
            c.SetValue(0.0);
        }

        if(m_format == MatrixFormat::matrixFormatSparseBlockCol || m_format == MatrixFormat::matrixFormatSparseBlockRow) 
        {
            int blocksPerGrid = m_blockSize;    
            bool isBlockCol = (m_format == MatrixFormat::matrixFormatSparseBlockCol);
            size_t len = isBlockCol ? GetNumRows(): GetNumCols();
            hipEvent_t done;       
            CUDACALL(hipEventCreate(&done));        
            _normalGrad<ElemType><<<blocksPerGrid,threadsPerBlock>>>(
                isBlockCol,
                len,
                momentum,
                m_blockIds,
                m_blockVal,
                c.BufferPointer(),
                c.GetNumRows());                        
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));    
            CUDACALL(hipEventDestroy(done));
        } 
        else 
        {
            throw std::exception("GPUSparseMatrix:: NormalGrad() only support block sparse format");
        }
    }

    //-------------------------------------------------------------------------
    // End of new GPU Sparse Matrix code 
    //-------------------------------------------------------------------------

    template<class ElemType>
    void  GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, const bool transposeA, 
        const GPUMatrix<ElemType>& b, ElemType beta, GPUMatrix<ElemType>& c)
    {
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId()||(b.GetComputeDeviceId()!=a.GetComputeDeviceId()))
            throw std::exception("MultiplyAndWeightedAddStD: All matrices must be on the same GPU");
        a.PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descr = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseOperation_t oper = transposeA ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

        int m = (int)a.GetNumRows();
        int n = (int)b.GetNumCols();
        assert(n==(int)c.GetNumCols());
        int k = (int)a.GetNumCols();

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsrmm(cusparseHandle,oper,m,n,k,(int)a.GetNZElements(),reinterpret_cast <float*>(&alpha),descr,reinterpret_cast <const float*>(a.NzLocation()),
                a.RowLocation(), a.ColLocation(), reinterpret_cast <float*>(b.BufferPointer()),
                (int)b.GetNumRows(),reinterpret_cast <float*>(&beta),reinterpret_cast <float*>(c.BufferPointer()),(int)c.GetNumRows()));
        }
        else 
        {
            CUSPARSECALL(hipsparseDcsrmm(cusparseHandle,oper,m,n,k,(int)a.GetNZElements(),reinterpret_cast <double*>(&alpha),descr,reinterpret_cast <const double*>(a.NzLocation()),
                a.RowLocation(), a.ColLocation(), reinterpret_cast <double*>(b.BufferPointer()),
                (int)b.GetNumRows(),reinterpret_cast <double*>(&beta),reinterpret_cast <double*>(c.BufferPointer()),(int)c.GetNumRows()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));        
    }
       

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S, const GPUMatrix<ElemType>& D, GPUMatrix<ElemType>& C)
    {
        if (C.GetNumRows()!=S.GetNumRows() || C.GetNumCols()!=D.GetNumRows())
        {
            GPUMatrix<ElemType> tmp(S.GetNumRows(),D.GetNumCols(),S.GetComputeDeviceId());
            C=tmp;
        }
        MultiplyAndWeightedAdd(1,S,false,D,0,C);
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Multiply(const GPUMatrix<ElemType>& D, const GPUSparseMatrix<ElemType>& S, GPUMatrix<ElemType>& C)
    {   
        GPUMatrix<ElemType> Res(S.GetNumCols(),D.GetNumRows());
        MultiplyAndWeightedAdd(1,S,true,D.Transpose(),0,Res);
        C.AssignTransposeOf(Res);       
    }

    // ElemCountFromBufferSize - Return the elemCountAllocated for a particular buffersize
    // totalBufferSize - total buffer we have to use
    // return: size of allocated elements/index slots available
    template<class ElemType>
    size_t GPUSparseMatrix<ElemType>::ElemCountFromBufferSize(size_t totalBufferSize)
    {
        size_t elemSizeAllocated;
        if (m_format & matrixFormatCompressed)
        {
            elemSizeAllocated = (totalBufferSize-CompressedIndexSize())/(sizeof(int)+sizeof(ElemType));
        }
        else // uncompressed COO format
        {
            elemSizeAllocated = totalBufferSize/(2*sizeof(int)+sizeof(ElemType));
        }
        return elemSizeAllocated;
    }

    // PrepareBuffer - Get the dimensions start buffer, computes the starting row/column of each value
    // m - rows in the source
    // n - cols in the source
    // canReuseBuffer - target matrix can be reused for temporary space
    // func - function to call to count elements in the result (returns count, and fills csrRowPtr array)
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::PrepareBuffer(size_t m, size_t n, bool canReuseBuffer, std::function<size_t (int* csrRowPtrC)> func)
    {
        int* csrRowPtrC=NULL;
        GPUSparseMatrix<ElemType>& c = *this;
        size_t cSize = c.BufferSize();
        size_t rowBufferRequired = (m + 1)*sizeof(int);
        // determine the size of the buffer and align the final location of the row index buffer
        size_t nzBufSize = cSize-rowBufferRequired;
        nzBufSize -= nzBufSize%(sizeof(int)+sizeof(ElemType));
        bool allocatedBuffer = false;

        // do we have enough memory to store just the row buffer?
        if (cSize >= rowBufferRequired && c.NzLocation() != NULL && canReuseBuffer)
        {
            // determine the final location if we reuse the buffer
            csrRowPtrC = (int*)((byte*)c.NzLocation() + nzBufSize);
        }
        else
        {
            CUDACALL(hipMalloc((void **)&csrRowPtrC,(m+1)*sizeof(int)));
            allocatedBuffer = true;
        }

        // get the non-zero count from the function (and 
        size_t nnzC = func(csrRowPtrC);

        // now we know the number of Non-zeros in the result set, set the output size
        c.m_elemSizeAllocated = c.m_nz = nnzC;
        c.m_numRows = m;
        c.m_numCols = n;
        size_t requiredSize = c.BufferSize();
        // see if the buffer we already have is big enough
        if (cSize >= requiredSize)
        {
            // compute the allocated size, to take up any additional space in the memory block 
            c.m_elemSizeAllocated = c.ElemCountFromBufferSize(cSize);
            // copy the rowPtr array to the proper location
            CUDACALL(hipMemcpy(c.CompressedIndexLocation(),csrRowPtrC,c.CompressedIndexSize(),hipMemcpyDeviceToDevice));
        }
        else
        {
            void* oldBuffer = c.m_pArray;
            // allocate required array space
            CUDACALL(hipMalloc((void **)&c.m_pArray,requiredSize));      
            // copy over 
            CUDACALL(hipMemcpy(c.CompressedIndexLocation(),csrRowPtrC,c.CompressedIndexSize(),hipMemcpyDeviceToDevice));
            // release the previous buffer since we just reallocated it
            if (oldBuffer != NULL)
                CUDACALL(hipFree(oldBuffer));
        }
        // if we allocated the buffer, free it here
        if (allocatedBuffer)
            CUDACALL(hipFree(csrRowPtrC));
    }

    // Multiply - multiply one spares matrix by another sparse matrix
    // S1 - first sparse matrix
    // transposeS1 - transpose first matrix?
    // S2 - second sparse matrix
    // transposeS2 - tanspose second matrix?
    // c - result matrix
    // NOTE: if c has enough space allocated, it will be reused, otherwise it will be freed and a new memory block used
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S1, bool transposeS1, const GPUSparseMatrix<ElemType>& S2, bool transposeS2, GPUSparseMatrix<ElemType> &c)
    {
        if (S1.GetComputeDeviceId()!=S2.GetComputeDeviceId())
            throw std::exception("Sparse matrix multiply: both matrices must be on the same device");

        S1.PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descrA)); CUSPARSECALL(hipsparseCreateMatDescr(&descrB)); CUSPARSECALL(hipsparseCreateMatDescr(&descrC));        
        hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrC,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrB,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseOperation_t operA = transposeS1 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t operB = transposeS2 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

        int m = int(transposeS1 ? S1.GetNumCols() : S1.GetNumRows());
        int n = int(transposeS2 ? S2.GetNumRows() : S2.GetNumCols());
        int k = int(transposeS1 ? S1.GetNumRows() : S1.GetNumCols());
        int l = int(transposeS2 ? S2.GetNumCols() : S2.GetNumRows());
        if (k!=l)
            throw std::exception("Sparse matrix multiply: dimensionality mismatch");

        int nnzA = (int)S1.GetNZElements();
        int nnzB = (int)S2.GetNZElements();

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        //Step 1 
        c.PrepareBuffer(m, n, true, // true means we can reuse the "c" buffer if it exists for temporaries
            [&](int* csrRowPtrC) -> size_t
        {
            int nnzTotal = -1; 
            CUSPARSECALL(hipsparseXcsrgemmNnz(cusparseHandle,operA,operB,m,n,k,descrA,nnzA,S1.RowLocation(),S1.ColLocation(),descrB,nnzB,
                S2.RowLocation(),S2.ColLocation(),descrC,csrRowPtrC,&nnzTotal));
            return nnzTotal;
        });


        //Step 2
        if (sizeof(float)==sizeof(ElemType))
        {
            CUSPARSECALL(hipsparseScsrgemm(cusparseHandle,operA,operB,m,n,k,descrA,nnzA,(const float*)S1.NzLocation(),S1.RowLocation(),S1.ColLocation(),
                descrB,nnzB,(const float*)S2.NzLocation(),S2.RowLocation(),S2.ColLocation(),
                descrC,(float*)c.NzLocation(),c.RowLocation(),c.ColLocation()));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsrgemm(cusparseHandle,operA,operB,m,n,k,descrA,nnzA,(const double*)S1.NzLocation(),S1.RowLocation(),S1.ColLocation(),
                descrB,nnzB,(const double*)S2.NzLocation(),S2.RowLocation(),S2.ColLocation(),
                descrC,(double*)c.NzLocation(),c.RowLocation(),c.ColLocation()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        hipsparseDestroy(cusparseHandle);   
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignProductOf(const GPUSparseMatrix<ElemType>& a, const bool transposeA, const GPUSparseMatrix<ElemType>& b, const bool transposeB)
    {
        Multiply(a,transposeA,b,transposeB,*this);
        return *this;
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha,const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUSparseMatrix<ElemType>& c)
    {
        if (a.GetNumCols()!=b.GetNumCols() || a.GetNumRows()!=b.GetNumRows())
            throw new std::exception("Dimensions mismatch in ScaleAndAdd");
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId())
            throw new std::exception("ScaleAndAdd: matrices must be on the same device");

        int m = (int)a.GetNumRows();
        int n = (int)a.GetNumCols();
        int nnzA = (int)a.GetNZElements();
        int nnzB = (int)b.GetNZElements();

        a.PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descrA)); CUSPARSECALL(hipsparseCreateMatDescr(&descrB)); CUSPARSECALL(hipsparseCreateMatDescr(&descrC));
        hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrB,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ZERO);

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        //Step 1 
        bool inOutParameter = (&b == &c);
        c.PrepareBuffer(m, n, !inOutParameter, [&] (int* csrRowPtrC) -> size_t
        {
            int nnzTotal = -1;
            CUSPARSECALL(hipsparseXcsrgeamNnz(cusparseHandle,m,n,descrA,nnzA,a.RowLocation(),a.ColLocation(),descrB,nnzB,b.RowLocation(),b.ColLocation(),descrC,csrRowPtrC,&nnzTotal));
            return nnzTotal;
        });

        //Step 2
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsrgeam(cusparseHandle,m,n,reinterpret_cast <const float*>(&alpha),descrA,nnzA,reinterpret_cast <const float*>(a.NzLocation()),a.RowLocation(),a.ColLocation(),
                reinterpret_cast <const float*>(&beta),descrB,nnzB,reinterpret_cast <const float*>(b.NzLocation()),b.RowLocation(),b.ColLocation(),descrC,reinterpret_cast <float*>(c.NzLocation()),c.RowLocation(),c.ColLocation()));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsrgeam(cusparseHandle,m,n,reinterpret_cast <const double*>(&alpha),descrA,nnzA,reinterpret_cast <const double*>(a.NzLocation()),a.RowLocation(),a.ColLocation(),
                reinterpret_cast <const double*>(&beta),descrB,nnzB,reinterpret_cast <const double*>(b.NzLocation()),b.RowLocation(),b.ColLocation(),descrC,reinterpret_cast <double*>(c.NzLocation()),c.RowLocation(),c.ColLocation()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        hipsparseDestroy(cusparseHandle);   
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha,const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        if (a.GetNumRows()!=b.GetNumRows()||a.GetNumRows()!=c.GetNumRows()||a.GetNumCols()!=b.GetNumCols()||a.GetNumCols()!=c.GetNumCols())
            throw std::logic_error("ScaleAndAdd: dimension mismatch");
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId()||a.GetComputeDeviceId()!=c.GetComputeDeviceId())
            throw std::exception("ScaleAndAdd: matrices must be on the same device");
        b.PrepareDevice();
        //copy b to c
        CUDACALL(hipMemcpy(c.BufferPointer(),b.BufferPointer(),sizeof(ElemType)*b.GetNumElements(),hipMemcpyDeviceToDevice));
        if (beta!=1)
        {
            c*=beta;
        }
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        long M=(long)a.GetNumRows();
        int blocksPerGrid =(int)ceil(1.0*M/threadsPerBlock);        
        _sparsePlusDense<ElemType><<<blocksPerGrid,threadsPerBlock>>>(alpha,a.NzLocation(),a.RowLocation(),a.ColLocation(),c.BufferPointer(),M);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha,const GPUMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        ScaleAndAdd(beta,b,alpha,a,c);
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Scale(ElemType alpha, GPUSparseMatrix<ElemType>& a)
    {
        if (a.IsEmpty())
            return;

        long N=(long)a.GetNZElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _scaleArray<ElemType><<<blocksPerGrid,threadsPerBlock>>>(alpha,a.NzLocation(),N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));        
        CUDACALL(hipEventDestroy(done));        
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ElementWisePower (ElemType alpha, const GPUSparseMatrix<ElemType>& a, GPUSparseMatrix<ElemType>& c)
    {
        if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }
        else 
        {
            if (a.IsEmpty())
                throw std::logic_error("ElementWisePower:  The input matrix a is empty.");
            if (a.GetNumRows()!=c.GetNumRows() || a.GetNumCols()!=c.GetNumCols() || a.GetNZElements()!=c.GetNZElements())
                c.ResizeAs(a);

            hipEvent_t done;
            CUDACALL(hipEventCreate(&done));
            a.PrepareDevice();
            long N=(long)a.GetNZElements();
            int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
            _elementWisePowerOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(alpha,a.NzLocation(),c.NzLocation(),N);             
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));   
        }
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId())
            throw std::exception("a and b must be on the same device");

        //This implementation requires additional memory
        //need to put a in ColumnMajor format
        int m = (int)a.GetNumRows();
        int n = (int)a.GetNumCols();
        int nnz = (int)a.GetNZElements();
        hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
        ElemType* cscValA = NULL;
        int* cscRowIndA = NULL;
        int* cscColPtrA = NULL;
        a.PrepareDevice();
        CUDACALL(hipMalloc((void **)&cscValA,nnz*sizeof(ElemType)));
        CUDACALL(hipMalloc((void **)&cscRowIndA,nnz*sizeof(int)));        
        CUDACALL(hipMalloc((void **)&cscColPtrA,(n+1)*sizeof(int)));
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const float*>(a.NzLocation()),a.RowLocation(),a.ColLocation(),reinterpret_cast<float*>(cscValA),cscRowIndA,cscColPtrA,cpVals,idxBase));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const double*>(a.NzLocation()),a.RowLocation(),a.ColLocation(),reinterpret_cast<double*>(cscValA),cscRowIndA,cscColPtrA,cpVals,idxBase));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        CUDACALL(hipEventDestroy(done));

        //Given sparse matrix in column major format, calculate indices for corresponding sparse vector
        int* vectArray=NULL;
        CUDACALL(hipMalloc((void**)&vectArray,sizeof(int)*a.m_nz));
        long M=n;
        long N=m;
        //int* h_vectArray= new int[a.m_nz];
        int blocksPerGrid =(int)ceil(1.0*M/threadsPerBlock);   
        CUDACALL(hipEventCreate(&done));
        _getSparseVectorRepresntationForMatrix<ElemType><<<blocksPerGrid,threadsPerBlock>>>(cscColPtrA,cscRowIndA,vectArray,M,N);        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        CUDACALL(hipFree(cscRowIndA));
        CUDACALL(hipFree(cscColPtrA));
        //CUDACALL(hipMemcpy(h_vectArray,vectArray,sizeof(int)*a.m_nz,hipMemcpyDeviceToHost));    

        //Actual dot product
        ElemType res=0;
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseSdoti(cusparseHandle,(int)a.m_nz,reinterpret_cast<float*>(cscValA),vectArray,
                reinterpret_cast<float*>(b.BufferPointer()),
                reinterpret_cast<float*>(&res),idxBase));
        }
        else
        {
            CUSPARSECALL(hipsparseDdoti(cusparseHandle,(int)a.m_nz,reinterpret_cast<double*>(cscValA),vectArray,
                reinterpret_cast<double*>(b.BufferPointer()),
                reinterpret_cast<double*>(&res),idxBase));
        }       
        CUDACALL(hipFree(vectArray));
        CUDACALL(hipFree(cscValA));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));   
        return res;        
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
    {
        return GPUSparseMatrix<ElemType>::InnerProductOfMatrices(b,a);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b, 
        const ElemType threshold)
    {
        if (a.GetNZElements()!=b.GetNZElements() || a.GetNumRows()  != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            return false;

        a.PrepareDevice();
        long *res = new long[3];
        res[0]=1;
        res[1]=1;
        res[2]=1;
        long *d_res = NULL;
        CUDACALL(hipMalloc((void**)&d_res,sizeof(long)*3)); 
        CUDACALL(hipMemcpy(d_res,res,sizeof(long)*3,hipMemcpyHostToDevice));

        int blocksPerGrid =(int)ceil(1.0*a.GetNZElements()/threadsPerBlock); 
        _areEqual<ElemType><<<blocksPerGrid,threadsPerBlock>>>(a.NzLocation(),b.NzLocation(),(long)a.GetNZElements(),threshold,d_res);        
        _areEqual<int><<<blocksPerGrid,threadsPerBlock>>>(a.ColLocation(),b.ColLocation(),(long)a.GetNZElements(),(int)threshold,d_res+1);
        blocksPerGrid =(int)ceil((1.0*a.GetNumRows()+1.0)/threadsPerBlock); 
        _areEqual<int><<<blocksPerGrid,threadsPerBlock>>>(a.RowLocation(),b.RowLocation(),(long)a.GetNumRows()+1,(int)threshold,d_res+2);

        CUDACALL(hipMemcpy(res,d_res,sizeof(long)*3,hipMemcpyDeviceToHost));        
        if (res[0]*res[1]*res[2]==1)
            return true;
        else
            return false;
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::AreEqual(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b, 
        const ElemType threshold)
    {
        if (a.GetNumElements()!=b.GetNZElements() || a.GetNumRows()  != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            return false;
        GPUSparseMatrix<ElemType> c;
        c.SetValue(a);
        return AreEqual(c,b,threshold);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, 
        const ElemType threshold)
    {
        if (a.GetNZElements()!=b.GetNumElements() || a.GetNumRows()  != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            return false;
        GPUSparseMatrix<ElemType> c;
        c.SetValue(b);
        return AreEqual(a,c,threshold);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUSparseMatrix<ElemType>& a, const ElemType threshold) const
    {
        return AreEqual(*this,a,threshold);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUMatrix<ElemType>& a, const ElemType threshold) const
    {
        return AreEqual(*this,a,threshold);
    }
#pragma endregion Static BLAS Functions

#pragma region Member BLAS Functions

    template<class ElemType>
    int GPUSparseMatrix<ElemType>::GetComputeDeviceId() const 
    {
        // for externally managed memory the CUDA context will have the current device
        if (m_computeDevice == MANAGEDEXTERN)
        {
            int devId;
            assert(m_externalBuffer);
            CUDACALL(hipGetDevice(&devId));
            return devId;
        }
        return m_computeDevice;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf (const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.GetNumRows()!=b.GetNumRows()||a.GetNumCols()!=b.GetNumCols())
            throw std::logic_error("ElementProductOf: matrix dimensions mismatch");

        b.PrepareDevice();        
        GPUMatrix<ElemType> c(b.GetNumRows(),b.GetNumCols(),b.GetComputeDeviceId());

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        long M=(long)a.GetNumRows();
        int blocksPerGrid =(int)ceil(1.0*M/threadsPerBlock);        
        _sparseMulDense<ElemType><<<blocksPerGrid,threadsPerBlock>>>(a.NzLocation(),a.RowLocation(),a.ColLocation(),b.BufferPointer(),c.BufferPointer(),M);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf (const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
    {
        return GPUSparseMatrix<ElemType>::ElementProductOf(b,a);        
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator+ (const GPUSparseMatrix<ElemType>& a) const
    {
        GPUSparseMatrix<ElemType> res;
        GPUSparseMatrix<ElemType>::ScaleAndAdd(1,*this,1,a,res);
        return res;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator- (const GPUSparseMatrix<ElemType>& a) const
    {
        GPUSparseMatrix<ElemType> res;
        GPUSparseMatrix<ElemType>::ScaleAndAdd(1,*this,-1,a,res);
        return res;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator^=(ElemType alpha)
    {
        auto& us = *this;
        ElementWisePower(alpha, us, us);
        return us;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator^ (ElemType alpha) const
    {
        GPUSparseMatrix<ElemType> c;
        c.ResizeAs(*this);
        ElementWisePower(alpha, *this, c);
        return c;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator*=(ElemType alpha)
    {
        auto& us = *this;
        if (alpha!=1)            
            Scale(alpha,us);
        return us;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator* (ElemType alpha) const
    {
        GPUSparseMatrix<ElemType> c(*this);
        if (alpha!=1)
            Scale(alpha, c);
        return c;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementPowerOf(const GPUSparseMatrix<ElemType>& a, const ElemType power)
    {
        ElementWisePower(power, a, *this);
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::Transpose() const
    {
        int m = (int)this->GetNumRows();
        int n = (int)this->GetNumCols();
        int nnz = (int)this->GetNZElements();
        hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;

        assert(GetFormat()&matrixFormatCompressed); // for now this only supports compressed formats
        PrepareDevice();
        GPUSparseMatrix c(n, m, nnz, NULL, GetFormat(), GetComputeDeviceId(), m_elemSizeAllocated);
        CUDACALL(hipMalloc((void **)&c.m_pArray,c.BufferSize()));

        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const float*>(this->NzLocation()),this->CompressedIndexLocation(),this->IndexLocation(),
                reinterpret_cast<float*>(c.NzLocation()),c.IndexLocation(),c.CompressedIndexLocation(),cpVals,idxBase));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const double*>(this->NzLocation()),this->CompressedIndexLocation(),this->IndexLocation(),
                reinterpret_cast<double*>(c.NzLocation()),c.IndexLocation(),c.CompressedIndexLocation(),cpVals,idxBase));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        CUDACALL(hipEventDestroy(done));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));        
        return c;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTransposeOf(const GPUSparseMatrix<ElemType>& a)
    {
        if (this == &a)
            throw std::logic_error("AssignTransposeOf: a is the same as [this]. Does not support inplace transpose.");

        if (a.IsEmpty())
            throw std::logic_error("AssignTransposeOf: Matrix a is empty.");

        *this = a.Transpose();
        return *this;
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::InplaceTranspose()
    {
        if (this->IsEmpty())
            return;
        // transfer converted block over to this pointer
        *this = std::move(this->Transpose());
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::SumOfAbsElements() const
    {
        if (this->IsEmpty())
            throw std::logic_error("SumOfAbsElements: Matrix is empty");

        hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(this->GetComputeDeviceId());
        if (sizeof(ElemType)==sizeof(float))
        {
            float res=0;
            hipblasSasum(cuHandle,(int)GetNZElements(),reinterpret_cast<float*>(m_pArray),1,&res);
            return res;
        }
        else
        {
            double res=0;
            hipblasDasum(cuHandle,(int)GetNZElements(),reinterpret_cast<double*>(m_pArray),1,&res);
            return ElemType(res);
        }         
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::SumOfElements() const
    {
        if (this->IsEmpty())
            throw std::logic_error("SumOfElements: Matrix is empty");

        PrepareDevice();
        ElemType* d_sum = NULL;
        ElemType h_sum;
        CUDACALL(hipMalloc((void**)&d_sum,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSum<ElemType><<<1,1024>>>(m_pArray,d_sum,(LONG64)this->GetNZElements());
        CUDACALL(hipMemcpy(&h_sum,d_sum,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDACALL(hipFree(d_sum));               
        return h_sum;        
    }


    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::FrobeniusNorm() const 
    {
        if (IsEmpty())
            throw std::logic_error("FrobeniusNorm: Matrix is empty.");

        ElemType* d_sum = NULL;
        ElemType h_sum=0;
        CUDACALL(hipMalloc((void**)&d_sum,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSum2<ElemType><<<1,1024>>>(m_pArray,d_sum,(int)this->GetNZElements());
        CUDACALL(hipMemcpy(&h_sum,d_sum,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDACALL(hipFree(d_sum));               
        if (sizeof(ElemType)==sizeof(float))
            return (ElemType)sqrtf(h_sum);
        else
            return (ElemType)sqrt(h_sum);
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::MatrixNormInf() const
    {
        if (IsEmpty())
            throw std::logic_error("MatrixNorm1: Matrix is empty.");

        ElemType* d_maxAbs = NULL;
        ElemType h_maxAbs=0;
        CUDACALL(hipMalloc((void**)&d_maxAbs,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionMatrixNormInf<ElemType><<<1,1024>>>(m_pArray,d_maxAbs,(int)this->GetNZElements());
        CUDACALL(hipMemcpy(&h_maxAbs,d_maxAbs,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDACALL(hipFree(d_maxAbs));               
        if (sizeof(ElemType)==sizeof(float))
            return h_maxAbs;
        else
            return h_maxAbs; 
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::MatrixNorm1() const
    {
        if (IsEmpty())
            throw std::logic_error("MatrixNorm1: Matrix is empty.");
        return this->SumOfAbsElements();              
    }

#pragma endregion Member BLAS Functions

#pragma region Other Functions

    template<class ElemType>    
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::ElementInverse ()
    {
        if (IsEmpty())
            throw std::logic_error("ElementInverse: Matrix is empty.");

        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _elemInverse<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);                        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementInverseOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        return this->ElementInverse();
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSigmoid()
    {
        performInplaceFunction(0);                    
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSigmoidOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceSigmoid();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLinearRectifierDerivative()
    {
        performInplaceFunction(6);                    
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLinearRectifierDerivativeOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceLinearRectifierDerivative();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTanh()
    {
        performInplaceFunction(1);
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTanhOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceTanh();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSqrt()
    {
        performInplaceFunction(2);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSqrtOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceSqrt();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceExp()
    {
        performInplaceFunction(3);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignExpOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceExp();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLog()
    {
        performInplaceFunction(4);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLogOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceLog();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceAbs()
    {
        performInplaceFunction(5);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignAbsOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceAbs();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateBottom (const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("InplaceTruncateBottom: Matrix is empty.");
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _inplaceTruncateBottom<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,threshold,N);                        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateBottomOf (const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignTruncateBottomOf: Matrix a is empty.");

        if (this!=&a)
        {
            //Resize(a.GetNumRows(), a.GetNumCols());           
            ResizeAs(a);  
        }
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _assignTruncateBottom<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,a.NzLocation(),threshold,N);                        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        return *this;
    }   

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateTop (const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("InplaceTruncateTop: Matrix is empty.");
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _inplaceTruncateTop<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,threshold,N);                        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        return *this;        
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateTopOf (const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignTruncateTopOf: Matrix a is empty.");

        if (this!=&a)
        {
            ResizeAs(a);
        }

        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _assignTruncateTop<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,a.NzLocation(),threshold,N);                        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        return *this;        
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::SetToZeroIfAbsLessThan (const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("SetToZeroIfAbsLessThan: Matrix is empty.");
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _setToZeroIfAbsLessThan<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,threshold,N);                        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        return *this;  
    }
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Unrolling (//GPUSparseMatrix<ElemType>& debugMatrix, 
        GPUMatrix<ElemType>& /*UnrolledMatrix*/, const GPUMatrix<ElemType>& /*InMatrix*/, GPUSparseMatrix<ElemType>& /*UnrollMapping*/, 
        const int /*inputWidth*/, const int /*inputHeight*/, const int /*inputChannelNum*/,
        const int /*FltWidth*/,const int /*FltHeight*/, const int /*FltChannel*/,
        const int /*FltStepW*/,  const int /*FltStepH*/)
    {
        ////if ((UnrolledMatrix.m_computeDevice!=InMatrix.m_computeDevice) ||(InMatrix.m_computeDevice!=UnrollMapping.m_computeDevice)) //different GPUs
        ////{
        ////    throw std::invalid_argument("All matrices must be on the same GPU");
        ////}
        ////else
        ////{ 
        //    //m_computeDevice = deviceId;

        //    const int inPatchSize = inputWidth * inputHeight;// * inputChannelNum;
        //    const int inRowHeight = InMatrix.GetNumRows();//m_inSampleNum;
        //    const int inColWidth = InMatrix.GetNumCols();
        //    const int inChannelNum = inputChannelNum;//column as sample VS column as channel//inColWidth;
        //    const int inSampleNum = inColWidth;// //inRowHeight / inPatchSize ;
        //    const int filterPatchSize = FltWidth * FltHeight;
        //    const int outWidth = inputWidth + 2 * (FltWidth - 1); // - FltWidth + 1; // Filter Width Step = 1; with padding
        //    const int outHeight = inputHeight + 2 * (FltHeight -1);//inputHeight - FltHeight + 1; 
        //    const int outWidthFltNum = ceil( double(outWidth - FltWidth + 1) / FltStepW);
        //    const int outHeightFltNum = ceil( double(outHeight - FltHeight + 1) /FltStepH);
        //    //const int convNum = outWidth * outHeight;
        //    //auto& UnrolledMatrix=*this;

        //    const int unrolledRowNum = outHeightFltNum * outWidthFltNum * inChannelNum;//Number of Filters Per Sample//outHeightFltNum * outWidthFltNum;
        //    const int unrolledColNum = filterPatchSize * inSampleNum;//filterPatchSize * inChannelNum;
        //    if (UnrolledMatrix.IsEmpty())
        //        UnrolledMatrix = GPUMatrix<ElemType>::Zeros(unrolledRowNum, unrolledColNum);//UnrolledMatrix.ZeroInit();
        //    //UnrollMapping.SetValue(-1);
        //    long N = inRowHeight * inColWidth; //total number of threads
        //    int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        //    //CUDA_CALL(hipSetDevice(InMatrix.m_computeDevice));
        //    ElemType* d_unrolledMatrix;
        //    ElemType* d_unrollMapping;
        //    const int outArraySize = unrolledRowNum * unrolledColNum;
        //    UnrollMapping.ZeroInit();

        //    //GPUSparseMatrix<ElemType>UnrollMapping;// = ZeroInit();//GPUSparseMatrix(InMatrix.GetNumElements(), UnrolledMatrix.GetNumElements());

        //    //const int _debugSize = unrolledRowNum * unrolledColNum;

        //    //int* d_debugArray; 
        //    //CUDA_CALL(hipMalloc((void**)&d_debugArray, _debugSize * sizeof(int)));
        //    //CUDA_CALL(hipMemcpy(d_debugArray, debugMatrix, _debugSize *sizeof(int),hipMemcpyHostToDevice)); 



        //    if (FltStepW == 1 && FltStepH == 1)
        //        _unrollElem_noStride<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
        //        UnrolledMatrix.BufferPointer(), InMatrix.BufferPointer(), UnrollMapping.m_pArray,
        //        inRowHeight, inColWidth, 
        //        inputWidth, inputHeight, inputChannelNum,
        //        FltWidth,FltHeight, FltChannel,
        //        inPatchSize, outWidthFltNum,outHeightFltNum,
        //        unrolledRowNum, unrolledColNum);
        //    else
        //    {
        //        _unrollElem_Stride<ElemType><<<blocksPerGrid, threadsPerBlock>>> (
        //            UnrolledMatrix.BufferPointer(), InMatrix.BufferPointer(), UnrollMapping.m_pArray,
        //            inRowHeight, inColWidth, 
        //            inputWidth, inputHeight, inputChannelNum,
        //            FltWidth,FltHeight, FltChannel,
        //            outWidthFltNum, outHeightFltNum,
        //            FltStepW,  FltStepH,
        //            unrolledRowNum, unrolledColNum);
        //    }           
        //    //CUDA_CALL(hipMemcpy(debugMatrix, d_debugArray, _debugSize *sizeof(int),hipMemcpyDeviceToHost)); 

        ////}
    }

#pragma endregion

#pragma region Helper Functions

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::performInplaceFunction(int kind)
    {        
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        switch (kind)
        {
        case 0:
            _inplaceSigmoidOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);            
            break;
        case 1:
            _inplaceTanhOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);   
            break;
        case 2:
            _inplaceSqrtOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);   
            break;
        case 3:
            _inplaceExpOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);   
            break;
        case 4:
            _inplaceLogOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);   
            break;
        case 5:
            _inplaceAbsOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);   
            break;
        case 6:
            _inplaceLinRectDerivative<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,N);
        } 
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));        
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetMatrixFromCSRFormat(int *h_CSRRow, int *h_Col, ElemType *h_Val, size_t nz, size_t numRows, size_t numCols, bool IsOnDevice, int devId)
    {
        m_computeDevice = devId;
        m_elemSizeAllocated = m_nz = nz;
        m_numCols=numCols;
        m_numRows=numRows;  
        m_format=matrixFormatSparseCSR;
        m_externalBuffer = false;

        if (OwnBuffer() && m_pArray != nullptr)
        {
            CUDACALL(hipFree(m_pArray));            
        }

        PrepareDevice();
        CUDACALL(hipMalloc((void **)&m_pArray,BufferSize()));

        hipMemcpyKind kind = IsOnDevice?hipMemcpyDeviceToDevice:hipMemcpyHostToDevice;
        CUDACALL(hipMemcpy(RowLocation(),h_CSRRow,RowSize(),kind));
        CUDACALL(hipMemcpy(ColLocation(),h_Col,ColSize(),kind));
        CUDACALL(hipMemcpy(NzLocation(),h_Val,NzSize(),kind));
    }

    // NOTE: we should change this to just use a single buffer, and return pointers into it
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::GetMatrixFromCSRFormat(int*& h_CSRRow, int*& h_Col, ElemType*& h_Val, size_t &nz, size_t &numRows, size_t &numCols) const
    {
        if (h_CSRRow!=NULL || h_Col!=NULL || h_Val!=NULL)
            throw std::exception("Passed pointers must be NULL");
        nz = this->GetNZElements();
        numRows = this->GetNumRows();
        numCols = this->GetNumCols();

        if (this->IsEmpty())
            return;
        else
        {
            PrepareDevice();
            h_Val = new ElemType[nz];
            h_CSRRow = new int[m_numRows + 1];
            h_Col = new int[nz];

            CUDACALL(hipMemcpy(h_CSRRow,RowLocation(),RowSize(),hipMemcpyDeviceToHost));
            CUDACALL(hipMemcpy(h_Col,   ColLocation(),ColSize(),hipMemcpyDeviceToHost));
            CUDACALL(hipMemcpy(h_Val,   NzLocation(), NzSize(), hipMemcpyDeviceToHost));
        }
    }

#pragma endregion Helper Functions

    template class GPUSparseMatrix<float>; 
    template class GPUSparseMatrix<double>;    

    template <class ElemType>
    MATH_API File& operator>>(File& stream, GPUSparseMatrix<ElemType>& us)
    {
        stream.GetMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
        size_t elsize;
        stream>>elsize;
        if (sizeof(ElemType)!=elsize)
            throw std::exception("Template argument size doesn't match those in file");
        std::wstring matrixName;

        // save off the buffer size being passed in
        ElemType* deviceBuffer = us.m_pArray;
        size_t deviceBufferSize = us.BufferSize();

        // now prepare this header to receive the data being read
        // Once CPUSpareMatrix uses same format, should use that class
        size_t nz, colnum, rownum;
        int format;

        // read in the header information
        stream>>matrixName>>format>>nz>>colnum>>rownum;
        us.m_format = (MatrixFormat)format;
        us.m_numCols = colnum;
        us.m_numRows = rownum;
        us.m_elemSizeAllocated = us.m_nz = nz;
        us.m_externalBuffer = false;

        // temporarily allocate a CPU side array here (could use CPUSparseMatrix when has same format)
        ElemType* hostBuffer = new ElemType[us.BufferSize()];
        us.m_pArray = hostBuffer;
        ElemType *dVal=us.NzLocation();
        int* idx=us.IndexLocation();
        int* cidx=us.CompressedIndexLocation();
        size_t ncidx = us.CompressedIndexCount();

        // read in the sparse matrix info
        for (int i=0;i<nz;++i)
        {
            stream>>dVal[i];
        }
        for (int i=0;i<nz;++i)
        {
            stream>>idx[i];
        }
        for (int i=0;i<ncidx;++i)
        {
            stream>>cidx[i];
        }  

        // decide if we have enough room in the current buffer
        if (deviceBufferSize >= us.BufferSize())
        {
            us.m_elemSizeAllocated = us.ElemCountFromBufferSize(deviceBufferSize);
        }
        else
        {
            us.PrepareDevice();
            if (deviceBufferSize > 0)
                CUDACALL(hipFree((void **)&deviceBuffer));
            CUDACALL(hipMalloc((void **)&us.m_pArray, us.BufferSize()));
        }

        // copy over the different sections data
        CUDACALL(hipMemcpy(us.NzLocation(),dVal,us.NzSize(),hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(us.IndexLocation(),idx,us.IndexSize(),hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(us.CompressedIndexLocation(),cidx,us.CompressedIndexSize(),hipMemcpyHostToDevice));

        // copy over the name if necessary
        if (us.m_matrixName != NULL)
            delete us.m_matrixName;
        us.m_matrixName = new wchar_t[matrixName.length()+1];
        wmemcpy(us.m_matrixName,matrixName.c_str(),matrixName.length()+1);

        return stream;
    }

    template MATH_API File& operator>>(File& stream, GPUSparseMatrix<float>& us);
    template MATH_API File& operator>>(File& stream, GPUSparseMatrix<double>& us);

    template <class ElemType>
    MATH_API File& operator<<(File& stream, const GPUSparseMatrix<ElemType>& us)
    {
        stream.PutMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
        stream<<sizeof(ElemType);
        if (us.GetMatrixName()==NULL)
        {
            std::wstring s(L"nnmatrix");
            stream<<s;
        }
        else
        {
            stream<<us.GetMatrixName();
        }

        // What we would like to do here, is transfer to CPUSparse and save, do that when the format is the same
        byte* hostBuffer = new byte[us.BufferSize()];
        GPUSparseMatrix<ElemType> hostSide(us.GetNumRows(), us.GetNumCols(), us.NzCount(), (ElemType*)hostBuffer, us.GetFormat());
        CUDACALL(hipMemcpy(hostBuffer, us.NzLocation(),us.BufferSize(),hipMemcpyDeviceToHost));

        // now setup all the stuff pointing to the CPU side info
        const ElemType *dVal=hostSide.NzLocation();
        int* idx=hostSide.IndexLocation();
        int* cidx=hostSide.CompressedIndexLocation();
        size_t nz=us.NzCount();
        size_t ncidx=us.CompressedIndexCount();
        int format = us.GetFormat();
        stream<<format<<nz<<us.GetNumCols()<<us.GetNumRows();
        for (int i=0;i<nz;++i)
        {
            stream<<dVal[i];
        }
        for (int i=0;i<nz;++i)
        {
            stream<<idx[i];
        }
        for (int i=0;i<ncidx;++i)
        {
            stream<<cidx[i];
        }
        stream.PutMarker(fileMarkerEndSection, std::wstring(L"EMAT"));

        // now release the hostSide buffer
        delete hostBuffer;
        hostSide.m_pArray = NULL;

        return stream;
    }
    template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<float>& us);
    template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<double>& us);

}}}