#include "hip/hip_runtime.h"
//
// <copyright file="GPUSparseMatrix.cu" company="Microsoft">
//     Copyright (c) Microsoft Corporation.  All rights reserved.
// </copyright>
//
#pragma once
#include "GPUSparseMatrix.cuh"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "hipblas.h"
#include "GPUMatrixCUDAKernels.cu"
#include <functional>
#include "CommonMatrix.h"
#include <iostream>
#include <ostream>
#include <stdexcept>

#ifdef	LINUX
#define	stdException(x) std::exception()
#else
#define	stdException(x) std::exception(x)
#endif

// thread local storage to access the current stream, initalize to default stream
#ifndef	LINUX
extern __declspec( thread ) 
#endif
	hipStream_t t_stream;

void CUDACALL(hipError_t x) 
{
    if(x!=hipSuccess) 
    { 
        const char* errmsg = hipGetErrorString(x);
        std::cout<<"!!!!!!!!CUDA EXCEPTION: "<<errmsg<<std::endl;

        throw stdException(errmsg);
    }    
}

void CUSPARSECALL(hipsparseStatus_t x) 
{
    if(x!= HIPSPARSE_STATUS_SUCCESS) 
    {         
        std::cout<<"!!!!!!!!CUSPARSE EXCEPTION: "<<std::endl;
        throw stdException("CUSPARSE EXCEPTION");
    }    
}

namespace Microsoft { namespace MSR { namespace CNTK {
    void PrepareDevice(short deviceId);

#pragma region Constructors and Destructor

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix()
    {
        ZeroInit();
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ZeroInit()
    {
        this->m_legacy = true;
        this->m_computeDevice=0; //current GPU device Id
        this->m_numRows=0;  
        this->m_numCols=0;
        this->m_elemSizeAllocated = this->m_nz = 0; //Number of non-zero elements
        this->m_format = matrixFormatSparseCSR;
        this->m_externalBuffer = false;
        this->m_pArray=NULL; 
        this->m_matrixName=NULL;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUMatrix<ElemType>& deepCopy)
    {
        ZeroInit();
        if (!deepCopy.IsEmpty()) 
            SetValue(deepCopy);
    }


    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        this->m_legacy = true;
        DeepCopy(deepCopy);
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const size_t numRows, const size_t numCols, const size_t nz, ElemType* pArray, 
        const size_t matrixFlags /*=matrixFormatSparseCSR*/, int deviceId /*=MANAGEDEXTERN*/, const size_t elemSizeAllocated /*=0*/)
    {
        this->m_legacy = true;
        this->m_computeDevice=deviceId;
        this->m_numRows=numRows;  
        this->m_numCols=numCols;
        this->m_nz=nz; 
        this->m_elemSizeAllocated=elemSizeAllocated?elemSizeAllocated:nz; 
        this->m_pArray = pArray;
        this->m_format = (MatrixFormat)(matrixFormatMask&matrixFlags);
        this->m_externalBuffer = true;
    }

    // legacy code
    /*template<class ElemType>
    void GPUSparseMatrix<ElemType>::Resize(const size_t nR, const size_t nC)
    {
        if (!this->IsEmpty())
        {
            Clear();
        }
        this->m_numRows=nR;  
        this->m_numCols=nC;
        this->m_nz=0; 
        this->m_elemSizeAllocated=m_nz; 
        this->m_pArray = NULL;
    }*/

    // PrepareDevice - Setup the correct cuda context for an operation
    // deviceId - the device on which the operation will take place
    //            defaults to -1, which means use matrices current device
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::PrepareDevice(short deviceId /*=-1*/) const
    {
        // if default value use current compute device
        if (deviceId == -1)
            deviceId = this->m_computeDevice;
        Microsoft::MSR::CNTK::PrepareDevice(deviceId);
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::DeepCopy(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        this->m_computeDevice=deepCopy.m_computeDevice;
        this->m_numRows=deepCopy.m_numRows;  
        this->m_numCols=deepCopy.m_numCols;
        this->m_nz=deepCopy.m_nz; 
        this->m_elemSizeAllocated=deepCopy.m_elemSizeAllocated; 
        this->m_format = deepCopy.m_format;

        deepCopy.PrepareDevice();

        // about to overwrite this buffer, so free it if we own it
        if (this->OwnBuffer() && this->m_pArray!=NULL)
        {
            CUDACALL(hipFree(this->m_pArray));
        }
        else if (!deepCopy.OwnBuffer())
        {
            // just copy over the pointer, this assumses duplicate non-owned buffers are valid
            this->m_pArray = deepCopy.m_pArray;
        }
        else if (deepCopy.m_pArray!=NULL)
        {
            CUDACALL(hipMalloc((void **)&this->m_pArray,BufferSize()));
            CUDACALL(hipMemcpy(this->m_pArray,deepCopy.m_pArray,BufferSize(),hipMemcpyDeviceToDevice));
        }
        else
            this->m_pArray = NULL;
        this->m_externalBuffer = deepCopy.m_externalBuffer;

        if (deepCopy.m_matrixName!=NULL)
        {
            this->m_matrixName = new wchar_t[wcslen(deepCopy.m_matrixName)+1];
            wmemcpy(this->m_matrixName,deepCopy.m_matrixName,wcslen(deepCopy.m_matrixName)+1);
        }
        else
            this->m_matrixName=NULL;
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetValue(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        if (!this->IsEmpty())
        {
            Clear();
        }

        DeepCopy(deepCopy);
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::CopyToDenseMatrix()
    {
        GPUMatrix<ElemType> res;
        if (this->IsEmpty())
            return res;

        PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descr = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

        ElemType* pArrayDev = NULL;
        CUDACALL(hipMalloc((void**)&pArrayDev,sizeof(ElemType)*this->m_numCols*this->m_numRows));
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        CUSPARSECALL(hipsparseSetStream(cusparseHandle, t_stream));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsr2dense(cusparseHandle,int(this->m_numRows),int(this->m_numCols),descr,(float*)NzLocation(),RowLocation(),ColLocation(),(float*)pArrayDev,int(this->m_numRows)));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsr2dense(cusparseHandle,int(this->m_numRows),int(this->m_numCols),descr,(double*)NzLocation(),RowLocation(),ColLocation(),(double*)pArrayDev,int(this->m_numRows)));
        }        
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));
        res.SetValue(this->m_numRows,this->m_numCols,pArrayDev,(matrixFlagNormal|matrixFlagSetValueOnDevice));
        if (pArrayDev!=NULL)
            CUDACALL(hipFree(pArrayDev));
        res.SetMatrixName(this->m_matrixName);        
        return res;            
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& denseMatrix)
    {
        if (!this->IsEmpty())
        {
            Clear();
        }

        PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descr = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);

        this->m_numRows = denseMatrix.GetNumRows(); //m
        this->m_numCols = denseMatrix.GetNumCols(); //n
        this->m_format = matrixFormatSparseCSR;

        int *nnzPerRow = NULL;
        CUDACALL(hipMalloc((void**)&nnzPerRow,sizeof(int)*this->m_numCols));            

        int nnzTotalDevHostPtr = -1;

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseSnnz(cusparseHandle,(this->m_format&matrixFormatRowMajor)?HIPSPARSE_DIRECTION_ROW:HIPSPARSE_DIRECTION_COLUMN,(int)this->m_numRows,(int)this->m_numCols,descr,
                reinterpret_cast<float*>(denseMatrix.BufferPointer()), (int)this->m_numRows,nnzPerRow,&nnzTotalDevHostPtr));
        }
        else
        {
            CUSPARSECALL(hipsparseDnnz(cusparseHandle,(this->m_format&matrixFormatRowMajor)?HIPSPARSE_DIRECTION_ROW:HIPSPARSE_DIRECTION_COLUMN,(int)this->m_numRows,(int)this->m_numCols,descr,
                reinterpret_cast<double*>(denseMatrix.BufferPointer()), (int)this->m_numRows,nnzPerRow,&nnzTotalDevHostPtr));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));

        // about to overwrite this buffer, so free it if we own it
        if (this->OwnBuffer() && this->m_pArray!=NULL)
        {
            CUDACALL(hipFree(this->m_pArray));
        }

        //allocate memory for sparse matrix
        this->m_elemSizeAllocated = this->m_nz = nnzTotalDevHostPtr;
        CUDACALL(hipMalloc((void**)&this->m_pArray,BufferSize()));
        this->m_externalBuffer = false;

        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseSdense2csr(cusparseHandle,(int)this->m_numRows,(int)this->m_numCols,descr,reinterpret_cast<float*>(denseMatrix.BufferPointer()),
                (int)this->m_numRows,nnzPerRow,reinterpret_cast<float*>(NzLocation()),RowLocation(),ColLocation()));
        }
        else
        {
            CUSPARSECALL(hipsparseDdense2csr(cusparseHandle,(int)this->m_numRows,(int)this->m_numCols,descr,reinterpret_cast<double*>(denseMatrix.BufferPointer()),
                (int)this->m_numRows,nnzPerRow,reinterpret_cast<double*>(NzLocation()),RowLocation(),ColLocation()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        this->SetMatrixName(denseMatrix.GetMatrixName());
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(const GPUSparseMatrix<ElemType>& deepCopy)
    {
        if (this != &deepCopy)
        {
            SetValue(deepCopy);
        }
        return *this;       
    }

#ifndef	LINUX
    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(GPUSparseMatrix<ElemType>&& moveFrom)
    {
        this->m_computeDevice=moveFrom.m_computeDevice;
        this->m_numRows=moveFrom.m_numRows;  
        this->m_numCols=moveFrom.m_numCols;
        this->m_nz=moveFrom.m_nz; 
        this->m_elemSizeAllocated = moveFrom.m_elemSizeAllocated;
        this->m_pArray = moveFrom.m_pArray;
        this->m_format = moveFrom.m_format;
        this->m_externalBuffer = moveFrom.m_externalBuffer;
        this->m_matrixName=moveFrom.m_matrixName;

        moveFrom.ZeroInit();
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator=(GPUSparseMatrix<ElemType>&& moveFrom)
    {
        Clear();
        this->m_computeDevice=moveFrom.m_computeDevice;
        this->m_numRows=moveFrom.m_numRows;
        this->m_numCols=moveFrom.m_numCols;
        this->m_nz=moveFrom.m_nz;
        this->m_elemSizeAllocated = moveFrom.m_elemSizeAllocated;
        this->m_pArray = moveFrom.m_pArray;
        this->m_format = moveFrom.m_format;
        this->m_externalBuffer = moveFrom.m_externalBuffer;

        this->m_matrixName=moveFrom.m_matrixName;

        moveFrom.m_pArray = NULL;
        moveFrom.m_matrixName=NULL;
        return *this;
    }
#endif /* LINUX */

    template<class ElemType>
    GPUSparseMatrix<ElemType>::~GPUSparseMatrix()
    {
        if(this->m_legacy) 
        {
            Clear();
        }
        else 
        {
            ClearNew();
        }
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClearNew()
    {
        if (this->m_matrixName!=NULL) 
        {
            delete[] this->m_matrixName;
            this->m_matrixName = NULL;
        }
        if(this->m_format == matrixFormatSparseCSC || this->m_format == matrixFormatSparseCSR) 
        {
            if(this->m_val != NULL) 
                CUDACALL(hipFree(this->m_val));
            if(this->m_row != NULL) 
                CUDACALL(hipFree(this->m_row));
            if(this->m_pb != NULL)
                CUDACALL(hipFree(this->m_pb));
        }  
        else if (this->m_format == matrixFormatSparseBlockCol || this->m_format == matrixFormatSparseBlockRow) 
        {
            if(this->m_blockVal != NULL) 
                CUDACALL(hipFree(this->m_blockVal));
            if(this->m_blockIds != NULL) 
                CUDACALL(hipFree(this->m_blockIds));
        }
    }


    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Clear()
    {
        if (this->m_pArray!=NULL)
            CUDACALL(hipFree(this->m_pArray));
        if (this->m_matrixName!=NULL)
            delete[] this->m_matrixName;
        ZeroInit();
    }

    //ResizeAs - Resize this sparse matrix to have the same element structure as the passed matrix
    // a - sparse matrix whose structure we want to clone
    // remark: this was done for element wise operations where the structure will be identical after an operation
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ResizeAs(const GPUSparseMatrix<ElemType>& a)
    {
        bool reallocate = (BufferSize() != a.BufferSize());

        this->m_numRows=a.m_numRows;
        this->m_numCols=a.m_numCols;
        this->m_nz=a.m_nz; 
        this->m_elemSizeAllocated = a.m_elemSizeAllocated;
        this->m_format = a.m_format;

        if (reallocate)
        {
            if (!this->OwnBuffer())
                throw std::runtime_error("cannot reallocate a buffer not owned by the matrix");
            if (this->m_pArray!=NULL)
                CUDACALL(hipFree(this->m_pArray));
            CUDACALL(hipMalloc((void **)&this->m_pArray,BufferSize()));                  
        }

        // copy over the non-zero locations from the source matrix
        CUDACALL(hipMemcpy(ColLocation(),a.ColLocation(),ColSize(),hipMemcpyDeviceToDevice));
        CUDACALL(hipMemcpy(RowLocation(),a.RowLocation(),RowSize(),hipMemcpyDeviceToDevice));
    }

    //-------------------------------------------------------------------------
    // Start of new GPU Sparse Matrix code 
    //-------------------------------------------------------------------------

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Init()
    {
        this->m_legacy = false;
        this->m_numRows = 0;
        this->m_numCols = 0;
        this->m_elemSizeAllocated = 0;
        this->m_externalBuffer = false;
        this->m_pArray = NULL;        
        PrepareDevice();
        this->m_nz = 0;
        this->m_matrixName = NULL;   

        if(this->m_format == matrixFormatSparseCSC || this->m_format == matrixFormatSparseCSR) 
        {
            this->m_colIdx = -1;
            this->m_val = NULL;
            this->m_row = NULL;
            this->m_pb = NULL;
            this->m_rowIdx = NULL;
            this->m_col = NULL;

            this->m_block2Id = NULL;
            this->m_block2UniqId = NULL;
        } 
        else if (this->m_format == matrixFormatSparseBlockCol || this->m_format == matrixFormatSparseBlockRow) 
        {
            this->m_blockSize = 0;      
            this->m_blockVal = NULL;
            this->m_blockIds = NULL;
        }
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>::GPUSparseMatrix(const MatrixFormat format, const int deviceId)
    {
        if(format != matrixFormatSparseCSC && format != matrixFormatSparseCSR && format != matrixFormatSparseBlockCol && format != matrixFormatSparseBlockRow) 
        {
            throw std::logic_error("GPUSparseMatrix:  unsupported sparse matrix format");
        }
        this->m_format = format;
        this->m_computeDevice = deviceId;
        Init();
    }

    template<class ElemType>
    ElemType* GPUSparseMatrix<ElemType>::BufferPointer() const
    {
        if(this->m_format == matrixFormatSparseCSC || this->m_format == matrixFormatSparseCSR) 
        {
            return this->m_val;
        }  
        else
        {
            return this->m_blockVal;
        }
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, int size)
    {               
        this->m_nz = 0; 
        this->m_colIdx = -1;
        this->m_numRows = numRows;
        this->m_numCols = numCols; 
        if(this->m_elemSizeAllocated < size) 
        {    
            this->m_elemSizeAllocated = size;
            if(this->m_format == matrixFormatSparseCSC || this->m_format == matrixFormatSparseCSR) 
            {
                if(this->m_val != NULL) 
                    CUDACALL(hipFree(this->m_val));
                if(this->m_row != NULL) 
                    CUDACALL(hipFree(this->m_row));
                if(this->m_pb != NULL) 
                    CUDACALL(hipFree(this->m_pb));                
                if(this->m_rowIdx != NULL) 
                    CUDACALL(hipFree(this->m_rowIdx));
                if(this->m_col != NULL) 
                    CUDACALL(hipFree(this->m_col));
                if(this->m_block2Id != NULL) 
                    CUDACALL(hipFree(this->m_block2Id));
                if(this->m_block2UniqId != NULL) 
                    CUDACALL(hipFree(this->m_block2UniqId));

                PrepareDevice();
                CUDACALL(hipMalloc((void **)&this->m_val,sizeof(ElemType)*size));
                CUDACALL(hipMalloc((void **)&this->m_row,sizeof(size_t)*size));
                int len = this->m_format == matrixFormatSparseCSC ? numCols : numRows;
                CUDACALL(hipMalloc((void **)&this->m_pb,sizeof(size_t)*(len+1)));
                CUDACALL(hipMalloc((void **)&this->m_rowIdx,sizeof(size_t)*size));
                CUDACALL(hipMalloc((void **)&this->m_col,sizeof(size_t)*size));                
                CUDACALL(hipMalloc((void **)&this->m_block2Id,sizeof(size_t)*(numCols*2)));
                CUDACALL(hipMalloc((void **)&this->m_block2UniqId,sizeof(size_t)*(numCols*2)));
            } 
            else if(this->m_format == matrixFormatSparseBlockCol || this->m_format == matrixFormatSparseBlockRow) 
            {
                if(this->m_blockVal != NULL) 
                    CUDACALL(hipFree(this->m_blockVal));
                if(this->m_blockIds != NULL) 
                    CUDACALL(hipFree(this->m_blockIds));
                PrepareDevice();
                CUDACALL(hipMalloc((void **)&this->m_blockVal,sizeof(ElemType)*size));
                int max = numCols > numRows ? numCols : numRows;
                CUDACALL(hipMalloc((void **)&this->m_blockIds,sizeof(size_t)*max));
            }
        }
    }

    //Reset matrix so it can be reused
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Reset()
    {                
        this->m_nz = 0;
        this->m_colIdx = -1;
        this->m_blockSize = 0;
    }

#pragma endregion Constructors and Destructor

#pragma region Static BLAS Functions
    
    // copy features to GPU matrix 
     template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetMatrixFromCSCFormat(size_t *h_row, size_t *h_rowIdx, size_t size, size_t blockSize)
    {
        if(this->m_format != matrixFormatSparseCSC) 
        {
            throw std::logic_error("CPUSparseMatrix: unsupported SetValue() call.");
        }

        if(this->m_elemSizeAllocated < size) 
        {
            throw std::logic_error("CPUSparseMatrix:  allocated size is too small.");
        }

        Reset();
        this->m_nz = size;
        this->m_blockSize = blockSize;
        PrepareDevice();
        CUDACALL(hipMemcpy(this->m_row, h_row, sizeof(size_t)*size,hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(this->m_rowIdx, h_rowIdx, sizeof(size_t)*size,hipMemcpyHostToDevice));   
    }
       
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetMatrixFromLabelAndClass(size_t *h_row, size_t *h_block2Id, size_t *h_block2UniqId, size_t labelSize, size_t expandedSize, size_t blockSize)
    {
        if(this->m_format != matrixFormatSparseCSC) 
        {
            throw std::logic_error("CPUSparseMatrix: unsupported SetValue() call.");
        }

        if(this->m_elemSizeAllocated < labelSize) 
        {
            throw std::logic_error("CPUSparseMatrix:  allocated size is too small.");
        }
        
        Reset();
        this->m_nz = labelSize;
        this->m_expandedSize = expandedSize;
        this->m_blockSize = blockSize;
        PrepareDevice();
        
        CUDACALL(hipMemcpy(this->m_row, h_row, sizeof(size_t)*labelSize,hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(this->m_block2Id, h_block2Id, sizeof(size_t)*labelSize,hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(this->m_block2UniqId, h_block2UniqId, sizeof(size_t)*labelSize,hipMemcpyHostToDevice));   
    }

    // forward pass from feature to hidden layer
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA, 
        const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, ElemType beta, GPUMatrix<ElemType>& c)

    {
        if (lhs.GetComputeDeviceId()!=rhs.GetComputeDeviceId()||(lhs.GetComputeDeviceId()!=c.GetComputeDeviceId()))
            throw stdException("MultiplyAndWeightedAddStD: All matrices must be on the same GPU");

        if (lhs.IsEmpty() || rhs.IsEmpty())
            throw std::logic_error("LeftMultiplyAndAdd:  one of the input matrix is empty.");

        int m = transposeA? (int)lhs.GetNumCols(): (int)lhs.GetNumRows();
        int k = transposeA? (int)lhs.GetNumRows(): (int)lhs.GetNumCols();
        int l = transposeB? (int)rhs.GetNumCols(): (int)rhs.GetNumRows();
        int n = transposeB? (int)rhs.GetNumRows(): (int)rhs.GetNumCols();

        assert (m>0 && k>0 && l>0 && n>0);  //converting from size_t to int may cause overflow
        assert (k == l);
        if (k != l) 
        {
            throw std::invalid_argument("CPUSparseMatrix::MultiplyAndAdd: The inner dimensions of a and b must match.");
        }

        if (c.GetNumRows() != m || c.GetNumCols() != n) 
        {
            c.Resize(m,n);
        }         

        if (beta == 0)
        {
            c.SetValue(0.0);
        }
        else 
        {
            c *= beta;
        }

        int blocksPerGrid = rhs.m_nz;
        int p = (threadsPerBlock < lhs.GetNumRows())? threadsPerBlock : lhs.GetNumRows();
        
        if (!transposeA && !transposeB)
        {
            hipEvent_t done; 
            CUDACALL(hipEventCreate(&done));
            _denseMulSparseToDense<ElemType><<<blocksPerGrid, p>>>(
                alpha,
                reinterpret_cast<ElemType*>(lhs.BufferPointer()),
                m,
                k,
                rhs.m_row,
                reinterpret_cast<ElemType*>(c.BufferPointer()));
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));
            CUDACALL(hipEventDestroy(done));
        }
        else if (!transposeA && transposeB)
        {           
            NOT_IMPLEMENTED;
        }
        else if (transposeA && !transposeB)
        {
            NOT_IMPLEMENTED;
        }
        else 
        {
            NOT_IMPLEMENTED;
        }
    }

    // backward pass from hidden layer to feature weight
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::MultiplyAndAdd(ElemType alpha, const GPUMatrix<ElemType>& lhs, const bool transposeA, 
        const GPUSparseMatrix<ElemType>& rhs, const bool transposeB, GPUSparseMatrix<ElemType>& c)
    {
        if (lhs.GetComputeDeviceId()!=rhs.GetComputeDeviceId())
            throw stdException("GPUSparseMatrix::MultiplyAndAdd: All matrices must be on the same GPU");
        
        int m = transposeA? (int)lhs.GetNumCols(): (int)lhs.GetNumRows();
        int k = transposeA? (int)lhs.GetNumRows(): (int)lhs.GetNumCols();
        int l = transposeB? (int)rhs.GetNumCols(): (int)rhs.GetNumRows();
        int n = transposeB? (int)rhs.GetNumRows(): (int)rhs.GetNumCols();

        assert (m>0 && k>0 && l>0 && n>0);  //converting from size_t to int may cause overflow
        assert (k == l);
        if (k != l) 
        {
            throw std::invalid_argument("GPUSparseMatrix::MultiplyAndAdd: The inner dimensions of a and b must match.");
        }

        c.SetFormat(matrixFormatSparseBlockCol);  
        size_t nz = rhs.m_blockSize * c.GetNumRows();        
        //allocate enough memory
        if(c.m_elemSizeAllocated < nz) 
        {
            c.Resize(c.GetNumRows(), c.GetNumCols(), nz);
        }
        c.m_blockSize = rhs.m_blockSize;      
        c.m_nz = nz;
        CUDACALL(hipMemset(c.m_blockVal,0,sizeof(ElemType)*(c.m_nz)));
        CUDACALL(hipMemset(c.m_blockIds,0,sizeof(size_t)*(c.m_blockSize)));
                
        if (!transposeA && !transposeB)
        {
            NOT_IMPLEMENTED;
        }
        else if (!transposeA && transposeB)
        {   
            hipEvent_t done;       
            CUDACALL(hipEventCreate(&done));
            int blocksPerGrid =rhs.GetNZElements();  
            _denseMulSparseToSparse<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
                lhs.BufferPointer(),
                lhs.GetNumRows(),
                rhs.m_row,
                rhs.m_rowIdx,          
                c.m_blockVal, 
                c.m_blockIds);
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));
            CUDACALL(hipEventDestroy(done));
        }
        else if (transposeA && !transposeB)
        {
            NOT_IMPLEMENTED;
        }
        else 
        {
            NOT_IMPLEMENTED;
        }
    }

    // used for gradients udpate
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(const ElemType alpha, const GPUSparseMatrix<ElemType>& lhs, GPUMatrix<ElemType>& rhs)
    {
        if (lhs.GetComputeDeviceId()!=rhs.GetComputeDeviceId())
            throw stdException("GPUSparseMatrix::ScaleAndAdd: All matrices must be on the same GPU");

        if (lhs.m_format == matrixFormatSparseBlockCol || lhs.m_format == matrixFormatSparseBlockRow) 
        {
            size_t len = (lhs.m_format == matrixFormatSparseBlockCol) ? lhs.GetNumRows(): lhs.GetNumCols();
            bool blockCol = (lhs.m_format == matrixFormatSparseBlockCol);

            hipEvent_t done;       
            CUDACALL(hipEventCreate(&done));
            int blocksPerGrid =lhs.m_blockSize;  
            _scaleAndAdd<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
                alpha,
                blockCol,
                lhs.m_blockVal,
                lhs.m_blockIds,
                len,
                rhs.BufferPointer(),
                rhs.GetNumRows());
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));
            CUDACALL(hipEventDestroy(done));
        } 
        else 
        {
            throw stdException("GPUSparseMatrix:: ScaleAndAdd() Not implemented");
        }
    }

    // a: H x No: H is hidden layer size and No is mini-batch size
    // weight: V x H, V is vocab size
    // label: V x No
    // cls: 2 x Nc, Nc is number of classes, each col is start and end word ids of a class
    // idx2cls: V x 1, mapping from word to class id
    // etp: V x No, stores predicted values
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropy(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& weight,
        const GPUSparseMatrix<ElemType> & label, const GPUMatrix<ElemType>& cls, 
        const GPUMatrix<ElemType>& idx2cls, GPUSparseMatrix<ElemType>& etp, GPUMatrix<ElemType>& entropyScore)
    {
        int deviceId = a.GetComputeDeviceId();
        if (weight.GetComputeDeviceId()!=deviceId || label.GetComputeDeviceId()!=deviceId || cls.GetComputeDeviceId()!=deviceId 
            || idx2cls.GetComputeDeviceId()!=deviceId || etp.GetComputeDeviceId()!=deviceId )
            throw stdException("GPUSparseMatrix:: ClassEntropy() All matrices must be on the same GPU");  

        size_t nC = cls.GetNumCols();
        size_t nV = label.GetNumRows() - nC;

        if (nV != idx2cls.GetNumRows() || idx2cls.GetNumCols() != 1 || cls.GetNumCols() + idx2cls.GetNumRows() != label.GetNumRows())
            throw std::logic_error("ClassEntropy: check matrix dimension");        
        
        //allocate enough memory
        if(etp.m_elemSizeAllocated < label.m_expandedSize) 
        {
            etp.Resize(etp.GetNumRows(), etp.GetNumCols(), label.m_expandedSize);
        }
        etp.m_nz = label.m_expandedSize;
        CUDACALL(hipMemset(etp.m_val,0,sizeof(ElemType)*(etp.m_nz)));
        entropyScore.SetValue((ElemType)0);     

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        int blocksPerGrid = label.m_expandedSize;

        //_computePrediction<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
        _computePrediction<ElemType><<<blocksPerGrid, 20>>>(
            idx2cls.GetNumRows(),
            a.BufferPointer(),
            a.GetNumRows(),
            weight.BufferPointer(),
            weight.GetNumRows(),
            label.m_nz,
            label.m_row,
            label.m_block2Id,
            cls.BufferPointer(),
            idx2cls.BufferPointer(),            
            etp.m_val,
            etp.m_row,
            etp.m_pb);

        blocksPerGrid = label.m_nz;
        _normalizePrediction<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
            label.m_nz,
            label.m_expandedSize,
            label.m_row,
            label.m_block2Id, 
            etp.m_row,
            etp.m_val,
            entropyScore.BufferPointer());

        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
   }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropyError(GPUSparseMatrix<ElemType>& a)
    {
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));

        int N = a.m_nz;
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock); 

        _computePredictionError<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
            a.m_val,
            N);

        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropyGradientOfInput(const GPUSparseMatrix<ElemType>& error, const GPUMatrix<ElemType>& weight,  GPUMatrix<ElemType>& grd)
    {
        int deviceId = error.GetComputeDeviceId();
        if (weight.GetComputeDeviceId()!=deviceId || grd.GetComputeDeviceId()!=deviceId )
            throw stdException("GPUSparseMatrix::ClassEntropyGradientOfInput() All matrices must be on the same GPU");

        grd.SetValue((ElemType)0); 
        hipEvent_t done; 
        CUDACALL(hipEventCreate(&done));

        int blocksPerGrid =grd.GetNumElements();
        //_computeGradientOfInput<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
        _computeGradientOfInput<ElemType><<<blocksPerGrid, 20>>>(
            error.m_val,
            error.m_row,
            error.m_pb,
            weight.BufferPointer(),
            weight.GetNumRows(),
            grd.BufferPointer(), 
            grd.GetNumRows());
        CUDACALL(hipEventRecord(done));  
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }
    
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ClassEntropyGradientOfWeight(const GPUSparseMatrix<ElemType>& error,  const GPUMatrix<ElemType>& input, const GPUSparseMatrix<ElemType> & label, const GPUMatrix<ElemType>& cls, 
        const GPUMatrix<ElemType>& idx2cls, GPUSparseMatrix<ElemType>& grd)
    {
        int deviceId = error.GetComputeDeviceId();
        if (input.GetComputeDeviceId()!=deviceId || label.GetComputeDeviceId()!=deviceId || cls.GetComputeDeviceId()!=deviceId  || idx2cls.GetComputeDeviceId()!=deviceId || grd.GetComputeDeviceId()!=deviceId )
            throw stdException("GPUSparseMatrix::ClassEntropyGradientOfWeight() All matrices must be on the same GPU");

        grd.SetFormat(matrixFormatSparseBlockRow);  
        size_t nz = label.m_blockSize * grd.GetNumCols();        
        //allocate enough memory
        if(grd.m_elemSizeAllocated < nz) 
        {
            grd.Resize(grd.GetNumRows(), grd.GetNumCols(), nz);
        }
        grd.m_blockSize = label.m_blockSize;      
        grd.m_nz = nz;
        CUDACALL(hipMemset(grd.m_blockVal,0,sizeof(ElemType)*(grd.m_nz)));
        CUDACALL(hipMemset(grd.m_blockIds,0,sizeof(size_t)*(grd.m_blockSize)));

        hipEvent_t done;  
        CUDACALL(hipEventCreate(&done));

        int blocksPerGrid =error.m_nz; 
        _computeGradientOfWeight<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
            error.m_val,
            error.m_row,
            error.m_pb,
            input.GetNumCols(),
            idx2cls.GetNumRows(),
            label.m_row,
            label.m_block2UniqId,
            cls.BufferPointer(),
            idx2cls.BufferPointer(),              
            input.BufferPointer(),
            input.GetNumRows(),
            grd.m_blockVal, 
            grd.m_blockIds);
        CUDACALL(hipEventRecord(done)); 
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncate (const ElemType threshold)
    {
        if(this->m_format == matrixFormatSparseBlockCol || this->m_format == matrixFormatSparseBlockRow) 
        {
            long N=(long)GetNZElements();
            int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
            hipEvent_t done;       
            CUDACALL(hipEventCreate(&done));        
            _inplaceTruncate<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_blockVal,threshold,N);
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));   
            CUDACALL(hipEventDestroy(done));
        } 
        else 
        {
            throw stdException("GPUSparseMatrix:: InplaceTruncate() only support block based sparse matrix");
        }
        return *this;
    } 

    // normal update for smoothed gradients c and current gradients (this)
    template<class ElemType> 
    void GPUSparseMatrix<ElemType>::NormalGrad(GPUMatrix<ElemType>& c, const ElemType momentum)
    {
        if (c.IsEmpty())
        {
            c.Resize(this->GetNumRows(), this->GetNumCols());
            c.SetValue(0.0);
        }

        if(this->m_format == matrixFormatSparseBlockCol || this->m_format == matrixFormatSparseBlockRow) 
        {
            int blocksPerGrid = this->m_blockSize;    
            bool isBlockCol = (this->m_format == matrixFormatSparseBlockCol);
            size_t len = isBlockCol ? this->GetNumRows(): this->GetNumCols();
            hipEvent_t done;       
            CUDACALL(hipEventCreate(&done));        
            _normalGrad<ElemType><<<blocksPerGrid,threadsPerBlock>>>(
                isBlockCol,
                len,
                momentum,
                this->m_blockIds,
                this->m_blockVal,
                c.BufferPointer(),
                c.GetNumRows());                        
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));    
            CUDACALL(hipEventDestroy(done));
        } 
        else 
        {
            throw stdException("GPUSparseMatrix:: NormalGrad() only support block sparse format");
        }
    }

    //-------------------------------------------------------------------------
    // End of new GPU Sparse Matrix code 
    //-------------------------------------------------------------------------

    template<class ElemType>
    void  GPUSparseMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUSparseMatrix<ElemType>& a, const bool transposeA, 
        const GPUMatrix<ElemType>& b, ElemType beta, GPUMatrix<ElemType>& c)
    {
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId()||(b.GetComputeDeviceId()!=a.GetComputeDeviceId()))
            throw stdException("MultiplyAndWeightedAddStD: All matrices must be on the same GPU");
        a.PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descr = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descr));
        hipsparseSetMatType(descr,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descr,HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseOperation_t oper = transposeA ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

        int m = (int)a.GetNumRows();
        int n = (int)b.GetNumCols();
        assert(n==(int)c.GetNumCols());
        int k = (int)a.GetNumCols();

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsrmm(cusparseHandle,oper,m,n,k,(int)a.GetNZElements(),reinterpret_cast <float*>(&alpha),descr,reinterpret_cast <const float*>(a.NzLocation()),
                a.RowLocation(), a.ColLocation(), reinterpret_cast <float*>(b.BufferPointer()),
                (int)b.GetNumRows(),reinterpret_cast <float*>(&beta),reinterpret_cast <float*>(c.BufferPointer()),(int)c.GetNumRows()));
        }
        else 
        {
            CUSPARSECALL(hipsparseDcsrmm(cusparseHandle,oper,m,n,k,(int)a.GetNZElements(),reinterpret_cast <double*>(&alpha),descr,reinterpret_cast <const double*>(a.NzLocation()),
                a.RowLocation(), a.ColLocation(), reinterpret_cast <double*>(b.BufferPointer()),
                (int)b.GetNumRows(),reinterpret_cast <double*>(&beta),reinterpret_cast <double*>(c.BufferPointer()),(int)c.GetNumRows()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));        
    }
       

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S, const GPUMatrix<ElemType>& D, GPUMatrix<ElemType>& C)
    {
        if (C.GetNumRows()!=S.GetNumRows() || C.GetNumCols()!=D.GetNumRows())
        {
            GPUMatrix<ElemType> tmp(S.GetNumRows(),D.GetNumCols(),S.GetComputeDeviceId());
            C=tmp;
        }
        MultiplyAndWeightedAdd(1,S,false,D,0,C);
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Multiply(const GPUMatrix<ElemType>& D, const GPUSparseMatrix<ElemType>& S, GPUMatrix<ElemType>& C)
    {   
        GPUMatrix<ElemType> Res(S.GetNumCols(),D.GetNumRows());
        MultiplyAndWeightedAdd(1,S,true,D.Transpose(),0,Res);
        C.AssignTransposeOf(Res);       
    }

    // ElemCountFromBufferSize - Return the elemCountAllocated for a particular buffersize
    // totalBufferSize - total buffer we have to use
    // return: size of allocated elements/index slots available
    template<class ElemType>
    size_t GPUSparseMatrix<ElemType>::ElemCountFromBufferSize(size_t totalBufferSize)
    {
        size_t elemSizeAllocated;
        if (this->m_format & matrixFormatCompressed)
        {
            elemSizeAllocated = (totalBufferSize-CompressedIndexSize())/(sizeof(int)+sizeof(ElemType));
        }
        else // uncompressed COO format
        {
            elemSizeAllocated = totalBufferSize/(2*sizeof(int)+sizeof(ElemType));
        }
        return elemSizeAllocated;
    }

    // PrepareBuffer - Get the dimensions start buffer, computes the starting row/column of each value
    // m - rows in the source
    // n - cols in the source
    // canReuseBuffer - target matrix can be reused for temporary space
    // func - function to call to count elements in the result (returns count, and fills csrRowPtr array)
    template<class ElemType>
#ifndef	LINUX
    void GPUSparseMatrix<ElemType>::PrepareBuffer(size_t m, size_t n, bool canReuseBuffer, std::function<size_t (int* csrRowPtrC)> func)
#else
    void GPUSparseMatrix<ElemType>::PrepareBuffer(size_t m, size_t n, bool canReuseBuffer, size_t (*func)(int *csRowPtrC))
#endif	/* LINUX */
    {
        int* csrRowPtrC=NULL;
        GPUSparseMatrix<ElemType>& c = *this;
        int cSize = c.BufferSize();
        int rowBufferRequired = (m+1)*sizeof(int);
        // determine the size of the buffer and align the final location of the row index buffer
        int nzBufSize = cSize-rowBufferRequired;
        nzBufSize -= nzBufSize%(sizeof(int)+sizeof(ElemType));
        bool allocatedBuffer = false;

        // do we have enough memory to store just the row buffer?
        if (cSize >= rowBufferRequired && c.NzLocation() != NULL && canReuseBuffer)
        {
            // determine the final location if we reuse the buffer
#ifndef	LINUX
            csrRowPtrC = (int*)((byte*)c.NzLocation() + nzBufSize);
#else
            csrRowPtrC = (int*)((char*)c.NzLocation() + nzBufSize);
#endif
        }
        else
        {
            CUDACALL(hipMalloc((void **)&csrRowPtrC,(m+1)*sizeof(int)));
            allocatedBuffer = true;
        }

        // get the non-zero count from the function (and 
        int nnzC = func(csrRowPtrC);

        // now we know the number of Non-zeros in the result set, set the output size
        c.m_elemSizeAllocated = c.m_nz = nnzC;
        c.m_numRows = m;
        c.m_numCols = n;
        size_t requiredSize = c.BufferSize();
        // see if the buffer we already have is big enough
        if (cSize >= requiredSize)
        {
            // compute the allocated size, to take up any additional space in the memory block 
            c.m_elemSizeAllocated = c.ElemCountFromBufferSize(cSize);
            // copy the rowPtr array to the proper location
            CUDACALL(hipMemcpy(c.CompressedIndexLocation(),csrRowPtrC,c.CompressedIndexSize(),hipMemcpyDeviceToDevice));
        }
        else
        {
            void* oldBuffer = c.m_pArray;
            // allocate required array space
            CUDACALL(hipMalloc((void **)&c.m_pArray,requiredSize));      
            // copy over 
            CUDACALL(hipMemcpy(c.CompressedIndexLocation(),csrRowPtrC,c.CompressedIndexSize(),hipMemcpyDeviceToDevice));
            // release the previous buffer since we just reallocated it
            if (oldBuffer != NULL)
                CUDACALL(hipFree(oldBuffer));
        }
        // if we allocated the buffer, free it here
        if (allocatedBuffer)
            CUDACALL(hipFree(csrRowPtrC));
    }

#ifdef	LINUXxx
    size_t PrepareBufferMultiply(int* csrRowPtrC)
        {
            int nnzTotal = -1; 
            CUSPARSECALL(hipsparseXcsrgemmNnz(cusparseHandle,operA,operB,m,n,k,descrA,nnzA,S1.RowLocation(),S1.ColLocation(),descrB,nnzB,
                S2.RowLocation(),S2.ColLocation(),descrC,csrRowPtrC,&nnzTotal));
            return nnzTotal;
        }
#endif

    // Multiply - multiply one spares matrix by another sparse matrix
    // S1 - first sparse matrix
    // transposeS1 - transpose first matrix?
    // S2 - second sparse matrix
    // transposeS2 - tanspose second matrix?
    // c - result matrix
    // NOTE: if c has enough space allocated, it will be reused, otherwise it will be freed and a new memory block used
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Multiply(const GPUSparseMatrix<ElemType>& S1, bool transposeS1, const GPUSparseMatrix<ElemType>& S2, bool transposeS2, GPUSparseMatrix<ElemType> &c)
    {
        if (S1.GetComputeDeviceId()!=S2.GetComputeDeviceId())
            throw stdException("Sparse matrix multiply: both matrices must be on the same device");

        S1.PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descrA)); CUSPARSECALL(hipsparseCreateMatDescr(&descrB)); CUSPARSECALL(hipsparseCreateMatDescr(&descrC));        
        hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrC,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrB,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ZERO);
        hipsparseOperation_t operA = transposeS1 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t operB = transposeS2 ? HIPSPARSE_OPERATION_TRANSPOSE : HIPSPARSE_OPERATION_NON_TRANSPOSE;

        int m = int(transposeS1 ? S1.GetNumCols() : S1.GetNumRows());
        int n = int(transposeS2 ? S2.GetNumRows() : S2.GetNumCols());
        int k = int(transposeS1 ? S1.GetNumRows() : S1.GetNumCols());
        int l = int(transposeS2 ? S2.GetNumCols() : S2.GetNumRows());
        if (k!=l)
            throw stdException("Sparse matrix multiply: dimensionality mismatch");

        int nnzA = (int)S1.GetNZElements();
        int nnzB = (int)S2.GetNZElements();

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        //Step 1 
        c.PrepareBuffer(m, n, true, // true means we can reuse the "c" buffer if it exists for temporaries
#ifndef	LINUX
            [&](int* csrRowPtrC) -> size_t
        {
            int nnzTotal = -1; 
            CUSPARSECALL(hipsparseXcsrgemmNnz(cusparseHandle,operA,operB,m,n,k,descrA,nnzA,S1.RowLocation(),S1.ColLocation(),descrB,nnzB,
                S2.RowLocation(),S2.ColLocation(),descrC,csrRowPtrC,&nnzTotal));
            return nnzTotal;
        }
#else
	NULL		// PrepareBufferMultiply
#endif
	);


        //Step 2
        if (sizeof(float)==sizeof(ElemType))
        {
            CUSPARSECALL(hipsparseScsrgemm(cusparseHandle,operA,operB,m,n,k,descrA,nnzA,(const float*)S1.NzLocation(),S1.RowLocation(),S1.ColLocation(),
                descrB,nnzB,(const float*)S2.NzLocation(),S2.RowLocation(),S2.ColLocation(),
                descrC,(float*)c.NzLocation(),c.RowLocation(),c.ColLocation()));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsrgemm(cusparseHandle,operA,operB,m,n,k,descrA,nnzA,(const double*)S1.NzLocation(),S1.RowLocation(),S1.ColLocation(),
                descrB,nnzB,(const double*)S2.NzLocation(),S2.RowLocation(),S2.ColLocation(),
                descrC,(double*)c.NzLocation(),c.RowLocation(),c.ColLocation()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        hipsparseDestroy(cusparseHandle);   
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignProductOf(const GPUSparseMatrix<ElemType>& a, const bool transposeA, const GPUSparseMatrix<ElemType>& b, const bool transposeB)
    {
        Multiply(a,transposeA,b,transposeB,*this);
        return *this;
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha,const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUSparseMatrix<ElemType>& c)
    {
        if (a.GetNumCols()!=b.GetNumCols() || a.GetNumRows()!=b.GetNumRows())
            throw new stdException("Dimensions mismatch in ScaleAndAdd");
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId())
            throw new stdException("ScaleAndAdd: matrices must be on the same device");

        int m = (int)a.GetNumRows();
        int n = (int)a.GetNumCols();
        int nnzA = (int)a.GetNZElements();
        int nnzB = (int)b.GetNZElements();

        a.PrepareDevice();
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipsparseMatDescr_t descrA = 0, descrB = 0, descrC = 0;
        CUSPARSECALL(hipsparseCreateMatDescr(&descrA)); CUSPARSECALL(hipsparseCreateMatDescr(&descrB)); CUSPARSECALL(hipsparseCreateMatDescr(&descrC));
        hipsparseSetMatType(descrA,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL); hipsparseSetMatType(descrB,HIPSPARSE_MATRIX_TYPE_GENERAL);
        hipsparseSetMatIndexBase(descrA,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrB,HIPSPARSE_INDEX_BASE_ZERO); hipsparseSetMatIndexBase(descrC,HIPSPARSE_INDEX_BASE_ZERO);

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        //Step 1 
        bool inOutParameter = (&b == &c);
        c.PrepareBuffer(m, n, !inOutParameter, 
#ifndef	LINUX
	[&] (int* csrRowPtrC) -> size_t
        {
            int nnzTotal = -1;
            CUSPARSECALL(hipsparseXcsrgeamNnz(cusparseHandle,m,n,descrA,nnzA,a.RowLocation(),a.ColLocation(),descrB,nnzB,b.RowLocation(),b.ColLocation(),descrC,csrRowPtrC,&nnzTotal));
            return nnzTotal;
        }
#else
	NULL
#endif	// Linux
	);

        //Step 2
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsrgeam(cusparseHandle,m,n,reinterpret_cast <const float*>(&alpha),descrA,nnzA,reinterpret_cast <const float*>(a.NzLocation()),a.RowLocation(),a.ColLocation(),
                reinterpret_cast <const float*>(&beta),descrB,nnzB,reinterpret_cast <const float*>(b.NzLocation()),b.RowLocation(),b.ColLocation(),descrC,reinterpret_cast <float*>(c.NzLocation()),c.RowLocation(),c.ColLocation()));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsrgeam(cusparseHandle,m,n,reinterpret_cast <const double*>(&alpha),descrA,nnzA,reinterpret_cast <const double*>(a.NzLocation()),a.RowLocation(),a.ColLocation(),
                reinterpret_cast <const double*>(&beta),descrB,nnzB,reinterpret_cast <const double*>(b.NzLocation()),b.RowLocation(),b.ColLocation(),descrC,reinterpret_cast <double*>(c.NzLocation()),c.RowLocation(),c.ColLocation()));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        hipsparseDestroy(cusparseHandle);   
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha,const GPUSparseMatrix<ElemType>& a, ElemType beta, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        if (a.GetNumRows()!=b.GetNumRows()||a.GetNumRows()!=c.GetNumRows()||a.GetNumCols()!=b.GetNumCols()||a.GetNumCols()!=c.GetNumCols())
            throw std::logic_error("ScaleAndAdd: dimension mismatch");
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId()||a.GetComputeDeviceId()!=c.GetComputeDeviceId())
            throw stdException("ScaleAndAdd: matrices must be on the same device");
        b.PrepareDevice();
        //copy b to c
        CUDACALL(hipMemcpy(c.BufferPointer(),b.BufferPointer(),sizeof(ElemType)*b.GetNumElements(),hipMemcpyDeviceToDevice));
        if (beta!=1)
        {
            c*=beta;
        }
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        long M=(long)a.GetNumRows();
        int blocksPerGrid =(int)ceil(1.0*M/threadsPerBlock);        
        _sparsePlusDense<ElemType><<<blocksPerGrid,threadsPerBlock>>>(alpha,a.NzLocation(),a.RowLocation(),a.ColLocation(),c.BufferPointer(),M);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ScaleAndAdd(ElemType alpha,const GPUMatrix<ElemType>& a, ElemType beta, const GPUSparseMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        ScaleAndAdd(beta,b,alpha,a,c);
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Scale(ElemType alpha, GPUSparseMatrix<ElemType>& a)
    {
        if (a.IsEmpty())
            return;

        long N=(long)a.GetNZElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _scaleArray<ElemType><<<blocksPerGrid,threadsPerBlock>>>(alpha,a.NzLocation(),N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));        
        CUDACALL(hipEventDestroy(done));        
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::ElementWisePower (ElemType alpha, const GPUSparseMatrix<ElemType>& a, GPUSparseMatrix<ElemType>& c)
    {
        if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }
        else 
        {
            if (a.IsEmpty())
                throw std::logic_error("ElementWisePower:  The input matrix a is empty.");
            if (a.GetNumRows()!=c.GetNumRows() || a.GetNumCols()!=c.GetNumCols() || a.GetNZElements()!=c.GetNZElements())
                c.ResizeAs(a);

            hipEvent_t done;
            CUDACALL(hipEventCreate(&done));
            a.PrepareDevice();
            long N=(long)a.GetNZElements();
            int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
            _elementWisePowerOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(alpha,a.NzLocation(),c.NzLocation(),N);
            CUDACALL(hipEventRecord(done));        
            CUDACALL(hipEventSynchronize(done));   
        }
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId())
            throw stdException("a and b must be on the same device");

        //This implementation requires additional memory
        //need to put a in ColumnMajor format
        int m = (int)a.GetNumRows();
        int n = (int)a.GetNumCols();
        int nnz = (int)a.GetNZElements();
        hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;
        ElemType* cscValA = NULL;
        int* cscRowIndA = NULL;
        int* cscColPtrA = NULL;
        a.PrepareDevice();
        CUDACALL(hipMalloc((void **)&cscValA,nnz*sizeof(ElemType)));
        CUDACALL(hipMalloc((void **)&cscRowIndA,nnz*sizeof(int)));        
        CUDACALL(hipMalloc((void **)&cscColPtrA,(n+1)*sizeof(int)));
        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const float*>(a.NzLocation()),a.RowLocation(),a.ColLocation(),reinterpret_cast<float*>(cscValA),cscRowIndA,cscColPtrA,cpVals,idxBase));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const double*>(a.NzLocation()),a.RowLocation(),a.ColLocation(),reinterpret_cast<double*>(cscValA),cscRowIndA,cscColPtrA,cpVals,idxBase));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        CUDACALL(hipEventDestroy(done));

        //Given sparse matrix in column major format, calculate indices for corresponding sparse vector
        int* vectArray=NULL;
        CUDACALL(hipMalloc((void**)&vectArray,sizeof(int)*a.m_nz));
        long M=n;
        long N=m;
        //int* h_vectArray= new int[a.m_nz];
        int blocksPerGrid =(int)ceil(1.0*M/threadsPerBlock);   
        CUDACALL(hipEventCreate(&done));
        _getSparseVectorRepresntationForMatrix<ElemType><<<blocksPerGrid,threadsPerBlock>>>(cscColPtrA,cscRowIndA,vectArray,M,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        CUDACALL(hipFree(cscRowIndA));
        CUDACALL(hipFree(cscColPtrA));
        //CUDACALL(hipMemcpy(h_vectArray,vectArray,sizeof(int)*a.m_nz,hipMemcpyDeviceToHost));    

        //Actual dot product
        ElemType res=0;
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseSdoti(cusparseHandle,(int)a.m_nz,reinterpret_cast<float*>(cscValA),vectArray,
                reinterpret_cast<float*>(b.BufferPointer()),
                reinterpret_cast<float*>(&res),idxBase));
        }
        else
        {
            CUSPARSECALL(hipsparseDdoti(cusparseHandle,(int)a.m_nz,reinterpret_cast<double*>(cscValA),vectArray,
                reinterpret_cast<double*>(b.BufferPointer()),
                reinterpret_cast<double*>(&res),idxBase));
        }       
        CUDACALL(hipFree(vectArray));
        CUDACALL(hipFree(cscValA));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));   
        return res;        
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::InnerProductOfMatrices(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
    {
        return GPUSparseMatrix<ElemType>::InnerProductOfMatrices(b,a);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b, 
        const ElemType threshold)
    {
        if (a.GetNZElements()!=b.GetNZElements() || a.GetNumRows()  != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            return false;

        a.PrepareDevice();
        long *res = new long[3];
        res[0]=1;
        res[1]=1;
        res[2]=1;
        long *d_res = NULL;
        CUDACALL(hipMalloc((void**)&d_res,sizeof(long)*3)); 
        CUDACALL(hipMemcpy(d_res,res,sizeof(long)*3,hipMemcpyHostToDevice));

        int blocksPerGrid =(int)ceil(1.0*a.GetNZElements()/threadsPerBlock); 
        _areEqual<ElemType><<<blocksPerGrid,threadsPerBlock>>>(a.NzLocation(),b.NzLocation(),(long)a.GetNZElements(),threshold,d_res);
        _areEqual<int><<<blocksPerGrid,threadsPerBlock>>>(a.ColLocation(),b.ColLocation(),(long)a.GetNZElements(),(int)threshold,d_res+1);
        blocksPerGrid =(int)ceil((1.0*a.GetNumRows()+1.0)/threadsPerBlock); 
        _areEqual<int><<<blocksPerGrid,threadsPerBlock>>>(a.RowLocation(),b.RowLocation(),(long)a.GetNumRows()+1,(int)threshold,d_res+2);

        CUDACALL(hipMemcpy(res,d_res,sizeof(long)*3,hipMemcpyDeviceToHost));        
        if (res[0]*res[1]*res[2]==1)
            return true;
        else
            return false;
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::AreEqual(const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b, 
        const ElemType threshold)
    {
        if (a.GetNumElements()!=b.GetNZElements() || a.GetNumRows()  != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            return false;
        GPUSparseMatrix<ElemType> c;
        c.SetValue(a);
        return AreEqual(c,b,threshold);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::AreEqual(const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, 
        const ElemType threshold)
    {
        if (a.GetNZElements()!=b.GetNumElements() || a.GetNumRows()  != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            return false;
        GPUSparseMatrix<ElemType> c;
        c.SetValue(b);
        return AreEqual(a,c,threshold);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUSparseMatrix<ElemType>& a, const ElemType threshold) const
    {
        return AreEqual(*this,a,threshold);
    }

    template<class ElemType>
    bool GPUSparseMatrix<ElemType>::IsEqualTo(const GPUMatrix<ElemType>& a, const ElemType threshold) const
    {
        return AreEqual(*this,a,threshold);
    }
#pragma endregion Static BLAS Functions

#pragma region Member BLAS Functions

    template<class ElemType>
    int GPUSparseMatrix<ElemType>::GetComputeDeviceId() const 
    {
        // for externally managed memory the CUDA context will have the current device
        if (this->m_computeDevice == MANAGEDEXTERN)
        {
            int devId;
            assert(this->m_externalBuffer);
            CUDACALL(hipGetDevice(&devId));
            return devId;
        }
        return this->m_computeDevice;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf (const GPUSparseMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.GetNumRows()!=b.GetNumRows()||a.GetNumCols()!=b.GetNumCols())
            throw std::logic_error("ElementProductOf: matrix dimensions mismatch");

        b.PrepareDevice();        
        GPUMatrix<ElemType> c(b.GetNumRows(),b.GetNumCols(),b.GetComputeDeviceId());

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        long M=(long)a.GetNumRows();
        int blocksPerGrid =(int)ceil(1.0*M/threadsPerBlock);        
        _sparseMulDense<ElemType><<<blocksPerGrid,threadsPerBlock>>>(a.NzLocation(),a.RowLocation(),a.ColLocation(),b.BufferPointer(),c.BufferPointer(),M);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        CUDACALL(hipEventDestroy(done));
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUSparseMatrix<ElemType>::ElementProductOf (const GPUMatrix<ElemType>& a, const GPUSparseMatrix<ElemType>& b)
    {
        return GPUSparseMatrix<ElemType>::ElementProductOf(b,a);        
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator+ (const GPUSparseMatrix<ElemType>& a) const
    {
        GPUSparseMatrix<ElemType> res;
        GPUSparseMatrix<ElemType>::ScaleAndAdd(1,*this,1,a,res);
        return res;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator- (const GPUSparseMatrix<ElemType>& a) const
    {
        GPUSparseMatrix<ElemType> res;
        GPUSparseMatrix<ElemType>::ScaleAndAdd(1,*this,-1,a,res);
        return res;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator^=(ElemType alpha)
    {
        GPUSparseMatrix<ElemType>& us = *this;
        ElementWisePower(alpha, us, us);
        return us;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator^ (ElemType alpha) const
    {
        GPUSparseMatrix<ElemType> c;
        c.ResizeAs(*this);
        ElementWisePower(alpha, *this, c);
        return c;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::operator*=(ElemType alpha)
    {
        GPUSparseMatrix<ElemType>& us = *this;
        if (alpha!=1)            
            Scale(alpha,us);
        return us;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::operator* (ElemType alpha) const
    {
        GPUSparseMatrix<ElemType> c(*this);
        if (alpha!=1)
            Scale(alpha, c);
        return c;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementPowerOf(const GPUSparseMatrix<ElemType>& a, const ElemType power)
    {
        ElementWisePower(power, a, *this);
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType> GPUSparseMatrix<ElemType>::Transpose() const
    {
        int m = (int)this->GetNumRows();
        int n = (int)this->GetNumCols();
        int nnz = (int)this->GetNZElements();
        hipsparseAction_t cpVals = HIPSPARSE_ACTION_NUMERIC;
        hipsparseIndexBase_t idxBase = HIPSPARSE_INDEX_BASE_ZERO;

        assert(this->GetFormat()&matrixFormatCompressed); // for now this only supports compressed formats
        PrepareDevice();
        GPUSparseMatrix c(n, m, nnz, NULL, this->GetFormat(), GetComputeDeviceId(), this->m_elemSizeAllocated);
        CUDACALL(hipMalloc((void **)&c.m_pArray,c.BufferSize()));

        hipsparseHandle_t cusparseHandle = 0;
        CUSPARSECALL(hipsparseCreate(&cusparseHandle));

        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CUSPARSECALL(hipsparseScsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const float*>(this->NzLocation()),this->CompressedIndexLocation(),this->IndexLocation(),
                reinterpret_cast<float*>(c.NzLocation()),c.IndexLocation(),c.CompressedIndexLocation(),cpVals,idxBase));
        }
        else
        {
            CUSPARSECALL(hipsparseDcsr2csc(cusparseHandle,m,n,nnz,reinterpret_cast<const double*>(this->NzLocation()),this->CompressedIndexLocation(),this->IndexLocation(),
                reinterpret_cast<double*>(c.NzLocation()),c.IndexLocation(),c.CompressedIndexLocation(),cpVals,idxBase));
        }
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        CUDACALL(hipEventDestroy(done));
        CUSPARSECALL(hipsparseDestroy(cusparseHandle));        
        return c;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTransposeOf(const GPUSparseMatrix<ElemType>& a)
    {
        if (this == &a)
            throw std::logic_error("AssignTransposeOf: a is the same as [this]. Does not support inplace transpose.");

        if (a.IsEmpty())
            throw std::logic_error("AssignTransposeOf: Matrix a is empty.");

        *this = a.Transpose();
        return *this;
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::InplaceTranspose()
    {
        if (this->IsEmpty())
            return;
        // transfer converted block over to this pointer
#ifndef	LINUX
        *this = std::move(this->Transpose());
#else	
	std::cerr << "Not sure how to do the InplaceTranspose()";
#endif
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::SumOfAbsElements() const
    {
        if (this->IsEmpty())
            throw std::logic_error("SumOfAbsElements: Matrix is empty");

        hipblasHandle_t cuHandle = GPUMatrix<ElemType>::GetCublasHandle(this->GetComputeDeviceId());
        if (sizeof(ElemType)==sizeof(float))
        {
            float res=0;
            hipblasSasum(cuHandle,(int)GetNZElements(),reinterpret_cast<float*>(this->m_pArray),1,&res);
            return res;
        }
        else
        {
            double res=0;
            hipblasDasum(cuHandle,(int)GetNZElements(),reinterpret_cast<double*>(this->m_pArray),1,&res);
            return ElemType(res);
        }         
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::SumOfElements() const
    {
        if (this->IsEmpty())
            throw std::logic_error("SumOfElements: Matrix is empty");

        PrepareDevice();
        ElemType* d_sum = NULL;
        ElemType h_sum;
        CUDACALL(hipMalloc((void**)&d_sum,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSum<ElemType><<<1,1024>>>(this->m_pArray,d_sum,(LONG64)this->GetNZElements());
        CUDACALL(hipMemcpy(&h_sum,d_sum,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDACALL(hipFree(d_sum));               
        return h_sum;        
    }


    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::FrobeniusNorm() const 
    {
        if (this->IsEmpty())
            throw std::logic_error("FrobeniusNorm: Matrix is empty.");

        ElemType* d_sum = NULL;
        ElemType h_sum=0;
        CUDACALL(hipMalloc((void**)&d_sum,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSum2<ElemType><<<1,1024>>>(this->m_pArray,d_sum,(int)this->GetNZElements());
        CUDACALL(hipMemcpy(&h_sum,d_sum,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDACALL(hipFree(d_sum));               
        if (sizeof(ElemType)==sizeof(float))
            return sqrtf(h_sum);
        else
            return sqrt(h_sum); 
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::MatrixNormInf() const
    {
        if (this->IsEmpty())
            throw std::logic_error("MatrixNorm1: Matrix is empty.");

        ElemType* d_maxAbs = NULL;
        ElemType h_maxAbs=0;
        CUDACALL(hipMalloc((void**)&d_maxAbs,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionMatrixNormInf<ElemType><<<1,1024>>>(this->m_pArray,d_maxAbs,(int)this->GetNZElements());
        CUDACALL(hipMemcpy(&h_maxAbs,d_maxAbs,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDACALL(hipFree(d_maxAbs));               
        if (sizeof(ElemType)==sizeof(float))
            return h_maxAbs;
        else
            return h_maxAbs; 
    }

    template<class ElemType>
    ElemType GPUSparseMatrix<ElemType>::MatrixNorm1() const
    {
        if (this->IsEmpty())
            throw std::logic_error("MatrixNorm1: Matrix is empty.");
        return this->SumOfAbsElements();              
    }

#pragma endregion Member BLAS Functions

#pragma region Other Functions

    template<class ElemType>    
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::ElementInverse ()
    {
        if (this->IsEmpty())
            throw std::logic_error("ElementInverse: Matrix is empty.");

        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _elemInverse<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignElementInverseOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        return this->ElementInverse();
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSigmoid()
    {
        performInplaceFunction(0);                    
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSigmoidOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceSigmoid();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLinearRectifierDerivative()
    {
        performInplaceFunction(6);                    
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLinearRectifierDerivativeOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceLinearRectifierDerivative();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTanh()
    {
        performInplaceFunction(1);
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTanhOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceTanh();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceSqrt()
    {
        performInplaceFunction(2);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignSqrtOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceSqrt();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceExp()
    {
        performInplaceFunction(3);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignExpOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceExp();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceLog()
    {
        performInplaceFunction(4);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignLogOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceLog();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceAbs()
    {
        performInplaceFunction(5);        
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignAbsOf (const GPUSparseMatrix<ElemType>& a)
    {
        this->SetValue(a);
        this->InplaceAbs();
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateBottom (const ElemType threshold)
    {
        if (this->IsEmpty())
            throw std::logic_error("InplaceTruncateBottom: Matrix is empty.");
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _inplaceTruncateBottom<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,threshold,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        return *this;
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateBottomOf (const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignTruncateBottomOf: Matrix a is empty.");

        if (this!=&a)
        {
            //Resize(a.GetNumRows(), a.GetNumCols());           
            ResizeAs(a);  
        }
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _assignTruncateBottom<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,a.NzLocation(),threshold,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        return *this;
    }   

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::InplaceTruncateTop (const ElemType threshold)
    {
        if (this->IsEmpty())
            throw std::logic_error("InplaceTruncateTop: Matrix is empty.");
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _inplaceTruncateTop<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,threshold,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        return *this;        
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::AssignTruncateTopOf (const GPUSparseMatrix<ElemType>& a, const ElemType threshold)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignTruncateTopOf: Matrix a is empty.");

        if (this!=&a)
        {
            ResizeAs(a);
        }

        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _assignTruncateTop<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,a.NzLocation(),threshold,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));
        return *this;        
    }

    template<class ElemType>
    GPUSparseMatrix<ElemType>& GPUSparseMatrix<ElemType>::SetToZeroIfAbsLessThan (const ElemType threshold)
    {
        if (this->IsEmpty())
            throw std::logic_error("SetToZeroIfAbsLessThan: Matrix is empty.");
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        _setToZeroIfAbsLessThan<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,threshold,N);
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done)); 
        return *this;  
    }
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::Unrolling (//GPUSparseMatrix<ElemType>& debugMatrix, 
        GPUMatrix<ElemType>& UnrolledMatrix, const GPUMatrix<ElemType>& InMatrix, GPUSparseMatrix<ElemType>& UnrollMapping, 
        const int inputWidth, const int inputHeight, const int inputChannelNum,
        const int FltWidth,const int FltHeight, const int FltChannel,
        const int FltStepW,  const int FltStepH)
    {
        ////if ((UnrolledMatrix.m_computeDevice!=InMatrix.m_computeDevice) ||(InMatrix.m_computeDevice!=UnrollMapping.m_computeDevice)) //different GPUs
        ////{
        ////    throw std::invalid_argument("All matrices must be on the same GPU");
        ////}
        ////else
        ////{ 
        //    //m_computeDevice = deviceId;

        //    const int inPatchSize = inputWidth * inputHeight;// * inputChannelNum;
        //    const int inRowHeight = InMatrix.GetNumRows();//m_inSampleNum;
        //    const int inColWidth = InMatrix.GetNumCols();
        //    const int inChannelNum = inputChannelNum;//column as sample VS column as channel//inColWidth;
        //    const int inSampleNum = inColWidth;// //inRowHeight / inPatchSize ;
        //    const int filterPatchSize = FltWidth * FltHeight;
        //    const int outWidth = inputWidth + 2 * (FltWidth - 1); // - FltWidth + 1; // Filter Width Step = 1; with padding
        //    const int outHeight = inputHeight + 2 * (FltHeight -1);//inputHeight - FltHeight + 1; 
        //    const int outWidthFltNum = ceil( double(outWidth - FltWidth + 1) / FltStepW);
        //    const int outHeightFltNum = ceil( double(outHeight - FltHeight + 1) /FltStepH);
        //    //const int convNum = outWidth * outHeight;
        //    //auto& UnrolledMatrix=*this;

        //    const int unrolledRowNum = outHeightFltNum * outWidthFltNum * inChannelNum;//Number of Filters Per Sample//outHeightFltNum * outWidthFltNum;
        //    const int unrolledColNum = filterPatchSize * inSampleNum;//filterPatchSize * inChannelNum;
        //    if (UnrolledMatrix.IsEmpty())
        //        UnrolledMatrix = GPUMatrix<ElemType>::Zeros(unrolledRowNum, unrolledColNum);//UnrolledMatrix.ZeroInit();
        //    //UnrollMapping.SetValue(-1);
        //    long N = inRowHeight * inColWidth; //total number of threads
        //    int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        //    //CUDA_CALL(hipSetDevice(InMatrix.m_computeDevice));
        //    ElemType* d_unrolledMatrix;
        //    ElemType* d_unrollMapping;
        //    const int outArraySize = unrolledRowNum * unrolledColNum;
        //    UnrollMapping.ZeroInit();

        //    //GPUSparseMatrix<ElemType>UnrollMapping;// = ZeroInit();//GPUSparseMatrix(InMatrix.GetNumElements(), UnrolledMatrix.GetNumElements());

        //    //const int _debugSize = unrolledRowNum * unrolledColNum;

        //    //int* d_debugArray; 
        //    //CUDA_CALL(hipMalloc((void**)&d_debugArray, _debugSize * sizeof(int)));
        //    //CUDA_CALL(hipMemcpy(d_debugArray, debugMatrix, _debugSize *sizeof(int),hipMemcpyHostToDevice)); 



        //    if (FltStepW == 1 && FltStepH == 1)
        //        _unrollElem_noStride<ElemType><<<blocksPerGrid, threadsPerBlock>>>(
        //        UnrolledMatrix.BufferPointer(), InMatrix.BufferPointer(), UnrollMapping.m_pArray,
        //        inRowHeight, inColWidth, 
        //        inputWidth, inputHeight, inputChannelNum,
        //        FltWidth,FltHeight, FltChannel,
        //        inPatchSize, outWidthFltNum,outHeightFltNum,
        //        unrolledRowNum, unrolledColNum);
        //    else
        //    {
        //        _unrollElem_Stride<ElemType><<<blocksPerGrid, threadsPerBlock>>> (
        //            UnrolledMatrix.BufferPointer(), InMatrix.BufferPointer(), UnrollMapping.m_pArray,
        //            inRowHeight, inColWidth, 
        //            inputWidth, inputHeight, inputChannelNum,
        //            FltWidth,FltHeight, FltChannel,
        //            outWidthFltNum, outHeightFltNum,
        //            FltStepW,  FltStepH,
        //            unrolledRowNum, unrolledColNum);
        //    }           
        //    //CUDA_CALL(hipMemcpy(debugMatrix, d_debugArray, _debugSize *sizeof(int),hipMemcpyDeviceToHost)); 

        ////}
    }

#pragma endregion

#pragma region Helper Functions

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::performInplaceFunction(int kind)
    {        
        long N=(long)GetNZElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done;       
        CUDACALL(hipEventCreate(&done));        
        switch (kind)
        {
        case 0:
            _inplaceSigmoidOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,N);
            break;
        case 1:
            _inplaceTanhOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,N);
            break;
        case 2:
            _inplaceSqrtOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,N);
            break;
        case 3:
            _inplaceExpOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,N);
            break;
        case 4:
            _inplaceLogOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,N);
            break;
        case 5:
            _inplaceAbsOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,N);
            break;
        case 6:
            _inplaceLinRectDerivative<ElemType><<<blocksPerGrid,threadsPerBlock>>>(this->m_pArray,N);
        } 
        CUDACALL(hipEventRecord(done));        
        CUDACALL(hipEventSynchronize(done));        
    }

    template<class ElemType>
    void GPUSparseMatrix<ElemType>::SetMatrixFromCSRFormat(int *h_CSRRow, int *h_Col, ElemType *h_Val, size_t nz, size_t numRows, size_t numCols, bool IsOnDevice, int devId)
    {
        this->m_computeDevice = devId;
        this->m_elemSizeAllocated = this->m_nz = nz;
        this->m_numCols=numCols;
        this->m_numRows=numRows;  
        this->m_format=matrixFormatSparseCSR;
        this->m_externalBuffer = false;

        if (this->OwnBuffer() && this->m_pArray != NULL)
        {
            CUDACALL(hipFree(this->m_pArray));            
        }

        PrepareDevice();
        CUDACALL(hipMalloc((void **)&this->m_pArray,BufferSize()));

        hipMemcpyKind kind = IsOnDevice?hipMemcpyDeviceToDevice:hipMemcpyHostToDevice;
        CUDACALL(hipMemcpy(RowLocation(),h_CSRRow,RowSize(),kind));
        CUDACALL(hipMemcpy(ColLocation(),h_Col,ColSize(),kind));
        CUDACALL(hipMemcpy(NzLocation(),h_Val,NzSize(),kind));
    }

    // NOTE: we should change this to just use a single buffer, and return pointers into it
    template<class ElemType>
    void GPUSparseMatrix<ElemType>::GetMatrixFromCSRFormat(int*& h_CSRRow, int*& h_Col, ElemType*& h_Val, size_t &nz, size_t &numRows, size_t &numCols) const
    {
        if (h_CSRRow!=NULL || h_Col!=NULL || h_Val!=NULL)
            throw stdException("Passed pointers must be NULL");
        nz = this->GetNZElements();
        numRows = this->GetNumRows();
        numCols = this->GetNumCols();

        if (this->IsEmpty())
            return;
        else
        {
            PrepareDevice();
            h_Val = new ElemType[nz];
            h_CSRRow = new int[this->m_numRows + 1];
            h_Col = new int[nz];

            CUDACALL(hipMemcpy(h_CSRRow,RowLocation(),RowSize(),hipMemcpyDeviceToHost));
            CUDACALL(hipMemcpy(h_Col,   ColLocation(),ColSize(),hipMemcpyDeviceToHost));
            CUDACALL(hipMemcpy(h_Val,   NzLocation(), NzSize(), hipMemcpyDeviceToHost));
        }
    }

#pragma endregion Helper Functions

    template class GPUSparseMatrix<float>; 
    template class GPUSparseMatrix<double>;    

    template <class ElemType>
    MATH_API File& operator>>(File& stream, GPUSparseMatrix<ElemType>& us)
    {
        stream.GetMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
        size_t elsize;
        stream>>elsize;
        if (sizeof(ElemType)!=elsize)
            throw stdException("Template argument size doesn't match those in file");
        std::wstring matrixName;

        // save off the buffer size being passed in
        ElemType* deviceBuffer = us.m_pArray;
        size_t deviceBufferSize = us.BufferSize();

        // now prepare this header to receive the data being read
        // Once CPUSpareMatrix uses same format, should use that class
        size_t nz, colnum, rownum;
        int format;

        // read in the header information
        stream>>matrixName>>format>>nz>>colnum>>rownum;
        us.m_format = (MatrixFormat)format;
        us.m_numCols = colnum;
        us.m_numRows = rownum;
        us.m_elemSizeAllocated = us.m_nz = nz;
        us.m_externalBuffer = false;

        // temporarily allocate a CPU side array here (could use CPUSparseMatrix when has same format)
        ElemType* hostBuffer = new ElemType[us.BufferSize()];
        us.m_pArray = hostBuffer;
        ElemType *dVal=us.NzLocation();
        int* idx=us.IndexLocation();
        int* cidx=us.CompressedIndexLocation();
        size_t ncidx = us.CompressedIndexCount();

        // read in the sparse matrix info
        for (int i=0;i<nz;++i)
        {
            stream>>dVal[i];
        }
        for (int i=0;i<nz;++i)
        {
            stream>>idx[i];
        }
        for (int i=0;i<ncidx;++i)
        {
            stream>>cidx[i];
        }  

        // decide if we have enough room in the current buffer
        if (deviceBufferSize >= us.BufferSize())
        {
            us.m_elemSizeAllocated = us.ElemCountFromBufferSize(deviceBufferSize);
        }
        else
        {
            us.PrepareDevice();
            if (deviceBufferSize > 0)
                CUDACALL(hipFree((void **)&deviceBuffer));
            CUDACALL(hipMalloc((void **)&us.m_pArray, us.BufferSize()));
        }

        // copy over the different sections data
        CUDACALL(hipMemcpy(us.NzLocation(),dVal,us.NzSize(),hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(us.IndexLocation(),idx,us.IndexSize(),hipMemcpyHostToDevice));
        CUDACALL(hipMemcpy(us.CompressedIndexLocation(),cidx,us.CompressedIndexSize(),hipMemcpyHostToDevice));

        // copy over the name if necessary
        if (us.m_matrixName != NULL)
            delete us.m_matrixName;
        us.m_matrixName = new wchar_t[matrixName.length()+1];
        wmemcpy(us.m_matrixName,matrixName.c_str(),matrixName.length()+1);

        return stream;
    }

    template MATH_API File& operator>>(File& stream, GPUSparseMatrix<float>& us);
    template MATH_API File& operator>>(File& stream, GPUSparseMatrix<double>& us);

    template <class ElemType>
    MATH_API File& operator<<(File& stream, const GPUSparseMatrix<ElemType>& us)
    {
        stream.PutMarker(fileMarkerBeginSection, std::wstring(L"BMAT"));
        stream<<sizeof(ElemType);
        if (us.GetMatrixName()==NULL)
        {
            std::wstring s(L"nnmatrix");
            stream<<s;
        }
        else
        {
            stream<<us.GetMatrixName();
        }

        // What we would like to do here, is transfer to CPUSparse and save, do that when the format is the same
        byte* hostBuffer = new byte[us.BufferSize()];
        GPUSparseMatrix<ElemType> hostSide(us.GetNumRows(), us.GetNumCols(), us.NzCount(), (ElemType*)hostBuffer, us.GetFormat());
        CUDACALL(hipMemcpy(hostBuffer, us.NzLocation(),us.BufferSize(),hipMemcpyDeviceToHost));

        // now setup all the stuff pointing to the CPU side info
        const ElemType *dVal=hostSide.NzLocation();
        int* idx=hostSide.IndexLocation();
        int* cidx=hostSide.CompressedIndexLocation();
        size_t nz=us.NzCount();
        size_t ncidx=us.CompressedIndexCount();
        int format = us.GetFormat();
        stream<<format<<nz<<us.GetNumCols()<<us.GetNumRows();
        for (int i=0;i<nz;++i)
        {
            stream<<dVal[i];
        }
        for (int i=0;i<nz;++i)
        {
            stream<<idx[i];
        }
        for (int i=0;i<ncidx;++i)
        {
            stream<<cidx[i];
        }
        stream.PutMarker(fileMarkerEndSection, std::wstring(L"EMAT"));

        // now release the hostSide buffer
        delete hostBuffer;
        hostSide.m_pArray = NULL;

        return stream;
    }
    template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<float>& us);
    template MATH_API File& operator<<(File& stream, const GPUSparseMatrix<double>& us);

}}}
