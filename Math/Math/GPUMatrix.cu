#include "hip/hip_runtime.h"
//
// <copyright file="GPUMatrix.cu" company="Microsoft">
//     Copyright (c) Microsoft Corporation.  All rights reserved.
// </copyright>
//

#include "stdafx.h"
#include "BestGpu.h"

#ifndef CPUONLY

#include "hipblas.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include ""
#include "GPUMatrix.h"
#include "GPUMatrixCUDAKernels.cu"
#include "GPUSparseMatrix.h"
#include <iostream> // for cout

#pragma comment (lib, "cudart.lib")     // instruct linker to reference these libs
#pragma comment (lib, "cublas.lib")
#pragma comment (lib, "cusparse.lib")
#pragma comment (lib, "hiprand.lib")

#pragma warning (disable: 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning (disable: 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this
#pragma warning (disable: 4702) // unreachable code; triggered for unknown reasons

#ifdef NO_SYNC
bool do_sync = false;
#else
bool do_sync = true;
#endif

#define DEFAULT_THREAD_PER_DIM		16

#ifdef _WIN32
// thread local storage to access the current stream, initalize to default stream
__declspec (thread) 
#endif
hipStream_t t_stream = hipStreamDefault;

#define DEFAULT_THREAD_PER_DIM		16

extern int _ConvertSMVer2Cores(int major, int minor);   // forward declaration

// SetStream - set the stream that will be used by the GPU routines
void MATH_API SetStream(hipStream_t stream)
{
    t_stream = stream;
}

// GetStream - get the stream that will be used by the GPU routines
hipStream_t MATH_API GetStream()
{
    return t_stream;
}


void CURAND_CALL(hiprandStatus x)
{
    if(x!=HIPRAND_STATUS_SUCCESS) 
    { 
        throw std::runtime_error("CURAND fail");
    }        
}

void CUBLAS_CALL(hipblasStatus_t x)
{
    if(x!=HIPBLAS_STATUS_SUCCESS) 
    { 
        throw std::runtime_error("CUBLAS fail");
    }
}

void CUDA_CALL(hipError_t x) 
{
    if(x!=hipSuccess) 
    { 
        const char* errmsg = hipGetErrorString(x);
        std::cerr << "!!!!!!!!CUDA EXCEPTION: " << errmsg << std::endl;
        hipDeviceSynchronize();
        throw std::runtime_error(errmsg);
    }    
}

namespace Microsoft { namespace MSR { namespace CNTK {

    // PrepareDevice - Setup the correct cuda context for an operation
    // deviceId - the device on which the operation will take place
    void PrepareDevice(DEVICEID_TYPE deviceId)
    {
        static DEVICEID_TYPE currentDevice = AUTOPLACEMATRIX; // set to anything valid
        // externally managed matrices are guaranteed to be on the right device
        if (deviceId == MANAGEDEXTERN)
            return;
        // and if we last set the device to be this device we are good
        if (deviceId == currentDevice)
            return;
        CUDA_CALL(hipSetDevice(deviceId));
        currentDevice=deviceId;
    }

#pragma region DeviceBoundNumber class

    template<class ElemType>
    DeviceBoundNumber<ElemType>::DeviceBoundNumber(const DeviceBoundNumber<ElemType> &/*deepCopy*/)
    {
        NOT_IMPLEMENTED;
    }

    template<class ElemType>
    DeviceBoundNumber<ElemType>::DeviceBoundNumber(DeviceBoundNumber<ElemType> &&shallowCopy)
    {
        ShallowCopyFrom(shallowCopy.m_data,shallowCopy.m_computeDevice);
        shallowCopy.m_data=NULL;
    }

    template<class ElemType>
    void DeviceBoundNumber<ElemType>::ShallowCopyFrom(ElemType* newVal,int newValsDevceId)
    {
        m_computeDevice = newValsDevceId;
        m_data = newVal;
    }

    template<class ElemType>
    DeviceBoundNumber<ElemType>::~DeviceBoundNumber()
    {
        if (m_data!=NULL)
        {
            if (m_computeDevice<0)
            {
                delete m_data;
                m_data = NULL;
            }
            else if (m_computeDevice != MANAGEDEXTERN)
                CUDA_CALL(hipFree(m_data));
        }
    }

#pragma endregion DeviceBoundNumber class

#pragma region Helper functions
    template<class ElemType>    
    hipblasHandle_t _initCUBLAS(int devId)
    {
        PrepareDevice((DEVICEID_TYPE)devId);
        hipblasHandle_t cuHandle;
        CUBLAS_CALL(hipblasCreate(&cuHandle));
        return cuHandle;
    }

    // GetBestGPUDeviceId - Get the best GPU DeviceId, based on cuda information
    //  TODO: should be replaced by BestGpu class instead, it's much better
    template<class ElemType>
    DEVICEID_TYPE GPUMatrix<ElemType>::GetBestGPUDeviceId() //returns -1 if no GPUs can be used
    {      
        // currently there is little point in giving out different device IDs each time ask for a matrix, 
        // we really want them all on the same device eventually
        static int chosenDeviceId = AUTOPLACEMATRIX;
        if (chosenDeviceId != AUTOPLACEMATRIX)
            return chosenDeviceId;

        __try
        {
            // stash previous device state
            // if there was one on entry:
            int nPrevDev = -1;
            hipError_t ePrevDev = hipGetDevice(&nPrevDev);
 
            int deviceCount = -1;
            hipError_t error_id = hipGetDeviceCount(&deviceCount);
            if (error_id != hipSuccess || deviceCount==0) 
            { 
                return -1;            
            }

            int setDev = -1;
            int curDev=0;
            long curPower = 0;
            for (DEVICEID_TYPE dev = 0; dev < deviceCount; ++dev)
            {
                CUDA_CALL(hipSetDevice(dev));
                setDev = dev;
                hipDeviceProp_t deviceProp;
                hipGetDeviceProperties(&deviceProp, dev);
                long power = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor) * deviceProp.multiProcessorCount;
                //long power = _GetFreeMemoryOnCUDADevice(dev);
                if (power>curPower)
                {
                    curPower=power;
                    curDev = dev;
                }
            }

            if(nPrevDev >= 0 && ePrevDev == hipSuccess && 
                setDev >= 0 && setDev != nPrevDev) {
                // restore current context to the one we entered with
                // if there was one the caller might want unchanged.
                hipSetDevice(nPrevDev);
            }
            chosenDeviceId = curDev;
            return curDev;
        }
        __except (1)
        {
            return -1; // CPU
        }
    }

    // PrepareDevice - Setup the correct cuda context for an operation
    // deviceId - the device on which the operation will take place
    //            defaults to -1, which means use matrices current device
    template<class ElemType>
    DEVICEID_TYPE GPUMatrix<ElemType>::PrepareDevice(DEVICEID_TYPE deviceId /*=-1*/) const
    {
        // if default value use current compute device
        DEVICEID_TYPE newId = deviceId >= 0 ? deviceId : m_computeDevice;

        Microsoft::MSR::CNTK::PrepareDevice(newId);
        return newId;
    }

    template<class ElemType>
    ElemType* GPUMatrix<ElemType>::CopyToArray() const
    {
        size_t numElements = GetNumElements();
        if (numElements != 0)
        {
            PrepareDevice();
            ElemType* pArray = new ElemType[numElements];                    
            CUDA_CALL(hipMemcpy(pArray,m_pArray,sizeof(ElemType)*m_numRows*m_numCols,hipMemcpyDeviceToHost));
            return pArray;
        }
        else
        {
            return NULL;
        }
    }

    //memory will be allocated by the callee if not enough but need to be deleted by the caller after it's done
    //return number of elements copied
    template<class ElemType>
    size_t  GPUMatrix<ElemType>::CopyToArray(ElemType*& arrayCopyTo, size_t& currentArraySize) const
    {
        size_t numElements = GetNumElements();

        if (numElements > currentArraySize)
        {
            delete arrayCopyTo;
            arrayCopyTo = new ElemType[numElements];  
            currentArraySize = numElements;
        }

        if (numElements != 0)
        {
            PrepareDevice();
            CUDA_CALL(hipMemcpy(arrayCopyTo, m_pArray, sizeof(ElemType)*numElements, hipMemcpyDeviceToHost));
        }

        return numElements;
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::ChangeDeviceTo(DEVICEID_TYPE to_id)
    {
        if (!OwnBuffer())
            throw std::logic_error("Cannot change device on Managed external matrix");
        if (to_id == CPUDEVICE)
            throw std::logic_error("to_id must be valid GPU");
        if (m_computeDevice==to_id) 
            return;

        PrepareDevice((DEVICEID_TYPE)to_id);       
        ElemType* d_dst=NULL;
        CUDA_CALL(hipMalloc((void**)&d_dst,sizeof(ElemType)*m_numRows*m_numCols));

        m_elemSizeAllocated = m_numRows*m_numCols;

        // check to make sure we have something to copy (on init we often have zero sized allocations)
        if (m_elemSizeAllocated > 0)
        {
            // first try peer access
            int canAccessPeer = false;
            CUDA_CALL(hipDeviceCanAccessPeer(&canAccessPeer, to_id, m_computeDevice));
            if (canAccessPeer)
            {
                CUDA_CALL(hipDeviceEnablePeerAccess(m_computeDevice, 0));
                CUDA_CALL(hipMemcpyPeer(d_dst,to_id,m_pArray,m_computeDevice,sizeof(ElemType)*m_numRows*m_numCols));  
            }
            else
            {
                // peer access didn't work, just copy normal
                // make this more efficient by keeping some buffers available for each copy
                ElemType* h_dst=NULL;
                PrepareDevice();
                CUDA_CALL(hipHostMalloc((void**)&h_dst,sizeof(ElemType)*m_numRows*m_numCols));
                CUDA_CALL(hipMemcpy(h_dst,m_pArray,sizeof(ElemType)*m_numRows*m_numCols, hipMemcpyDeviceToHost));  
                PrepareDevice((DEVICEID_TYPE)to_id);       
                CUDA_CALL(hipMemcpy(d_dst,h_dst,sizeof(ElemType)*m_numRows*m_numCols, hipMemcpyHostToDevice)); 
                CUDA_CALL(hipHostFree(h_dst));  
            }
        }
        PrepareDevice();
        CUDA_CALL(hipFree(m_pArray));
        m_pArray=d_dst;

        PrepareDevice((DEVICEID_TYPE)to_id);       
        m_computeDevice=to_id;
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::performInplaceFunction(int kind)    
    {        
        PrepareDevice();
        LONG64 N= (LONG64) GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        switch (kind)
        {
        case 0:
            _inplaceSigmoidOnCuda<ElemType><<<blocksPerGrid, threadsPerBlock, 0, t_stream>>>(m_pArray, N);
            break;
        case 1:
            _inplaceTanhOnCuda<ElemType><<<blocksPerGrid, threadsPerBlock, 0, t_stream>>>(m_pArray, N);
            break;
        case 2:
            _inplaceSqrtOnCuda<ElemType><<<blocksPerGrid, threadsPerBlock, 0, t_stream>>>(m_pArray, N);
            break;
        case 3:
            _inplaceExpOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,N);
            break;
        case 4:
            _inplaceLogOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,N);
            break;
        case 5:
            _inplaceAbsOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,N);
            break;
        case 6:
            _inplaceLinRectDerivative<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,N);
            break;
        case 7:
            _inplaceCosineOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,N);
            break;
        case 8:
            _inplaceNegativeSineOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,N);
            break;
        } 
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));       
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }


#pragma endregion Helper functions

#pragma region Constructors and Destructor

   //should only be used by constructors.
    template<class ElemType>
    void GPUMatrix<ElemType>::ZeroInit(int deviceId)
    {
        m_computeDevice = deviceId;
        m_pArray = nullptr;
        m_numRows = 0;
        m_numCols = 0;
        m_elemSizeAllocated = 0;
        m_matrixName=NULL;
        m_format = matrixFormatDense; 
        m_externalBuffer = false;
    }

    template<class ElemType>
    GPUMatrix<ElemType>::GPUMatrix(int deviceId) 
    {
        if (deviceId == MANAGEDEXTERN)
            throw std::logic_error("Basic constructor cannot be used with Managed Extern types");

        ZeroInit(deviceId);
    };

    //matrixName is used to verify that correct matrix is read.
    template<class ElemType>
    GPUMatrix<ElemType>::GPUMatrix(FILE* f, const char * matrixName, int deviceId)
    {
        if (deviceId == MANAGEDEXTERN)
            throw std::logic_error("File constructor cannot be used with Managed Extern types");

        ReadFromFile(f, matrixName);
    }

    template<class ElemType>
    GPUMatrix<ElemType>::GPUMatrix(const size_t numRows, const size_t numCols,int deviceId)
    {
        if (deviceId == MANAGEDEXTERN)
            throw std::logic_error("constructor cannot be used with Managed Extern types");
        ZeroInit(deviceId);
        m_numRows = numRows;
        m_numCols = numCols;
        m_elemSizeAllocated = GetNumElements();

        if (m_elemSizeAllocated != 0)
        {
            PrepareDevice();        
            CUDA_CALL(hipMalloc((void**)&m_pArray,sizeof(ElemType)*m_elemSizeAllocated));      
        CUDA_CALL(hipMemset(m_pArray,0,sizeof(ElemType)*m_elemSizeAllocated));  
        }
    };

    template<class ElemType>
    GPUMatrix<ElemType>::GPUMatrix(const size_t numRows, const size_t numCols, ElemType *pArray, const size_t matrixFlags, int deviceId)
    {
        ZeroInit(deviceId);
        SetValue(numRows, numCols, pArray, matrixFlags, deviceId);
    };               

    template<class ElemType>
    GPUMatrix<ElemType>::GPUMatrix(const GPUMatrix<ElemType>& deepCopyFrom)
    {
        ZeroInit(deepCopyFrom.m_computeDevice);
        SetValue(deepCopyFrom);
        SetMatrixName(deepCopyFrom.m_matrixName);       
    }

    template<class ElemType>
    GPUMatrix<ElemType>::GPUMatrix(GPUMatrix<ElemType>&& moveFrom)
    {
        m_numRows = moveFrom.m_numRows;
        m_numCols = moveFrom.m_numCols;
        m_computeDevice = moveFrom.m_computeDevice;
        m_pArray = moveFrom.m_pArray;  //shallow copy the pointer       
        m_matrixName=moveFrom.m_matrixName;
        m_elemSizeAllocated = moveFrom.m_elemSizeAllocated;
        m_format = moveFrom.m_format;
        m_externalBuffer = moveFrom.m_externalBuffer;

        //release the pointer from the source object so that the destructor won't release it twice
        moveFrom.ZeroInit(0);       
    }

    //assignment operator, deep copy
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator=(const GPUMatrix<ElemType>& deepCopyFrom)  
    {
        if (this != &deepCopyFrom)
        {
            SetValue(deepCopyFrom);
            SetMatrixName(deepCopyFrom.m_matrixName);       
        }
        return *this;
    }

    //move assignment operator, shallow copy
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator=(GPUMatrix<ElemType>&& moveFrom)  
    {
        if (this != &moveFrom)
        {
            if (OwnBuffer() && m_pArray!=NULL)
            {
                CUDA_CALL(hipFree(m_pArray));  
            }

            m_numRows = moveFrom.m_numRows;
            m_numCols = moveFrom.m_numCols;
            m_elemSizeAllocated =  moveFrom.m_elemSizeAllocated;
            m_pArray = moveFrom.m_pArray;
            m_computeDevice = moveFrom.m_computeDevice;
            m_format = moveFrom.m_format;
            m_externalBuffer = moveFrom.m_externalBuffer;

            //release the pointer from the source object so that the destructor won't release it twice
            moveFrom.ZeroInit(0);
        }
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>::~GPUMatrix(void)
    {
        Clear();
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::Clear()
    {
        if (OwnBuffer() && m_pArray!=NULL)
        {
            if (m_computeDevice>=0)
            {            
                PrepareDevice();
                hipFree(m_pArray);
                m_pArray = NULL;
                m_elemSizeAllocated = 0;
            }        
        }
        BaseMatrix<ElemType>::Clear();

        ZeroInit(m_computeDevice);
    }
#pragma endregion Constructors and Destructor 

    template<class ElemType>
    int GPUMatrix<ElemType>::GetComputeDeviceId() const 
    {
        // for externally managed memory the CUDA context will have the current device
        if (m_computeDevice == MANAGEDEXTERN)
        {
            int devId;
            assert(m_externalBuffer);
            CUDA_CALL(hipGetDevice(&devId));
            return devId;
        }
        return m_computeDevice;
    }

#pragma region Basic Operators
    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::ColumnSlice(size_t startColumn, size_t numCols) const
    {
        if (numCols == 0)
            throw std::logic_error("The slice cannot have 0 columns.");

        if (startColumn + numCols > m_numCols)
            throw std::logic_error("The slice is out of range of the source matrix.");
            
        GPUMatrix<ElemType> slice(m_numRows, numCols, m_pArray + startColumn * m_numRows, matrixFlagDontOwnBuffer, m_computeDevice);

        return slice;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignColumnSlice(const GPUMatrix<ElemType>& fromMatrix, size_t startColumn, size_t numCols)
    {
        if (numCols == 0)
            throw std::logic_error("The slice cannot have 0 columns.");

        if (startColumn + numCols > m_numCols)
            throw std::logic_error("The slice is out of range of the source matrix.");
        
        Clear();

        m_computeDevice=fromMatrix.m_computeDevice;
        m_externalBuffer=true;
        m_numRows = fromMatrix.m_numRows;
        m_pArray=fromMatrix.m_pArray + startColumn * m_numRows;

        m_elemSizeAllocated = GetNumElements();
        m_matrixName=NULL;
        m_format = fromMatrix.m_format;

        return *this;
    }     


    //for each column of a, we assign numRows starting from startIndex to this
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignRowSliceValuesOf(const GPUMatrix<ElemType>& a, const size_t startIndex, const size_t numRows)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignRowSliceValuesOf: input matrix a is empty.");

        if (startIndex + numRows > a.GetNumRows())
            throw std::logic_error("AssignRowSliceValuesOf: startIndex + numRows exceeds a.GetNumRows().");

        Resize(numRows, a.GetNumCols());

        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _assignRowSliceValuesOf<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray, a.m_pArray, N, (long)startIndex, (long)numRows, (long)a.GetNumRows());
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    //for the row slice of this starting from startIndex we add a to it.
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddToRowSliceValuesOf(const GPUMatrix<ElemType>& a, const size_t startIndex, const size_t numRows)
    {
        if (a.IsEmpty())
            throw std::logic_error("AddToRowSliceValuesOf: input matrix a is empty.");

        if (a.GetNumRows() != numRows)
            throw std::logic_error("AddToRowSliceValuesOf: a.GetNumRows() != numRows.");

        if (startIndex + numRows > GetNumRows())
            throw std::logic_error("AddToRowSliceValuesOf: startIndex + numRows exceeds GetNumRows().");

        if (a.GetNumCols() != GetNumCols())
            throw std::logic_error("AddToRowSliceValuesOf: columns does not match.");

        LONG64 N=(LONG64)a.GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _addToRowSliceValuesOf<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray, a.m_pArray, N, (long)startIndex, (long)GetNumRows(), (long)a.GetNumRows());
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    //for each column of this, we add row slice of a starting from startIndex
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddWithRowSliceValuesOf(const GPUMatrix<ElemType>& a, const size_t startIndex, const size_t numRows)
    {
        if (a.IsEmpty())
            throw std::logic_error("AddWithRowSliceValuesOf: input matrix a is empty.");

        if (GetNumRows() != numRows)
            throw std::logic_error("AddWithRowSliceValuesOf: GetNumRows() != numRows.");

        if (startIndex + numRows > a.GetNumRows())
            throw std::logic_error("AddWithRowSliceValuesOf: startIndex + numRows exceeds a.GetNumRows().");

        if (a.GetNumCols() != GetNumCols())
            throw std::logic_error("AddWithRowSliceValuesOf: columns does not match.");

        LONG64 N = (LONG64)GetNumElements();
        int blocksPerGrid = (int)ceil(1.0*N / threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _addWithRowSliceValuesOf<ElemType> << <blocksPerGrid, threadsPerBlock, 0, t_stream >> >(m_pArray, a.m_pArray, N, (long)startIndex, (long)GetNumRows(), (long)a.GetNumRows());
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    /// c = c - 1.0 for a specific position
    template<class ElemType>
    void GPUMatrix<ElemType>::MinusOneAt(GPUMatrix<ElemType>& c, const size_t position)
    {
        assert(position < c.GetNumElements());

        hipEvent_t done = nullptr;
        LONG64 n = (LONG64)c.GetNumElements();
        LONG64 p = (LONG64)position;

        int blocksPerGrid = (int)ceil(1.0*n / threadsPerBlock);
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _minusOneAt<ElemType> << <blocksPerGrid, threadsPerBlock, 0, t_stream >> >(c.m_pArray, p, n);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    template<class ElemType>
    GPUMatrix<ElemType>&  GPUMatrix<ElemType>::AssignRepeatOf(const GPUMatrix<ElemType>& a, const size_t numRowRepeats, const size_t numColRepeats)
    {
        if (this == &a)
            throw std::logic_error("AssignRepeatOf: a is the same as [this]. Does not support inplace repeat.");

        if (a.IsEmpty())
            throw std::logic_error("AssignRepeatOf: Matrix a is empty.");

        Resize(a.GetNumRows() * numRowRepeats, a.GetNumCols() * numColRepeats);

        LONG64 N = (LONG64)GetNumElements();
        long n = (long)a.GetNumCols(), m = (long)a.GetNumRows();
        int blocksPerGrid = (int)ceil(1.0*N / threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _assignRepeatOf<ElemType> << <blocksPerGrid, threadsPerBlock, 0, t_stream >> >(m_pArray, a.m_pArray, N, m, n, (long)GetNumRows());
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }
        
    template<class ElemType>
    GPUMatrix<ElemType>&  GPUMatrix<ElemType>::AssignPositiveAndShiftedNegSample(const GPUMatrix<ElemType>& a, const size_t posNumber, const size_t negNumber, const size_t shiftNumber)
    {
        if (this == &a)
            throw std::logic_error("AssignPositiveAndShiftedNegSample: a is the same as [this]. Does not support inplace assignment.");

        if (a.IsEmpty())
            throw std::logic_error("AssignPositiveAndShiftedNegSample: Matrix a is empty.");

        Resize(a.GetNumRows() * (posNumber + negNumber), a.GetNumCols());

        LONG64 N = (LONG64)GetNumElements();
        long n = (long)a.GetNumCols(), m = (long)a.GetNumRows();
        int blocksPerGrid = (int)ceil(1.0*N / threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _assignPositiveAndShiftedNegSample<ElemType> << <blocksPerGrid, threadsPerBlock, 0, t_stream >> >(m_pArray, a.m_pArray, N, m, n, (long)GetNumRows(), posNumber, shiftNumber);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }
    
    template<class ElemType>
    GPUMatrix<ElemType>&  GPUMatrix<ElemType>::AddFoldedPositiveAndShiftedNegSample(const GPUMatrix<ElemType>& a, const size_t posNumber, const size_t negNumber, const size_t shiftNumber)
    {
        if (this == &a)
            throw std::logic_error("AddFoldedPositiveAndShiftedNegSample: a is the same as [this]. Does not support inplace assignment.");

        if (a.IsEmpty())
            throw std::logic_error("AddFoldedPositiveAndShiftedNegSample: Matrix a is empty.");

        if (a.GetNumRows() != GetNumRows() * (posNumber + negNumber) || a.GetNumCols() != GetNumCols())
            throw std::logic_error("AddFoldedPositiveAndShiftedNegSample: dimensions mismatch.");

        LONG64 N = (LONG64)a.GetNumElements();
        long n = (long)a.GetNumCols(), m = (long)a.GetNumRows();
        int blocksPerGrid = (int)ceil(1.0*N / threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _addFoldedPositiveAndShiftedNegSample<ElemType> << <blocksPerGrid, threadsPerBlock, 0, t_stream >> >(m_pArray, a.m_pArray, N, m, n, (long)GetNumRows(), posNumber, shiftNumber);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }


    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::Transpose() const
    {
        if (IsEmpty())
            throw std::logic_error("Transpose: Matrix is empty.");

        GPUMatrix<ElemType> c(GetComputeDeviceId());
        c.AssignTransposeOf(*this);
        return c;
    }

    // GetCublasHandle - get a cublas handle for the given GPU, should only need one per GPU
    // computeDevice - The compute device for which the cublas handle is desired
    // returns: cublas handle
    // NOTE: we currently don't bother to ever free the CUBLAS handle, it will be freed automatically by CUDA when the process ends
    template<class ElemType>
    hipblasHandle_t GPUMatrix<ElemType>::GetCublasHandle(int computeDevice/*=-1*/)
    {
        // if the compute device is not passed, get the current device from CUDA
        if (computeDevice < 0)
            hipGetDevice(&computeDevice);

        if (computeDevice < 0 || computeDevice >= MaxGpus)
            throw std::logic_error("GetCublasHandle: Maximum GPU exceeded");
        hipblasHandle_t cuHandle = s_cuHandle[computeDevice];
        if (cuHandle == NULL)
        {
            s_cuHandle[computeDevice] = cuHandle = _initCUBLAS<ElemType>(computeDevice);
        }
        CUBLAS_CALL(hipblasSetStream(cuHandle, t_stream));

        return cuHandle;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignTransposeOf (const GPUMatrix<ElemType>& a)
    {
        if (this == &a)
            throw std::logic_error("AssignTransposeOf: a is the same as [this]. Does not support inplace transpose.");

        if (a.IsEmpty())
            throw std::logic_error("AssignTransposeOf: Matrix a is empty.");

        if (GetNumRows()!=a.GetNumCols() || GetNumCols()!=a.GetNumRows())
            Resize(a.GetNumCols(), a.GetNumRows());

        hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
        hipblasOperation_t transA =  HIPBLAS_OP_T;
        hipblasOperation_t transB =  HIPBLAS_OP_T;
        int m = (int)a.m_numCols;
        int n = (int)a.m_numRows;                
        ElemType alpha=1;
        ElemType beta=0;
        hipblasStatus_t st;
        if (sizeof(ElemType)==sizeof(float))
        {
            st = hipblasSgeam(cuHandle,transA,transB,m,n,reinterpret_cast<float*>(&alpha),reinterpret_cast<float*>(a.m_pArray),(int)a.m_numRows,reinterpret_cast<float*>(&beta),reinterpret_cast<float*>(a.m_pArray),(int)a.m_numRows,reinterpret_cast<float*>(m_pArray),(int)m_numRows);
        }
        else if (sizeof(ElemType)==sizeof(double))
        {            
            st = hipblasDgeam(cuHandle,transA,transB,m,n,reinterpret_cast<double*>(&alpha),reinterpret_cast<double*>(a.m_pArray),(int)a.m_numRows,reinterpret_cast<double*>(&beta),reinterpret_cast<double*>(a.m_pArray),(int)a.m_numRows,reinterpret_cast<double*>(m_pArray),(int)m_numRows);
        }
        else  
        {
            throw std::runtime_error("Unsupported template argument in GPUMatrix"); 
        }
        if (st!=HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error("AssignTransposeOf failed");     
        }
        m_numRows=a.m_numCols;
        m_numCols=a.m_numRows;
        SetMatrixName(a.GetMatrixName());
        return *this;
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::SetValue(const ElemType v)
    {
        if (IsEmpty())
            throw std::logic_error("SetValue: Matrix is empty.");

        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _setValue<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,v,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::SetValue(const ElemType* d_v) //d_v is pointer to the the value in GPU memory
    {
        if (IsEmpty())
            throw std::logic_error("SetValue: Matrix is empty.");

        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _setValue<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,d_v,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done)); 
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::SetColumn(const ElemType* colPointer, size_t colInd)
    {
        if (IsEmpty())
            throw std::logic_error("SetValue: Matrix is empty.");
        if (colPointer==NULL)
            return;
        CUDA_CALL(hipMemcpy(m_pArray+LocateColumn(colInd),colPointer,sizeof(ElemType)*m_numRows,hipMemcpyHostToDevice));
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::SetValue(const GPUMatrix<ElemType>& deepCopyFrom)
    {
        if (this == &deepCopyFrom)
            return;

        Resize(deepCopyFrom.GetNumRows(), deepCopyFrom.GetNumCols());
        m_format = deepCopyFrom.m_format; // copy the format over just to be sure
        size_t cpSize = deepCopyFrom.GetNumRows() * deepCopyFrom.GetNumCols();
        if (cpSize != 0)
            CUDA_CALL(hipMemcpy(m_pArray,deepCopyFrom.m_pArray,cpSize*sizeof(ElemType),hipMemcpyDeviceToDevice));        
    }

    template<class ElemType>    
    void GPUMatrix<ElemType>::SetValue(const size_t numRows, const size_t numCols, ElemType *pArray, size_t matrixFlags, int deviceId)
    {
        // handle externally managed case
        if (matrixFlags&matrixFlagDontOwnBuffer)
        {
            // free the existing array if it used to be an owned array
            if (OwnBuffer() && m_pArray!=NULL)
            {
                PrepareDevice();
                CUDA_CALL(hipFree(m_pArray));
            }
            m_numRows = numRows;
            m_numCols = numCols;
            m_pArray = pArray;
            m_elemSizeAllocated = GetNumElements();
            m_matrixName = NULL;
            m_format = matrixFormatDense;
            m_externalBuffer = true;
            m_computeDevice = deviceId;
        }
        else 
        {
            // if didn't previously own the buffer, wipe it clean 
            if (!OwnBuffer())
            {
                ZeroInit(deviceId);
            }

            // if the devices are different move it now
            if (m_computeDevice != deviceId && deviceId >= 0)
            {
                Clear();
                ZeroInit(deviceId);
            }

            // now resize/allocate as necessary
            Resize(numRows, numCols);
            m_externalBuffer = false;

            // copy over the content to the buffer
            PrepareDevice();
            if (pArray!=NULL) 
            {
                if (!(matrixFlags&matrixFormatRowMajor))
                {
                    CUDA_CALL(hipMemcpy(m_pArray, pArray, sizeof(ElemType)*GetNumElements(), 
                        (matrixFlags&matrixFlagSetValueOnDevice)?hipMemcpyDeviceToDevice:hipMemcpyHostToDevice));
                }
                else
                {
                    throw std::runtime_error("Row major isn't implemented");
                }
            }
        }
        m_format = matrixFormatDense;
    }


    template<class ElemType>
    void GPUMatrix<ElemType>::SetDiagonalValue(const ElemType v)
    {
        unsigned long N=(unsigned long)GetNumRows();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _setDiagonalValue<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,v,N,(unsigned long)GetNumRows());
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::SetDiagonalValue(GPUMatrix<ElemType>& vector)
    {
        if (IsEmpty() || vector.IsEmpty())
            throw std::logic_error("SetDiagonalValue: Matrix is empty.");

        if (GetNumRows() != GetNumCols())
            throw std::logic_error("SetDiagonalValue: NumRows and NumCols do not agree.");

        if (vector.GetNumRows() != 1 && vector.GetNumCols() != 1)
            throw std::logic_error("SetDiagonalValue: input vector must be a vector.");

        if (vector.GetNumElements() == 1) //reduce to simple form
            SetDiagonalValue(vector.m_pArray[0]);

        else if (vector.GetNumRows() != GetNumRows())
            throw std::logic_error("SetDiagonalValue: input vector's dimension does not agree with [this].");
        else
        {
            long N=(long)GetNumRows();
            int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
            PrepareDevice();
            hipEvent_t done = nullptr;
            if (do_sync)    CUDA_CALL(hipEventCreate(&done));
            _setDiagonalValueFromVector<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,vector.m_pArray,N);
            if (do_sync)    CUDA_CALL(hipEventRecord(done));        
            if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
            if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        }
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::SetUniformRandomValue(const ElemType low, const ElemType high, unsigned long seed)
    {
        PrepareDevice();
        if (s_curandGenerator==NULL)
        {            
            s_curandGenerator = new hiprandGenerator_t;
            /* Create pseudo-random number generator */        
            CURAND_CALL(hiprandCreateGenerator(&(((hiprandGenerator_t*)s_curandGenerator)[0]),HIPRAND_RNG_PSEUDO_XORWOW));        
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(((hiprandGenerator_t*)s_curandGenerator)[0], seed==USE_TIME_BASED_SEED ? time(NULL) : seed));       
            CURAND_CALL(hiprandSetGeneratorOrdering(((hiprandGenerator_t*)s_curandGenerator)[0],HIPRAND_ORDERING_PSEUDO_SEEDED));
        }

        hipEvent_t done = nullptr;
        CUDA_CALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CURAND_CALL(hiprandGenerateUniform(((hiprandGenerator_t*)s_curandGenerator)[0], reinterpret_cast<float*>(m_pArray), GetNumElements()));
        }
        else
        {
            CURAND_CALL(hiprandGenerateUniformDouble(((hiprandGenerator_t*)s_curandGenerator)[0], reinterpret_cast<double*>(m_pArray), GetNumElements()));
        }
        CUDA_CALL(hipEventRecord(done));        
        CUDA_CALL(hipEventSynchronize(done)); 
        //CURAND_CALL(hiprandDestroyGenerator(gen));
        CUDA_CALL(hipEventDestroy(done));

        size_t N=GetNumElements();
        size_t blocksPerGrid = (size_t)ceil(N/(double)threadsPerBlock);

        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _rescaleToRange<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,N,low,high);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::SetGaussianRandomValue(const ElemType mean, const ElemType sigma, unsigned long seed)
    {
        PrepareDevice();
        if (s_curandGenerator==NULL)
        {            
            s_curandGenerator = new hiprandGenerator_t;
            /* Create pseudo-random number generator */        
            CURAND_CALL(hiprandCreateGenerator(&(((hiprandGenerator_t*)s_curandGenerator)[0]),HIPRAND_RNG_PSEUDO_XORWOW));        
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(((hiprandGenerator_t*)s_curandGenerator)[0], seed==USE_TIME_BASED_SEED ? time(NULL) : seed));       
            CURAND_CALL(hiprandSetGeneratorOrdering(((hiprandGenerator_t*)s_curandGenerator)[0],HIPRAND_ORDERING_PSEUDO_SEEDED));
        }

        if (sizeof(ElemType)==sizeof(float))
        {
            CURAND_CALL(hiprandGenerateNormal(((hiprandGenerator_t*)s_curandGenerator)[0], reinterpret_cast<float*>(m_pArray), GetNumElements(), (float)mean, (float)sigma));
        }
        else
        {
            CURAND_CALL(hiprandGenerateNormalDouble(((hiprandGenerator_t*)s_curandGenerator)[0], reinterpret_cast<double*>(m_pArray), GetNumElements(), (double)mean, (double)sigma));
        }
        //CURAND_CALL(hiprandDestroyGenerator(gen));
    }

    //maskRate: percentage of values masked out (similar to dropout rate)
    //scaleValue: which scale value to set to the left ones (unmasked items).
    template<class ElemType>
    void GPUMatrix<ElemType>::SetUniformRandomMask(const ElemType maskRate, const ElemType scaleValue, unsigned long seed)
    {
        PrepareDevice();
        if (s_curandGenerator==NULL)
        {            
            s_curandGenerator = new hiprandGenerator_t;
            /* Create pseudo-random number generator */        
            CURAND_CALL(hiprandCreateGenerator(&(((hiprandGenerator_t*)s_curandGenerator)[0]),HIPRAND_RNG_PSEUDO_XORWOW));        
            CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(((hiprandGenerator_t*)s_curandGenerator)[0], seed==USE_TIME_BASED_SEED ? time(NULL) : seed));       
            CURAND_CALL(hiprandSetGeneratorOrdering(((hiprandGenerator_t*)s_curandGenerator)[0],HIPRAND_ORDERING_PSEUDO_SEEDED));
        }

        hipEvent_t done = nullptr;
        CUDA_CALL(hipEventCreate(&done));
        if (sizeof(ElemType)==sizeof(float))
        {
            CURAND_CALL(hiprandGenerateUniform((((hiprandGenerator_t*)s_curandGenerator)[0]), reinterpret_cast<float*>(m_pArray), GetNumElements()));
        }
        else
        {
            CURAND_CALL(hiprandGenerateUniformDouble((((hiprandGenerator_t*)s_curandGenerator)[0]), reinterpret_cast<double*>(m_pArray), GetNumElements()));
        }
        CUDA_CALL(hipEventRecord(done));        
        CUDA_CALL(hipEventSynchronize(done)); 
        CUDA_CALL(hipEventDestroy(done));
        //CURAND_CALL(hiprandDestroyGenerator(gen));

        size_t N=GetNumElements();
        size_t blocksPerGrid = (size_t)ceil(N/(double)threadsPerBlock);        
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _setMaskAndScale<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,N,maskRate,scaleValue);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::Adagrad(GPUMatrix<ElemType>& gradients, const bool needAveMultiplier)
    {
        size_t numColsNeeded = gradients.GetNumCols();
        if (needAveMultiplier)
            numColsNeeded += gradients.GetNumCols();

        if (IsEmpty() || GetNumCols() < numColsNeeded)
        {
            Resize(gradients.GetNumRows(), numColsNeeded);
            SetValue(0.0);
        }

        assert(GetNumRows() == gradients.GetNumRows() && GetNumCols() == numColsNeeded);

        size_t n = gradients.GetNumElements();

        ElemType *multipliers = nullptr;
        if (needAveMultiplier)
            multipliers = m_pArray + n; // temp memory used to store multipliers,

        int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
        _adagrad<ElemType> << <blocksPerGrid, threadsPerBlock >> >(m_pArray, gradients.m_pArray, GetNumElements(), multipliers);

        if (!needAveMultiplier)
            return 1;

        hipblasHandle_t cuHandle = GetCublasHandle(GetComputeDeviceId());
        if (sizeof(ElemType) == sizeof(float))
        {
            float aveMultiplier = 0;
            CUBLAS_CALL(hipblasSasum(cuHandle, (LONG64)n, reinterpret_cast<float*>(multipliers), 1, &aveMultiplier));
            return aveMultiplier / n;
        }
        else
        {
            double aveMultiplier = 0;
            CUBLAS_CALL(hipblasDasum(cuHandle, (LONG64)n, reinterpret_cast<double*>(multipliers), 1, &aveMultiplier));
            return aveMultiplier / n;
        }
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::RmsProp(GPUMatrix<ElemType>& gradients,
        ElemType RMS_GAMMA,
        ElemType RMS_WGT_INC,
        ElemType RMS_WGT_MAX,
        ElemType RMS_WGT_DEC,
        ElemType RMS_WGT_MIN,
        const bool needAveMultiplier
        )
    {
        const ElemType floor = 1e-6f;
        static ElemType *upd_gpu = (ElemType*)0;

        size_t n = gradients.GetNumElements();
        int blocksPerGrid = (GetNumElements() + threadsPerBlock -1 )/threadsPerBlock;

        size_t numColsNeeded = gradients.GetNumCols() * 3;
        if (needAveMultiplier)
            numColsNeeded += gradients.GetNumCols();

        if (IsEmpty() || GetNumCols() < numColsNeeded)
        {
            Resize(gradients.GetNumRows(), numColsNeeded);
            SetValue(0.0);

            ElemType *avars=m_pArray; // accumulated variances for RMS scaling
            ElemType *signs=m_pArray+n; // sign of previous gradient
            ElemType *steps=m_pArray+2*n; // current step size
            //m_pArray+3*n is temp memory used to store multipliers, no need to initialize

            _rmsprop_init<ElemType><<<blocksPerGrid, threadsPerBlock>>>(avars,signs,steps,gradients.m_pArray,n);

        }
        assert(GetNumRows() == gradients.GetNumRows() && GetNumCols() == numColsNeeded);

        ElemType *avars=m_pArray; // accumulated variances for RMS scaling
        ElemType *signs=m_pArray+n; // sign of previous gradient
        ElemType *steps=m_pArray+2*n; // current step size

        ElemType *multipliers = nullptr;
        if (needAveMultiplier)
            multipliers = m_pArray + 3 * n; // temp memory used to store multipliers,

        if( !upd_gpu )
        {
            ElemType upd[] = {
                2,2,0,
                2,2,0,
                1,1,1,
                2,2,0,
                1,2,1,
                0,2,2,
                1,1,1,
                0,2,2,
                0,2,2,
            };

            CUDA_CALL(hipMalloc((void**)&upd_gpu,sizeof(ElemType)*27));
            CUDA_CALL(hipMemcpy(upd_gpu,upd,sizeof(ElemType)*27,hipMemcpyHostToDevice));
        }

        _rmsprop<ElemType> << <blocksPerGrid, threadsPerBlock >> >(avars, signs, steps, gradients.m_pArray, n,
            RMS_GAMMA, RMS_WGT_INC, RMS_WGT_MAX, RMS_WGT_DEC, RMS_WGT_MIN,
            floor, upd_gpu, multipliers);

        if (!needAveMultiplier)
            return 1;

        hipblasHandle_t cuHandle = GetCublasHandle(GetComputeDeviceId());
        if (sizeof(ElemType) == sizeof(float))
        {
            float aveMultiplier = 0;
            CUBLAS_CALL(hipblasSasum(cuHandle, (LONG64)n, reinterpret_cast<float*>(multipliers), 1, &aveMultiplier));
            return aveMultiplier / n;
        }
        else
        {
            double aveMultiplier = 0;
            CUBLAS_CALL(hipblasDasum(cuHandle, (LONG64)n, reinterpret_cast<double*>(multipliers), 1, &aveMultiplier));
            return aveMultiplier / n;
        }
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::Reshape(const size_t numRows, const size_t numCols)
    {
        assert (numRows*numCols == GetNumElements());
        if (numRows*numCols != GetNumElements())
            throw std::invalid_argument("Reshape: total number of elements does not match.");

        m_numRows = numRows;
        m_numCols = numCols;
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::Resize(const size_t numRows, const size_t numCols, bool growOnly)
    {
        if (m_numRows==numRows && m_numCols==numCols)
            return;   

        m_numRows = numRows;
        m_numCols = numCols;

        size_t numElements = GetNumElements();
        if (numElements > m_elemSizeAllocated || (!growOnly && numElements != m_elemSizeAllocated))
        {
            if (IsEmpty())
            {
                m_elemSizeAllocated = 0;
                m_pArray = NULL;
            }
            else
            {            
                if (!OwnBuffer())
                    throw std::invalid_argument("Can't resize a externally managed matrix");
                PrepareDevice();
                if (m_pArray!=NULL)
                    CUDA_CALL(hipFree(m_pArray)); //delete and reallocate                            
                m_elemSizeAllocated = numElements;
                CUDA_CALL(hipMalloc((void**)&m_pArray,sizeof(ElemType)*m_elemSizeAllocated));
                CUDA_CALL(hipMemset(m_pArray,0,sizeof(ElemType)*m_elemSizeAllocated));
            }
        }
    }

    template<class ElemType>
    size_t GPUMatrix<ElemType>::LocateElement (const size_t row, const size_t col) const 
    { 
        assert (row < m_numRows && col < m_numCols); 
        return col * m_numRows  + row;  // matrix in column-wise storage
    }  

    template<class ElemType>
    size_t GPUMatrix<ElemType>::LocateColumn (const size_t col) const 
    { 
        assert (col < m_numCols); 
        return col * m_numRows;  // matrix in column-wise storage
    }  

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::Get00Element() const 
    {        
        ElemType res=0;        
        CUDA_CALL(hipMemcpy(&res,m_pArray,sizeof(ElemType),hipMemcpyDeviceToHost));
        return res;
    }
#pragma endregion Basic Operators

#pragma region Member BLAS Functions
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator+= (ElemType alpha) 
    {
        if (IsEmpty())
            throw std::logic_error("operator+=: Matrix is empty.");
        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _addValue<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,alpha,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::operator+ (ElemType alpha) const
    {
        if (IsEmpty())
            throw std::logic_error("operator+: Matrix is empty.");

        const GPUMatrix<ElemType>& us=*this;
        GPUMatrix<ElemType> c(us);
        c+=alpha;
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSumOf(const ElemType alpha, const GPUMatrix<ElemType>& a)
    {
        SetValue(a);
        (*this)+=alpha;
        return (*this);
    }


    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator+= (const GPUMatrix<ElemType>& a) 
    {
        //if (a.GetNumElements()==1)
        //{
        //    //*this += a.Get00Element();
        //    LONG64 N=(LONG64)GetNumElements();
        //    int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        //    hipEvent_t done = nullptr;
        //    if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        //    _addValue<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,N);
        //    if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        //    if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        //    if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        //}
        //else 
        //{
            ScaleAndAdd(1, a, *this);
        //}
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::operator+ (const GPUMatrix<ElemType>& a) const
    {
        if (GetNumElements()==1)
        {
            GPUMatrix<ElemType> c(a);
            c+=Get00Element();
            return c;
        }
        else if (a.GetNumElements()==1)
        {
            GPUMatrix<ElemType> c(*this);
            c+=a.Get00Element();
            return c;
        }
        else
        {
            GPUMatrix<ElemType> c(*this); //this implementation will introduce a copy overhead. but make resue of the code
            c += a;
            return c;
        }
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSumOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        SetValue(a);
        (*this)+=b;
        return (*this);
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator-= (ElemType alpha) 
    {
        if (IsEmpty())
            throw std::logic_error("operato-=: Matrix is empty.");
        return operator+=(-1*alpha);        
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::operator- (ElemType alpha) const
    {
        if (IsEmpty())
            throw std::logic_error("operator-: Matrix is empty.");
        return operator+(-1*alpha);
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignDifferenceOf(const ElemType alpha, const GPUMatrix<ElemType>& a)
    {
        Resize(a.m_numRows,a.m_numCols);
        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _assignDifferenceOf1<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,alpha,a.m_pArray,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
        /*Resize(a.m_numRows,a.m_numCols);
        SetValue(alpha);
        (*this)-=a;
        return *this;*/
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignDifferenceOf(const GPUMatrix<ElemType>& a, const ElemType alpha)
    {
        Resize(a.m_numRows,a.m_numCols);
        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _assignDifferenceOf2<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,alpha,a.m_pArray,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
        /*SetValue(a);
        (*this)-=alpha;
        return *this;*/
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator-= (const GPUMatrix<ElemType>& a)
    {
        //if (a.GetNumElements() == 1)
        //    AssignDifferenceOf(*this, a.Get00Element());
        //else if (GetNumElements() == 1)
        //    AssignDifferenceOf(Get00Element(), a);
        //else
            ScaleAndAdd(-1, a, *this);

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::operator- (const GPUMatrix<ElemType>& a) const
    {
        GPUMatrix<ElemType> c(*this); //this implementation will introduce a copy overhead. but make resue of the code
        c -= a;
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignDifferenceOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (this != &a)
        {
            Resize(a.GetNumRows(), a.GetNumCols());
            SetValue(a);
        }
        (*this) -= b;
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator*= (ElemType alpha)
    {
        Scale(alpha, *this);
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::operator* (ElemType alpha) const
    {
        GPUMatrix<ElemType> c(GetNumRows(), GetNumCols());
        Scale(alpha, *this, c);
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignProductOf(const ElemType alpha, const GPUMatrix<ElemType>& a)
    {
        Scale(alpha, a, *this);
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignProductOf (const GPUMatrix<ElemType>& a, const bool transposeA, const GPUMatrix<ElemType>& b, const bool transposeB)
    {
        if (a.GetNumElements() == 1)
        {  
            if (transposeB)
                AssignTransposeOf(b);
            (*this) *= a.Get00Element();
        }
        else if (b.GetNumElements() == 1)
        { 
            if (transposeA)
                AssignTransposeOf(a);
            (*this) *= b.Get00Element();
        }
        else
            Multiply(a, transposeA, b, transposeB, *this);
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::operator* (const GPUMatrix<ElemType>& a) const
    {
        const GPUMatrix<ElemType>& us = *this;
        if (GetNumElements() == 1)
        {
            GPUMatrix<ElemType> c(GetComputeDeviceId());
            c.AssignProductOf(Get00Element(), a);
            return c;
        }
        else if (a.GetNumElements() == 1)
        {
            GPUMatrix<ElemType> c(GetComputeDeviceId());
            c.AssignProductOf(a.Get00Element(), us);
            return c;
        }
        else
        {
            GPUMatrix<ElemType> c(GetNumRows(),a.GetNumCols(),GetComputeDeviceId());
            Multiply(*this, a, c);
            return c;
        }
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator/= (ElemType alpha)
    {
        (*this) *= 1/alpha;
        return (*this);
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::operator/ (ElemType alpha) const
    {
        return ((*this) * (1/alpha));
    }

    //element-wise power
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::operator^= (ElemType alpha)
    {
        GPUMatrix<ElemType>& us = *this;
        ElementWisePower(alpha, us, us);
        return us;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::operator^ (ElemType alpha) const
    {
        GPUMatrix<ElemType> c(GetNumRows(), GetNumCols());
        ElementWisePower(alpha, *this, c);
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementPowerOf(const GPUMatrix<ElemType>& a, const ElemType power)
    {
        ElementWisePower(power, a, *this);
        return *this;
    }


    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddElementProductOf (const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.IsEmpty() || b.IsEmpty())
            throw std::logic_error("AddElementProductOf: Matrix is empty.");

        assert (a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());
        if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
            throw std::invalid_argument("The input matrix dimensions do not match.");

        if (!(a.GetNumRows() == GetNumRows() && a.GetNumCols() == GetNumCols()))
            throw std::invalid_argument("The input matrix dimensions do not match [this].");

        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);    
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _addElementProductOf<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,b.m_pArray,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));      
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::ColumnElementMultiplyWith(const GPUMatrix<ElemType>& a)
    {
        if (a.IsEmpty() || IsEmpty())
            throw std::logic_error("ColumnElementMultiplyWith: Matrix is empty.");

        if (!(a.GetNumRows() == GetNumRows() && a.GetNumCols() == 1))
            throw std::invalid_argument("ColumnElementMultiplyWith: The input matrix should be a col vector and match [this]'s rows.");

        long N=(long)a.GetNumRows();
        long M=(long)GetNumCols();        
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);  
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _columnElementMultiplyWith<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,N,M);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));      
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::RowElementMultiplyWith(const GPUMatrix<ElemType>& a)
    {
        if (a.IsEmpty() || IsEmpty())
            throw std::logic_error("RowElementMultiplyWith: Matrix is empty.");

        if (!(a.GetNumRows() == 1 && a.GetNumCols() == GetNumCols()))
            throw std::invalid_argument("RowElementMultiplyWith: The input matrix should be a row vector and match [this]'s columns.");

        long N = (long)GetNumRows();
        long M = (long)a.GetNumCols();
        int blocksPerGrid = (int)ceil(1.0*M / threadsPerBlock);
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _rowElementMultiplyWith<ElemType><<<blocksPerGrid,threadsPerBlock>>>(m_pArray,a.m_pArray,N,M);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));      
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::RowElementDivideBy(const GPUMatrix<ElemType>& a)
    {
        if (a.IsEmpty() || IsEmpty())
            throw std::logic_error("RowElementDivideBy: Matrix is empty.");

        if (!(a.GetNumRows() == 1 && a.GetNumCols() == GetNumCols()))
            throw std::invalid_argument("RowElementDivideBy: The input matrix should be a row vector and match [this]'s columns.");

        long N = (long)GetNumRows();
        long M = (long)a.GetNumCols();
        int blocksPerGrid = (int)ceil(1.0*M / threadsPerBlock);
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _rowElementDivideBy<ElemType> << <blocksPerGrid, threadsPerBlock >> >(m_pArray, a.m_pArray, N, M);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::ColumnElementDivideBy(const GPUMatrix<ElemType>& a)
    {
        if (a.IsEmpty() || IsEmpty())
            throw std::logic_error("ColumnElementDivideBy: Matrix is empty.");

        if (!(a.GetNumRows() == GetNumRows() && a.GetNumCols() == 1))
            throw std::invalid_argument("ColumnElementDivideBy: The input matrix should be a col vector and match [this]'s rows.");

        long N = (long)a.GetNumRows();
        long M = (long)GetNumCols();
        int blocksPerGrid = (int)ceil(1.0*N / threadsPerBlock);
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _ColumnElementDivideBy<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,N,M);                        
        if (do_sync)    CUDA_CALL(hipEventRecord(done));      
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::ElementInverse ()
    {
        if (IsEmpty())
            throw std::logic_error("ElementInverse: Matrix is empty.");

        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);  
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _elemInverse<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));     
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementInverseOf (const GPUMatrix<ElemType>& a)
    {
        SetValue(a);
        return ElementInverse();
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceSigmoid()
    {
        performInplaceFunction(0);                    
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSigmoidOf (const GPUMatrix<ElemType>& a)
    {
        Resize(a.GetNumRows(),a.GetNumCols());
        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _assignSigmoidOf<<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(a.m_pArray,m_pArray,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        /*SetValue(a);
        InplaceSigmoid();*/
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceSigmoidDerivative()
    {
        AssignSigmoidDerivativeOf(*this);                    
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSigmoidDerivativeOf (const GPUMatrix<ElemType>& a)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignSigmoidDerivativeOf: Matrix a is empty.");

        //auto& us=*this;
        if (this != &a)
            Resize(a.GetNumRows(), a.GetNumCols());

        PrepareDevice();
        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        

        _assignSigmoidDerivative<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(a.m_pArray, m_pArray, N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }


    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceTanh()
    {
        performInplaceFunction(1);
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignTanhOf (const GPUMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceTanh();
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceLogSoftmax (const bool isColWise)
    {
        if (IsEmpty())
            throw std::logic_error("InplaceLogSoftmax: Matrix is empty.");

        PrepareDevice();
        if (isColWise)
        {
            long N=(long)GetNumCols(); //one kernel per column
            int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);             
            hipEvent_t done = nullptr;
            if (do_sync)    CUDA_CALL(hipEventCreate(&done));
            _logSoftMaxColWise<<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,(long)m_numCols,(long)m_numRows);
            if (do_sync)    CUDA_CALL(hipEventRecord(done));        
            if (do_sync)    CUDA_CALL(hipEventSynchronize(done));  
            if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        }
        else
        {
            long N=(long)GetNumRows(); //one kernel per column
            int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);                
            hipEvent_t done = nullptr;
            if (do_sync)    CUDA_CALL(hipEventCreate(&done));
            _logSoftMaxRowWise<<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,(long)m_numCols,(long)m_numRows);
            if (do_sync)    CUDA_CALL(hipEventRecord(done));        
            if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
            if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        }
        return *this; 
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignLogSoftmaxOf (const GPUMatrix<ElemType>& a, const bool isColWise)
    {
        Resize(a.GetNumRows(),a.GetNumCols());        
        if (isColWise)
        {            
            PrepareDevice();
            long N = (long)GetNumCols();
            long M = (long)GetNumRows();
            hipEvent_t done = nullptr;
            if (do_sync)    CUDA_CALL(hipEventCreate(&done));
            _assignColumnwiseLogSoftmaxOf<<<N,512,0,t_stream>>>(a.m_pArray,m_pArray,N,M);
            if (do_sync)    CUDA_CALL(hipEventRecord(done));        
            if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
            if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        }
        else
        {
            NOT_IMPLEMENTED;
        }

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceSqrt()
    {
        performInplaceFunction(2);        
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSqrtOf (const GPUMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceSqrt();
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceExp()
    {
        performInplaceFunction(3);        
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignExpOf (const GPUMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceExp();
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceLog()
    {
        performInplaceFunction(4);        
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignLogOf (const GPUMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceLog();
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceAbs()
    {
        performInplaceFunction(5);        
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignAbsOf (const GPUMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceAbs();
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceLinearRectifierDerivative()
    {
        performInplaceFunction(6);                    
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignLinearRectifierDerivativeOf (const GPUMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceLinearRectifierDerivative();
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceCosine()
    {
        performInplaceFunction(7);        
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignCosineOf (const GPUMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceCosine();
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceNegativeSine()
    {
        performInplaceFunction(8);        
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignNegativeSineOf (const GPUMatrix<ElemType>& a)
    {
        SetValue(a);
        InplaceNegativeSine();
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceTruncateBottom (const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("InplaceTruncateBottom: Matrix is empty.");    

        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock); 
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _inplaceTruncateBottom<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,threshold,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignTruncateBottomOf (const GPUMatrix<ElemType>& a, const ElemType threshold)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignTruncateBottomOf: Matrix a is empty.");

        if (this!=&a)
        {
            Resize(a.GetNumRows(), a.GetNumCols());
        }

        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);      
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _assignTruncateBottom<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,threshold,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceTruncateTop (const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("InplaceTruncateTop: Matrix is empty.");
        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock);      
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _inplaceTruncateTop<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,threshold,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;        
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignTruncateTopOf (const GPUMatrix<ElemType>& a, const ElemType threshold)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignTruncateTopOf: Matrix a is empty.");

        if (this!=&a)
        {
            Resize(a.GetNumRows(), a.GetNumCols());
        }

        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock); 
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _assignTruncateTop<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,threshold,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;        
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceTruncate(const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("InplaceTruncate: Matrix is empty.");

        LONG64 N = (LONG64)GetNumElements();
        int blocksPerGrid = (int)ceil(N*1.0 / threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _inplaceTruncate<ElemType> << <blocksPerGrid, threadsPerBlock, 0, t_stream >> >(m_pArray, threshold, N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::InplaceSoftThreshold(const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("InplaceSoftThreshold: Matrix is empty.");

        LONG64 N = (LONG64)GetNumElements();
        int blocksPerGrid = (int)ceil(N*1.0 / threadsPerBlock);
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _inplaceSoftThreshold<ElemType> << <blocksPerGrid, threadsPerBlock, 0, t_stream >> >(m_pArray, threshold, N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::SetToZeroIfAbsLessThan (const ElemType threshold)
    {
        if (IsEmpty())
            throw std::logic_error("SetToZeroIfAbsLessThan: Matrix is empty.");
        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(N*1.0/threadsPerBlock); 
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _setToZeroIfAbsLessThan<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,threshold,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;  
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::SumOfAbsElements() const
    {
        if (IsEmpty())
            throw std::logic_error("SumOfAbsElements: Matrix is empty");

        hipblasHandle_t cuHandle = GetCublasHandle(GetComputeDeviceId());          
        if (sizeof(ElemType)==sizeof(float))
        {
            float res=0;
            CUBLAS_CALL(hipblasSasum(cuHandle, (LONG64)GetNumElements(), reinterpret_cast<float*>(m_pArray), 1, &res));
            return res;
        }
        else
        {
            double res=0;
            CUBLAS_CALL(hipblasDasum(cuHandle, (LONG64)GetNumElements(), reinterpret_cast<double*>(m_pArray), 1, &res));
            return ElemType(res);
        }         
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::SumOfElements() const
    {
        if (IsEmpty())
            throw std::logic_error("SumOfElements: Matrix is empty");

        PrepareDevice();
        ElemType* d_sum = NULL;
        ElemType h_sum;
        CUDA_CALL(hipMalloc((void**)&d_sum,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSum<ElemType><<<1,1024,0,t_stream>>>(m_pArray,d_sum,(LONG64)GetNumElements());
        CUDA_CALL(hipMemcpy(&h_sum,d_sum,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDA_CALL(hipFree(d_sum));               
        return h_sum;        
    }

    
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSumOfElements(const GPUMatrix<ElemType>& a)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignSumOfElements: Matrix a is empty");

        Resize(1,1);

        PrepareDevice();     
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSumAndAssign<ElemType><<<1,1024>>>(m_pArray,a.m_pArray,(LONG64)a.GetNumElements(),(LONG64)GetNumElements());
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return (*this);
    }

    template<class ElemType>
    DeviceBoundNumber<ElemType> GPUMatrix<ElemType>::Sum_AsDeviceBoundNum() const
    {
        if (IsEmpty())
            throw std::logic_error("Matrix is empty");
        PrepareDevice();
        ElemType* d_sum = NULL;        
        CUDA_CALL(hipMalloc((void**)&d_sum,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSum<ElemType><<<1,1024,0,t_stream>>>(m_pArray,d_sum,(LONG64)GetNumElements());
        DeviceBoundNumber<ElemType> result;
        result.ShallowCopyFrom(d_sum,GetComputeDeviceId());
        return result;
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::Max() const
    {
        hipblasHandle_t cuHandle = GetCublasHandle(GetComputeDeviceId());   
        ElemType res;
        if (sizeof(ElemType)==sizeof(float))
        {
            int resInd=0;
            hipblasIsamax(cuHandle,(LONG64)GetNumElements(),reinterpret_cast<float*>(m_pArray),1,&resInd); 
            resInd--;
            CUDA_CALL(hipMemcpy(reinterpret_cast<float*>(&res),reinterpret_cast<float*>(m_pArray+resInd),sizeof(float),hipMemcpyDeviceToHost));
            return res;
        }
        else
        {
            int resInd=0;
            hipblasIdamax(cuHandle,(LONG64)GetNumElements(),reinterpret_cast<double*>(m_pArray),1,&resInd);
            resInd--;
            CUDA_CALL(hipMemcpy(reinterpret_cast<double*>(&res),m_pArray+resInd,sizeof(float),hipMemcpyDeviceToHost));
            return res;
        }        
    }


    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::ElementMultiplyWith (const GPUMatrix<ElemType>& a)
    {
        if (IsEmpty() || a.IsEmpty())
            throw std::logic_error("ElementMultiplyWith: Matrix is empty.");

        GPUMatrix<ElemType>& us=*this;
        assert (us.GetNumRows() == a.GetNumRows() && us.GetNumCols() == a.GetNumCols());
        if (us.GetNumRows() != a.GetNumRows() || us.GetNumCols() != a.GetNumCols())
            throw std::invalid_argument("The matrix dimensions do not match.");

        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(((double)N)/threadsPerBlock); 
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _elemMul<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementProductOf (const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.IsEmpty() || b.IsEmpty())
            throw std::logic_error("AssignElementProductOf: Matrix is empty.");

        assert (a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());
        if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
            throw std::invalid_argument("The input matrix dimensions do not match.");

        Resize(a.GetNumRows(), a.GetNumCols());
        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(((double)N)/threadsPerBlock);  
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _assignElementProductOf<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,b.m_pArray,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::ElementDivideBy(const GPUMatrix<ElemType>& a)
    {
        return AssignElementDivisionOf(*this, a);
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementDivisionOf (const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.IsEmpty() || b.IsEmpty())
            throw std::logic_error("AssignElementDivisionOf: Matrix is empty.");

        assert (a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());
        if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
            throw std::invalid_argument("The input matrix dimensions do not match.");

        Resize(a.GetNumRows(), a.GetNumCols());
        LONG64 N=(LONG64)GetNumElements();
        int blocksPerGrid =(int)ceil(((double)N)/threadsPerBlock);  
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _assignElementDivisionOf<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,b.m_pArray,N);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

    template<class ElemType>
    bool GPUMatrix<ElemType>::IsEqualTo(const GPUMatrix<ElemType>& a, const ElemType threshold /*= 1e-8*/) const
    {
        return AreEqual(*this, a, threshold);
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::VectorSum(const GPUMatrix<ElemType>& a, GPUMatrix<ElemType>& c, const bool isColWise)
    {
        if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }

        a.PrepareDevice();

        if (a.IsEmpty())
            throw std::logic_error("VectorSum:  Input matrix is empty.");

        const long n = (long)a.GetNumRows();
        const long m = (long)a.GetNumCols();
        assert(m>0 && n>0); //converting from size_t to int may cause overflow

        hipEvent_t done = nullptr;

        int blocksPerGrid = 0;
        if (isColWise)  //col-wise
        {
            c.Resize(1, m);
            blocksPerGrid = (int)ceil(1.0*m / threadsPerBlock);
        }
        else
        {
            c.Resize(n, 1);
            blocksPerGrid = (int)ceil(1.0*n / threadsPerBlock);
        }

        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _vectorSum<ElemType> << <blocksPerGrid, threadsPerBlock, 0, t_stream >> >(a.m_pArray, c.m_pArray, n, m, isColWise);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }
    template<class ElemType>
    void GPUMatrix<ElemType>::VectorNorm1(GPUMatrix<ElemType>& c, const bool isColWise) const
    {
        if (IsEmpty())
            throw std::logic_error("VectorNorm1: Matrix is empty.");

        const long n = (long)GetNumRows();
        const long m = (long)GetNumCols();
        assert (m>0 && n>0); //converting from size_t to int may cause overflow

        hipEvent_t done = nullptr;  
        PrepareDevice();
        c.ChangeDeviceTo(GetComputeDeviceId());

        int blocksPerGrid=0;
        if (isColWise)  //col-wise
        {
            c.Resize(1,m);   
            blocksPerGrid =(int)ceil(1.0*m/threadsPerBlock);                                        
        }
        else
        {
            c.Resize(n, 1);
            blocksPerGrid =(int)ceil(1.0*n/threadsPerBlock);                        
        }       

        if (do_sync)    CUDA_CALL(hipEventCreate(&done));  
        _vectorNorm1<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(c.m_pArray, m_pArray,n,m,isColWise);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignVectorNorm1Of(GPUMatrix<ElemType>& a, const bool isColWise)
    {
        a.VectorNorm1(*this, isColWise);
        return *this;
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::VectorNorm2(GPUMatrix<ElemType>& c, const bool isColWise) const
    {
        if (IsEmpty())
            throw std::logic_error("VectorNorm2: Matrix is empty.");

        const long n = (long)GetNumRows();
        const long m = (long)GetNumCols();
        assert (m>0 && n>0); //converting from size_t to int may cause overflow

        hipEvent_t done = nullptr;  
        PrepareDevice();
        c.ChangeDeviceTo(GetComputeDeviceId());

        int blocksPerGrid=0;
        if (isColWise)  //col-wise
        {
            c.Resize(1,m);   
            blocksPerGrid =(int)ceil(1.0*m/threadsPerBlock);                                        
        }
        else
        {
            c.Resize(n, 1);
            c.ChangeDeviceTo(GetComputeDeviceId());
            blocksPerGrid =(int)ceil(1.0*n/threadsPerBlock);                        
        }       

        if (do_sync)    CUDA_CALL(hipEventCreate(&done));  
        _vectorNorm2<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(c.m_pArray, m_pArray,n,m,isColWise);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignVectorNorm2Of(GPUMatrix<ElemType>& a, const bool isColWise)
    {
        a.VectorNorm2(*this, isColWise);
        return *this;
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::VectorNormInf(GPUMatrix<ElemType>& c, const bool isColWise) const
    {
        if (IsEmpty())
            throw std::logic_error("VectorMax: Matrix is empty.");

        //this implementation is not efficient
        GPUMatrix<ElemType> tmp;
        GPUMatrix<ElemType> tmp1;
        tmp.AssignAbsOf((*this));
        tmp.VectorMax(tmp1,c,isColWise);
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignVectorNormInfOf(GPUMatrix<ElemType>& a, const bool isColWise)
    {
        a.VectorNormInf(*this, isColWise);
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignInnerProductOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const bool isColWise)
    {
        InnerProduct (a, b, *this,isColWise);
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignKhatriRaoProductOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.IsEmpty() || b.IsEmpty())
            throw std::logic_error("AssignKhatriRaoProductOf: Matrix is empty.");

        long cols = a.GetNumCols();
        assert (cols == b.GetNumCols());
        if (!(cols == b.GetNumCols()))
            throw std::invalid_argument("AssignKhatriRaoProductOf: The input matrix dimensions do not match.");

        long rowsA = (long)a.GetNumRows();
        long rowsB = (long)b.GetNumRows();
        Resize(rowsA * rowsB, cols);
        float N=(float)GetNumElements();
        int blocksPerGrid =(int)ceil(N/threadsPerBlock);  
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _assignKhatriRaoProductOf<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,b.m_pArray,rowsA, rowsB, cols);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    //column-wise reshaped product. Used to compute KhatriRaoProduct Gradient
    //   this = reshape each column of a from (K1xK2,1) to (K1, K2) 
    //   if each column of a is not transposed, each (K1, K2) times each column of b (K2, frames).
    //   the output is a (K1, frames) matrix
    //   if each column of a is tranposed, each (K1, K2)^T times each column of b(K1, frames) and output is (K2, frames)
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddColumnReshapeProductOf(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const bool transposeAColumn)
    {
        if (a.IsEmpty() || b.IsEmpty())
            throw std::logic_error("AddColumnReshapeProductOf: Matrix is empty.");

        long cols = a.GetNumCols();
        assert (cols == b.GetNumCols());
        if (!(cols == b.GetNumCols()))
            throw std::invalid_argument("AddColumnReshapeProductOf: The input matrix dimensions do not match.");

        long rowsA = (long)a.GetNumRows();
        long rowsB = (long)b.GetNumRows();
        if (rowsA % rowsB != 0)
            throw std::invalid_argument("AddColumnReshapeProductOf: number of rows in a should be multiples of that in b.");

        long rowsC = rowsA / rowsB;
        if (rowsC != GetNumRows() || cols != GetNumCols())
            throw  std::invalid_argument("AddColumnReshapeProductOf: This matrix does not have the right size.");

        float N=(float)GetNumElements();
        int blocksPerGrid =(int)ceil(N/threadsPerBlock);  
        a.PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _addColumnReshapeProductOf<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray,a.m_pArray,b.m_pArray, rowsB, rowsC, cols, transposeAColumn);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddWithScaleOf(ElemType alpha, const GPUMatrix<ElemType>& a)
    {
        ScaleAndAdd(alpha, a, *this);
        return *this;
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::FrobeniusNorm() const
    {
        if (IsEmpty())
            throw std::logic_error("FrobeniusNorm: Matrix is empty.");

        PrepareDevice();
        ElemType* d_sum = NULL;
        ElemType h_sum=0;
        CUDA_CALL(hipMalloc((void**)&d_sum,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSum2<ElemType><<<1,1024,0,t_stream>>>(m_pArray,d_sum,(LONG64)GetNumElements(), true);
        CUDA_CALL(hipMemcpy(&h_sum,d_sum,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDA_CALL(hipFree(d_sum));               

        return (h_sum); 
    }
    
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignFrobeniusNormOf (const GPUMatrix<ElemType>& a)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignFrobeniusNormOf: Matrix a is empty.");

        Resize(1,1);        
    
        PrepareDevice();
        //WARNING: THIS kernel is not the most efficient way!
        _reductionSum2<ElemType><<<1,1024,0,t_stream>>>(a.m_pArray,m_pArray,(LONG64)a.GetNumElements(), true);

        return *this;
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::MatrixNormInf() const
    {
        if (IsEmpty())
            throw std::logic_error("MatrixNorm1: Matrix is empty.");

        PrepareDevice();
        ElemType* d_maxAbs = NULL;
        ElemType h_maxAbs=0;
        CUDA_CALL(hipMalloc((void**)&d_maxAbs,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionMatrixNormInf<ElemType><<<1,1024,0,t_stream>>>(m_pArray,d_maxAbs,(LONG64)GetNumElements());
        CUDA_CALL(hipMemcpy(&h_maxAbs,d_maxAbs,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDA_CALL(hipFree(d_maxAbs));               
        return h_maxAbs; 
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::MatrixNorm1() const
    {
        if (IsEmpty())
            throw std::logic_error("MatrixNorm1: Matrix is empty.");
        return SumOfAbsElements();              
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::MatrixNorm0() const
    {
        if (IsEmpty())
            throw std::logic_error("MatrixNorm0: Matrix is empty.");

        PrepareDevice();
        ElemType* d_nz = NULL;
        ElemType h_nz=0;
        CUDA_CALL(hipMalloc((void**)&d_nz,sizeof(ElemType)));
        //WARNING: THIS kernel is not the most efficient way!
        _reductionMatrixNorm0<ElemType><<<1,1024,0,t_stream>>>(m_pArray,d_nz,(LONG64)GetNumElements());
        CUDA_CALL(hipMemcpy(&h_nz,d_nz,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDA_CALL(hipFree(d_nz));               
        return h_nz; 
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignSignOf(const GPUMatrix<ElemType>& a)
    {
        if (a.IsEmpty())
            throw std::logic_error("AssignSignOf: Matrix a is empty.");

        if (this != &a)
            Resize(a.GetNumRows(), a.GetNumCols());

        PrepareDevice();
        hipEvent_t done = nullptr;
        int blocksPerGrid=(int)ceil(1.0*GetNumElements()/threadsPerBlock);  
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _assignSignOf<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray, a.m_pArray, (long)GetNumElements());
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));    
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AddSignOf(const GPUMatrix<ElemType>& a)
    {
        if (a.IsEmpty())
            throw std::logic_error("AddSignOf: Matrix a is empty.");

        if (this != &a)
            Resize(a.GetNumRows(), a.GetNumCols());

        PrepareDevice();
        hipEvent_t done = nullptr;
        int blocksPerGrid=(int)ceil(1.0*GetNumElements()/threadsPerBlock);  
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _addSignOf<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(m_pArray, a.m_pArray, (LONG64)GetNumElements());
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));    
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::VectorMax(GPUMatrix<ElemType>& maxIndexes, GPUMatrix<ElemType>& maxValues, const bool isColWise) const
    {
        if (IsEmpty())
            throw std::logic_error("VectorMax: Matrix is empty.");

        const GPUMatrix<ElemType>& us=*this;
        const long m = (long)GetNumRows();
        const long n = (long)GetNumCols();
        assert (m>0 && n>0); //converting from size_t to int may cause overflow
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)     CUDA_CALL(hipEventCreate(&done));                
        if (isColWise)
        {
            maxValues.Resize(1, n);
            maxIndexes.Resize(1, n);

            int blocksPerGrid = n; //we'll have 1 block processing 1 column
            _vectorMaxMinReduce<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(us.m_pArray,maxIndexes.m_pArray,maxValues.m_pArray,m,n,true);

            /*int blocksPerGrid=(int)ceil(1.0*n/threadsPerBlock);  
            _vectorMax<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(us.m_pArray,maxIndexes.m_pArray,maxValues.m_pArray,m,n,isColWise);*/
        }
        else
        {
            maxValues.Resize(m, 1);
            maxIndexes.Resize(m, 1);
            int blocksPerGrid=(int)ceil(1.0*m/threadsPerBlock);  
            _vectorMax<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(us.m_pArray,maxIndexes.m_pArray,maxValues.m_pArray,m,n,isColWise);
        }
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::VectorMin(GPUMatrix<ElemType>& minIndexes, GPUMatrix<ElemType>& minValues, const bool isColWise) const
    {
        if (IsEmpty())
            throw std::logic_error("VectorMax: Matrix is empty.");

        const GPUMatrix<ElemType>& us=*this;
        const int m = (int)GetNumRows();
        const int n = (int)GetNumCols();

        assert (m>0 && n>0); //converting from size_t to int may cause overflow
        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));                
        if (isColWise)
        {
            minValues.Resize(1, n);
            minIndexes.Resize(1, n);

            int blocksPerGrid = n; //we'll have 1 block processing 1 column
            _vectorMaxMinReduce<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(us.m_pArray,minIndexes.m_pArray,minValues.m_pArray,m,n,false);

            /*
            int blocksPerGrid=(int)ceil(1.0*n/threadsPerBlock);  
            _vectorMin<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(us.m_pArray,minIndexes.m_pArray,minValues.m_pArray,m,n,isColWise);*/
        }
        else
        {
            minValues.Resize(m, 1);
            minIndexes.Resize(m, 1);
            int blocksPerGrid=(int)ceil(1.0*m/threadsPerBlock);  
            _vectorMin<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(us.m_pArray,minIndexes.m_pArray,minValues.m_pArray,m,n,isColWise);
        }
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    template<class ElemType>
    GPUMatrix<ElemType>&  GPUMatrix<ElemType>::AssignNumOfDiff(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.GetNumRows() != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            throw std::invalid_argument ("AssignNumOfDiff: a and b must have same dimension.");

        Resize(1,1); //result should be one element

        PrepareDevice();
        hipEvent_t done = nullptr;
        //int blocksPerGrid=(int)ceil(1.0*a.GetNumElements()/threadsPerBlock);  
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        //_assignNumOfDiff<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(a.m_pArray, b.m_pArray, m_pArray, a.GetNumElements());
        _assignNumOfDiff<ElemType><<<1,1024,0,t_stream>>>(a.m_pArray, b.m_pArray, m_pArray, (LONG64)a.GetNumElements());
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));  
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

#pragma endregion Member BLAS Functions    

#pragma region Other helper functions
    template<class ElemType>
    void GPUMatrix<ElemType>::Print(const char* /*matrixName*/, size_t /*rowStart*/, size_t /*rowEnd*/, size_t /*colStart*/, size_t /*colEnd*/) const
    {
        NOT_IMPLEMENTED;
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::Print(const char* matrixName /*=nullptr*/) const
    {
        Print(matrixName, 0, GetNumRows()-1, 0, GetNumCols()-1);
    }

    // file I/O
    //matrixName is used to verify that correct matrix is read.
    template<class ElemType>
    void GPUMatrix<ElemType>::ReadFromFile(FILE*, const char * /*matrixName*/)
    {
        NOT_IMPLEMENTED;
    }

    //matrixName is used to verify that correct matrix is read.
    template<class ElemType>
    void GPUMatrix<ElemType>::WriteToFile(FILE*, const char * /*matrixName*/)
    {
        NOT_IMPLEMENTED;
    }

    //helpfer function used for convolution neural network 
    template<class ElemType>
    GPUMatrix<ElemType>&  GPUMatrix<ElemType>::AssignPackedConvolutionInput(const GPUMatrix<ElemType>& inputSubBatch, 
                                            const size_t inputWidth, const size_t inputHeight, const size_t inputChannels,
                                            const size_t outputWidth, const size_t outputHeight, const size_t outputChannels,
                                            const size_t kernelWidth, const size_t kernelHeight, const size_t horizontalSubsample, const size_t verticalSubsample, 
                                            const bool zeroPadding)
    {
        assert (verticalSubsample <= kernelHeight && horizontalSubsample <= kernelWidth);

        size_t packedInputRows = kernelWidth * kernelHeight * inputChannels;
        size_t packedInputColsPerSample = outputWidth * outputHeight;
        size_t smallBatchSize = inputSubBatch.GetNumCols();
        Resize(packedInputRows, packedInputColsPerSample * smallBatchSize);
        if (zeroPadding) 
            SetValue((ElemType)0);

        PrepareDevice();
        int numThreadPerBlock = threadsPerBlock; 
#if 1
        int blocksPerGrid = (smallBatchSize * inputWidth*inputHeight*inputChannels + numThreadPerBlock - 1)/numThreadPerBlock; 
#else
        dim3 blocksPerGrid((inputWidth*inputHeight*inputChannels + numThreadPerBlock - 1)/numThreadPerBlock, smallBatchSize);
#endif
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _assignPackedConvolutionInput<<<blocksPerGrid, numThreadPerBlock,0,t_stream>>>(m_pArray, 
                                            inputSubBatch.m_pArray, 
                                            smallBatchSize,
                                            inputWidth, inputHeight, inputChannels,
                                            outputWidth, outputHeight, outputChannels,
                                            kernelWidth, kernelHeight, horizontalSubsample, verticalSubsample, zeroPadding);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    //helpfer function used for convolution neural network 
    template<class ElemType>
    GPUMatrix<ElemType>&  GPUMatrix<ElemType>::UnpackConvolutionInput(GPUMatrix<ElemType>& inputSubBatch, 
                                            const size_t inputWidth, const size_t inputHeight, const size_t inputChannels,
                                            const size_t outputWidth, const size_t outputHeight, const size_t outputChannels,
                                            const size_t kernelWidth, const size_t kernelHeight, const size_t horizontalSubsample, const size_t verticalSubsample, 
                                            const bool zeroPadding) const
    {
        assert (verticalSubsample <= kernelHeight && horizontalSubsample <= kernelWidth);

        size_t smallBatchSize = inputSubBatch.GetNumCols();

        PrepareDevice();
        int numThreadPerBlock = threadsPerBlock; 
#if 1
        int blocksPerGrid = (smallBatchSize * inputWidth*inputHeight*inputChannels + numThreadPerBlock - 1)/numThreadPerBlock; 
#else
        dim3 blocksPerGrid((inputWidth*inputHeight*inputChannels + numThreadPerBlock - 1)/numThreadPerBlock, smallBatchSize);
#endif
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _unpackConvolutionInput<<<blocksPerGrid, numThreadPerBlock,0,t_stream>>>(m_pArray, 
                                            inputSubBatch.m_pArray, 
                                            smallBatchSize,
                                            inputWidth, inputHeight, inputChannels,
                                            outputWidth, outputHeight, outputChannels,
                                            kernelWidth, kernelHeight, horizontalSubsample, verticalSubsample, zeroPadding);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return inputSubBatch;
    }

    template<class ElemType>
    GPUMatrix<ElemType>&  GPUMatrix<ElemType>::AssignMaxPoolingResult(const GPUMatrix<ElemType>& inputBatch, const size_t channels, 
                                                const size_t inputWidth, const size_t inputHeight, const size_t inputSizePerSample, 
                                                const size_t outputWidth, const size_t outputHeight, const size_t outputSizePerSample, 
                                                const size_t windowWidth, const size_t windowHeight, const size_t horizontalSubsample, const size_t verticalSubsample)
    {
        assert (verticalSubsample <= windowHeight && horizontalSubsample <= windowWidth);

        unsigned int batchSize = inputBatch.GetNumCols();
        Resize(outputSizePerSample, batchSize);

        int numThreadPerBlock = threadsPerBlock; 
        int blocksPerGrid = (batchSize * outputSizePerSample + numThreadPerBlock - 1)/numThreadPerBlock; 

        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _assignMaxPoolingResult<<<blocksPerGrid, numThreadPerBlock,0,t_stream>>>(m_pArray, inputBatch.m_pArray, batchSize, channels,
                                                 inputWidth, inputHeight,inputSizePerSample, 
                                                 outputWidth, outputHeight, outputSizePerSample, 
                                                 windowWidth, windowHeight, horizontalSubsample, verticalSubsample);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>&  GPUMatrix<ElemType>::AddMaxPoolingGradient(const GPUMatrix<ElemType>& outputGradientBatch, const GPUMatrix<ElemType>& inputBatch, const GPUMatrix<ElemType>& outputBatch, 
                                                const size_t channels, 
                                                const size_t inputWidth, const size_t inputHeight, const size_t inputSizePerSample, 
                                                const size_t outputWidth, const size_t outputHeight, const size_t outputSizePerSample, 
                                                const size_t windowWidth, const size_t windowHeight, const size_t horizontalSubsample, const size_t verticalSubsample)
    {
        assert (verticalSubsample <= windowHeight && horizontalSubsample <= windowWidth);

        unsigned int batchSize = outputGradientBatch.GetNumCols();
        int numThreadPerBlock = threadsPerBlock; 

        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));

        int blocksPerGrid = (batchSize * inputSizePerSample + numThreadPerBlock - 1)/numThreadPerBlock; 
        _addMaxPoolingGradient<<<blocksPerGrid, numThreadPerBlock,0,t_stream>>>(m_pArray, outputGradientBatch.m_pArray, inputBatch.m_pArray, outputBatch.m_pArray, batchSize, channels,
                                                 inputWidth, inputHeight,inputSizePerSample, 
                                                 outputWidth, outputHeight,  outputSizePerSample, 
                                                 windowWidth, windowHeight, horizontalSubsample, verticalSubsample);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>&  GPUMatrix<ElemType>::AssignAveragePoolingResult(const GPUMatrix<ElemType>& inputBatch, const size_t channels, 
                                                const size_t inputWidth, const size_t inputHeight, const size_t inputSizePerSample, 
                                                const size_t outputWidth, const size_t outputHeight, const size_t outputSizePerSample, 
                                                const size_t windowWidth, const size_t windowHeight, const size_t horizontalSubsample, const size_t verticalSubsample)
    {
        assert (verticalSubsample <= windowHeight && horizontalSubsample <= windowWidth);

        unsigned int batchSize = inputBatch.GetNumCols();
        Resize(outputSizePerSample, batchSize);

        int numThreadPerBlock = threadsPerBlock; 
        int blocksPerGrid = (batchSize * outputSizePerSample + numThreadPerBlock - 1)/numThreadPerBlock; 

        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _assignAveragePoolingResult<<<blocksPerGrid, numThreadPerBlock,0,t_stream>>>(m_pArray, inputBatch.m_pArray, batchSize, channels,
                                                 inputWidth, inputHeight,inputSizePerSample, 
                                                 outputWidth, outputHeight, outputSizePerSample, 
                                                 windowWidth, windowHeight, horizontalSubsample, verticalSubsample);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

    template<class ElemType>
    GPUMatrix<ElemType>&  GPUMatrix<ElemType>::AddAveragePoolingGradient(const GPUMatrix<ElemType>& outputGradientBatch, 
                                                const size_t channels, 
                                                const size_t inputWidth, const size_t inputHeight, const size_t inputSizePerSample, 
                                                const size_t outputWidth, const size_t outputHeight, const size_t outputSizePerSample, 
                                                const size_t windowWidth, const size_t windowHeight, const size_t horizontalSubsample, const size_t verticalSubsample)
    {
        assert (verticalSubsample <= windowHeight && horizontalSubsample <= windowWidth);

        size_t batchSize = outputGradientBatch.GetNumCols();
        int numThreadPerBlock = threadsPerBlock; 

        PrepareDevice();
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));

        size_t blocksPerGrid = (batchSize * inputSizePerSample + numThreadPerBlock - 1)/numThreadPerBlock; 
        _addAveragePoolingGradient<<<blocksPerGrid, numThreadPerBlock,0,t_stream>>>(m_pArray, outputGradientBatch.m_pArray, (long)batchSize, channels,
                                                 inputWidth, inputHeight,inputSizePerSample, 
                                                 outputWidth, outputHeight,  outputSizePerSample, 
                                                 windowWidth, windowHeight, horizontalSubsample, verticalSubsample);
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));

        return *this;
    }

#pragma endregion Other helper functions

#pragma region Static BLAS Functions
    template<class ElemType>
    void GPUMatrix<ElemType>::MultiplyAndWeightedAdd(ElemType alpha, const GPUMatrix<ElemType>& a, const bool transposeA, const GPUMatrix<ElemType>& b, const bool transposeB, 
        ElemType beta, GPUMatrix<ElemType>& c)
    {
        a.PrepareDevice();
        if ((a.GetComputeDeviceId()!=b.GetComputeDeviceId()) || (b.GetComputeDeviceId()!=c.GetComputeDeviceId())) //different GPUs
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }
        else
        {  
            hipblasHandle_t cuHandle = GetCublasHandle(b.GetComputeDeviceId());
            hipblasOperation_t transA =  transposeA ? HIPBLAS_OP_T : HIPBLAS_OP_N;
            hipblasOperation_t transB =  transposeB ? HIPBLAS_OP_T : HIPBLAS_OP_N;
            int m = int(transposeA ? a.m_numCols : a.m_numRows);
            int n = int(transposeB ? b.m_numRows : b.m_numCols);
            int k = int(transposeA ? a.m_numRows : a.m_numCols);
            int l = int(transposeB ? b.m_numCols : b.m_numRows);
            c.Resize(m,n);

            if (!(m>0 && k>0 && l>0 && n>0)) 
            {
                throw std::runtime_error("!(m>0 && k>0 && l>0 && n>0)");  //converting from size_t to int may cause overflow
            }
            if (k!=l) 
            {
                throw std::runtime_error("matrix dim mismatch in MultiplyAndWeightedAdd");
            }
            if (sizeof(ElemType)==sizeof(float))
            {
                CUBLAS_CALL(hipblasSgemm(cuHandle,transA,transB,m,n,k,reinterpret_cast<float*>(&alpha),reinterpret_cast<float*>(a.m_pArray),(int)a.m_numRows,reinterpret_cast<float*>(b.m_pArray),(int)b.m_numRows,reinterpret_cast<float*>(&beta),reinterpret_cast<float*>(c.m_pArray),(int)c.m_numRows));
            }
            else if (sizeof(ElemType)==sizeof(double))
            {            
                CUBLAS_CALL(hipblasDgemm(cuHandle,transA,transB,m,n,k,reinterpret_cast<double*>(&alpha),reinterpret_cast<double*>(a.m_pArray),(int)a.m_numRows,reinterpret_cast<double*>(b.m_pArray),(int)b.m_numRows,reinterpret_cast<double*>(&beta),reinterpret_cast<double*>(c.m_pArray),(int)c.m_numRows));
            }
            else 
            {
                throw std::runtime_error("Unsupported template argument in GPUMatrix");             
            }
            c.m_numRows=m;
            c.m_numCols=n;
        }
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::MultiplyAndAdd(const GPUMatrix<ElemType>& a, const bool transposeA, const GPUMatrix<ElemType>& b, const bool transposeB, GPUMatrix<ElemType>& c)
    {
        return GPUMatrix<ElemType>::MultiplyAndWeightedAdd(1, a, transposeA, b, transposeB, 1, c);
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::Multiply(const GPUMatrix<ElemType>& a, const bool transposeA, const GPUMatrix<ElemType>& b, const bool transposeB, GPUMatrix<ElemType>& c)
    {    
        return GPUMatrix<ElemType>::MultiplyAndWeightedAdd(1, a, transposeA, b, transposeB, 0, c);
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::Multiply(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        return GPUMatrix<ElemType>::MultiplyAndWeightedAdd(1, a, false, b, false, 0, c);
    }

    /// <summary>Matrix-scalar multiply with col-major matrices: c = alpha * a + c</summary>
    /// if a is a column vector, add to all columns of c 
    /// if a is a row vector, add to all rows of c    
    /// if a is a scalar, add to all elements of c
    /// <param name="alpha">Scalar</param>
    /// <param name="a">Input matrix</param>
    /// <param name="c">Resulting matrix, user is responsible for allocating this</param>
    template<class ElemType>
    void GPUMatrix<ElemType>::ScaleAndAdd(ElemType alpha,const GPUMatrix<ElemType>& a, GPUMatrix<ElemType>& c)
    {
        if (a.GetComputeDeviceId()!=c.GetComputeDeviceId())
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }
        else
        {
            a.PrepareDevice();
            if (a.IsEmpty() || c.IsEmpty())
                throw std::logic_error("ScaleAndAdd:  one of the input matrices is empty.");
            //if (a.GetNumRows() != 1 && a.GetNumCols() != 1) // a is not a col or row vector
            if (a.GetNumRows()==c.GetNumRows() && a.GetNumCols()==c.GetNumCols()) // dimensions match
            {
                const int m = (int)a.GetNumRows();
                const int n = (int)a.GetNumCols();
                const int len = m * n;
                const int incx = 1;
                const int incy = 1;

                assert (m>0 && n>0 && len>0); //converting from size_t to int may cause overflow
                assert ((int)c.GetNumRows() == m && (int)c.GetNumCols() == n);
                if ((int)c.GetNumRows() != m || (int)c.GetNumCols() != n)
                    throw std::invalid_argument("Dimention of matrix c does not match dimention of matrix a.");

                hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
                if (sizeof(ElemType) == sizeof(float))
                {
                    CUBLAS_CALL(hipblasSaxpy(cuHandle,len,reinterpret_cast <float*>(&alpha),reinterpret_cast <float*>(a.m_pArray),incx,reinterpret_cast <float*>(c.m_pArray) ,incy));                
                }
                else if (sizeof(ElemType) == sizeof(double))
                {   
                    CUBLAS_CALL(hipblasDaxpy(cuHandle,len,reinterpret_cast <double*>(&alpha),reinterpret_cast <double*>(a.m_pArray),incx,reinterpret_cast <double*>(c.m_pArray) ,incy)); 
                }
                else 
                {
                    throw std::runtime_error("Unsupported template argument in GPUMatrix"); 
                }
            }
            else if (a.GetNumElements() == 1)
            {
                LONG64 N=(LONG64)c.GetNumElements();
                int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
                c.PrepareDevice();
                hipEvent_t done = nullptr;
                if (do_sync)    CUDA_CALL(hipEventCreate(&done));
                _scaleAndAddScalar<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(c.m_pArray, N, alpha, a.m_pArray);
                if (do_sync)    CUDA_CALL(hipEventRecord(done));        
                if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
                if (do_sync)    CUDA_CALL(hipEventDestroy(done));
            }
            else if (a.GetNumCols() == 1) //col vector, add it to all columns
            {                
                long m = (long)c.GetNumRows();
                long n = (long)c.GetNumCols();                
                if (m != (long)a.GetNumRows())
                    throw std::invalid_argument("To add column vector, rows should match.");

                hipEvent_t done = nullptr;
                int blocksPerGrid = (int)(ceil(1.0*m*n / threadsPerBlock));
                if (do_sync)    CUDA_CALL(hipEventCreate(&done));   
#ifdef VALIDATION
                printf(">>>> CUDA compute device is %d\n", a.GetComputeDeviceId());
                printf(">>>> a.m_pArray = %p, c.m_pArray = %p, alpha = %f, m = %ld, n = %ld\n", a.m_pArray,c.m_pArray,alpha,m,n);   
                for (int i=0; i < 2; i++)
                {
                    ElemType buffer[10] = {-1.234f};
                    hipError_t error = hipMemcpy(buffer, !i?a.m_pArray:c.m_pArray, sizeof(buffer), hipMemcpyKind::hipMemcpyDeviceToHost);
                    if (error == hipError_t::hipSuccess)
                        printf("buffer valid\n"); 
                }
#endif

                _matrixVectorColumnWiseAddWithThreadPerElem<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(a.m_pArray,c.m_pArray,alpha,m,n);


                if (do_sync)    CUDA_CALL(hipEventRecord(done));
                if (do_sync)    CUDA_CALL(hipEventSynchronize(done));   
                if (do_sync)    CUDA_CALL(hipEventDestroy(done));                
            }
            else  if (a.GetNumRows()==1)  //row vector, add it to all rows
            {
                hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
                int m = (int)c.GetNumRows();
                int n = (int)c.GetNumCols();
                assert (n == (int)a.GetNumCols());
                if (n != (int)a.GetNumCols())
                    throw std::invalid_argument("To add row vector, cols should match.");

                if (sizeof(ElemType) == sizeof(double))
                {
                    foreach_row(i,c)
                    {
                        CUBLAS_CALL(hipblasDaxpy(cuHandle,n,reinterpret_cast <double*>(&alpha),reinterpret_cast <double*>(a.m_pArray),1,reinterpret_cast <double*>(c.m_pArray+i),m));
                    }                    
                }
                else
                {
                    foreach_row(i,c)
                    {
                        CUBLAS_CALL(hipblasSaxpy(cuHandle,n,reinterpret_cast <float*>(&alpha),reinterpret_cast <float*>(a.m_pArray),1,reinterpret_cast <float*>(c.m_pArray+i),m));
                    }                    
                }
            }
            else
                throw std::invalid_argument("Dimention of matrix c does not match dimention of matrix a.");
        }
    }

    /// <summary>c += alpha * (a-b)</summary>
    /// if a, b, c  must have same dim 
    /// <param name="alpha">Scalar</param>
    /// <param name="a">Input matrix</param>
    /// <param name="b">Input matrix</param>
    /// <param name="c">Resulting matrix, user is responsible for allocating this</param>
    template<class ElemType>
    void GPUMatrix<ElemType>::AddScaledDifference(const ElemType alpha, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        if (a.GetComputeDeviceId()!=c.GetComputeDeviceId())
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }
        else
        {
            a.PrepareDevice();

            assert(a.GetNumRows() == b.GetNumRows() && a.GetNumRows() == c.GetNumRows() &&
                a.GetNumCols() == b.GetNumCols() && a.GetNumCols() == c.GetNumCols());

            if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumRows() == c.GetNumRows() &&
                a.GetNumCols() == b.GetNumCols() && a.GetNumCols() == c.GetNumCols()))
            {
                throw std::invalid_argument("AddScaledDifference:  a, b, and c must have same dimension.");
            }

            if (a.IsEmpty())
                throw std::logic_error("AddScaledDifference:  Input matrix a is empty.");

            hipEvent_t done = nullptr;
            LONG64 n=(LONG64)a.GetNumElements();            
            int blocksPerGrid=(int)ceil(1.0*n/threadsPerBlock);  
            if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
            _addScaledDifference<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(alpha, a.m_pArray, b.m_pArray, c.m_pArray, n);
            if (do_sync)    CUDA_CALL(hipEventRecord(done));        
            if (do_sync)    CUDA_CALL(hipEventSynchronize(done));   
            if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        }
    }

    /// <summary> c = alpha * (a-b)</summary>
    /// if a, b, c  must have same dim 
    /// <param name="alpha">Scalar</param>
    /// <param name="a">Input matrix</param>
    /// <param name="b">Input matrix</param>
    /// <param name="c">Resulting matrix, user is responsible for allocating this</param>
    template<class ElemType>    
    void GPUMatrix<ElemType>::AssignScaledDifference(const ElemType alpha, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        if (a.GetComputeDeviceId()!=c.GetComputeDeviceId())
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }
        else
        {
            a.PrepareDevice();

            assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols() );

            if (!(a.GetNumRows() == b.GetNumRows()  && a.GetNumCols() == b.GetNumCols()))
            {
                throw std::invalid_argument("AssignScaledDifference:  a, b must have same dimension.");
            }

            if (a.IsEmpty())
                throw std::logic_error("AssignScaledDifference:  Input matrix a is empty.");

            if (&c != &a && &c != &b)
                c.Resize(a.GetNumRows(), a.GetNumCols());

            hipEvent_t done = nullptr;
            LONG64 n=(LONG64)a.GetNumElements();            
            int blocksPerGrid=(int)ceil(1.0*n/threadsPerBlock);  
            if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
            _assignScaledDifference<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(alpha, a.m_pArray, b.m_pArray, c.m_pArray, n);
            if (do_sync)    CUDA_CALL(hipEventRecord(done));        
            if (do_sync)    CUDA_CALL(hipEventSynchronize(done));   
            if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        }
    }

    /// <summary>c += alpha * (a-b)</summary>
    /// if a, b, c  must have same dim 
    /// <param name="alpha">1X1 matrix</param>
    /// <param name="a">Input matrix</param>
    /// <param name="b">Input matrix</param>
    /// <param name="c">Resulting matrix, user is responsible for allocating this</param>
    template<class ElemType>
    void GPUMatrix<ElemType>::AddScaledDifference(const GPUMatrix<ElemType>& alpha, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        assert(alpha.GetNumElements() == 1);
        if (!(alpha.GetNumElements() == 1))
            throw std::invalid_argument("AddScaledDifference:  alpha must be a 1X1 matrix.");

        if (a.GetComputeDeviceId()!=c.GetComputeDeviceId())
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }
        else
        {
            a.PrepareDevice();

            assert(a.GetNumRows() == b.GetNumRows() && a.GetNumRows() == c.GetNumRows() &&
                a.GetNumCols() == b.GetNumCols() && a.GetNumCols() == c.GetNumCols());

            if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumRows() == c.GetNumRows() &&
                a.GetNumCols() == b.GetNumCols() && a.GetNumCols() == c.GetNumCols()))
            {
                throw std::invalid_argument("AddScaledDifference:  a, b, and c must have same dimension.");
            }

            if (a.IsEmpty())
                throw std::logic_error("AddScaledDifference:  Input matrix a is empty.");

            hipEvent_t done = nullptr;
            LONG64 n=(LONG64)a.GetNumElements();            
            int blocksPerGrid=(int)ceil(1.0*n/threadsPerBlock);  
            if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
            _addScaledDifference<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(alpha.m_pArray, a.m_pArray, b.m_pArray, c.m_pArray, n);
            if (do_sync)    CUDA_CALL(hipEventRecord(done));        
            if (do_sync)    CUDA_CALL(hipEventSynchronize(done));   
            if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        }
    }

    /// <summary> c = alpha * (a-b)</summary>
    /// if a, b, c  must have same dim 
    /// <param name="alpha">Scalar</param>
    /// <param name="a">Input matrix</param>
    /// <param name="b">Input matrix</param>
    /// <param name="c">Resulting matrix, user is responsible for allocating this</param>
    template<class ElemType>    
    void GPUMatrix<ElemType>::AssignScaledDifference(const GPUMatrix<ElemType>& alpha, const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c)
    {
        assert(alpha.GetNumElements() == 1);
        if (!(alpha.GetNumElements() == 1))
            throw std::invalid_argument("AddScaledDifference:  alpha must be a 1X1 matrix.");

        if (a.GetComputeDeviceId()!=c.GetComputeDeviceId())
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }
        else
        {
            a.PrepareDevice();

            assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols() );

            if (!(a.GetNumRows() == b.GetNumRows()  && a.GetNumCols() == b.GetNumCols()))
            {
                throw std::invalid_argument("AssignScaledDifference:  a, b must have same dimension.");
            }

            if (a.IsEmpty())
                throw std::logic_error("AssignScaledDifference:  Input matrix a is empty.");

            c.Resize(a.GetNumRows(), a.GetNumCols());

            hipEvent_t done = nullptr;
            LONG64 n=(LONG64)a.GetNumElements();            
            int blocksPerGrid=(int)ceil(1.0*n/threadsPerBlock);  
            if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
            _assignScaledDifference<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(alpha.m_pArray, a.m_pArray, b.m_pArray, c.m_pArray, n);
            if (do_sync)    CUDA_CALL(hipEventRecord(done));        
            if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
            if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        }
    }

    //c[ci,cj] += a[ai,aj]
    template<class ElemType>
    void GPUMatrix<ElemType>::AddElementToElement(const GPUMatrix<ElemType>& a, const size_t ai, const size_t aj, GPUMatrix<ElemType>& c, const size_t ci, const size_t cj)
    {
        if (ai >= a.GetNumRows() || aj >=a.GetNumCols() ||
            ci >= c.GetNumRows() || cj >=c.GetNumCols())
            throw std::invalid_argument("AddElementToElement:  index out of range.");

        a.PrepareDevice();
        hipEvent_t done = nullptr;
        int blocksPerGrid=1;  //only one element
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));        
        _addElementToElement<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(a.m_pArray, (LONG64)a.LocateElement(ai, aj), c.m_pArray, (LONG64)c.LocateElement(ci, cj));
        if (do_sync)    CUDA_CALL(hipEventRecord(done));        
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));  
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    template<class ElemType>
    void GPUMatrix<ElemType>::Scale(ElemType alpha, GPUMatrix<ElemType>& a)
    {   
        hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
        if (sizeof(ElemType)==sizeof(float))
        {
            float alph = (float)alpha;            
            CUBLAS_CALL(hipblasSscal(cuHandle,int(a.m_numRows*a.m_numCols),&alph,(float*)a.m_pArray,1));
        }
        else if (sizeof(ElemType)==sizeof(double))
        {
            double alph = alpha;
            CUBLAS_CALL(hipblasDscal(cuHandle,int(a.m_numRows*a.m_numCols),&alph,(double*)a.m_pArray,1));
        }
        else 
        {
            throw std::runtime_error("Unsupported template argument in GPUMatrix");            
        }
    }


    template<class ElemType>
    void GPUMatrix<ElemType>::Scale(GPUMatrix<ElemType>& alpha, GPUMatrix<ElemType>& a)
    {           
        if (alpha.GetNumElements()!=1)
        {
            throw std::runtime_error("Matrix alpha must be 1x1");
        }
        hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
        hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_DEVICE);
        if (sizeof(ElemType)==sizeof(float))
        {
            CUBLAS_CALL(hipblasSscal(cuHandle,int(a.m_numRows*a.m_numCols),(float*)alpha.m_pArray,(float*)a.m_pArray,1));
        }
        else if (sizeof(ElemType)==sizeof(double))
        {            
            CUBLAS_CALL(hipblasDscal(cuHandle,int(a.m_numRows*a.m_numCols),(double*)alpha.m_pArray,(double*)a.m_pArray,1));
        }
        else 
        {
            hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_HOST);
            throw std::runtime_error("Unsupported template argument in GPUMatrix");            
        }
        hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_HOST);
    }

    template<class ElemType> //c = alpha * a
    void GPUMatrix<ElemType>::Scale(ElemType alpha, const GPUMatrix<ElemType>& a, GPUMatrix<ElemType>& c)
    {
        if (a.IsEmpty())
            throw std::logic_error("Scale:  Input matrix a is empty.");

        c=a;
        Scale(alpha,c);
    }


    template<class ElemType>
    void GPUMatrix<ElemType>::InnerProduct (const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c, const bool isColWise)
    {
        if (a.GetComputeDeviceId()!=b.GetComputeDeviceId() || b.GetComputeDeviceId()!=c.GetComputeDeviceId()) //different GPUs
            throw std::invalid_argument("All matrices must be on the same GPU");

        if (a.IsEmpty() || b.IsEmpty())
            throw std::logic_error("Scale:  one of the input matrices is empty.");

        const int m = (int)a.GetNumRows();
        const int n = (int)a.GetNumCols();
        const int k = (int)b.GetNumRows();
        const int l = (int)b.GetNumCols();

        assert (m>0 && n>0 && k>0 && l>0); //converting from size_t to int may cause overflow
        assert (m==k && n==l); //converting from size_t to int may cause overflow
        if (m!=k || n!=l)
            throw std::invalid_argument("Matrices a and b should have same dimension.");

        if (isColWise)
            c.Resize(1,n);
        else
            c.Resize(m,1);

        if ((isColWise && m == 1) || !isColWise && n == 1)  //in this case it's equivalent to element-wise product
        {
            c.AssignElementProductOf(a, b);
        }
        else 
        {
            hipEvent_t done = nullptr;  
            c.PrepareDevice();

            int blocksPerGrid=0;
            if (isColWise)  //col-wise
            {
                c.Resize(1,n);   
                blocksPerGrid =(int)ceil(1.0*n/threadsPerBlock);                                        
            }
            else
            {
                c.Resize(m, 1);
                blocksPerGrid =(int)ceil(1.0*m/threadsPerBlock);                        
            }       

            if (do_sync)    CUDA_CALL(hipEventCreate(&done));  
            _innerProduct<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(c.m_pArray, a.m_pArray,b.m_pArray,m,n,isColWise);
            if (do_sync)    CUDA_CALL(hipEventRecord(done));        
            if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
            if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        }             
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::InnerProductOfMatrices(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.IsEmpty() || b.IsEmpty())
            throw std::logic_error("InnerProductOfMatrices:  one of the input matrices is empty.");

        const int m = (int)a.GetNumRows();
        const int n = (int)a.GetNumCols();
        const int k = (int)b.GetNumRows();
        const int l = (int)b.GetNumCols();

        assert (m>0 && n>0 && k>0 && l>0); //converting from size_t to int may cause overflow
        assert (m==k && n==l); //converting from size_t to int may cause overflow
        if (m!=k || n!=l)
            throw std::invalid_argument("InnerProductOfMatrices: Matrices a and b should have same dimension.");

        hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
        if (sizeof(ElemType) == sizeof(double))
        {
            double tmp=0;                        
            CUBLAS_CALL(hipblasDdot(cuHandle,m*n, reinterpret_cast <double*>(a.m_pArray), 1, reinterpret_cast <double*>(b.m_pArray), 1,&tmp));
            return ElemType(tmp);
            //return (ElemType)ddot((int)a.GetNumElements(), reinterpret_cast <double*>(a.m_pArray), 1, reinterpret_cast <double*>(b.m_pArray), 1);
        }
        else
        {
            float tmp=0;                        
            CUBLAS_CALL(hipblasSdot(cuHandle,m*n, reinterpret_cast <float*>(a.m_pArray), 1, reinterpret_cast <float*>(b.m_pArray), 1,&tmp));
            return tmp;
            //return (ElemType)sdot((int)a.GetNumElements(), reinterpret_cast <float*>(a.m_pArray), 1, reinterpret_cast <float*>(b.m_pArray), 1);
        }
    }


    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignInnerProductOfMatrices(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b)
    {
        if (a.IsEmpty() || b.IsEmpty())
            throw std::logic_error("InnerProductOfMatrices:  one of the input matrices is empty.");        

        Resize(1,1);

        const int m = (int)a.GetNumRows();
        const int n = (int)a.GetNumCols();
        const int k = (int)b.GetNumRows();
        const int l = (int)b.GetNumCols();

        assert (m>0 && n>0 && k>0 && l>0); //converting from size_t to int may cause overflow
        assert (m==k && n==l); //converting from size_t to int may cause overflow
        if (m!=k || n!=l)
            throw std::invalid_argument("InnerProductOfMatrices: Matrices a and b should have same dimension.");

        hipblasHandle_t cuHandle = GetCublasHandle(a.GetComputeDeviceId());
        hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_DEVICE);
        if (sizeof(ElemType) == sizeof(double))
        {   
            CUBLAS_CALL(hipblasDdot(cuHandle,m*n, reinterpret_cast <double*>(a.m_pArray), 1, reinterpret_cast <double*>(b.m_pArray), 1,reinterpret_cast <double*>(m_pArray)));                    
        }
        else
        {   
            CUBLAS_CALL(hipblasSdot(cuHandle,m*n, reinterpret_cast <float*>(a.m_pArray), 1, reinterpret_cast <float*>(b.m_pArray), 1,reinterpret_cast <float*>(m_pArray)));                      
        }
        hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_HOST);
        return *this;
    }


    template<class ElemType>
    void GPUMatrix<ElemType>::ElementWisePower(ElemType alpha, const GPUMatrix<ElemType>& a, GPUMatrix<ElemType>& c)
    {
        if (a.GetComputeDeviceId() != c.GetComputeDeviceId())
        {
            throw std::invalid_argument("All matrices must be on the same GPU");
        }
        else 
        {
            if (a.IsEmpty())
                throw std::logic_error("ElementWisePower:  The input matrix a is empty.");
            if (a.GetNumRows()!=c.GetNumRows() || a.GetNumCols()!=c.GetNumCols())
                throw std::logic_error("ElementWisePower: matrices must be of the same size");

            hipEvent_t done = nullptr;
            a.PrepareDevice();
            if (do_sync)    CUDA_CALL(hipEventCreate(&done));            
            LONG64 N=(LONG64)a.GetNumElements();
            int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);                
            _elementWisePowerOnCuda<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(alpha,a.m_pArray,c.m_pArray,N);
            if (do_sync)    CUDA_CALL(hipEventRecord(done));        
            if (do_sync)    CUDA_CALL(hipEventSynchronize(done)); 
            if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        }
    }

    template<class ElemType>
    bool GPUMatrix<ElemType>::AreEqual(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const ElemType threshold /*= 1e-8*/)
    {
        if (a.IsEmpty() || b.IsEmpty())
            throw std::logic_error("AreEqual: one of the input matrices is empty.");

        if (a.GetNumRows()  != b.GetNumRows() || a.GetNumCols() != b.GetNumCols())
            return false;

        a.PrepareDevice();
        long *res = new long[1];
        res[0]=1;
        long *d_res = NULL;
        CUDA_CALL(hipMalloc((void**)&d_res,sizeof(long)*1));
        CUDA_CALL(hipMemcpy(d_res,res,sizeof(long)*1,hipMemcpyHostToDevice));
        long N=(long)a.GetNumElements();
        int blocksPerGrid =(int)ceil(1.0*N/threadsPerBlock);
        _areEqual<ElemType><<<blocksPerGrid,threadsPerBlock,0,t_stream>>>(a.m_pArray,b.m_pArray,N,threshold,d_res);
        CUDA_CALL(hipMemcpy(res,d_res,sizeof(long)*1,hipMemcpyDeviceToHost));
        if (res[0]!=0)
            return true;
        else
            return false;
    }

    template<class ElemType>
    GPUMatrix<ElemType>  GPUMatrix<ElemType>::Ones(const size_t rows, const size_t cols)
    {
        GPUMatrix<ElemType> c(rows, cols); //will initialize to 0
        c.SetValue(1);
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType>  GPUMatrix<ElemType>::Zeros(const size_t rows, const size_t cols)
    {
        GPUMatrix<ElemType> c(rows, cols); //will initialize to 0
        //c.SetValue(0);
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType>  GPUMatrix<ElemType>::Eye(const size_t rows)
    {
        GPUMatrix<ElemType> c(rows, rows); //will initialize to 0
        c.SetDiagonalValue(1);
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType>  GPUMatrix<ElemType>::RandomUniform(const size_t rows, const size_t cols, const ElemType low, const ElemType high, unsigned long seed)
    {
        GPUMatrix<ElemType> c(rows, cols); //will initialize to 0
        c.SetUniformRandomValue(low, high, seed);
        return c;
    }

    template<class ElemType>
    GPUMatrix<ElemType> GPUMatrix<ElemType>::RandomGaussian(const size_t rows, const size_t cols, const ElemType mean, const ElemType sigma, unsigned long seed)
    {
        GPUMatrix<ElemType> c(rows, cols); //will initialize to 0
        c.SetGaussianRandomValue(mean, sigma, seed);
        return c;
    }

    template<class ElemType>
    ElemType GPUMatrix<ElemType>::GetLearnRateForBlock_Helper(const GPUMatrix<ElemType> &Gradients, const GPUMatrix<ElemType> &SmoothedGradients)
    {                
        Gradients.PrepareDevice();
        ElemType* d_res=NULL;
        CUDA_CALL(hipMalloc((void**)&d_res,sizeof(ElemType))); //we allocate memory on the device

        //Compute inner product of matrices and keep it on device
        const int m = (int)Gradients.GetNumRows();
        const int n = (int)Gradients.GetNumCols();
        const int k = (int)SmoothedGradients.GetNumRows();
        const int l = (int)SmoothedGradients.GetNumCols();
        assert (m>0 && n>0 && k>0 && l>0); //converting from size_t to int may cause overflow
        assert (m==k && n==l); //converting from size_t to int may cause overflow
        if (m!=k || n!=l) throw std::invalid_argument("InnerProductOfMatrices: Matrices a and b should have same dimension.");

        if (sizeof(ElemType) == sizeof(double))
        {                 
            hipblasHandle_t cuHandle = GetCublasHandle(Gradients.GetComputeDeviceId());
            hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_DEVICE);
            CUBLAS_CALL(hipblasDdot(cuHandle,m*n, reinterpret_cast <double*>(Gradients.m_pArray), 1, reinterpret_cast <double*>(SmoothedGradients.m_pArray), 1,reinterpret_cast <double*>(d_res)));
            hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_HOST);
        }
        else
        {            
            hipblasHandle_t cuHandle = GetCublasHandle(Gradients.GetComputeDeviceId());
            hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_DEVICE);
            CUBLAS_CALL(hipblasSdot(cuHandle,m*n, reinterpret_cast <float*>(Gradients.m_pArray), 1, reinterpret_cast <float*>(SmoothedGradients.m_pArray), 1,reinterpret_cast <float*>(d_res)));
            hipblasSetPointerMode(cuHandle, HIPBLAS_POINTER_MODE_HOST);
        }
        // d_res[0] should now contain inner product of matrices
        // Compute squared Frobenius norms (squared sums of elements)       
        _lrHelper<ElemType><<<1,512,0,t_stream>>>(Gradients.m_pArray,SmoothedGradients.m_pArray, (LONG64)Gradients.GetNumElements(), d_res);
        ElemType res;
        CUDA_CALL(hipMemcpy(&res,d_res,sizeof(ElemType),hipMemcpyDeviceToHost));
        CUDA_CALL(hipFree(d_res));
        return res;
    }
    // The inputs are two row vectors [a1 a2 a3 a4] [b1 b2 b3 b4]
    // The outputs are one matrix of size (nt+1)*4
    // The first row is just element multiplication
    // The rest rows will be with shift
    template<class ElemType>
    GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementProductOfWithShiftNeg(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const size_t shift, const size_t nt)
    {
        if (a.IsEmpty() || b.IsEmpty())
            throw std::logic_error("AssignElementProductOf: Matrix is empty.");

        assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());
        if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
            throw std::invalid_argument("The input matrix dimensions do not match.");

        if (!(a.GetNumRows() == 1))
            throw std::invalid_argument("The input matrix must be a row vector.");

        Resize(nt + 1, a.GetNumCols());
        int BS = a.GetNumCols();

        // the output matrix is of size (nt+1, BS)
        dim3 thread_tail(DEFAULT_THREAD_PER_DIM, DEFAULT_THREAD_PER_DIM);
        dim3 block_tail((nt + 1 + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM, (BS + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM);

        a.PrepareDevice();
        hipEvent_t done = nullptr;;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        _assignElementProductOfWithShiftNeg<ElemType> << < block_tail, thread_tail, 0, t_stream >> >(m_pArray, a.m_pArray, b.m_pArray, shift, nt + 1, BS);
        //		_assignElementProductOf<ElemType> << <block_tail, thread_tail, 0, t_stream >> >(m_pArray, a.m_pArray, b.m_pArray, nt);

        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
        return *this;
    }

	template<class ElemType>
	void GPUMatrix<ElemType>::InnerProductWithShiftNeg(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c, const size_t shift, const size_t nt)
	{
		if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || b.GetComputeDeviceId() != c.GetComputeDeviceId()) //different GPUs
			throw std::invalid_argument("All matrices must be on the same GPU");

		if (a.IsEmpty() || b.IsEmpty())
			throw std::logic_error("Scale:  one of the input matrices is empty.");

		const int m = (int)a.GetNumRows();
		const int n = (int)a.GetNumCols();
		const int k = (int)b.GetNumRows();
		const int l = (int)b.GetNumCols();

		assert(m>0 && n>0 && k>0 && l>0); //converting from size_t to int may cause overflow
		assert(m == k && n == l); //converting from size_t to int may cause overflow
		if (m != k || n != l)
			throw std::invalid_argument("Matrices a and b should have same dimension.");

		c.Resize(nt + 1, n);

		if (true)
		{
            hipEvent_t done = nullptr;;
			c.PrepareDevice();

			dim3 thread_tail(DEFAULT_THREAD_PER_DIM, DEFAULT_THREAD_PER_DIM);
			dim3 block_tail((nt + 1 + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM, (n + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM);


			if (do_sync)    CUDA_CALL(hipEventCreate(&done));
			_innerProductWithShiftNeg<ElemType> << <block_tail, thread_tail, 0, t_stream >> >(c.m_pArray, a.m_pArray, b.m_pArray, m, n, shift, nt + 1);
			if (do_sync)    CUDA_CALL(hipEventRecord(done));
			if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
			if (do_sync)    CUDA_CALL(hipEventDestroy(done));
		}
	}

	template<class ElemType>
	GPUMatrix<ElemType>& GPUMatrix<ElemType>::GetARowByIndex(const GPUMatrix<ElemType>& a, const size_t m)
	{
		if (a.IsEmpty())
			throw std::logic_error("GetARowByIndex: Matrix is empty.");

		Resize(1, a.GetNumCols());

		int n = a.GetNumRows();
		int P = a.GetNumCols();

		if (m >= n)
			throw std::logic_error("GetARowByIndex: m is out of range.");


		int blocksPerGrid = (int)ceil(((double)P) / threadsPerBlock);

		a.PrepareDevice();
        hipEvent_t done = nullptr;;
		if (do_sync)    CUDA_CALL(hipEventCreate(&done));
		_getARowByIndex<ElemType> << <blocksPerGrid, threadsPerBlock, 0, t_stream >> >(m_pArray, a.m_pArray, n, P, m);
		//		_assignElementProductOf<ElemType> << <block_tail, thread_tail, 0, t_stream >> >(m_pArray, a.m_pArray, b.m_pArray, nt);

		if (do_sync)    CUDA_CALL(hipEventRecord(done));
		if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
		if (do_sync)    CUDA_CALL(hipEventDestroy(done));
		return *this;
	}


	template<class ElemType>
	void GPUMatrix<ElemType>::ConductRowElementMultiplyWithShift(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, GPUMatrix<ElemType>& c, const size_t shift, const bool isafixed)
	{
		if (a.GetComputeDeviceId() != b.GetComputeDeviceId() || b.GetComputeDeviceId() != c.GetComputeDeviceId()) //different GPUs
			throw std::invalid_argument("All matrices must be on the same GPU");

		if (a.IsEmpty() || b.IsEmpty())
			throw std::logic_error("Scale:  one of the input matrices is empty.");

		const int m = (int)a.GetNumRows();
		const int n = (int)a.GetNumCols();
		const int O = (int)b.GetNumRows();
		const int P = (int)b.GetNumCols();

		assert(m>0 && n>0 && O>0 && P>0); //converting from size_t to int may cause overflow
		if (m != 1 || n != P)
			throw std::invalid_argument("Matrices a and b should have same dimension.");

		c.Resize(O, P);

		if (true)
		{
            hipEvent_t done = nullptr;;
			c.PrepareDevice();

			dim3 thread_tail(DEFAULT_THREAD_PER_DIM, DEFAULT_THREAD_PER_DIM);
			dim3 block_tail((O + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM, (P + DEFAULT_THREAD_PER_DIM - 1) / DEFAULT_THREAD_PER_DIM);


			if (do_sync)    CUDA_CALL(hipEventCreate(&done));
			_conductRowElementMultiplyWithShift<ElemType> << <block_tail, thread_tail, 0, t_stream >> >(c.m_pArray, a.m_pArray, b.m_pArray, O, P, shift, isafixed);
			if (do_sync)    CUDA_CALL(hipEventRecord(done));
			if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
			if (do_sync)    CUDA_CALL(hipEventDestroy(done));
		}
	}



	template<class ElemType>
	GPUMatrix<ElemType>& GPUMatrix<ElemType>::AssignElementProductOfWithShift(const GPUMatrix<ElemType>& a, const GPUMatrix<ElemType>& b, const size_t shift)
	{
		if (a.IsEmpty() || b.IsEmpty())
			throw std::logic_error("AssignElementProductOfWithShift: Matrix is empty.");

		assert(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols());
		if (!(a.GetNumRows() == b.GetNumRows() && a.GetNumCols() == b.GetNumCols()))
			throw std::invalid_argument("The input matrix dimensions do not match.");

		//int O = a.GetNumRows();
		int P = a.GetNumCols();

		Resize(1, P);
		LONG64 N = (LONG64)GetNumElements();
		int blocksPerGrid = (int)ceil(((double)N) / threadsPerBlock);
		a.PrepareDevice();
        hipEvent_t done = nullptr;;
		if (do_sync)    CUDA_CALL(hipEventCreate(&done));
		_assignElementProductOfWithShift<ElemType> << <blocksPerGrid, threadsPerBlock, 0, t_stream >> >(m_pArray, a.m_pArray, b.m_pArray, shift, N);
		if (do_sync)    CUDA_CALL(hipEventRecord(done));
		if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
		if (do_sync)    CUDA_CALL(hipEventDestroy(done));
		return *this;
	}



#pragma endregion Static BLAS Functions


    template class GPUMatrix<float>; 
    template class GPUMatrix<double>;
    template class DeviceBoundNumber<float>;
    template class DeviceBoundNumber<double>;

    template<class ElemType>
    hipblasHandle_t GPUMatrix<ElemType>::s_cuHandle[GPUMatrix<ElemType>::MaxGpus]={0};

    template<class ElemType>
    void* GPUMatrix<ElemType>::s_curandGenerator=NULL;    
}}}

// !!!!This is from hip/hip_runtime_api.h which comes with CUDA samples!!!! Consider if it is beneficial to just include all hip/hip_runtime_api.h
// TODO: This is duplicated in BestGpu.cpp
// Beginning of GPU Architecture definitions
int _ConvertSMVer2Cores(int major, int minor)
{
    // Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
    typedef struct
    {
        int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
        int Cores;
    } sSMtoCores;

    sSMtoCores nGpuArchCoresPerSM[] =
    {
        { 0x10,  8 }, // Tesla Generation (SM 1.0) G80 class
        { 0x11,  8 }, // Tesla Generation (SM 1.1) G8x class
        { 0x12,  8 }, // Tesla Generation (SM 1.2) G9x class
        { 0x13,  8 }, // Tesla Generation (SM 1.3) GT200 class
        { 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
        { 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
        { 0x30, 192}, // Kepler Generation (SM 3.0) GK10x class
        { 0x35, 192}, // Kepler Generation (SM 3.5) GK11x class
        {   -1, -1 }
    };

    int index = 0;

    while (nGpuArchCoresPerSM[index].SM != -1)
    {
        if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
        {
            return nGpuArchCoresPerSM[index].Cores;
        }

        index++;
    }
    return nGpuArchCoresPerSM[7].Cores;
};
// end of GPU Architecture definitions

//inline long _GetFreeMemoryOnCUDADevice(int devId)
//{   
//    hipDevice_t cudaDevice;  
//    hipError_t result = hipDeviceGet(&cudaDevice, devId);  
//    if(result!= hipSuccess)  
//    {          
//        return 0;         
//    }  
//  
//    //create cuda context  
//    hipCtx_t cudaContext;    
//    result = hipCtxCreate(&cudaContext, hipDeviceScheduleAuto, cudaDevice);  
//    if(result != hipSuccess)  
//    {          
//        return 0;         
//    }  
//  
//    //get the amount of free memory on the graphics card  
//    size_t free;  
//    size_t total;  
//    result = hipMemGetInfo(&free, &total);  
//    if (result!=hipSuccess)
//    {
//        return 0;
//    }
//    else
//        return (long)free;
//}

#endif // CPUONLY
