#include "hip/hip_runtime.h"
//
// <copyright file="GPUMatrixCUDAKernels.cpp" company="Microsoft">
//     Copyright (c) Microsoft Corporation.  All rights reserved.
// </copyright>
//
#include <float.h>
#include <hip/hip_runtime.h>
#include "CommonMatrix.h"

#ifndef LONG64  //we would like to use 64-bit long to support large matrices. However, CUDA seems to support only 32-bit long
#define LONG64  long
#endif

#define IDX2C(i,j,ld) (((j)*(ld))+(i)) // 0 based indexing
#define threadsPerBlock 512

#define LZERO  -10e10
#define MINLOGEXP -9.2103
#define LSMALL -0.5E10

//CUDA Kernels code
template<class ElemType>
__global__ void _elementWisePowerOnCuda(
    ElemType alpha,     
    const ElemType *a, 
    ElemType* c,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (alpha==0)
    {
        c[id]=1;
    }
    else if (alpha==1)
    {
        c[id]=a[id];
    }
    else if (alpha==2)
    {
        c[id]=a[id]*a[id];
    }
    else if (alpha==3)
    {
        c[id]=a[id]*a[id]*a[id];
    }
    else
    {
        if (sizeof(ElemType)==sizeof(double))
        {
            c[id]=pow(a[id],alpha);
        }
        else
        {
            c[id]=powf(a[id],alpha);
        }
    }    
};

template<class ElemType>
__global__ void _inplaceSigmoidOnCuda(    
    ElemType* c,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (sizeof(ElemType)==sizeof(double))
    {
        if (c[id]>=0)
        {
            double e = exp(-1*c[id]);
            c[id]=1/(1+e);
        }
        else
        {
            double e = exp(c[id]);
            c[id]=e/(1+e);
        }
    }
    else
    {
        if (c[id]>=0)
        {
            float e = expf(-1*c[id]);
            c[id]=1/(1+e);
        }
        else
        {
            float e = exp(c[id]);
            c[id]=e/(1+e);
        }
    }
};

template<class ElemType>
__global__ void _assignSigmoidOf(    
    const ElemType* a,
    ElemType* res,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (sizeof(ElemType)==sizeof(double))
    {
        if (a[id]>=0)
        {
            double e = exp(-1*a[id]);
            res[id]=1/(1+e);
        }
        else
        {
            double e = exp(a[id]);
            res[id]=e/(1+e);
        }
    }
    else
    {
        if (a[id]>=0)
        {
            float e = expf(-1*a[id]);
            res[id]=1/(1+e);
        }
        else
        {
            float e = exp(a[id]);
            res[id]=e/(1+e);
        }
    }
};

template<class ElemType>
__global__ void _inplaceLinRectDerivative(    
    ElemType* c,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (c[id]<=0)
        c[id]=0;
    else
        c[id]=1;
}

template<class ElemType>
__global__ void _assignSigmoidDerivative( 
    ElemType *a,
    ElemType *c,
    LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    c[id] = a[id] * (1-a[id]);
}

template<class ElemType>
__global__ void _inplaceTanhOnCuda(    
    ElemType* c,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (sizeof(ElemType)==sizeof(double))
    {
        c[id]=tanh(c[id]);
    }
    else
    {
        c[id]=tanhf(c[id]);
    }

};

//to prevent negative values caused by floating operations, we force inputs to be >=0
//this may, however, hide problems in the caller.
template<class ElemType>
__global__ void _inplaceSqrtOnCuda(    
    ElemType* c,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (sizeof(ElemType)==sizeof(double))
    {
        c[id]=sqrt(max((ElemType)0, c[id]));
    }
    else
    {
        c[id]=sqrtf(max(ElemType(0), c[id]));
    }
};

template<class ElemType>
__global__ void _inplaceExpOnCuda(    
    ElemType* c,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (sizeof(ElemType)==sizeof(double))
    {
        c[id]=exp(c[id]);
    }
    else
    {
        c[id]=expf(c[id]);
    }
};

template<class ElemType>
__global__ void _inplaceLogOnCuda(    
    ElemType* c,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (c[id]<EPS_IN_LOG)
    {
        c[id]=LOG_OF_EPS_IN_LOG;
    }
    else
    {
        if (sizeof(ElemType)==sizeof(double))
        {
            c[id]=log(c[id]);
        }
        else
        {
            c[id]=logf(c[id]);
        }
    }
};

template<class ElemType>
__global__ void _inplaceAbsOnCuda(    
    ElemType* c,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (sizeof(ElemType)==sizeof(double))
    {
        c[id]=fabs(c[id]);
    }
    else
    {
        c[id]=fabsf(c[id]);
    }
};

template<class ElemType>
__global__ void _inplaceCosineOnCuda(    
    ElemType* c,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (sizeof(ElemType)==sizeof(double))
    {
        c[id]=cos(c[id]);
    }
    else
    {
        c[id]=cosf(c[id]);
    }
};

template<class ElemType>
__global__ void _inplaceNegativeSineOnCuda(    
    ElemType* c,    
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (sizeof(ElemType)==sizeof(double))
    {
        c[id]=-sin(c[id]);
    }
    else
    {
        c[id]=-sinf(c[id]);
    }
};


template<class ElemType>
__global__ void _setValue(    
    ElemType* a,
    const ElemType v,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    a[id]=v;
};

template<class ElemType>
__global__ void _setValue(    
    ElemType* a,
    const ElemType* d_v,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    a[id]=d_v[0];
};

template<class ElemType>
__global__ void _assignRowSliceValuesOf(ElemType * dest, ElemType * src, const LONG64 N, const long startIndex, const long destRows, const long srcRows)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;

    long col = id / destRows;
    long row = id - (col * destRows);

    //dest[id] = src[col*srcRows + row + startIndex];
    dest[id] = src[IDX2C(row + startIndex, col, srcRows)];
}

template<class ElemType>
__global__ void _addToRowSliceValuesOf(ElemType * dest, ElemType * src, const LONG64 N, const long startIndex, const long destRows, const long srcRows)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;

    long col = id / srcRows;  //src is the full matrix, rowslice is taken from the dest
    long row = id - (col * srcRows);

    //dest[col*destRows + row + startIndex] += src[id];
    dest[IDX2C(row + startIndex, col, destRows)] += src[id];
}

template<class ElemType>
__global__ void _addWithRowSliceValuesOf(ElemType * dest, ElemType * src, const LONG64 N, const long startIndex, const long destRows, const long srcRows)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= N)
        return;

    long col = id / destRows;  //dest is the full matrix, rowslice is taken from the src
    long row = id - (col * destRows);

    dest[id] += src[IDX2C(row + startIndex, col, srcRows)];
}

template<class ElemType>
__global__ void _assignRepeatOf(ElemType * dest, ElemType * src, const LONG64 N, const long srcRows, const long srcCols, const long destRows)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= N)
        return;

    long destCol = id / destRows;
    long destRow = id - (destCol * destRows);
    long srcRow = destRow % srcRows;
    long srcCol = destCol % srcCols;

    dest[id] = src[IDX2C(srcRow,srcCol,srcRows)];
}

template<class ElemType>
__global__ void _assignDifferenceOf1(
    ElemType* us,
    const ElemType alpha,
    const ElemType* a,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    us[id]=alpha-a[id];
};

template<class ElemType>
__global__ void _assignDifferenceOf2(
    ElemType* us,
    const ElemType alpha,
    const ElemType* a,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    us[id]=a[id]-alpha;
};

///a is a scalar
template<class ElemType>
__global__ void _scaleAndAddScalar(
    ElemType* c,
    const LONG64 N,
    const ElemType alpha,
    const ElemType* a
)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    c[id] += alpha*a[0];
};

template<class ElemType>
__global__ void _addValue(    
    ElemType* a,
    const ElemType v,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    a[id]+=v;
};

template<class ElemType>
__global__ void _addValue(    
    ElemType* a,
    const ElemType* d_v,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    a[id]+=d_v[0];
};


template<class ElemType>
__global__ void _elemMul(    
    ElemType* a,
    const ElemType* b,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    a[id]*=b[id];
};

template<class ElemType>
__global__ void _assignElementProductOf(
    ElemType* us,
    const ElemType* a,
    const ElemType* b,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    us[id]=a[id]*b[id];
}

template<class ElemType>
__global__ void _assignKhatriRaoProductOf(
    ElemType* us,
    const ElemType* a,
    const ElemType* b,
    const long rowsA, 
    const long rowsB, 
    const long cols)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;

    const long rows = rowsA * rowsB;
    const long col = id / rows;
    if (col >= cols) 
        return; 

    const long row = id % rows;
    const long rowB = row / rowsA; 
    const long rowA = row % rowsA;

    us[id] = a[rowA + col * rowsA] * b[rowB + col * rowsB];
}

template<class ElemType>
__global__ void _addColumnReshapeProductOf(
    ElemType* us,
    const ElemType* a,
    const ElemType* b,
    const long rowsB, 
    const long rowsC, 
    const long cols,
    const bool transposeAColumn)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;

    const long col = id / rowsC;
    if (col >= cols) 
        return; 

    const long row = id % rowsC;
    long bBase = col * rowsB;
    long aBase = bBase * rowsC;
    ElemType v = 0;

    if (transposeAColumn)
    {
        aBase += row * rowsB;
        for (long i=0; i<rowsB; i++)
        {
            v += a[aBase++] * b[bBase++];
        }
    }
    else
    {
        aBase += row;
        for (long i=0; i<rowsB; i++)
        {
            v += a[aBase] * b[bBase++];
            aBase += rowsC;
        }
    }
    us[row + col * rowsC] += v;
}

template<class ElemType>
__global__ void _assignElementDivisionOf(
    ElemType* us,
    const ElemType* a,
    const ElemType* b,
    const LONG64 N)
{
    ElemType smallValue = EPS_IN_INVERSE;

    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;

    ElemType v = b[id];

    if (v <0 && v > -smallValue)
        us[id] = a[id]/(-smallValue);
    else if (v >=0 && v < smallValue)
        us[id] = a[id]/smallValue;
    else
        us[id]=a[id]/v;
}

template<class ElemType>
__global__ void _elemInverse(
    ElemType* us,
    const LONG64 N)
{
    ElemType smallValue = EPS_IN_INVERSE;

    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;

    if (us[id] <0 && us[id] > -smallValue)
        us[id] = 1/-smallValue;
    else if (us[id] >=0 && us[id] < smallValue)
        us[id] = 1/smallValue;
    else
        us[id]=1/us[id];
}

template<class ElemType>
__global__ void _logSoftMaxColWise(
    ElemType *a,
    const long m_numCols,
    const long m_numRows) //ld
{
    int col_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (col_id>=m_numCols)
        return;

    __shared__ ElemType maxV[threadsPerBlock];
    __shared__ ElemType Sum[threadsPerBlock];
    maxV[threadIdx.x]=a[IDX2C(0,col_id,m_numRows)];
    Sum[threadIdx.x]=0;

    for (long i=0;i<m_numRows;++i)
    {
        if (a[IDX2C(i,col_id,m_numRows)]>maxV[threadIdx.x])
        {
            maxV[threadIdx.x]=a[IDX2C(i,col_id,m_numRows)];
        }
    }

    for (long i=0;i<m_numRows;++i)
    {
		ElemType tmp = a[IDX2C(i,col_id,m_numRows)]-maxV[threadIdx.x];
		Sum[threadIdx.x] += (sizeof(ElemType)==sizeof(float) ? expf(tmp) : exp(tmp));
	}
	Sum[threadIdx.x] = maxV[threadIdx.x] + (sizeof(ElemType)==sizeof(float)?logf(Sum[threadIdx.x]):log(Sum[threadIdx.x]));
    for (long i=0;i<m_numRows;++i)
    {
        a[IDX2C(i,col_id,m_numRows)] -= Sum[threadIdx.x] ;
    }
}

//template<class ElemType>
//__global__ void _assignColumnwiseSoftmaxOf(
//    const ElemType *a,
//    ElemType* us,
//    const long m_numCols,
//    const long m_numRows) //thead per column
//{
//    int col_id = blockDim.x * blockIdx.x + threadIdx.x;
//    if (col_id>=m_numCols)
//        return;
//
//    __shared__ ElemType maxV[threadsPerBlock];
//    __shared__ ElemType Sum[threadsPerBlock];
//    maxV[threadIdx.x]=a[IDX2C(0,col_id,m_numRows)];
//    Sum[threadIdx.x]=0;
//
//    for (long i=0;i<m_numRows;++i)
//    {
//        if (a[IDX2C(i,col_id,m_numRows)]>maxV[threadIdx.x])
//        {
//            maxV[threadIdx.x]=a[IDX2C(i,col_id,m_numRows)];
//        }
//    }
//
//    for (long i=0;i<m_numRows;++i)
//    {
//        if (sizeof(ElemType)==sizeof(float))
//        {
//            us[IDX2C(i,col_id,m_numRows)] = expf(a[IDX2C(i,col_id,m_numRows)]-maxV[threadIdx.x]);
//        }
//        else
//        {
//            us[IDX2C(i,col_id,m_numRows)] = exp(a[IDX2C(i,col_id,m_numRows)]-maxV[threadIdx.x]);
//        }
//        Sum[threadIdx.x] +=  us[IDX2C(i,col_id,m_numRows)];
//    }
//
//    for (long i=0;i<m_numRows;++i)
//    {
//        us[IDX2C(i,col_id,m_numRows)] /= Sum[threadIdx.x] ;
//    }
//}

template<class ElemType>
__global__ void _assignColumnwiseLogSoftmaxOf(
    const ElemType *a,
    ElemType* us,
    const long m_numCols,
    const long m_numRows) // each block processes one column. There must be 512 threads in a block
{
    //we first find max per column
    __shared__ ElemType colMax[1];
    __shared__ ElemType partials[512];    
    colMax[0]=-10000000;
    partials[threadIdx.x]=-10000000;

    //int id = blockDim.x * blockIdx.x + threadIdx.x;
    int loadPerThread = m_numRows/blockDim.x; 

    for (int i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? m_numRows : (threadIdx.x+1)*loadPerThread);++i)
    {
        partials[threadIdx.x]=max(partials[threadIdx.x],a[IDX2C(i,blockIdx.x,m_numRows)]);
    }
    __syncthreads();

    //256
    if (threadIdx.x<256)
    {
        partials[threadIdx.x]=max(partials[threadIdx.x+256],partials[threadIdx.x]);
    }
    __syncthreads();

    //128
    if (threadIdx.x<128)
    {
        partials[threadIdx.x]=max(partials[threadIdx.x+128],partials[threadIdx.x]);
    }
    __syncthreads();

    //64
    if (threadIdx.x<64)
    {
        partials[threadIdx.x]=max(partials[threadIdx.x+64],partials[threadIdx.x]);
    }
    __syncthreads();

    //32
    if (threadIdx.x<32)
    {
        partials[threadIdx.x]=max(partials[threadIdx.x+32],partials[threadIdx.x]);
    }
    __syncthreads();

    //16
    if (threadIdx.x<16)
    {
        partials[threadIdx.x]=max(partials[threadIdx.x+16],partials[threadIdx.x]);
    }
    __syncthreads();

    //8
    if (threadIdx.x<8)
    {
        partials[threadIdx.x]=max(partials[threadIdx.x+8],partials[threadIdx.x]);
    }
    __syncthreads();

    //4
    if (threadIdx.x<4)
    {
        partials[threadIdx.x]=max(partials[threadIdx.x+4],partials[threadIdx.x]);
    }
    __syncthreads();

    if (threadIdx.x==0)
    {
        colMax[0] = max(max(partials[0],partials[1]),max(partials[2],partials[3]));        
    }
    partials[threadIdx.x]=0.0f;
    __syncthreads();
    //end of finding max
    //now start finding sums
    __shared__ ElemType colSum[1];
    colSum[0]=0.0f;
    for (int i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? m_numRows : (threadIdx.x+1)*loadPerThread);++i)
    {
        ElemType tmp=a[IDX2C(i,blockIdx.x,m_numRows)]-colMax[0];
		us[IDX2C(i,blockIdx.x,m_numRows)]=tmp;
		partials[threadIdx.x]+=(sizeof(ElemType)==sizeof(float)?expf(tmp):exp(tmp));
    }
    __syncthreads();

    //256
    if (threadIdx.x<256)
    {
        partials[threadIdx.x]+=partials[threadIdx.x+256];
    }
    __syncthreads();

    //128
    if (threadIdx.x<128)
    {
        partials[threadIdx.x]+=partials[threadIdx.x+128];
    }
    __syncthreads();

    //64
    if (threadIdx.x<64)
    {
        partials[threadIdx.x]+=partials[threadIdx.x+64];
    }
    __syncthreads();

    //32
    if (threadIdx.x<32)
    {
        partials[threadIdx.x]+=partials[threadIdx.x+32];
    }
    __syncthreads();

    //16
    if (threadIdx.x<16)
    {
        partials[threadIdx.x]+=partials[threadIdx.x+16];
    }
    __syncthreads();

    //8
    if (threadIdx.x<8)
    {
        partials[threadIdx.x]+=partials[threadIdx.x+8];
    }
    __syncthreads();

    //4
    if (threadIdx.x<4)
    {
        partials[threadIdx.x]+=partials[threadIdx.x+4];
    }
    __syncthreads();

    if (threadIdx.x==0)
    {
        colSum[0] = partials[0]+partials[1]+partials[2]+partials[3];
		colSum[0] = (sizeof(ElemType)==sizeof(float)?logf(colSum[0]):log(colSum[0]));
    }
    __syncthreads();
    //end of finding sums
    for (int i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? m_numRows : (threadIdx.x+1)*loadPerThread);++i)
    {        
        us[IDX2C(i,blockIdx.x,m_numRows)]-=colSum[0];        
    }
}

template<class ElemType>
__global__ void _logSoftMaxRowWise(
    ElemType *a,
    const long m_numCols,
    const long m_numRows) //ld
{
    int row_id = blockDim.x * blockIdx.x + threadIdx.x;
    if (row_id>=m_numRows)
        return;

    __shared__ ElemType maxV[threadsPerBlock];
    __shared__ ElemType Sum[threadsPerBlock];
    maxV[threadIdx.x]=a[IDX2C(row_id,0,m_numRows)];
    Sum[threadIdx.x]=0;

    for (long j=0;j<m_numCols;++j)
    {
        if (a[IDX2C(row_id,j,m_numRows)]>maxV[threadIdx.x])
        {
            maxV[threadIdx.x]=a[IDX2C(row_id,j,m_numRows)];
        }
    }

    for (long j=0;j<m_numCols;++j)
    {
		ElemType tmp = a[IDX2C(row_id,j,m_numRows)]-maxV[threadIdx.x];
		Sum[threadIdx.x] += sizeof(ElemType)==sizeof(float) ? expf(tmp) : exp(tmp);
    }
	Sum[threadIdx.x] = maxV[threadIdx.x]+(sizeof(ElemType)==sizeof(float)?logf(Sum[threadIdx.x]):log(Sum[threadIdx.x]));
    for (long j=0;j<m_numCols;++j)
    {
        a[IDX2C(row_id,j,m_numRows)] -= Sum[threadIdx.x] ;
    }
}

template<class ElemType>
__global__ void _inplaceTruncateBottom(
    ElemType* a,
    const ElemType threshold,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (a[id]<threshold)
        a[id]=threshold;
}

template<class ElemType>
__global__ void _assignTruncateBottom(
    ElemType* us,
    const ElemType* a,
    const ElemType threshold,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (a[id]<threshold)
        us[id]=threshold;
    else
        us[id]=a[id];
}

template<class ElemType>
__global__ void _inplaceTruncateTop(
    ElemType* a,
    const ElemType threshold,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (a[id]>threshold)
        a[id]=threshold;
}

template<class ElemType>
__global__ void _assignTruncateTop(
    ElemType* us,
    const ElemType* a,
    const ElemType threshold,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (a[id]>threshold)
        us[id]=threshold;
    else
        us[id]=a[id];
}

template<class ElemType>
__global__ void _setToZeroIfAbsLessThan(
    ElemType* a,
    const ElemType threshold,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    if (sizeof(ElemType)==sizeof(float))
    {
        if (fabsf(a[id])<threshold)
            a[id]=0;
    }
    else
    {
        if (fabs(a[id])<threshold)
            a[id]=0;
    }
}

template<class ElemType>
__global__ void _areEqual(
    const ElemType* a,
    const ElemType* b,
    const LONG64 N,
    const ElemType threshold,
    long *d_res)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;

    if (sizeof(ElemType)==sizeof(float))
    {
        if (fabsf(a[id]-b[id]) > threshold) 
        {
            d_res[0]=0;
        }
    }
    else
    {
        if (fabs(1.0*a[id]-1.0*b[id]) > threshold) 
        {
            d_res[0]=0;
        }
    }

}

template<class ElemType>
__global__ void _setDiagonalValue(
    ElemType* a,
    const ElemType v,
    const unsigned long N,
    const unsigned long ld)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;    
    a[IDX2C(id,id,ld)]=v;

}

template<class ElemType>
__global__ void _setDiagonalValueFromVector(
    ElemType* a,
    const ElemType* b,
    const long N)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return; 
    a[IDX2C(id,id,N)]=b[id];
}

template<class ElemType>
__global__ void _adagrad(
    ElemType* a,
    ElemType* d_v,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= N)
        return;

    const ElemType floor = 1e-16f;

    a[id] += d_v[id] * d_v[id];
    d_v[id] /= sqrt(a[id]+floor);
}

template<class ElemType>
__global__ void _rmsprop_init(
	ElemType* avars, ElemType* signs, ElemType* steps,
	ElemType* curr_grad,
	const LONG64 N
	)
{
    LONG64 i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= N)
        return;

	ElemType tmp = curr_grad[i];
	avars[i] = tmp * tmp;
	signs[i] = ElemType(0.0);
	steps[i] = ElemType(0.02);
}

template<class ElemType>
__global__ void _rmsprop(
	ElemType* avars, ElemType* signs, ElemType* steps,
	ElemType* curr_grad,
	const LONG64 N,
	ElemType RMS_GAMMA,ElemType RMS_WGT_INC,ElemType RMS_WGT_MAX,ElemType RMS_WGT_DEC,ElemType RMS_WGT_MIN,
	ElemType floor,
	ElemType *upd_gpu
	)
{
    LONG64 i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i >= N)
        return;

	avars[i] = RMS_GAMMA * avars[i] + (ElemType(1.0)-RMS_GAMMA)* (curr_grad[i] * curr_grad[i]);

	//// grad sign base 3: 0->neg, 1->zero, 2->pos
	//const int grad_sign = 1 + (ElemType(0) < curr_grad[i]) - (curr_grad[i] < ElemType(0));

	//// signs[i] contains three consecutive grad_sign
	//signs[i]  = 3*(int(signs[i]) % 9) + grad_sign;

	//// update according to the following table:
	//// (!pos,!pos,!pos) or (!neg,!neg,!neg): RMS_WGT_INC
	//// (!neg,!neg,neg) or (!pos,!pos,pos): RMS_WGT_DEC
	//// otherwise: no action

	//switch(int(upd_gpu[int(signs[i])]))
	//{
	//case 0:
	//	steps[i] = max(steps[i] * RMS_WGT_DEC, RMS_WGT_MIN);
	//	break;
	//case 2:
	//	steps[i] = min(steps[i] * RMS_WGT_INC, RMS_WGT_MAX);
	//	break;
	//}
	//curr_grad[i] *= steps[i] / sqrt(avars[i] + floor);

	const int grad_sign = (ElemType(0) < curr_grad[i]) - (curr_grad[i] < ElemType(0));

	if( signs[i] * grad_sign > 0 )
		steps[i] = min(steps[i] * RMS_WGT_INC, RMS_WGT_MAX);
	else
		steps[i] = max(steps[i] * RMS_WGT_DEC, RMS_WGT_MIN);

	curr_grad[i] *= steps[i] / sqrt(avars[i] + floor);
	signs[i] = grad_sign;

}

template<class ElemType>
__global__ void _rescaleToRange(
    ElemType* a,
    const LONG64 N,
    const ElemType low,
    const ElemType high)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;    
    a[id]=a[id]*(high-low)+low;
}

template<class ElemType>
__global__ void _setMaskAndScale(
    ElemType* a,
    const LONG64 N,
    const ElemType maskRate,
    const ElemType scaleValue)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;    
    a[id]=a[id]<=maskRate? 0 : scaleValue;
}

template<class ElemType>
__global__ void _vectorNorm1(
    ElemType* c, //output
    const ElemType* a, //input
    const long n, //a.numRows
    const long m, //a.numCols
    const bool isColWise)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if ((isColWise && id>=m)||(!isColWise && id>=n))
        return;

    ElemType sum = 0;

    if (isColWise)
    {
        for (long i=0;i<n;++i)
        {
            if (sizeof(ElemType)==sizeof(float))
            {
                sum+=fabsf(a[IDX2C(i,id,n)]);
            }
            else
            {
                sum+=fabs(a[IDX2C(i,id,n)]);
            }
        }
    }
    else
    {
        for (long j=0;j<m;++j)
        {
            if (sizeof(ElemType)==sizeof(float))
            {
                sum+=fabsf(a[IDX2C(id,j,n)]);
            }
            else
            {
                sum+=fabs(a[IDX2C(id,j,n)]);
            }
        }
    }
    c[id]=sum;
}


//one column per thread
template<class ElemType>
__global__ void _vectorNorm2(
    ElemType* c,  //output
    const ElemType* a, //input
    const long N, //a.GetNumRows();
    const long M, //a.GetNumCols();
    const bool isColWise) 
{
    long id = blockDim.x * blockIdx.x + threadIdx.x;
    if ((isColWise && id>=M) || (!isColWise && id>=N))
        return;

    ElemType sum = 0;
    if (isColWise)
    {
        for (long i=0;i<N;++i)
        {
            ElemType v = a[IDX2C(i,id,N)];
            sum += v * v;
        }
    }
    else
    {
        for (long j=0;j<M;++j)
        {
            ElemType v = a[IDX2C(id,j,N)];
            sum += v * v;
        }
    }

    if (sizeof(ElemType) == sizeof(float))
        c[id] = sqrtf(sum);
    else
        c[id] = sqrt(sum);
}

template<class ElemType>
__global__ void _convertInd2ValsAdjustInd(
    ElemType* inds,
    const ElemType* M,
    ElemType* vals,    
    const long n, //number of cols
    const long m, //number of rows
    const bool isColWise)
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if ((isColWise && id>=n)||(!isColWise && id>=m))
        return;
    inds[id]--;
    if (isColWise)
    {
        vals[id]=M[IDX2C((int)inds[id],id,m)];
    }
    else
    {
        vals[id]=M[IDX2C(id,(int)inds[id],m)];
    }
}


    //assume each column is an input sample. Each sample is stored in [channel, row, col]  (r00, g00, b00, r01, g01, b01, r10, g10, b10, r11, g11, b11)
template<class ElemType>
__global__ void _assignPackedConvolutionInput(ElemType * packedMatrix, const ElemType * inputSubBatch, const long batchSize,
                                                 const long inputWidth, const long inputHeight, const long inputChannels,
                                                 const long outputWidth, const long outputHeight, const long outputChannels,
                                                 const long kernelWidth, const long kernelHeight, const long horizontalSubsample, const long verticalSubsample, const bool zeroPadding)
{
    const long inputHeightTimesChannel = inputHeight * inputChannels; 
    const size_t inputDim = inputWidth*inputHeightTimesChannel;

    const long idall = blockIdx.x * blockDim.x + threadIdx.x; 
    const long sample = idall / inputDim;
    if (sample >= batchSize) 
        return; 

    const long id = idall % inputDim;
    const long y = id / inputHeightTimesChannel; //inputCol

    const size_t packedInputRows = kernelWidth * kernelHeight * inputChannels;
    const size_t packedInputColsPerSample = outputWidth * outputHeight;  //output size per channel

    // IN_ELEM_ROWPOS(channel, row, col) = (channel + (row + col * inputHeight) * inputChannels)
    // IN_ELEM_COLPOS = sample

    const long nXC = id % inputHeightTimesChannel; //channel + inputRow*inputChannels
    const long x = nXC / inputChannels; //inputRow
    const long c = nXC % inputChannels; //channel

    ElemType currentInputValue = inputSubBatch[id + sample*inputDim]; 

    long x0 = 0, y0 = 0, x1 = 0, y1 = 0;
    if (zeroPadding)
    {
        const long halfKernelWidth = kernelWidth/2; 
        const long halfKernelHeight = kernelHeight/2; 

        x0 = max(0.0f, ceil((x-(ElemType)kernelHeight+1.0f+halfKernelHeight)/ (ElemType)verticalSubsample));  //row : first wrow in which x is in
        x1 = x+halfKernelHeight-x0*verticalSubsample;    //first posxInKernel
        y0 = max(0.0f, ceil((y-(ElemType)kernelWidth+1.0f+halfKernelWidth)/(ElemType)horizontalSubsample));  //col : first wcol in which y is in
        y1 = y+halfKernelWidth-y0*horizontalSubsample;  //first posyInKernel
    }
    else
    {
        x0 = max(0.0f, ceil((x-(ElemType)kernelHeight+1)/ (ElemType)verticalSubsample));  //row : first wrow in which x is in
        x1 = x-x0*verticalSubsample;    //first posxInKernel
        y0 = max(0.0f, ceil((y-(ElemType)kernelWidth+1)/(ElemType)horizontalSubsample));  //col : first wcol in which y is in
        y1 = y-y0*horizontalSubsample;  //first posyInKernel
    }

    // PACK_ELEM_ROWPOS(channel, posxInKernel, posyInKernel) = (channel * kernelWidth * kernelHeight + posxInKernel + posyInKernel * kernelHeight)
    // PACK_ELEM_COLPOS(sample, wrow, wcol) = (sample*packedInputColsPerSample + outputHeight*wcol + wrow

    long packColBase = sample*packedInputColsPerSample + y0*outputHeight; 
    for (long wcol = y0, posyInKernel = y1; wcol < outputWidth && posyInKernel>=0; wcol++, posyInKernel -= horizontalSubsample) 
    {
        long packRowBase = c * kernelWidth * kernelHeight + posyInKernel * kernelHeight;
        for (long wrow = x0, posxInKernel = x1; wrow < outputHeight && posxInKernel>=0; wrow++, posxInKernel -= verticalSubsample) 
        {
            const long packRow = packRowBase + posxInKernel; 
            const long packCol = packColBase + wrow; 
            packedMatrix[packRow + packCol*packedInputRows] = currentInputValue; 
        }
        packColBase += outputHeight; 
    }
}

    //assume each column is an input sample. Each sample is stored in [channel, row, col]  (r00, g00, b00, r01, g01, b01, r10, g10, b10, r11, g11, b11)
template<class ElemType>
__global__ void _unpackConvolutionInput(const ElemType * packedMatrix, ElemType * inputSubBatch, const long batchSize,
                                                 const long inputWidth, const long inputHeight, const long inputChannels,
                                                 const long outputWidth, const long outputHeight, const long outputChannels,
                                                 const long kernelWidth, const long kernelHeight, const long horizontalSubsample, const long verticalSubsample, const bool zeroPadding)
{
    const long inputHeightTimesChannel = inputHeight * inputChannels; 
    const size_t inputDim = inputWidth*inputHeightTimesChannel;

    const long idall = blockIdx.x * blockDim.x + threadIdx.x; 
    const long sample = idall / inputDim;
    if (sample >= batchSize) 
        return; 

    const long id = idall % inputDim;
    const long y = id / inputHeightTimesChannel; //inputCol

    const size_t packedInputRows = kernelWidth * kernelHeight * inputChannels;
    const size_t packedInputColsPerSample = outputWidth * outputHeight;  //output size per channel

    // IN_ELEM_ROWPOS(channel, row, col) = (channel + (row + col * inputHeight) * inputChannels)
    // IN_ELEM_COLPOS = sample

    const long nXC = id % inputHeightTimesChannel; //channel + inputRow*inputChannels
    const long x = nXC / inputChannels; //inputRow
    const long c = nXC % inputChannels; //channel

    long x0 = 0, y0 = 0, x1 = 0, y1 = 0;
    if (zeroPadding)
    {
        const long halfKernelWidth = kernelWidth/2; 
        const long halfKernelHeight = kernelHeight/2; 

        x0 = max(0.0f, ceil((x-(ElemType)kernelHeight+1.0f+halfKernelHeight)/ (ElemType)verticalSubsample));  //row : first wrow in which x is in
        x1 = x+halfKernelHeight-x0*verticalSubsample;    //first posxInKernel
        y0 = max(0.0f, ceil((y-(ElemType)kernelWidth+1.0f+halfKernelWidth)/(ElemType)horizontalSubsample));  //col : first wcol in which y is in
        y1 = y+halfKernelWidth-y0*horizontalSubsample;  //first posyInKernel
    }
    else
    {
        x0 = max(0.0f, ceil((x-(ElemType)kernelHeight+1)/ (ElemType)verticalSubsample));  //row : first wrow in which x is in
        x1 = x-x0*verticalSubsample;    //first posxInKernel
        y0 = max(0.0f, ceil((y-(ElemType)kernelWidth+1)/(ElemType)horizontalSubsample));  //col : first wcol in which y is in
        y1 = y-y0*horizontalSubsample;  //first posyInKernel
    }

    // PACK_ELEM_ROWPOS(channel, posxInKernel, posyInKernel) = (channel * kernelWidth * kernelHeight + posxInKernel + posyInKernel * kernelHeight)
    // PACK_ELEM_COLPOS(sample, wrow, wcol) = (sample*packedInputColsPerSample + outputHeight*wcol + wrow

    ElemType currentInputValue = inputSubBatch[id + sample*inputDim]; 
    long packColBase = sample*packedInputColsPerSample + y0*outputHeight; 
    for (long wcol = y0, posyInKernel = y1; wcol < outputWidth && posyInKernel>=0; wcol++, posyInKernel -= horizontalSubsample) 
    {
        long packRowBase = c * kernelWidth * kernelHeight + posyInKernel * kernelHeight;
        for (long wrow = x0, posxInKernel = x1; wrow < outputHeight && posxInKernel>=0; wrow++, posxInKernel -= verticalSubsample) 
        {
            const long packRow = packRowBase + posxInKernel; 
            const long packCol = packColBase + wrow; 
            currentInputValue += packedMatrix[packRow + packCol*packedInputRows]; 
        }
        packColBase += outputHeight; 
    }

    inputSubBatch[id + sample*inputDim] = currentInputValue; 
}

template<class ElemType>
__global__ void _assignMaxPoolingResult(ElemType * outputBatch, const ElemType * inputBatch, const long batchSize, const long channels,
                                                const long inputWidth, const long inputHeight,  const long inputSizePerSample, 
                                                const long outputWidth, const long outputHeight, const long outputSizePerSample, 
                                                const long windowWidth, const long windowHeight, const long horizontalSubsample, const long verticalSubsample)
{
    const long outputIndex = blockIdx.x * blockDim.x + threadIdx.x; 
    const long sample = outputIndex / outputSizePerSample; 
    if (sample >= batchSize) 
        return; 

    const long outputIndexWithinSample = outputIndex % outputSizePerSample; 
    const long inputHeightTimesChannel = inputHeight * channels; 
    const long outputHeightTimesChannel = outputHeight * channels; 


    // IN_ELEM_ROWPOS(channel, row, col) = (channel + (row + col * inputHeight) * channels)
    // IN_ELEM_COLPOS = sample

    // OUT_ELEM_ROWPOS(channel, wrow, wcol) = (channel + (wrow + wcol * outputHeight) * channels)
    // OUT_ELEM_COLPOS = sample

    const long y = outputIndexWithinSample / outputHeightTimesChannel; //wcol
    const long nXC = outputIndexWithinSample % outputHeightTimesChannel; //channel + wrow*channels
    const long x = nXC / channels; //wrow
    const long c = nXC % channels; //channel

    const ElemType *inputBatchBase4Sample = inputBatch + sample*inputSizePerSample;
    register ElemType maxVal = -FLT_MAX; 
    const long rowInWindowBase = (x*verticalSubsample + y*horizontalSubsample*inputHeight)*channels+c;
    for (long colInWindow=0; colInWindow<windowWidth; colInWindow++) 
    {   
        long rowInInput = rowInWindowBase + colInWindow * inputHeightTimesChannel;
        for (long rowInWindow=0; rowInWindow<windowHeight; rowInWindow++)
        {
            const ElemType val = inputBatchBase4Sample[rowInInput]; 
            maxVal = max(maxVal, val); 
            rowInInput += channels;
        }
    }
    outputBatch[outputIndexWithinSample + sample*outputSizePerSample] = maxVal; 
}

template<class ElemType>
__global__ void _addMaxPoolingGradient(ElemType * inputGradientBatch, const ElemType * outputGradientBatch, const ElemType * inputBatch, const ElemType * outputBatch, 
                                                const long batchSize, const long channels, 
                                                const long inputWidth, const long inputHeight, const long inputSizePerSample, 
                                                const long outputWidth, const long outputHeight, const long outputSizePerSample, 
                                                const long windowWidth, const long windowHeight, const long horizontalSubsample, const long verticalSubsample)
{
    const long inputIndex = blockIdx.x * blockDim.x + threadIdx.x; 
    const long sample = inputIndex / inputSizePerSample; 
    if (sample >= batchSize) 
        return; 
   
    const long inputIndexWithinSample = inputIndex % inputSizePerSample; 

    const long inputHeightTimesChannel = inputHeight * channels; 
    const long outputHeightTimesChannel = outputHeight * channels; 

    // IN_ELEM_ROWPOS(channel, row, col) = (channel + (row + col * inputHeight) * channels)
    // IN_ELEM_COLPOS = sample

    // OUT_ELEM_ROWPOS(channel, wrow, wcol) = (channel + (wrow + wcol * outputHeight) * channels)
    // OUT_ELEM_COLPOS = sample

    const long y = inputIndexWithinSample / inputHeightTimesChannel; //col in input
    const long nXC = inputIndexWithinSample % inputHeightTimesChannel; //channel + row*chanels
    const long x = nXC / channels; //row in input
    const long c = nXC % channels; //channel

    long startOutX = max(0.0f, ceil((x-(ElemType)windowHeight+1)/ (ElemType)verticalSubsample));  //inclusive start
    long endOutX = (x/verticalSubsample < outputHeight-1)? x/verticalSubsample : outputHeight-1; //inclusive end
    long startOutY = max(0.0f, ceil((y-(ElemType)windowWidth+1)/(ElemType)horizontalSubsample));  //inclusive start
    long endOutY = (x/horizontalSubsample < outputWidth-1)? x/horizontalSubsample : outputWidth-1; //inclusive end


    ElemType *inputGradientBatchBase4Sample = inputGradientBatch + sample*inputSizePerSample;
    const ElemType *outputGradientBatchBase4Sample = outputGradientBatch + sample*outputSizePerSample;
    const ElemType * outputBatchBase4Sample = outputBatch + sample*outputSizePerSample;

    ElemType inputValue = inputBatch[inputIndexWithinSample + sample*inputSizePerSample];
    for (long outY=startOutY; outY<=endOutY; outY++)
    {
        for (long outX=startOutX; outX<=endOutX; outX++)
        {
            long outputIndex = outY * outputHeightTimesChannel + outX * channels + c; 
            if (inputValue == outputBatchBase4Sample[outputIndex])
                inputGradientBatchBase4Sample[inputIndexWithinSample] += outputGradientBatchBase4Sample[outputIndex];
        }
    }  
}
template<class ElemType>
__global__ void _assignAveragePoolingResult(ElemType * outputBatch, const ElemType * inputBatch, const long batchSize, const long channels,
                                                const long inputWidth, const long inputHeight,  const long inputSizePerSample, 
                                                const long outputWidth, const long outputHeight, const long outputSizePerSample, 
                                                const long windowWidth, const long windowHeight, const long horizontalSubsample, const long verticalSubsample)
{
    const long outputIndex = blockIdx.x * blockDim.x + threadIdx.x; 
    const long sample = outputIndex / outputSizePerSample; 
    if (sample >= batchSize) 
        return; 

    const long outputIndexWithinSample = outputIndex % outputSizePerSample; 
    const long inputHeightTimesChannel = inputHeight * channels; 
    const long outputHeightTimesChannel = outputHeight * channels; 


    // IN_ELEM_ROWPOS(channel, row, col) = (channel + (row + col * inputHeight) * channels)
    // IN_ELEM_COLPOS = sample

    // OUT_ELEM_ROWPOS(channel, wrow, wcol) = (channel + (wrow + wcol * outputHeight) * channels)
    // OUT_ELEM_COLPOS = sample

    const long y = outputIndexWithinSample / outputHeightTimesChannel; //wcol
    const long nXC = outputIndexWithinSample % outputHeightTimesChannel; //channel + wrow*channels
    const long x = nXC / channels; //wrow
    const long c = nXC % channels; //channel

    const ElemType *inputBatchBase4Sample = inputBatch + sample*inputSizePerSample;

    register ElemType average = 0; 
    const long rowInWindowBase = (x*verticalSubsample + y*horizontalSubsample*inputHeight)*channels+c;
    for (long colInWindow=0; colInWindow<windowWidth; colInWindow++) 
    {   
        long rowInInput = rowInWindowBase + colInWindow * inputHeightTimesChannel;
        for (long rowInWindow=0; rowInWindow<windowHeight; rowInWindow++)
        {
            average += inputBatchBase4Sample[rowInInput]; 
            rowInInput += channels;
        }
    }

    outputBatch[outputIndexWithinSample + sample*outputSizePerSample] = average/windowWidth/windowHeight; 
}

template<class ElemType>
__global__ void _addAveragePoolingGradient(ElemType * inputGradientBatch, const ElemType * outputGradientBatch, 
                                                const long batchSize, const long channels, 
                                                const long inputWidth, const long inputHeight, const long inputSizePerSample, 
                                                const long outputWidth, const long outputHeight, const long outputSizePerSample, 
                                                const long windowWidth, const long windowHeight, const long horizontalSubsample, const long verticalSubsample)
{
    const long inputIndex = blockIdx.x * blockDim.x + threadIdx.x; 
    const long sample = inputIndex / inputSizePerSample; 
    if (sample >= batchSize) 
        return; 
   
    const long inputIndexWithinSample = inputIndex % inputSizePerSample; 

    const long inputHeightTimesChannel = inputHeight * channels; 
    const long outputHeightTimesChannel = outputHeight * channels; 
    const long windowSize = windowWidth * windowHeight;

    // IN_ELEM_ROWPOS(channel, row, col) = (channel + (row + col * inputHeight) * channels)
    // IN_ELEM_COLPOS = sample

    // OUT_ELEM_ROWPOS(channel, wrow, wcol) = (channel + (wrow + wcol * outputHeight) * channels)
    // OUT_ELEM_COLPOS = sample

    const long y = inputIndexWithinSample / inputHeightTimesChannel; //col in input
    const long nXC = inputIndexWithinSample % inputHeightTimesChannel; //channel + row*chanels
    const long x = nXC / channels; //row in input
    const long c = nXC % channels; //channel

    long startOutX = max(0.0f, ceil((x-(ElemType)windowHeight+1)/ (ElemType)verticalSubsample));  //inclusive start
    long endOutX = (x/verticalSubsample < outputHeight-1)? x/verticalSubsample : outputHeight-1; //inclusive end
    long startOutY = max(0.0f, ceil((y-(ElemType)windowWidth+1)/(ElemType)horizontalSubsample));  //inclusive start
    long endOutY = (x/horizontalSubsample < outputWidth-1)? x/horizontalSubsample : outputWidth-1; //inclusive end

    ElemType *inputGradientBatchBase4Sample = inputGradientBatch + sample*inputSizePerSample;
    const ElemType *outputGradientBatchBase4Sample = outputGradientBatch + sample*outputSizePerSample;

    for (long outY=startOutY; outY<=endOutY; outY++)
    {
        for (long outX=startOutX; outX<=endOutX; outX++)
        {
            long outputIndex = outY * outputHeightTimesChannel + outX * channels + c; 
            inputGradientBatchBase4Sample[inputIndexWithinSample] += outputGradientBatchBase4Sample[outputIndex]/windowSize;
        }
    }  
}

template<class ElemType>
__global__ void _addMaxPoolingGradientLoopOut(ElemType * inputGradientBatch, const ElemType * outputGradientBatch, const ElemType * inputBatch, const ElemType * outputBatch, 
                                                const long batchSize, const long channels, 
                                                const long inputWidth, const long inputHeight, const long inputSizePerSample, 
                                                const long outputWidth, const long outputHeight, const long outputSizePerSample, 
                                                const long windowWidth, const long windowHeight, const long horizontalSubsample, const long verticalSubsample)
{
    const long outputIndex = blockIdx.x * blockDim.x + threadIdx.x; 
    const long sample = outputIndex / outputSizePerSample; 
    if (sample >= batchSize) 
        return; 
   
    const long outputIndexWithinSample = outputIndex % outputSizePerSample; 
    const long inputWidthTimesChannel = inputWidth * channels; 
    const long outputWidthTimesChannel = outputWidth * channels; 
    const long y = outputIndexWithinSample / outputWidthTimesChannel; 
    const long nXC = outputIndexWithinSample % outputWidthTimesChannel; 
    const long x = nXC / channels; 
    const long c = nXC % channels; 

    const long offset0 = sample*inputSizePerSample + y*verticalSubsample*inputWidthTimesChannel + x*horizontalSubsample*channels;
    const ElemType *pCurWindow4Input = inputBatch + offset0; // pooling to current window's first input pixel 
    ElemType *pCurWindow4InGradient = inputGradientBatch + offset0; 
    for (long yy=0; yy<windowHeight; yy++) 
    {
        const long offset1 = yy*inputWidthTimesChannel + c; 
        const ElemType *pf0 = pCurWindow4Input + offset1; 
        ElemType *pf1 = pCurWindow4InGradient + offset1; 
        for (long xx=0; xx<windowWidth; xx++)
        {
            const long offset2 = xx*channels; 
            if (pf0[offset2] == outputBatch[outputIndex]) 
            {
                pf1[offset2] += outputGradientBatch[outputIndex]; //need to be atomic however atomicAdd on double is not supported.
            }
        }
    }
}

template<class ElemType>
__global__ void _addElementProductOf(
    ElemType* us,
    const ElemType* a,
    const ElemType* b,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    us[id]+=(a[id]*b[id]);
}

template<class ElemType>
__global__ void _columnElementMultiplyWith(
    ElemType* us,
    const ElemType* a,
    const long N, //a.GetNumRows();
    const long M) //us.GetNumCols();
{
    long id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;

    //__shared__ ElemType _a[threadsPerBlock];
    //_a[threadIdx.x]=a[id];
    ElemType mul=a[id];
    for (long j=0;j<M;++j)
    {
        us[IDX2C(id,j,N)]=us[IDX2C(id,j,N)]*mul;
    }
}

template<class ElemType>
__global__ void _rowElementMultiplyWith(
    ElemType* us,
    const ElemType* a,
    const long N, //us.GetNumRows();
    const long M) //a.GetNumCols();
{
    long id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=M)
        return;

    //__shared__ ElemType _a[threadsPerBlock];
    //_a[threadIdx.x]=a[id];
    ElemType mul=a[id];
    for (long i=0;i<N;++i)
    {
        us[IDX2C(i,id,N)]=us[IDX2C(i,id,N)]*mul;
    }
}

template<class ElemType>
__global__ void _rowElementDivideBy(
    ElemType* us,
    const ElemType* a,
    const long N, //us.GetNumRows();
    const long M) //a.GetNumCols();
{
    long id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id >= M)
        return;

    //__shared__ ElemType _a[threadsPerBlock];
    //_a[threadIdx.x]=a[id];
    ElemType v = a[id];
    if (v >= 0 && v < EPS_IN_INVERSE)
        v = EPS_IN_INVERSE;
    else if (v < 0 && v > -EPS_IN_INVERSE)
        v = (-EPS_IN_INVERSE);

    for (long i = 0; i<N; ++i)
    {
        us[IDX2C(i, id, N)] = us[IDX2C(i, id, N)] / v;
    }
}

template<class ElemType>
__global__ void _ColumnElementDivideBy(
    ElemType* us,
    const ElemType* a,
    const long N, //a.GetNumRows();
    const long M) //us.GetNumCols();
{
    long id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;

    ElemType smallValue = EPS_IN_INVERSE;

    //__shared__ ElemType _a[threadsPerBlock];
    //_a[threadIdx.x]=a[id];
    ElemType v=a[id];
    for (long j=0;j<M;++j)
    {
        if (v <0 && v > -smallValue)
            us[IDX2C(id,j,N)] /= (-smallValue);
        else if (v >=0 && v < smallValue)
            us[IDX2C(id,j,N)] /= smallValue;
        else
            us[IDX2C(id,j,N)] /= v;
    }

}


template<class ElemType>
__global__ void _innerProduct(
    ElemType* c,
    const ElemType* a,
    const ElemType* b,
    const long N, //a.GetNumRows();
    const long M, //a.GetNumCols();
    const bool isColWise) 
{
    long id = blockDim.x * blockIdx.x + threadIdx.x;
    if ((isColWise && id>=M) || (!isColWise && id>=N))
        return;

    ElemType sum = 0;
    long index;
    if (isColWise)
    {
        for (long i=0; i<N; ++i)
        {
            index = IDX2C(i,id,N);
            sum += a[index]* b[index];
        }
    }
    else
    {
        for (long j=0; j<M; ++j)
        {
            index = IDX2C(id,j, N);
            sum += a[index]* b[index];
        }
    }

    c[id] = sum;
}


template<class ElemType>
__global__ void _assignSignOf(
    ElemType* a,
    const ElemType* b,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    ElemType v = b[id];
    a[id] = (v == (ElemType)0? (ElemType)0 : (v > 0? (ElemType)1 : (ElemType)(-1)));
}

template<class ElemType>
__global__ void _addSignOf(
    ElemType* a,
    const ElemType* b,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    ElemType v = b[id];
    a[id] += (v == (ElemType)0? (ElemType)0 : (v > 0? (ElemType)1 : (ElemType)(-1)));
}

template<class ElemType>
__global__ void _vectorMaxMinReduce( //this function processes 1 column per block. this function needs 512 threads
                                 const ElemType* us,
                                 ElemType* Indexes,
                                 ElemType* Values,
                                 const long m,  //number of rows
                                 const long n,
                                 bool isMax)  //number of cols
{
    //we first find max per column    
    __shared__ ElemType partials[512];        
    __shared__ int partialsInd[512];
    if (isMax)
    {
        partials[threadIdx.x]=-10000000;
    }
    else
    {
        partials[threadIdx.x]=10000000;
    }
    partialsInd[threadIdx.x]=-1;

    //int id = blockDim.x * blockIdx.x + threadIdx.x;
    int loadPerThread = m/blockDim.x; 

    for (int i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? m : (threadIdx.x+1)*loadPerThread);++i)
    {
        if (( isMax ? us[IDX2C(i,blockIdx.x,m)]>partials[threadIdx.x] : us[IDX2C(i,blockIdx.x,m)]<partials[threadIdx.x]) || partialsInd[threadIdx.x]==-1)
        {
            partials[threadIdx.x]=us[IDX2C(i,blockIdx.x,m)];
            partialsInd[threadIdx.x]=i;       
        }
    }
    __syncthreads();

    //256
    if (threadIdx.x<256)
    {
        //partials[threadIdx.x]=max(partials[threadIdx.x+256],partials[threadIdx.x]);
        if ((isMax ? partials[threadIdx.x+256]>partials[threadIdx.x] : partials[threadIdx.x+256]<partials[threadIdx.x]) || partialsInd[threadIdx.x]==-1)
        {
            partials[threadIdx.x]=partials[threadIdx.x+256];
            partialsInd[threadIdx.x]=partialsInd[threadIdx.x+256];
        }
    }
    __syncthreads();

    //128
    if (threadIdx.x<128)
    {
        //partials[threadIdx.x]=max(partials[threadIdx.x+128],partials[threadIdx.x]);
        if ((isMax ? partials[threadIdx.x+128]>partials[threadIdx.x] : partials[threadIdx.x+128]<partials[threadIdx.x]) || partialsInd[threadIdx.x]==-1)
        {
            partials[threadIdx.x]=partials[threadIdx.x+128];
            partialsInd[threadIdx.x]=partialsInd[threadIdx.x+128];
        }
    }
    __syncthreads();

    //64
    if (threadIdx.x<64)
    {
        //partials[threadIdx.x]=max(partials[threadIdx.x+64],partials[threadIdx.x]);
        if ((isMax ? partials[threadIdx.x+64]>partials[threadIdx.x] : partials[threadIdx.x+64]<partials[threadIdx.x]) || partialsInd[threadIdx.x]==-1)
        {
            partials[threadIdx.x]=partials[threadIdx.x+64];
            partialsInd[threadIdx.x]=partialsInd[threadIdx.x+64];
        }
    }
    __syncthreads();

    //32
    if (threadIdx.x<32)
    {
        //partials[threadIdx.x]=max(partials[threadIdx.x+32],partials[threadIdx.x]);
        if ((isMax ? partials[threadIdx.x+32]>partials[threadIdx.x] : partials[threadIdx.x+32]<partials[threadIdx.x]) || partialsInd[threadIdx.x]==-1)
        {
            partials[threadIdx.x]=partials[threadIdx.x+32];
            partialsInd[threadIdx.x]=partialsInd[threadIdx.x+32];
        }
    }
    __syncthreads();

    //16
    if (threadIdx.x<16)
    {
        //partials[threadIdx.x]=max(partials[threadIdx.x+16],partials[threadIdx.x]);
        if ((isMax ? partials[threadIdx.x+16]>partials[threadIdx.x] : partials[threadIdx.x+16]<partials[threadIdx.x]) || partialsInd[threadIdx.x]==-1)
        {
            partials[threadIdx.x]=partials[threadIdx.x+16];
            partialsInd[threadIdx.x]=partialsInd[threadIdx.x+16];
        }
    }
    __syncthreads();

    //8
    if (threadIdx.x<8)
    {
        //partials[threadIdx.x]=max(partials[threadIdx.x+8],partials[threadIdx.x]);
        if ((isMax ? partials[threadIdx.x+8]>partials[threadIdx.x] : partials[threadIdx.x+8]<partials[threadIdx.x]) || partialsInd[threadIdx.x]==-1)
        {
            partials[threadIdx.x]=partials[threadIdx.x+8];
            partialsInd[threadIdx.x]=partialsInd[threadIdx.x+8];
        }
    }
    __syncthreads();

    //4
    if (threadIdx.x<4)
    {
        //partials[threadIdx.x]=max(partials[threadIdx.x+4],partials[threadIdx.x]);
        if ((isMax ? partials[threadIdx.x+4]>partials[threadIdx.x] : partials[threadIdx.x+4]<partials[threadIdx.x]) || partialsInd[threadIdx.x]==-1)
        {
            partials[threadIdx.x]=partials[threadIdx.x+4];
            partialsInd[threadIdx.x]=partialsInd[threadIdx.x+4];
        }
    }
    __syncthreads();

    if (threadIdx.x==0)
    {
        ElemType mx = partials[0];
        int ind = partialsInd[0];
        if ((isMax ? mx<partials[1] : mx>partials[1]) || ind ==-1)
        {
            mx = partials[1];
            ind = partialsInd[1];
        }
        if ((isMax ? mx<partials[2] : mx>partials[2]) || ind ==-1)
        {
            mx = partials[2];
            ind = partialsInd[2];
        }
        if ((isMax ? mx<partials[3] : mx>partials[3]) || ind ==-1)
        {
            mx = partials[3];
            ind = partialsInd[3];
        }
        Values[blockIdx.x] = mx;
        Indexes[blockIdx.x] = ind;
    }
}

template<class ElemType>
__global__ void _vectorMax(
    const ElemType* us,
    ElemType* maxIndexes,
    ElemType* maxValues,
    const long m,  //number of rows
    const long n,  //number of cols
    const bool isColWise) 
{
    long id = blockDim.x * blockIdx.x + threadIdx.x;
    long maxInd = -1;
    ElemType maxVal = -100000;

    if (isColWise)
    {
        if (id>=n)
            return;

        for (long i=0;i<m;i++)
        {
            if (maxInd==-1 || us[IDX2C(i,id,m)]>=maxVal)
            {
                maxInd = i;
                maxVal = us[IDX2C(i,id,m)];
            }
        }
    }
    else
    {
        if (id>=m)
            return;

        for (long j=0;j<n;j++)
        {
            if (maxInd==-1 || us[IDX2C(id,j,m)]>=maxVal)
            {
                maxInd = j;
                maxVal = us[IDX2C(id,j,m)];
            }
        }
    }
    maxIndexes[id]=maxInd;
    maxValues[id]=maxVal;
}

template<class ElemType>
__global__ void _vectorMin(
    const ElemType* us,
    ElemType* minIndexes,
    ElemType* minValues,
    const long m,  //number of rows
    const long n,  //number of cols
    const bool isColWise) 
{
    long id = blockDim.x * blockIdx.x + threadIdx.x;
    long minInd = -1;
    ElemType minVal = -100000;

    if (isColWise)
    {
        if (id>=n)
            return;

        for (long i=0;i<m;i++)
        {
            if (minInd==-1 || us[IDX2C(i,id,m)]<=minVal)
            {
                minInd = i;
                minVal = us[IDX2C(i,id,m)];
            }
        }
    }
    else
    {
        if (id>=m)
            return;

        for (long j=0;j<n;j++)
        {
            if (minInd==-1 || us[IDX2C(id,j,m)]<=minVal)
            {
                minInd = j;
                minVal = us[IDX2C(id,j,m)];
            }
        }
    }
    minIndexes[id]=minInd;
    minValues[id]=minVal;
}

template<class ElemType>
__global__ void _matrixVectorColumnWiseAdd(
    const ElemType* a,
    ElemType* us,
    ElemType alpha,
    const long m,  //number of rows
    const long n)  //number of cols     
{
#ifdef VALIDATION
    if (blockDim.x * blockIdx.x + threadIdx.x == 0)
    {
        printf("** _matrixVectorColumnWiseAdd on device:\na = %p, us = %p, alpha = %f, m = %ld, n = %ld\n", 
            a,us,alpha,m,n);
        printf("us[0] = %f\n", us[0]);
        printf("a[0] = %f\n", a[0]);
    }
#endif
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=m)
        return;
    ElemType tmp = a[id];
#ifdef VALIDATION
    printf("  a[%d] = %f\n", id, tmp);
#endif
    for (long j = 0; j < n; ++j )
    {
        us[j*m+id] += alpha*tmp;
    }
 
}

#ifdef OLD
template<class ElemType>
__global__ void _matrixVectorColumnWiseAdd(
    const ElemType* a,
    ElemType* us,
    ElemType alpha,
    const long m,  //number of rows
    const long n)  //number of cols     
{
    int id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=m)
        return;
    if (blockIdx.x == 0)
    {
        printf("_matrixVectorColumnWiseAdd: a=%p, us=%p\n", a, us);
    }
    ElemType tmp = a[id];
    for (long j = 0; j < n; ++j )
    {
        us[j*m+id] += alpha*tmp;
    }
}
#endif

template<class ElemType>
__global__ void _matrixVectorColumnWiseAddBlockPerRow(
    const ElemType* a,
    ElemType* us,
    ElemType alpha,
    const long m,  //number of rows
    const long n)  //number of cols     
{    
    ElemType tmp;

    if (threadIdx.x==0)
    {
        tmp = a[blockIdx.x];
    }
    __syncthreads();

    int loadPerThread = n/blockDim.x; 

    for (int i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? n : (threadIdx.x+1)*loadPerThread);++i)
    {
        us[m*blockIdx.x + i] += alpha*tmp;
    }
}



template<class ElemType>
__global__ void _addScaledDifference( 
    ElemType alpha,
    ElemType *a,
    ElemType *b,
    ElemType *c,
    LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    c[id] = c[id] + (a[id]-b[id]) * (alpha);
}

template<class ElemType>
__global__ void _assignScaledDifference( 
    ElemType alpha,
    ElemType *a,
    ElemType *b,
    ElemType *c,
    LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    c[id] = (a[id]-b[id]) * (alpha);
}

template<class ElemType>
__global__ void _addScaledDifference( 
    ElemType *alpha,
    ElemType *a,
    ElemType *b,
    ElemType *c,
    LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    c[id] = c[id] + (a[id]-b[id]) * alpha[0];
}

template<class ElemType>
__global__ void _assignScaledDifference( 
    ElemType *alpha,
    ElemType *a,
    ElemType *b,
    ElemType *c,
    LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    c[id] = (a[id]-b[id]) * alpha[0];
}

template<class ElemType>
__global__ void _addElementToElement( 
    const ElemType *a, LONG64 indexA,
    ElemType *c, LONG64 indexC)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>0)
        return;
    c[indexC] += a[indexA];
}

template<class ElemType>
__global__ void _assignNumOfDiff( 
    const ElemType *a,
    const ElemType *b,
    ElemType *c,
    LONG64 N)
{
    __shared__ ElemType partialSums[1024];
    partialSums[threadIdx.x]=0;
    //int id = blockDim.x * blockIdx.x + threadIdx.x;
    LONG64 loadPerThread = N/blockDim.x; 
    for (LONG64 i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? N : (threadIdx.x+1)*loadPerThread);++i)
    {
        partialSums[threadIdx.x]+=(a[i] != b[i]);
    }
    __syncthreads();

    //512
    if (threadIdx.x<512)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+512];
    }
    __syncthreads();

    //256
    if (threadIdx.x<256)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+256];
    }
    __syncthreads();

    //128
    if (threadIdx.x<128)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+128];
    }
    __syncthreads();

    //64
    if (threadIdx.x<64)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+64];
    }
    __syncthreads();

    //32
    if (threadIdx.x<32)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+32];
    }
    __syncthreads();

    //16
    if (threadIdx.x<16)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+16];
    }
    __syncthreads();

    //8
    if (threadIdx.x<8)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+8];
    }
    __syncthreads();

    //4
    if (threadIdx.x<4)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+4];
    }
    __syncthreads();

    if (threadIdx.x==0)
    {
        c[0] = partialSums[0]+partialSums[1]+partialSums[2]+partialSums[3];
    }
}


/*template<class ElemType>
__global__ void _assignNumOfDiff( 
ElemType *a,
ElemType *b,
ElemType *c,
long N)
{
//TO DO: replace atomic operation with reduction

__shared__ int totalSum;
if (threadIdx.x == 0) totalSum = 0;
__syncthreads();

int id = blockDim.x * blockIdx.x + threadIdx.x;
if (id>=N)
return;

int localVal = (a[id] != b[id]);
atomicAdd(&totalSum, localVal);
__syncthreads();

c[id] = totalSum;
}*/

template<class ElemType>
__global__ void _scaleArray(
    ElemType alpha,
    ElemType *us,
    LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    us[id]=us[id]*alpha;
}


template<class ElemType>
__global__ void _sparsePlusDense(
    ElemType alpha,
    const ElemType* m_dVal,
    const int* m_dRow,
    const int* m_dCol,
    ElemType* pArrayDev,
    LONG64 M)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=M)
        return;
    int start = m_dRow[id];
    int end = m_dRow[id+1];
    for (int _i=start;_i<end;++_i)  //_i is index in m_dVal and m_dCol
    {
        int j = m_dCol[_i];
        pArrayDev[IDX2C(id,j,M)]+=(alpha*m_dVal[_i]);
    }
}

template<class ElemType>
__global__ void _sparseMulDense(    
    const ElemType* m_dVal,
    const int* m_dRow,
    const int* m_dCol,
    const ElemType* b,
    ElemType* c,
    LONG64 M)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=M)
        return;
    int start = m_dRow[id];
    int end = m_dRow[id+1];
    for (int _i=start;_i<end;++_i)  //_i is index in m_dVal and m_dCol
    {
        int j = m_dCol[_i];
        c[IDX2C(id,j,M)]=b[IDX2C(id,j,M)]*m_dVal[_i];
    }
}

// forward pass from feature to hidden layer
template<class ElemType>
__global__ void _denseMulSparseToDense(
    ElemType alpha,
    const ElemType* lhs,
    int numrows,
    int numcols,
    const size_t* row,
    ElemType* c)
{
    int loadPerThread = (numrows+blockDim.x-1)/blockDim.x;
    int tStart = loadPerThread * threadIdx.x;
    int tEnd = min(numrows, loadPerThread + tStart);

    int p = blockIdx.x;
    int i = row[p];
    int j = blockIdx.x;

    for (int h = tStart; h < tEnd; h++) 
    {
        ElemType res = alpha * lhs[IDX2C(h, i, numrows)]; 
        atomicAdd(&c[IDX2C(h,j,numrows)], res);
    }
}

// backward pass from hidden layer to feature weight
template<class ElemType>
__global__ void _denseMulSparseToSparse(    
    ElemType* lhs,
    size_t nrs,
    const size_t* row,
    const size_t* rowIdx,
    ElemType* blockVal,
    size_t* blockIds)
{
    int p = blockIdx.x;
    int i = row[p];
    int ii = rowIdx[p];
    int j = blockIdx.x;

    int load = (nrs+blockDim.x-1)/blockDim.x;
    int pStart = load * threadIdx.x;
    int pEnd = min((int)nrs, load + pStart);

    for(int h = pStart; h < pEnd; h++) 
    {        
        ElemType temp = lhs[IDX2C(h, j, nrs)];    
        atomicAdd(&blockVal[ii*nrs+h], temp);
        blockIds[ii] = i;
    }
}

// gradients update
template<class ElemType>
__global__ void _scaleAndAdd(    
    ElemType alpha,
    bool blockCol,
    ElemType* blockVal,
    size_t* blockIds,
    size_t len,
    ElemType* rhs,
    size_t numrows)
{
    int ii = blockIdx.x;
    int i = blockIds[ii];
    int load = (len+blockDim.x-1)/blockDim.x;
    int pStart = load * threadIdx.x;
    int pEnd = min((int)len, load + pStart);

    for(int h = pStart; h < pEnd; h++) 
    {   ElemType temp = alpha*blockVal[ii*len + h];
        if(blockCol)
        {
            atomicAdd(&rhs[IDX2C(h, i, numrows)], temp);
        }
        else
        {
            atomicAdd(&rhs[IDX2C(i, h, numrows)], temp);
        }
    }
}

// compute predictions in cross entory node
template<class ElemType>
__global__ void _computePrediction(
    int nv,
    const ElemType* a,
    int numrows,
    const ElemType* weight,   
    int nrs,
    int labelSize,
    const size_t* labelRow,
    const size_t* block2Id,
    const ElemType* cls,
    const ElemType* idx2cls,    
    ElemType* val,
    size_t* row,
    size_t* pb)
{
    // get label block id
    int id = -1;
    int offset = -1;
    for(int i = 1; i < labelSize; i++) 
    {
        if (blockIdx.x < block2Id[i]) 
        {
            id = i-1;
            offset = blockIdx.x - block2Id[i-1];
            break;
        }
    }
    if( id == -1) 
    {
        id = labelSize-1;
        offset = blockIdx.x - block2Id[labelSize-1];
    }

    int t = labelRow[id];
    int iStt;
    int iEnd;
    if(t < nv) 
    {
        int clsid = idx2cls[t];
        iStt = cls[IDX2C(0, clsid, 2)];
        iEnd = cls[IDX2C(1, clsid, 2)];
    } 
    else 
    {
        iStt = nv;
        iEnd = nrs;
    }
    int i = iStt + offset;
    int j = id /2;
    
    int loadPerThread = (numrows+blockDim.x-1)/blockDim.x;
    int tStart = loadPerThread * threadIdx.x;
    int tEnd = min((int)numrows, loadPerThread + tStart);

    ElemType v = 0.0;
    for (int h = tStart; h < tEnd; h++)
    {
        v += weight[IDX2C(i,h,nrs)] * a[IDX2C(h,j,numrows)]; 
    }
    atomicAdd(&val[blockIdx.x], v);
    row[blockIdx.x] = i;

    if(blockIdx.x == 0 && threadIdx.x == 0) 
        pb[0] = 0;
    
    if((threadIdx.x == 0) && (i == iEnd-1) && (i >= nv)) 
        pb[j+1] = blockIdx.x+1;
}

// normalize predictions in cross entropy node
template<class ElemType>
__global__ void _normalizePrediction(
    const size_t labelSize,
    const size_t expandedLabelSize,
    const size_t* labelRow,
    const size_t* block2Id,    
    const size_t* row,
    ElemType* val,
    ElemType* entropyScore)
{    
    __shared__ ElemType partials[512];
    partials[threadIdx.x] = 0;

    int p = blockIdx.x;
    int t = labelRow[p];
    int start = block2Id[p];
    int end;
    if(p == labelSize -1) 
    {
        end = expandedLabelSize;
    } 
    else 
    {
        end = block2Id[p+1];
    }
    int len = end - start;

    int loadPerThread = (len+blockDim.x-1)/blockDim.x;
    int tStart = loadPerThread * threadIdx.x;
    int tLen = min((int)len, loadPerThread + tStart);

    for(int i = start + tStart; i < start + tLen; i++) 
    {
        partials[threadIdx.x] += exp(val[i]);
    }

    __syncthreads();

    // now sum up the objective function
    int nTotalThreads = blockDim.x;

    while (nTotalThreads >1)
    {
        int halfPoint = (nTotalThreads >> 1);

        if (threadIdx.x < halfPoint)
            partials[threadIdx.x] += partials[threadIdx.x+halfPoint];

        __syncthreads();

        nTotalThreads = (nTotalThreads>>1);
    }
    
    for(int i = start + tStart; i < start + tLen; i++) 
    {
        val[i] = log(exp(val[i])/partials[0]);
        if(row[i] == t) 
        {
            atomicAdd(entropyScore, -val[i]);
            val[i] *= -1;
        }
    }
}

// compute prediction error in cross entropy node
template<class ElemType>
__global__ void _computePredictionError(
    ElemType* val,
    int N)
{    
    int p = blockDim.x * blockIdx.x + threadIdx.x;
    if (p>=N)
        return;

    if(val[p] < 0) 
        val[p] = exp(val[p]); //negative;
    else 
        val[p] = exp(-val[p])-1; //positive
}

// compute gradients of input in cross entropy node
template<class ElemType>
__global__ void _computeGradientOfInput(
    const ElemType* val,
    const size_t* row,
    const size_t* pb,    
    ElemType* weight,
    size_t nrs,
    ElemType* grd,
    size_t numrows)
{        
    int h = blockIdx.x%numrows;
    int j = blockIdx.x/numrows;

    int start = pb[j];
    int end = pb[j+1];
    int len = end - start;
    
    int load = (len+blockDim.x-1)/blockDim.x;
    int pStart = start + load * threadIdx.x;
    int pEnd = start + min(len, load * (threadIdx.x+1));

    ElemType sum = 0;
    for(int p = pStart; p < pEnd; p++) 
    {
        int i = row[p];
        sum += val[p] * weight[IDX2C(i, h, nrs)]; 
    }    

    atomicAdd(&grd[IDX2C(h,j,numrows)], sum);
}

// compute gradients of weights in cross entropy node
template<class ElemType>
__global__ void _computeGradientOfWeight(
    const ElemType* val,
    const size_t* row,
    const size_t* pb,
    size_t mb,
    size_t nv,
    const size_t* labelRow,
    const size_t* labelBlock2UniqId,
    const ElemType* cls,
    const ElemType* idx2cls,
    ElemType* input,
    size_t nrs,
    ElemType* blockVal,
    size_t* blockIds)
{
    int p = blockIdx.x;
    ElemType v = val[p];
    int i = row[p];
    int j = -1;
    for(int k = 1; k < mb; k++) 
    {
        if( p < pb[k]) 
        {
            j = k-1;
            break;
        }
    }
    if( j == -1) 
    {
        j = mb-1;
    }

    //figure out blocks
    int bId = i < nv ? 2*j : 2*j+1;
    int t = labelRow[bId];
    int iStt;
    if(t < nv) 
    {
        int clsid = idx2cls[t];
        iStt = cls[IDX2C(0, clsid, 2)];
    } 
    else 
    {
        iStt = nv;
    }
    int offset = i - iStt;
    int ii = labelBlock2UniqId[bId] + offset;

    int load = (nrs+blockDim.x-1)/blockDim.x;
    int pStart = load * threadIdx.x;
    int pEnd = min((int)nrs, load + pStart);

    for(int h = pStart; h < pEnd; h++) 
    {        
        ElemType temp = v * input[IDX2C(h, j, nrs)];    
        atomicAdd(&blockVal[ii*nrs+h], temp);
        blockIds[ii] = i;
    }
}

// used in clipping gradients
template<class ElemType>
__global__ void _inplaceTruncate(
    ElemType* a,
    const ElemType threshold,
    const LONG64 N)
{
    LONG64 id = blockDim.x * blockIdx.x + threadIdx.x;
    if (id>=N)
        return;
    ElemType locThresholdPos = abs(threshold);
    ElemType locTHresholdNeg = -locThresholdPos; 
    if (a[id] > locThresholdPos)
    {
        a[id] = locThresholdPos;
    }
    else if(a[id] < locTHresholdNeg)
    {
        a[id] = locTHresholdNeg;
    }
}

template<class ElemType>
__global__ void _normalGrad(
    bool isBlockCol,
    size_t len,
    const ElemType momentum,
    size_t* blockIds,
    ElemType* blockVal,
    ElemType* c,
    size_t numrows)
{
    int j = blockIdx.x;
    int i = blockIds[j];
    int start = j * len;

    int load = (len+blockDim.x-1)/blockDim.x;
    int pStart = load * threadIdx.x;
    int pLen = min((int)len, load + pStart);

    for(int p = start+pStart; p < start+pLen; p++) 
    {
        int row = isBlockCol ? (p - start) : i;
        int col = isBlockCol ? i: (p - start);
        c[IDX2C(row, col, numrows)] = (1-momentum)*blockVal[p] + momentum*c[IDX2C(row, col, numrows)];
        blockVal[p] = c[IDX2C(row, col, numrows)];
    }
}

static __inline__ __device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull = (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed, __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);

    return __longlong_as_double(old);
}

template<class ElemType>
static __inline__ __device__ ElemType logadd(ElemType x, ElemType y)
{
    ElemType temp, diff, z; 

    if (x < y) 
    {
        temp = x; x = y; y = temp;
    }
    diff = y - x; 
    if (diff < MINLOGEXP)
    {
        return (x < LSMALL)?LZERO:x;
    }
    else
    {
        z = exp(diff);
        return x + log(1.0 + z);
    }
}

//This function should be called with 1024 threads per block and 1 block
//THIS IS NOT THE MOST EFFICIENT IMPLEMENTATION!!!
template<class ElemType>
__global__ void _reductionSum(
    const ElemType* data,
    ElemType *sum,
    LONG64 N)
{

    __shared__ ElemType partialSums[1024];
    partialSums[threadIdx.x]=0;
    //int id = blockDim.x * blockIdx.x + threadIdx.x;
    LONG64 loadPerThread = N/blockDim.x; 
    for (LONG64 i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? N : (threadIdx.x+1)*loadPerThread);++i)
    {
        partialSums[threadIdx.x]+=data[i];
    }
    __syncthreads();

    //512
    if (threadIdx.x<512)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+512];
    }
    __syncthreads();

    //256
    if (threadIdx.x<256)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+256];
    }
    __syncthreads();

    //128
    if (threadIdx.x<128)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+128];
    }
    __syncthreads();

    //64
    if (threadIdx.x<64)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+64];
    }
    __syncthreads();

    //32
    if (threadIdx.x<32)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+32];
    }
    __syncthreads();

    //16
    if (threadIdx.x<16)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+16];
    }
    __syncthreads();

    //8
    if (threadIdx.x<8)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+8];
    }
    __syncthreads();

    //4
    if (threadIdx.x<4)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+4];
    }
    __syncthreads();

    if (threadIdx.x==0)
    {
        sum[0] = partialSums[0]+partialSums[1]+partialSums[2]+partialSums[3];
    }
}

//This function should be called with 1024 threads per block and 1 block
//THIS IS NOT THE MOST EFFICIENT IMPLEMENTATION!!!
template<class ElemType>
__global__ void _reductionSumAndAssign(
    ElemType* toAssign,
    const ElemType* data,
    LONG64 N, //length of data
    LONG64 M) //length of toAssign
{
    __shared__ ElemType partialSums[1024];
    __shared__ ElemType res;
    partialSums[threadIdx.x]=0;
    //int id = blockDim.x * blockIdx.x + threadIdx.x;
    LONG64 loadPerThread = N/blockDim.x; 
    for (LONG64 i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? N : (threadIdx.x+1)*loadPerThread);++i)
    {
        partialSums[threadIdx.x]+=data[i];
    }
    __syncthreads();

    //512
    if (threadIdx.x<512)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+512];
    }
    __syncthreads();

    //256
    if (threadIdx.x<256)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+256];
    }
    __syncthreads();

    //128
    if (threadIdx.x<128)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+128];
    }
    __syncthreads();

    //64
    if (threadIdx.x<64)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+64];
    }
    __syncthreads();

    //32
    if (threadIdx.x<32)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+32];
    }
    __syncthreads();

    //16
    if (threadIdx.x<16)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+16];
    }
    __syncthreads();

    //8
    if (threadIdx.x<8)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+8];
    }
    __syncthreads();

    //4
    if (threadIdx.x<4)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+4];
    }
    __syncthreads();

    if (threadIdx.x==0)
    {
        res = partialSums[0]+partialSums[1]+partialSums[2]+partialSums[3];
        for (LONG64 i=0;i<M;++i)
            toAssign[i]=res;
    }
}

//This function should be called with 1024 threads per block and 1 block
//THIS IS NOT THE MOST EFFICIENT IMPLEMENTATION!!!
template<class ElemType>
__global__ void _reductionSum2(
    const ElemType* data,
    ElemType *sum,
    LONG64 N, 
    bool takeSqrt=false)
{

    __shared__ ElemType partialSums[1024];
    partialSums[threadIdx.x]=0;
    //int id = blockDim.x * blockIdx.x + threadIdx.x;
    LONG64 loadPerThread = N/blockDim.x; 
    for (LONG64 i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? N : (threadIdx.x+1)*loadPerThread);++i)
        //for (int i= threadIdx.x*loadPerThread; i<(threadIdx.x+1)*loadPerThread;++i)
    {
        partialSums[threadIdx.x]+=(data[i]*data[i]);
    }
    __syncthreads();

    //512
    if (threadIdx.x<512)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+512];
    }
    __syncthreads();

    //256
    if (threadIdx.x<256)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+256];
    }
    __syncthreads();

    //128
    if (threadIdx.x<128)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+128];
    }
    __syncthreads();

    //64
    if (threadIdx.x<64)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+64];
    }
    __syncthreads();

    //32
    if (threadIdx.x<32)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+32];
    }
    __syncthreads();

    //16
    if (threadIdx.x<16)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+16];
    }
    __syncthreads();

    //8
    if (threadIdx.x<8)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+8];
    }
    __syncthreads();

    //4
    if (threadIdx.x<4)
    {
        partialSums[threadIdx.x]+=partialSums[threadIdx.x+4];
    }
    __syncthreads();

    if (threadIdx.x==0)
    {
        sum[0] = partialSums[0]+partialSums[1]+partialSums[2]+partialSums[3];
        if (takeSqrt)
        {
            if (sizeof(ElemType)==sizeof(float))
                sum[0] = sqrtf(sum[0]);
            else
                sum[0] = sqrt(sum[0]); 
        }
    }
}


//This function should be called with 1024 threads per block and 1 block
//THIS IS NOT THE MOST EFFICIENT IMPLEMENTATION!!!
template<class ElemType>
__global__ void _reductionMatrixNormInf(
    const ElemType* data,
    ElemType *maxAbs,
    LONG64 N)
{

    __shared__ ElemType partialSums[1024];
    partialSums[threadIdx.x]=0;
    //int id = blockDim.x * blockIdx.x + threadIdx.x;
    int loadPerThread = N/blockDim.x; 
    for (int i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? N : (threadIdx.x+1)*loadPerThread);++i)    
    {
        if (sizeof(ElemType)==sizeof(float))
        {
            partialSums[threadIdx.x]=max(fabsf(data[i]),partialSums[threadIdx.x]);
        }
        else
        {
            partialSums[threadIdx.x]=max(fabs(data[i]),partialSums[threadIdx.x]);
        }
    }
    __syncthreads();

    //512
    if (threadIdx.x<512)
    {
        partialSums[threadIdx.x]=max(partialSums[threadIdx.x+512],partialSums[threadIdx.x]);        
    }
    __syncthreads();

    //256
    if (threadIdx.x<256)
    {
        partialSums[threadIdx.x]=max(partialSums[threadIdx.x+256],partialSums[threadIdx.x]);
    }
    __syncthreads();

    //128
    if (threadIdx.x<128)
    {
        partialSums[threadIdx.x]=max(partialSums[threadIdx.x+128],partialSums[threadIdx.x]);
    }
    __syncthreads();

    //64
    if (threadIdx.x<64)
    {
        partialSums[threadIdx.x]=max(partialSums[threadIdx.x+64],partialSums[threadIdx.x]);
    }
    __syncthreads();

    //32
    if (threadIdx.x<32)
    {
        partialSums[threadIdx.x]=max(partialSums[threadIdx.x+32],partialSums[threadIdx.x]);
    }
    __syncthreads();

    //16
    if (threadIdx.x<16)
    {
        partialSums[threadIdx.x]=max(partialSums[threadIdx.x+16],partialSums[threadIdx.x]);
    }
    __syncthreads();

    //8
    if (threadIdx.x<8)
    {
        partialSums[threadIdx.x]=max(partialSums[threadIdx.x+8],partialSums[threadIdx.x]);
    }
    __syncthreads();

    //4
    if (threadIdx.x<4)
    {
        partialSums[threadIdx.x]=max(partialSums[threadIdx.x+4],partialSums[threadIdx.x]);
    }
    __syncthreads();

    if (threadIdx.x==0)
    {
        maxAbs[0] = max(max(partialSums[0],partialSums[1]),max(partialSums[2],partialSums[3]));
    }
}

//This function should be called with 1024 threads per block and 1 block
//THIS IS NOT THE MOST EFFICIENT IMPLEMENTATION!!!
template<class ElemType>
__global__ void _reductionMatrixNorm0(
    const ElemType* data,
    ElemType *nz,
    LONG64 N)
{

    __shared__ ElemType partialSums[1024];
    partialSums[threadIdx.x]=0;
    //int id = blockDim.x * blockIdx.x + threadIdx.x;
    LONG64 loadPerThread = N/blockDim.x; 
    for (LONG64 i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? N : (threadIdx.x+1)*loadPerThread);++i)    
    {
        if (data[i]!=0)
            ++partialSums[threadIdx.x];
    }
    __syncthreads();

    //512
    if (threadIdx.x<512)
    {
        partialSums[threadIdx.x]=partialSums[threadIdx.x+512]+partialSums[threadIdx.x];        
    }
    __syncthreads();

    //256
    if (threadIdx.x<256)
    {
        partialSums[threadIdx.x]=partialSums[threadIdx.x+256]+partialSums[threadIdx.x];
    }
    __syncthreads();

    //128
    if (threadIdx.x<128)
    {
        partialSums[threadIdx.x]=partialSums[threadIdx.x+128]+partialSums[threadIdx.x];
    }
    __syncthreads();

    //64
    if (threadIdx.x<64)
    {
        partialSums[threadIdx.x]=partialSums[threadIdx.x+64]+partialSums[threadIdx.x];
    }
    __syncthreads();

    //32
    if (threadIdx.x<32)
    {
        partialSums[threadIdx.x]=partialSums[threadIdx.x+32]+partialSums[threadIdx.x];
    }
    __syncthreads();

    //16
    if (threadIdx.x<16)
    {
        partialSums[threadIdx.x]=partialSums[threadIdx.x+16]+partialSums[threadIdx.x];
    }
    __syncthreads();

    //8
    if (threadIdx.x<8)
    {
        partialSums[threadIdx.x]=partialSums[threadIdx.x+8]+partialSums[threadIdx.x];
    }
    __syncthreads();

    //4
    if (threadIdx.x<4)
    {
        partialSums[threadIdx.x]=partialSums[threadIdx.x+4]+partialSums[threadIdx.x];
    }
    __syncthreads();

    if (threadIdx.x==0)
    {
        nz[0] = partialSums[0]+partialSums[1]+partialSums[2]+partialSums[3];
    }
}


template<class ElemType>
__global__ void _getSparseVectorRepresntationForMatrix(
    const int* m_dRow,
    const int* m_dCol,    
    int* vectArray,    
    const long M,
    const long N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i>=M)
        return;
    int start = m_dRow[i];
    int end = m_dRow[i+1];
    for (int _i=start;_i<end;++_i)  //_i is index in m_dVal and m_dCol
    {
        int j = m_dCol[_i];
        vectArray[_i] = i*N + j;
    }
}


template<class ElemType>
__global__ void _lrHelper(
    const ElemType* data1,    
    const ElemType* data2,    
    const long N,
    ElemType* d_res)
{
    __shared__ ElemType partialSums1[512];
    __shared__ ElemType partialSums2[512];
    partialSums1[threadIdx.x]=0;
    partialSums2[threadIdx.x]=0;

    //int id = blockDim.x * blockIdx.x + threadIdx.x;
    int loadPerThread = N/blockDim.x;     
    for (int i= threadIdx.x*loadPerThread; i< (threadIdx.x == blockDim.x - 1 ? N : (threadIdx.x+1)*loadPerThread);++i)        
    {
        partialSums1[threadIdx.x]+=(data1[i]*data1[i]);
        partialSums2[threadIdx.x]+=(data2[i]*data2[i]);
    }
    __syncthreads();

    /*
    //512
    if (threadIdx.x<512)
    {
    partialSums1[threadIdx.x]+=partialSums1[threadIdx.x+512];
    partialSums2[threadIdx.x]+=partialSums2[threadIdx.x+512];
    }
    __syncthreads();*/

    //256
    if (threadIdx.x<256)
    {
        partialSums1[threadIdx.x]+=partialSums1[threadIdx.x+256];
        partialSums2[threadIdx.x]+=partialSums2[threadIdx.x+256];        
    }
    __syncthreads();

    //128
    if (threadIdx.x<128)
    {
        partialSums1[threadIdx.x]+=partialSums1[threadIdx.x+128];
        partialSums2[threadIdx.x]+=partialSums2[threadIdx.x+128];        
    }
    __syncthreads();

    //64
    if (threadIdx.x<64)
    {
        partialSums1[threadIdx.x]+=partialSums1[threadIdx.x+64];
        partialSums2[threadIdx.x]+=partialSums2[threadIdx.x+64];        
    }
    __syncthreads();

    //32
    if (threadIdx.x<32)
    {
        partialSums1[threadIdx.x]+=partialSums1[threadIdx.x+32];
        partialSums2[threadIdx.x]+=partialSums2[threadIdx.x+32];        
    }
    __syncthreads();

    //16
    if (threadIdx.x<16)
    {
        partialSums1[threadIdx.x]+=partialSums1[threadIdx.x+16];
        partialSums2[threadIdx.x]+=partialSums2[threadIdx.x+16];        
    }
    __syncthreads();

    //8
    if (threadIdx.x<8)
    {
        partialSums1[threadIdx.x]+=partialSums1[threadIdx.x+8];
        partialSums2[threadIdx.x]+=partialSums2[threadIdx.x+8];        
    }
    __syncthreads();

    //4
    if (threadIdx.x<4)
    {
        partialSums1[threadIdx.x]+=partialSums1[threadIdx.x+4];
        partialSums2[threadIdx.x]+=partialSums2[threadIdx.x+4];        
    }
    __syncthreads();

    if (threadIdx.x==0)
    {        
        ElemType fns1 = partialSums1[0]+partialSums1[1]+partialSums1[2]+partialSums1[3];
        ElemType fns2 = partialSums2[0]+partialSums2[1]+partialSums2[2]+partialSums2[3];
        if (sizeof(ElemType)==sizeof(float))
        {                    
            d_res[0] = max((ElemType)0, d_res[0]/max((ElemType)1.0e-10,sqrtf(fns1))/max((ElemType)1.0e-10,sqrtf(fns2)));            
        }
        else
        {            
            d_res[0] = max((ElemType)0, d_res[0]/max((ElemType)1.0e-10,sqrt(fns1))/max((ElemType)1.0e-10,sqrt(fns2)));              
        }   
    }
}

/*
template<class ElemType>
__global__ void _lrHelper(
ElemType* d_tmp)
{
if (sizeof(ElemType)==sizeof(float))
{
d_tmp[0] = max((ElemType)0, d_tmp[0]/max((ElemType)1.0e-10,sqrtf(d_tmp[1]))/max((ElemType)1.0e-10,sqrtf(d_tmp[2])));            
}
else
{
d_tmp[0] = max((ElemType)0, d_tmp[0]/max((ElemType)1.0e-10,sqrt(d_tmp[1]))/max((ElemType)1.0e-10,sqrt(d_tmp[2])));            
}
}
*/