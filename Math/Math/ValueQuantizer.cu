#include "hip/hip_runtime.h"
#ifndef __VALLUE_QUANTIZER_CUH__
#define __VALLUE_QUANTIZER_CUH__

#include "stdafx.h"
#include "ValueQuantizer.h"

namespace Microsoft { namespace MSR { namespace CNTK {

    template<class ElemType>
    cudasharedcode
    ValueQuantizer<ElemType>::ValueQuantizer(size_t ldNbits, ElemType lower, ElemType upper) 
    : ldNbits(ldNbits), Nbits(1 << ldNbits), rangeend(1 << Nbits), quantimin(lower), quantimax(upper)
    {
        // post-fix for incorrect shift for no-quant hack (Nbits=32): << arg is taken mod 32!
        // in this case, it's only used as (rangeend-1) which is now correct (before it was 0!)
        if (Nbits >= (8 * sizeof(rangeend)))
        {
            rangeend = 0;
        }

        // must protect against NaN: interval is 0 -> quantization is futile, just emit 0
        if (((quantimax - quantimin) < 1e-36f) || (rangeend == 0))
        {
            qfactor = ufactor = 0.0f;
        }
        else
        {
            // precompute this for quantize() (see comment there)
            qfactor = rangeend / (quantimax - quantimin);   
            // and for unquantize()
            ufactor = (quantimax - quantimin) / rangeend;   
        }
    #ifndef ZERO_THRESHOLD_FOR_1BIT
        // set the quantization threshold for the special case of 1-bit
        quantimid = 0.5f * (quantimax + quantimin);
    #endif
    }

    // quantize for 32-bits case (special case that allows to bypass quantization, for testing/debugging purposes)
    template<class ElemType>
    cudasharedcode unsigned int
    ValueQuantizer<ElemType>::Quantize32(ElemType u) const
    {
        assert ((Nbits == 32) && (sizeof(unsigned int) == 4));
        
        // we return the bit pattern that encodes the float value
        return *(unsigned int*)&u;  
    }

    // quantize one value --special version for 1 bit
    template<class ElemType>
    cudasharedcode bool
    ValueQuantizer<ElemType>::Quantize1(ElemType u) const
    {
        assert (Nbits == 1);
    #ifndef ZERO_THRESHOLD_FOR_1BIT
        return u >= quantimid;
    #else
        return u >= 0.0f;
    #endif
    }

    // quantize one value
    // TODO: we can optimize for 1 bit here very simply... use a template arg 'isonebit'
    template<class ElemType>
    cudasharedcode unsigned int
    ValueQuantizer<ElemType>::Quantize(ElemType u) const
    {
        // 32-bits case for hacking
        if (Nbits == 32)
        {
            return Quantize32(u);
        }
        // TODO: we may need to optimize this by a template arg
        else if (ldNbits == 0)
        {
            return Quantize1(u) ? 1 : 0;
        }
        else
        {
            int result = (int) ((u - quantimin) * qfactor);
            // (note: '(int)' rounds asymmetrically towards 0, but that's OK since we clip against 0
            if (result < 0)
            {
                return 0;
            }
            else if (((unsigned int)result) >= rangeend)
            {
                return rangeend - 1;
            }
            else
            {
                return (unsigned int)result;
            }
        }
    }

    // unquantize one value
    template<class ElemType>
    cudasharedcode  
    ElemType ValueQuantizer<ElemType>::Unquantize(unsigned int u) const
    {
        // 32-bits case for hacking
        if (Nbits == 32)
        {
            return *(ElemType *)&u;
        }
        
        // Note: in 1-bit case, we want 0.5 -> mean0, 1.5 -> mean1
        return (u + 0.5f) * ufactor + quantimin;
    }

    // unquantize one value  --special case for 1 bit
    template<class ElemType>
    cudasharedcode 
    ElemType ValueQuantizer<ElemType>::Unquantize1(bool u, ElemType val0, ElemType val1)
    {
        return u ? val1 : val0;
    }

    // helper: compute the binary log of a power of two (utility function to convert 'Nbits' into 'ldNbits'
    template<class ElemType>
    size_t ValueQuantizer<ElemType>::ld(size_t v)
    {
        if (v == 1)
        {
            return 0;
        }
        else if (v & 1) // not a power of two
        {
            throw std::runtime_error("ld: 'bits' must be a power of two");
        }
        else
        {
            return 1 + ld (v >> 1);
        }
    }
    
    // Explicit instantiation
    template class ValueQuantizer<float>;
    template class ValueQuantizer<double>;
}}}
#endif
