#include "hip/hip_runtime.h"
#ifndef __VALLUE_QUANTIZER_CUH__
#define __VALLUE_QUANTIZER_CUH__

#include "stdafx.h"
#include "ValueQuantizer.h"

namespace Microsoft { namespace MSR { namespace CNTK {

    template<class ElemType>
    cudasharedcode
    ValueQuantizer<ElemType>::ValueQuantizer(size_t ldNbits, ElemType lower, ElemType upper) 
    : ldNbits(ldNbits), Nbits(1 << ldNbits), quantimin(lower), quantimax(upper)
    {
        rangeend = ((QWordVal)1) << Nbits;

        // post-fix for incorrect shift for no-quant hack (Nbits=32): << arg is taken mod 32!
        // in this case, it's only used as (rangeend-1) which is now correct (before it was 0!)
        if (Nbits >= (8 * sizeof(rangeend)))
        {
            rangeend = 0;
        }

        // must protect against NaN: interval is 0 -> quantization is futile, just emit 0
        if (((quantimax - quantimin) < 1e-36f) || (rangeend == 0))
        {
            qfactor = ufactor = (ElemType)0.0;
        }
        else
        {
            // precompute this for quantize() (see comment there)
            qfactor = rangeend / (quantimax - quantimin);   
            // and for unquantize()
            ufactor = (quantimax - quantimin) / rangeend;   
        }

        // set the quantization threshold for the special case of 1-bit
        quantimid = 0.5f * (quantimax + quantimin);
    }

    // quantize for full ElemType size bits case (special case that allows to bypass quantization, for testing/debugging purposes)
    template<class ElemType>
    cudasharedcode ValueQuantizer<ElemType>::QWordVal
    ValueQuantizer<ElemType>::QuantizeToFullQWord(ElemType u) const
    {
        assert(Nbits == QWordNumBits);
        
        // we return the bit pattern that encodes the float value
        return *(QWordVal*)&u;
    }

    // quantize one value --special version for 1 bit
    template<class ElemType>
    template<bool ZeroThresholdFor1Bit>
    cudasharedcode bool
    ValueQuantizer<ElemType>::Quantize1(ElemType u) const
    {
        assert (Nbits == 1);
        if (!ZeroThresholdFor1Bit)
        {
            return u >= quantimid;
        }
        else
        {
            return u >= (ElemType)0.0;
        }
    }

    // quantize one value
    // TODO: we can optimize for 1 bit here - very simply use a template arg 'isonebit'
    template<class ElemType>
    template<bool ZeroThresholdFor1Bit>
    cudasharedcode ValueQuantizer<ElemType>::QWordVal
    ValueQuantizer<ElemType>::Quantize(ElemType u) const
    {
        if (Nbits == QWordNumBits)
        {
            return QuantizeToFullQWord(u);
        }
        // TODO: we may need to optimize this by a template arg
        else if (ldNbits == 0)
        {
            return Quantize1<ZeroThresholdFor1Bit>(u) ? 1 : 0;
        }
        else
        {
            if (u <= quantimin)
            {
                return 0;
            }
            else if (u >= quantimax)
            {
                return (rangeend - 1);
            }
            else
            {
                return (QWordVal)((QWordValSigned)((u - quantimin) * qfactor));
            }
        }
    }

    // unquantize one value
    template<class ElemType>
    cudasharedcode  
    ElemType ValueQuantizer<ElemType>::Unquantize(QWordVal u) const
    {
        if (Nbits == QWordNumBits)
        {
            return *(ElemType*)&u;
        }
        
        // Note: in 1-bit case, we want 0.5 -> mean0, 1.5 -> mean1
        return ((u + (ElemType)0.5) * ufactor) + quantimin;
    }

    // unquantize one value  --special case for 1 bit
    template<class ElemType>
    cudasharedcode 
    ElemType ValueQuantizer<ElemType>::Unquantize1(bool u, ElemType val0, ElemType val1)
    {
        return u ? val1 : val0;
    }

    // helper: compute the binary log of a power of two (utility function to convert 'Nbits' into 'ldNbits'
    template<class ElemType>
    size_t ValueQuantizer<ElemType>::ld(size_t v)
    {
        if (v == 1)
        {
            return 0;
        }
        else if (v & 1) // not a power of two
        {
            throw std::runtime_error("ld: 'bits' must be a power of two");
        }
        else
        {
            return 1 + ld (v >> 1);
        }
    }
    
    // Explicit instantiation
    template class ValueQuantizer<float>;
    template class ValueQuantizer<double>;
    template ValueQuantizer<float>::QWordVal ValueQuantizer<float>::Quantize<true>(float u) const;
    template ValueQuantizer<float>::QWordVal ValueQuantizer<float>::Quantize<false>(float u) const;
    template ValueQuantizer<double>::QWordVal ValueQuantizer<double>::Quantize<true>(double u) const;
    template ValueQuantizer<double>::QWordVal ValueQuantizer<double>::Quantize<false>(double u) const;
    template bool ValueQuantizer<float>::Quantize1<true>(float u) const;
    template bool ValueQuantizer<float>::Quantize1<false>(float u) const;
    template bool ValueQuantizer<double>::Quantize1<true>(double u) const;
    template bool ValueQuantizer<double>::Quantize1<false>(double u) const;
}}}
#endif
