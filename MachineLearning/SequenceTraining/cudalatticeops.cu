// .cu file --#includes all actual .cu files which we store as .cu.h so we get syntax highlighting (VS does not recognize .cu files)
//
// F. Seide, V-hansu

#include <stdexcept>

namespace msra { namespace cuda {

    // call this after all kernel launches
    // This is non-blocking. It catches launch failures, but not crashes during execution.
    static void checklaunch (const char * fn)
    {
        hipError_t rc = hipGetLastError();
        if (rc != hipSuccess)
        {
            char buf[1000];
            sprintf(buf, "%s: launch failure: %s (cuda error %d)", fn, hipGetErrorString (rc), rc);
            throw std::runtime_error (buf);
        }
    }

};};

// now include actual code which is in those files to allow for code highlighting etc.
#include "cudalatticeops.cu.h"
