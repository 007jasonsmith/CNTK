#include "hip/hip_runtime.h"
//
// <copyright file="GPUMatrix.cu" company="Microsoft">
//     Copyright (c) Microsoft Corporation.  All rights reserved.
// </copyright>
//

#include "stdafx.h"
#include "Basics.h"
#include "BestGpu.h"
//#include "DebugUtil.h"

#ifndef CPUONLY

#include "GPUTensor.h"
#include "GPUMatrix.h"
#include "GPUMatrixCUDAKernels.cuh"
#include "CommonMatrix.h"
#define TENSOR_OPS_DECL __device__ __host__
#include "TensorOps.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <assert.h>

#ifndef let
#define let const auto
#endif

#pragma comment (lib, "cudart.lib")     // instruct linker to reference these libs
#pragma comment (lib, "cublas.lib")

#pragma warning (disable: 4267) // conversion from 'size_t' to 'unsigned int'; happens in CUDA <<<a,b>>> syntax if a and b are size_t
#pragma warning (disable: 4127) // conditional expression is constant; "if (sizeof(ElemType)==sizeof(float))" triggers this
#pragma warning (disable: 4702) // unreachable code; triggered for unknown reasons

extern bool do_sync;

#ifdef _WIN32
// thread local storage to access the current stream, initalize to default stream
__declspec (thread)
#endif
extern hipStream_t t_stream;

namespace Microsoft { namespace MSR { namespace CNTK {

    // =======================================================================
    // TensorView support
    // =======================================================================

    // To save time, this makes extensive use of templates and macros.

    // -----------------------------------------------------------------------
    // simple fixed-size arrays for passing dimension information by value
    // since CUDA can't just take our std::array and std::vector
    // -----------------------------------------------------------------------

    template<typename T, size_t N>
    struct FixedArray
    {
        T m_data[N];
        __device__ __host__ size_t size() const { return N; }
        __device__ __host__ T & operator[](size_t n)       { return m_data[n]; }
        __device__ __host__ T   operator[](size_t n) const { return m_data[n]; }
        template<class VEC> FixedArray(const VEC & data)    // construct from CPU-side STL array or vector
        {
            assert(data.size() == N);
            for (size_t n = 0; n < N; n++)
            {
                m_data[n] = (T)data[n];
                if (m_data[n] != data[n])   // overflow check
                    InvalidArgument("FixedArray: Dimensions out of range, too few bits.");
            }
        }
    };
    template<typename T>        // specialized version for 0 elements
    struct FixedArray<T, 0>
    {
        __device__ __host__ size_t size() const { return 0; }
        template<class VEC> FixedArray(const VEC & data) { assert(data.size() == 0); UNUSED(data); }
    };

    template<typename T, size_t N, size_t K>    // N = which input/output; K = index depth
    struct FixedMatrix
    {
        T m_data[N][K];
        __device__ __host__ size_t getNumRows() const { return N; }
        __device__ __host__ size_t getNumCols() const { return K; }
        __device__ __host__ T & operator()(size_t n, size_t k)       { return m_data[n][k]; }
        __device__ __host__ T   operator()(size_t n, size_t k) const { return m_data[n][k]; }
        template<typename U> FixedMatrix(const array<SmallVector<U>, N> & data)  // construct from CPU-side array of vectors
        {
            assert(data.size() == N);
            for (size_t n = 0; n < N; n++)
            {
                assert(data[n].size() == K);
                for (size_t k = 0; k < K; k++)
                {
                    m_data[n][k] = (T)data[n][k];
                    if (m_data[n][k] != data[n][k])   // overflow check
                        InvalidArgument("FixedArray: Dimensions out of range, too few bits.");
                }
            }
        }
    };
    template<typename T, size_t N>        // specialized version for 0 elements
    struct FixedMatrix<T, N, 0>
    {
        __device__ __host__ size_t getNumRows() const { return N; }
        __device__ __host__ size_t getNumCols() const { return 0; }
        template<typename U> FixedMatrix(const array<SmallVector<U>, N> & data) { assert(data.size() == N); for (size_t n = 0; n < N; n++) assert(data[n].size() == 0); UNUSED(data); }
    };

    // -----------------------------------------------------------------------
    // function to actually compute a function of (N-1) inputs based on the opcode
    // -----------------------------------------------------------------------

    template<class ElemType>
    struct TensorOps
    {
        static __device__ ElemType Compute(const FixedArray<ElemType*, 1> & pointers, ElementWiseOperator op)
        {
#define CaseNullaryTensorOp(oper) case ElementWiseOperator::op ## oper: return Op ## oper<ElemType>()
            switch (op)
            {
            ForAllNullaryOps(CaseNullaryTensorOp);
            default: return OpConstOne<ElemType>();   // (failure--we only have one nullary op, so use the same, maybe it will eliminate the switch altogether)
            }
        }
        static __device__ ElemType Compute(const FixedArray<ElemType*, 2> & pointers, ElementWiseOperator op)
        {
            ElemType a = *(pointers[0]);
#define CaseUnaryTensorOp(oper) case ElementWiseOperator::op ## oper: return Op ## oper(a)
            switch (op)
            {
            ForAllUnaryOps(CaseUnaryTensorOp);
            default: return 0;  // (failure)
            }
        }
        static __device__ ElemType Compute(const FixedArray<ElemType*, 3> & pointers, ElementWiseOperator op)
        {
            ElemType a = *(pointers[0]);
            ElemType b = *(pointers[1]);
#define CaseBinaryTensorOp(oper) case ElementWiseOperator::op ## oper: return Op ## oper(a,b)
            switch (op)
            {
            ForAllBinaryOps(CaseBinaryTensorOp);    // note: this costs about 6% compared to having only a single case
            default: return 0;  // (failure)
            }
        }
        static __device__ ElemType Compute(const FixedArray<ElemType*, 4> & pointers, ElementWiseOperator op)
        {
            ElemType a = *(pointers[0]);
            ElemType b = *(pointers[1]);
            ElemType c = *(pointers[2]);
#define CaseTernaryTensorOp(oper) case ElementWiseOperator::op ## oper: return Op ## oper(a,b,c)
            switch (op)
            {
            ForAllTernaryOps(CaseTernaryTensorOp);
            default: return 0;  // (failure)
            }
        }
    };

    // -----------------------------------------------------------------------
    // function to compute the value for a given output location (this version performs reduction if needed)
    // -----------------------------------------------------------------------

    template<class ElemType, C_size_t N, C_int M, C_int m>
    struct TensorOpReduce
    {
        // this version for m >= 0
        static __device__ ElemType Compute(FixedArray<ElemType*, N> pointers, ElementWiseOperator op,
                                           const FixedArray<C_unsigned_int, M> & reducingOpDims, const FixedMatrix<C_int, N, M> & reducingStrides)
        {
            // start with index 0
            // Using 'double' since we are memory-bound anyway.
            double/*ElemType*/ aggregate = TensorOpReduce<ElemType, N, M, m - 1>::Compute(pointers, op, reducingOpDims, reducingStrides);
            // apply this index to the pointers
            C_size_t dim = reducingOpDims[m];
            for (C_size_t k = 1/*done with k=0 already*/; k < dim; k++)
            {
                // bump the pointers
                for (C_size_t i = 0; i < N; i++)
                    pointers[i] += reducingStrides(i,(C_size_t)m);
                ElemType val = TensorOpReduce<ElemType, N, M, m - 1>::Compute(pointers, op, reducingOpDims, reducingStrides);
                aggregate += val;
            }
            return (ElemType)aggregate;
        }
    };

    // this one terminates the template recursion over reduction dimensions
    // The pointers are pointing to the input element.
    template<class ElemType, C_size_t N, C_int M>
    struct TensorOpReduce<ElemType, N, M, /*m=*/-1>
    {
        // this version for m = -1
        // the pointers are pointing to the right location(s) to take the operation over
        static __device__ ElemType Compute(FixedArray<ElemType*, N> pointers, ElementWiseOperator op,
                                           const FixedArray<C_unsigned_int, M> & /*reducingOpDims*/, const FixedMatrix<C_int, N, M> & /*reducingStrides*/)
        {
            return TensorOps<ElemType>::Compute(pointers, op);   // finally computing something!
        }
    };

    // -----------------------------------------------------------------------
    // function to compute one constituent of the value for a given output location (this version has reduction done outside)
    // -----------------------------------------------------------------------

    template<class ElemType, C_size_t N, C_int M, C_int m>
    struct TensorOpParallelReduce
    {
        // this version for m >= 0
        static __device__ ElemType Compute(CUDA_LONG id, FixedArray<ElemType*, N> pointers, ElementWiseOperator op,
                                           const FixedArray<C_unsigned_int, M> & reducingOpDims, const FixedMatrix<C_int, N, M> & reducingStrides)
        {
            // map id (location on grid) to index[k]
            C_size_t stride = 1;                    // compute the stride. This seems expensive, but since we we only currently support M <= 2, this is just compile-time selection between 1 and reducingOpDims[0].
            for (int i = 0; i < m; i++)
                stride *= reducingOpDims[(C_size_t)i];
            C_size_t index = id / stride;           // this dimension. For m=0, the stride is 1 and hence the division will be removed at compile time.
            id = id % stride;                       // remaining dimensions inside this. For m=0 this value is ignored and hence not even computed.
            // apply this index to the pointers
            for (C_size_t i = 0; i < N; i++)
                pointers[i] += index * reducingStrides(i, (C_size_t)m);    // now this dimension is taken care of
            return TensorOpParallelReduce<ElemType, N, M, m - 1>::Compute(id, pointers, op, reducingOpDims, reducingStrides);
        }
    };

    // this one terminates the template recursion over reduction dimensions
    // The pointers are pointing to the input element.
    template<class ElemType, C_size_t N, C_int M>
    struct TensorOpParallelReduce<ElemType, N, M, /*m=*/-1>
    {
        // this version for m = -1
        // the pointers are pointing to the right location(s) to take the operation over
        static __device__ ElemType Compute(CUDA_LONG /*id*/, FixedArray<ElemType*, N> pointers, ElementWiseOperator op,
                                           const FixedArray<C_unsigned_int, M> & /*reducingOpDims*/, const FixedMatrix<C_int, N, M> & /*reducingStrides*/)
        {
            return TensorOps<ElemType>::Compute(pointers, op);   // finally computing something!
        }
    };

    // -----------------------------------------------------------------------
    // perform loop over regular index k for N-nary operations (N counting the output)
    // -----------------------------------------------------------------------

    // The canonical case, vector op without reduction, is this PTX function:
    // _ZN9Microsoft3MSR4CNTK15_launchTensorOpIfLi3ELi0ELi1EEEvT_NS1_10FixedArrayIPS3_XT0_EEES3_NS1_19ElementWiseOperatorENS4_IiXT2_EEENS1_11FixedMatrixIiXT0_EXT2_EEENS4_IiXT1_EEENS9_IiXT0_EXT1_EEEi
    //                                   float ^      ^ aggregate loop
    //                                      args? ^       ^ input dims
    // _ZN9Microsoft3MSR4CNTK15_launchTensorOpIfLi2ELi0ELi1EEEvT_NS1_10FixedArrayIPS3_XT0_EEES3_NS1_19ElementWiseOperatorENS4_IiXT2_EEENS1_11FixedMatrixIiXT0_EXT2_EEENS4_IiXT1_EEENS9_IiXT0_EXT1_EEEi

    // The 'pointers' only refer to a single element, so we will bump them in-place to perform indexing.
    template<class ElemType, C_size_t N, C_int M, C_int K, bool parallelReduce, C_int k>
    struct TensorOpElement
    {
        // template-recursive version loops over indices
        static __device__ void Compute(CUDA_LONG id, ElemType beta, FixedArray<ElemType*, N> & pointers, ElemType alpha, ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, K> & regularOpStrides,  const FixedMatrix<C_int, N, K> & regularStrides,
                                       const FixedArray<C_unsigned_int, M> & reducingOpDims, const FixedMatrix<C_int, N, M> & reducingStrides, CUDA_LONG reductionDim)
        {
            // map id (location on grid) to index[k]
            C_size_t stride = regularOpStrides[(C_size_t)k];
            C_size_t index = id / stride;             // this dimension
            id = id % stride;                       // remaining dimensions inside this
            // apply this index to the pointers
            for (C_size_t i = 0; i < N; i++)
                pointers[i] += index * regularStrides(i,(C_size_t)k);    // now this dimension is taken care of
            // process the previous index
            TensorOpElement<ElemType, N, M, K, parallelReduce, k - 1>::Compute(id, beta, pointers, alpha, op, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionDim);
        }
    };

    // specialization for k=0 where op stride is guaranteed to be 1
    template<class ElemType, C_size_t N, C_int M, C_int K, bool parallelReduce>
    struct TensorOpElement<ElemType, N, M, K, parallelReduce, /*k=*/0>
    {
        // template-recursive version loops over indices
        static __device__ void Compute(CUDA_LONG id, ElemType beta, FixedArray<ElemType*, N> & pointers, ElemType alpha, ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, K> & regularOpStrides,  const FixedMatrix<C_int, N, K> & regularStrides,
                                       const FixedArray<C_unsigned_int, M> & reducingOpDims,    const FixedMatrix<C_int, N, M> & reducingStrides, CUDA_LONG reductionDim)
        {
            // map id (location on grid) to index[k]
            C_size_t index = id;                      // this dimension
            // apply this index to the pointers
            for (C_size_t i = 0; i < N; i++)
                pointers[i] += index * regularStrides(i,0);    // now this dimension is taken care of
            // process the previous index
            TensorOpElement<ElemType, N, M, K, parallelReduce, -1>::Compute(/*id*/0, beta, pointers, alpha, op, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionDim);
        }
    };

    // apply beta and alpha and save
    template<class ElemType, class PointersType>
    static __device__ void SetFinalValue(ElemType val, ElemType beta, const PointersType & pointers, ElemType alpha)
    {
        // scale
        val *= alpha;
        // combine with previous value in target matrix, then write it out
        auto * pout = pointers[pointers.size() - 1];
        if (beta != 0)
            val += beta * *pout;
        // save
        *pout = val;
    }

    // specialization for k = -1 terminates the template recursion, and computes reductions in a for loop
    template<class ElemType, C_size_t N, C_int M, C_int K>
    struct TensorOpElement<ElemType, N, M, K, /*parallelReduce=*/false, /*k=*/-1>
    {
        // template-recursion-teminating version computes the actual value for this output location
        // now the output pointers point to the right element (input pointers may still iterate for reduction)
        static __device__ void Compute(CUDA_LONG /*id*/, ElemType beta, FixedArray<ElemType*, N> & pointers, ElemType alpha, ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, K> & /*regularOpStrides*/, const FixedMatrix<C_int, N, K> & /*regularStrides*/,
                                       const FixedArray<C_unsigned_int, M> & reducingOpDims,       const FixedMatrix<C_int, N, M> & reducingStrides, CUDA_LONG /*reductionDim*/)
        {
            // compute the operation for this output coordinate
            // This may still involve a reduction over inverse-broadcasting dimensions.
            ElemType val = TensorOpReduce<ElemType, N, M, M - 1>::Compute(pointers, op, reducingOpDims, reducingStrides);
            // and save the final value
            SetFinalValue(val, beta, pointers, alpha);
        }
    };

    // specialization for k = -1 terminates the template recursion, and computes reductions in parallel
    template<class ElemType, C_size_t N, C_int M, C_int K>
    struct TensorOpElement<ElemType, N, M, K, /*parallelReduce=*/true, /*k=*/-1>
    {
        // template-recursion-teminating version computes the actual value for this output location
        // now the output pointers point to the right element (input pointers may still iterate for reduction)
        static __device__ void Compute(CUDA_LONG /*id*/, ElemType beta, FixedArray<ElemType*, N> & pointers, ElemType alpha, ElementWiseOperator op,
                                       const FixedArray<C_unsigned_int, K> & /*regularOpStrides*/, const FixedMatrix<C_int, N, K> & /*regularStrides*/,
                                       const FixedArray<C_unsigned_int, M> & reducingOpDims,       const FixedMatrix<C_int, N, M> & reducingStrides, CUDA_LONG reductionDim)
        {
            CUDA_LONG redId = threadIdx.x;  // note: blockDim.x might be out of bounds w.r.t. redId in case we split reduction
            reductionDim = blockDim.x;

            // accumulator
            __shared__ double accumulators[GridDim::maxThreadsPerBlock];

            // compute the operation for this input coordinate
            if (redId < reductionDim)
               accumulators[redId] = TensorOpParallelReduce<ElemType, N, M, M - 1>::Compute(redId, pointers, op, reducingOpDims, reducingStrides);

            // reduce    --cf https://docs.nvidia.com/cuda/samples/6_Advanced/reduction/doc/reduction.pdf
            __syncthreads();
            static_assert(GridDim::maxThreadsPerBlock <= 512, "GridDim::maxThreadsPerBlock too large, need to add manually unrolled steps");
            if (redId < 256 && redId + 256 < reductionDim) accumulators[redId] += accumulators[redId + 256]; if (0 + 256 < reductionDim) __syncthreads();   // sync if condition true for at least one thread
            if (redId < 128 && redId + 128 < reductionDim) accumulators[redId] += accumulators[redId + 128]; if (0 + 128 < reductionDim) __syncthreads();
            if (redId < 64  && redId + 64  < reductionDim) accumulators[redId] += accumulators[redId + 64];  if (0 + 64  < reductionDim) __syncthreads();
            if (redId < 32  && redId + 32  < reductionDim) accumulators[redId] += accumulators[redId + 32];  if (0 + 32  < reductionDim) __syncthreads();   // somehow I still need to sync, contradicting the PDF
            if (redId < 16  && redId + 16  < reductionDim) accumulators[redId] += accumulators[redId + 16];  if (0 + 16  < reductionDim) __syncthreads();
            if (redId < 8   && redId + 8   < reductionDim) accumulators[redId] += accumulators[redId + 8];                               __syncthreads();
            if (redId < 4   && redId + 4   < reductionDim) accumulators[redId] += accumulators[redId + 4];                               __syncthreads();
            if (redId < 2   && redId + 2   < reductionDim) accumulators[redId] += accumulators[redId + 2];                               __syncthreads();
            if (redId < 1   && redId + 1   < reductionDim) accumulators[redId] += accumulators[redId + 1];

            // now set final value to output coordinate
            if (redId == 0)
            {
                ElemType val = (ElemType)accumulators[0];
                SetFinalValue(val, beta, pointers, alpha);
            }
        }
    };

    // -----------------------------------------------------------------------
    // kernel and launch
    // -----------------------------------------------------------------------

    // the top-level kernel
    template<class ElemType, C_size_t N, C_int M, C_int K>
    __global__ void _launchTensorOp(ElemType beta, FixedArray<ElemType*, N> pointers, ElemType alpha, ElementWiseOperator op,
                                    FixedArray<C_unsigned_int, K> regularOpStrides, FixedMatrix<C_int, N, K> regularStrides,  CUDA_LONG numElements,
                                    FixedArray<C_unsigned_int, M> reducingOpDims,   FixedMatrix<C_int, N, M> reducingStrides, CUDA_LONG reductionDim)
    {
        CUDA_LONG id = GridDim::GetLinearThreadId();
        if (id < numElements)       // note: there are no __syncthread() calls inside
            TensorOpElement<ElemType, N, M, K, false, K - 1>::Compute(id, beta, pointers, alpha, op, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionDim);
    }
    template<class ElemType, C_size_t N, C_int M, C_int K>
    __global__ void _launchTensorOpParallelReduction(ElemType beta, FixedArray<ElemType*, N> pointers, ElemType alpha, ElementWiseOperator op,
                                                     FixedArray<C_unsigned_int, K> regularOpStrides, FixedMatrix<C_int, N, K> regularStrides,  CUDA_LONG numElements,
                                                     FixedArray<C_unsigned_int, M> reducingOpDims,   FixedMatrix<C_int, N, M> reducingStrides, CUDA_LONG reductionDim)
    {
        CUDA_LONG id = gridDim.y * blockIdx.x + blockIdx.y;   // input dimensions are Y dimension of blocks in this case, so we can use thread dim for shared-memory/parallelization
        if (id < numElements)       // note: we have __syncthread() calls but only entire blocks in sync, so this is OK
            TensorOpElement<ElemType, N, M, K, true, K - 1>::Compute(id, beta, pointers, alpha, op, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, reductionDim);
    }

    // launch tensor op with CUDA
    // All dimensions (N-ariness, number of input dimensions K and number of reduction dimensions M) are bound to template parameters now.
    template<class ElemType, C_size_t N, C_int M, C_int K>
    static void LaunchTensorOp(ElemType beta, array<ElemType*, N> pointerVector, ElemType alpha, ElementWiseOperator op,
                               const SmallVector<size_t> & regularOpDims,       const array<SmallVector<ptrdiff_t>, N> & regularStrideVectors,
                               const SmallVector<size_t> & reducingOpDimVector, const array<SmallVector<ptrdiff_t>, N> & reducingStrideVectors)
    {
        // copy all parameters to CUDA-compatible data structures
        FixedArray<ElemType*, N> pointers(pointerVector);
        SmallVector<C_size_t> regularOpStrideVector;    // kernel needs the strides for converting thread index back to multi-dimensional tensor index
        C_size_t numElements = 1;
        for (C_size_t k = 0; k < regularOpDims.size(); k++)
        {
            regularOpStrideVector.push_back(numElements);
            numElements *= (C_size_t)regularOpDims[k];
        }
        FixedArray<C_unsigned_int, K> regularOpStrides(regularOpStrideVector);
        FixedMatrix<C_int, N, K> regularStrides(regularStrideVectors);
        FixedArray<C_unsigned_int, M> reducingOpDims(reducingOpDimVector);
        FixedMatrix<C_int, N, M> reducingStrides(reducingStrideVectors);

        // launch the kernel
        CUDA_LONG NN = (CUDA_LONG)numElements;      // linear space identifying each individual input element
        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));

        // do some optimization for reductions
        // Cases:
        //  - input elements >> GPU procs  -->  do reduction in inner loop
        //  - reduction dimension fits into a single kernel  -->  launch it that way
        //  - reduction dimension requires multiple kernels  -->  use atomic add, to avoid temp mem alloc  --is this any good?
        //     - PlusNode: reducing to a bias for small matrices
        //     - ScaleNode: big elementwise product reduced to a scalar (dot product)
#if 1
        C_size_t reductionDim = 1;  // number of elements to reduce over
        for (C_size_t k = 0; k < reducingOpDimVector.size(); k++)
            reductionDim *= (C_size_t)reducingOpDimVector[k];
        let & props = GridDim::GetDeviceProps();
        if (reductionDim > 1 && NN < props.multiProcessorCount * props.warpSize     && reductionDim > GridDim::maxThreadsPerBlock)
        {
            fprintf(stderr, "%d %d\n", (int)reductionDim, (int)props.multiProcessorCount * props.warpSize);
        }
        if (reductionDim > 1 && NN < props.multiProcessorCount * props.warpSize             && reductionDim <= GridDim::maxThreadsPerBlock)
        {
            if (reductionDim <= GridDim::maxThreadsPerBlock)
            {
                // one thread block per reduction is sufficient
                // TODO: In the special case where reduction dim <= 16 (half warp size), we could fit more than one reduction.
                GridDim grid(NN);
                let blocksPerGrid   = dim3(grid.m_blocksPerGrid, grid.m_threadsPerBlock); // block Y is element dimension
                let threadsPerBlock = dim3(reductionDim);       // X dimension is reduction dimension
                _launchTensorOpParallelReduction<ElemType, N, M, K> << <blocksPerGrid, threadsPerBlock, reductionDim * sizeof(double), t_stream >> >(beta, pointers, alpha, op, regularOpStrides, regularStrides, grid.m_N, reducingOpDims, reducingStrides, reductionDim);
            }
            else
            {
                // we need more than one block for each reduction
                // Temporary memory is required.
#if 0
                // We have too few elements to do reduction in inner loop. Need to be more clever.
                CUDA_LONG numReductionThreads = min(reductionDim, GridDim::maxThreadsPerBlock);
                // TODO: special case: <= half warp size: We can fit more than one.
                // round up to multiples of warp size
                numReductionThreads = (numReductionThreads + props.warpSize - 1) / props.warpSize * props.warpSize;

                // atomicAdd mode: need to zero out/pre-multiply first
                if (reductionDim > numReductionThreads)
                {
                    LaunchTensorOp<ElemType, /*N=*/1, /*M=*/0, K>(beta, array<ElemType*, 1> { pointerVector.back() }, /*alpha=*/0, ElementWiseOperator::opConstOne,
                                                                  regularOpDims, array<SmallVector<ptrdiff_t>, 1> { regularStrideVectors.back() },
                                                                  SmallVector<size_t>(), array<SmallVector<ptrdiff_t>, 1>());
                    beta = 1;       // and actual operation now adds with weight 1 since we already initialized/pre-multiplied
                }
                size_t numReductionDoubles = numReductionThreads;   // using 4k shared mem out of 48k, should be OK

                GridDim grid(NN);
                _launchTensorOp<ElemType, N, M, K> << <grid.m_blocksPerGrid, grid.m_threadsPerBlock, numReductionDoubles * sizeof(ElemType), t_stream >> >(beta, pointers, alpha, op, regularOpStrides, regularStrides, reducingOpDims, reducingStrides, grid.m_N);
#endif
            }
        }
        else
#endif
        {
            GridDim grid(NN);
            _launchTensorOp<ElemType, N, M, K> << <grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream >> >(beta, pointers, alpha, op, regularOpStrides, regularStrides, grid.m_N, reducingOpDims, reducingStrides, 1);
        }
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    // for linear unary ops, we need to define a functor for every function for use as a template parameter (lambda syntax doesn't work in CUDA 7)
    #define DefineUnaryTensorFunctor(oper) \
        struct Functor ## oper { template<class ElemType> static __device__ ElemType f(ElemType a) { return Op ## oper(a); } };
    ForAllUnaryOps(DefineUnaryTensorFunctor);

    // the top-level kernel for linear unary ops
    // Note: If we have a beta, we have 2 memory accesses, so this optimization may no longer be needed as we are memory-bound.
    template<class ElemType, class FN>
    __global__ void _launchUnaryTensorOp(ElemType beta, const ElemType * pa, ElemType * pb, ElemType alpha, CUDA_LONG numElements)
    {
        CUDA_LONG id = GridDim::GetLinearThreadId();
        if (id >= numElements)
            return;
        ElemType a = pa[id];
        ElemType val = FN::f(a);
        val *= alpha;
        if (beta != 0)
            val += beta * pb[id];
        pb[id] = val;
    }
    // version without beta and alpha
    template<class ElemType, class FN>
    __global__ void _launchUnaryTensorOp(const ElemType * pa, ElemType * pb, CUDA_LONG numElements)
    {
        CUDA_LONG id = GridDim::GetLinearThreadId();
        if (id >= numElements)
            return;
        ElemType a = pa[id];
        ElemType val = FN::f(a);
        pb[id] = val;
    }

    // special case of linear unary operation
    template<class ElemType>
    void LaunchUnaryTensorOp(ElemType beta, const ElemType * pa, ElemType * pb, ElemType alpha, ElementWiseOperator op, size_t regularOpDim)
    {
        CUDA_LONG NN = (CUDA_LONG)regularOpDim;

        #define CaseLaunchUnaryTensorOp(oper) case ElementWiseOperator::op ## oper: \
            if (beta == 0 && alpha == 1) \
                return _launchUnaryTensorOp<ElemType,Functor ## oper> << <grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream >> >(pa, pb, NN); \
            else \
                return _launchUnaryTensorOp<ElemType,Functor ## oper> << <grid.m_blocksPerGrid, grid.m_threadsPerBlock, 0, t_stream >> >(beta, pa, pb, alpha, NN);

        hipEvent_t done = nullptr;
        if (do_sync)    CUDA_CALL(hipEventCreate(&done));
        GridDim grid(NN);
        switch (op)
        {
        ForAllUnaryOps(CaseLaunchUnaryTensorOp);
        default: LogicError("LaunchTensorOp1: Unknown op code %d.", (int)op);
        }
        if (do_sync)    CUDA_CALL(hipEventRecord(done));
        if (do_sync)    CUDA_CALL(hipEventSynchronize(done));
        if (do_sync)    CUDA_CALL(hipEventDestroy(done));
    }

    // -----------------------------------------------------------------------
    // map runtime parameters N to template parameters
    // -----------------------------------------------------------------------

    // tensor operation with k+1 dimensions (-1 means scalar)
    template<class ElemType, C_size_t N, C_int K>
    static void TensorOpWithRegularLoop(ElemType beta, const array<ElemType*, N> & pointers, ElemType alpha, ElementWiseOperator op,
                                        const SmallVector<size_t> & regularOpDims,  const array<SmallVector<ptrdiff_t>, N> & regularStrides,
                                        const SmallVector<size_t> & reducingOpDims, const array<SmallVector<ptrdiff_t>, N> & reducingStrides)
    {
        size_t dims = reducingOpDims.size();
        switch (dims)
        {
        case 2: return LaunchTensorOp<ElemType, N, 2, K>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
        case 1: return LaunchTensorOp<ElemType, N, 1, K>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
        case 0: return LaunchTensorOp<ElemType, N, 0, K>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
        default: LogicError("TensorOp: %d non-flattened reduction dimensions are not supported.", (C_int)dims);
        }
    }

    // tensor operation, generalized in number of arguments
    // This function now expands into different k. It also eliminates the offsets by adding them to the pointers.
    template<class ElemType, C_size_t N>
    void TensorOpN(ElemType beta, array<ElemType*, N> pointers, ElemType alpha, ElementWiseOperator op,
                        const array<size_t, N> & offsets,
                        const SmallVector<size_t> & regularOpDims,  const array<SmallVector<ptrdiff_t>, N> & regularStrides,
                        const SmallVector<size_t> & reducingOpDims, const array<SmallVector<ptrdiff_t>, N> & reducingStrides)
    {
        for (C_size_t i = 0; i < N; i++)  // N = a small constant, this will be unrolled
            pointers[i] += offsets[i];
        size_t dims = regularOpDims.size();
        switch (dims)
        {
        case 4: return TensorOpWithRegularLoop<ElemType, N, 4>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
        case 3: return TensorOpWithRegularLoop<ElemType, N, 3>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
        case 2: return TensorOpWithRegularLoop<ElemType, N, 2>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
        case 1: return TensorOpWithRegularLoop<ElemType, N, 1>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
        case 0: return TensorOpWithRegularLoop<ElemType, N, 0>(beta, pointers, alpha, op, regularOpDims, regularStrides, reducingOpDims, reducingStrides);
        default: LogicError("TensorOp: %d non-flattened input dimensions are not supported.", (C_int)dims);
        }
    }

    //------------------------------------------------------------------------
    // explicit instantiations--these are being called from GPUMatrix.cu
    //------------------------------------------------------------------------

    template void TensorOpN<float,  2>(float beta, array<float*, 2> pointers, float alpha, ElementWiseOperator op,
                                       const array<size_t, 2> & offsets,
                                       const SmallVector<size_t> & regularOpDims,  const array<SmallVector<ptrdiff_t>, 2> & regularStrides,
                                       const SmallVector<size_t> & reducingOpDims, const array<SmallVector<ptrdiff_t>, 2> & reducingStrides);
    template void TensorOpN<float,  3>(float beta, array<float*, 3> pointers, float alpha, ElementWiseOperator op,
                                       const array<size_t, 3> & offsets,
                                       const SmallVector<size_t> & regularOpDims,  const array<SmallVector<ptrdiff_t>, 3> & regularStrides,
                                       const SmallVector<size_t> & reducingOpDims, const array<SmallVector<ptrdiff_t>, 3> & reducingStrides);
    template void TensorOpN<float,  4>(float beta, array<float*, 4> pointers, float alpha, ElementWiseOperator op,
                                       const array<size_t, 4> & offsets,
                                       const SmallVector<size_t> & regularOpDims,  const array<SmallVector<ptrdiff_t>, 4> & regularStrides,
                                       const SmallVector<size_t> & reducingOpDims, const array<SmallVector<ptrdiff_t>, 4> & reducingStrides);
    template void TensorOpN<double, 2>(double beta, array<double*, 2> pointers, double alpha, ElementWiseOperator op,
                                       const array<size_t, 2> & offsets,
                                       const SmallVector<size_t> & regularOpDims,  const array<SmallVector<ptrdiff_t>, 2> & regularStrides,
                                       const SmallVector<size_t> & reducingOpDims, const array<SmallVector<ptrdiff_t>, 2> & reducingStrides);
    template void TensorOpN<double, 3>(double beta, array<double*, 3> pointers, double alpha, ElementWiseOperator op,
                                       const array<size_t, 3> & offsets,
                                       const SmallVector<size_t> & regularOpDims,  const array<SmallVector<ptrdiff_t>, 3> & regularStrides,
                                       const SmallVector<size_t> & reducingOpDims, const array<SmallVector<ptrdiff_t>, 3> & reducingStrides);
    template void TensorOpN<double, 4>(double beta, array<double*, 4> pointers, double alpha, ElementWiseOperator op,
                                       const array<size_t, 4> & offsets,
                                       const SmallVector<size_t> & regularOpDims,  const array<SmallVector<ptrdiff_t>, 4> & regularStrides,
                                       const SmallVector<size_t> & reducingOpDims, const array<SmallVector<ptrdiff_t>, 4> & reducingStrides);

    template void LaunchUnaryTensorOp(float beta, const float * pa, float * pb, float alpha, ElementWiseOperator op, size_t regularOpDim);
    template void LaunchUnaryTensorOp(double beta, const double * pa, double * pb, double alpha, ElementWiseOperator op, size_t regularOpDim);

}}}

#endif // CPUONLY
